#include "hip/hip_runtime.h"
#include "ExchangeCUDA.h"

#if COMPILECUDA == 1

#ifdef MODULE_COMPILATION_EXCHANGE

#include "BorisCUDALib.cuh"

#include "MeshCUDA.h"
#include "MeshParamsControlCUDA.h"
#include "MeshDefs.h"

//////////////////////////////////////////////////////////////////////// UPDATE FIELD

__global__ void ExchangeCUDA_FM_UpdateField(ManagedMeshCUDA& cuMesh, ManagedModulesCUDA& cuModule, bool do_reduction)
{
	cuVEC_VC<cuReal3>& M = *cuMesh.pM;
	cuVEC<cuReal3>& Heff = *cuMesh.pHeff;

	int idx = blockIdx.x * blockDim.x + threadIdx.x;

	cuBReal energy_ = 0.0;

	if (idx < Heff.linear_size()) {

		cuReal3 Hexch = cuReal3();

		if (M.is_not_empty(idx)) {

			cuBReal Ms = *cuMesh.pMs;
			cuBReal A = *cuMesh.pA;
			cuMesh.update_parameters_mcoarse(idx, *cuMesh.pMs, Ms, *cuMesh.pA, A);

			if (*cuMesh.pbase_temperature > 0.0 && *cuMesh.pT_Curie > 0.0) {

				//for finite temperature simulations the magnetization length may have a spatial variation
				//this will not affect the transverse torque (mxH), but will affect the longitudinal term in the sLLB equation (m.H) and cannot be neglected when close to Tc.

				cuReal33 Mg = M.grad_neu(idx);
				cuReal3 dMdx = Mg.x, dMdy = Mg.y, dMdz = Mg.z;

				cuBReal delsq_Msq = 2 * M[idx] * (M.dxx_neu(idx) + M.dyy_neu(idx) + M.dzz_neu(idx)) + 2 * (dMdx * dMdx + dMdy * dMdy + dMdz * dMdz);
				cuBReal Mnorm = M[idx].norm();
				if (cuIsNZ(Mnorm)) Hexch = (2 * A / (MU0*Ms*Ms)) * (M.delsq_neu(idx) - M[idx] * delsq_Msq / (2 * Mnorm*Mnorm));
			}
			else {

				//zero temperature simulations : magnetization length could still vary but will only affect mxH term, so not needed for 0K simulations.
				Hexch = 2 * A * M.delsq_neu(idx) / ((cuBReal)MU0 * Ms * Ms);
			}

			if (do_reduction) {

				int non_empty_cells = M.get_nonempty_cells();
				if (non_empty_cells) energy_ = -(cuBReal)MU0 * M[idx] * Hexch / (2 * non_empty_cells);
			}

			if (do_reduction && cuModule.pModule_Heff->linear_size()) (*cuModule.pModule_Heff)[idx] = Hexch;
			if (do_reduction && cuModule.pModule_energy->linear_size()) (*cuModule.pModule_energy)[idx] = -(cuBReal)MU0 * (M[idx] * Hexch) / 2;
		}

		Heff[idx] += Hexch;
	}

	if (do_reduction) reduction_sum(0, 1, &energy_, *cuModule.penergy);
}

__global__ void ExchangeCUDA_AFM_UpdateField(ManagedMeshCUDA& cuMesh, ManagedModulesCUDA& cuModule, bool do_reduction)
{
	cuVEC_VC<cuReal3>& M = *cuMesh.pM;
	cuVEC<cuReal3>& Heff = *cuMesh.pHeff;

	cuVEC_VC<cuReal3>& M2 = *cuMesh.pM2;
	cuVEC<cuReal3>& Heff2 = *cuMesh.pHeff2;

	int idx = blockIdx.x * blockDim.x + threadIdx.x;

	cuBReal energy_ = 0.0;

	if (idx < Heff.linear_size()) {

		cuReal3 Hexch = cuReal3();
		cuReal3 Hexch2 = cuReal3();

		if (M.is_not_empty(idx)) {

			cuReal2 Ms_AFM = *cuMesh.pMs_AFM;
			cuReal2 A_AFM = *cuMesh.pA_AFM;
			cuReal2 Ah = *cuMesh.pAh;
			cuReal2 Anh = *cuMesh.pAnh;
			cuMesh.update_parameters_mcoarse(idx, *cuMesh.pMs_AFM, Ms_AFM, *cuMesh.pA_AFM, A_AFM, *cuMesh.pAh, Ah, *cuMesh.pAnh, Anh);

			cuReal3 delsq_M_A = M.delsq_neu(idx);
			cuReal3 delsq_M_B = M2.delsq_neu(idx);

			cuReal2 Mmag = cuReal2(M[idx].norm(), M2[idx].norm());
			if (cuIsNZ(Mmag.i)) Hexch = (2 * A_AFM.i / ((cuBReal)MU0*Ms_AFM.i*Ms_AFM.i)) * delsq_M_A + (-4 * Ah.i * (M[idx] ^ (M[idx] ^ M2[idx])) / (Mmag.i*Mmag.i) + Anh.i * delsq_M_B) / ((cuBReal)MU0*Ms_AFM.i*Ms_AFM.j);
			if (cuIsNZ(Mmag.j)) Hexch2 = (2 * A_AFM.j / ((cuBReal)MU0*Ms_AFM.j*Ms_AFM.j)) * delsq_M_B + (-4 * Ah.j * (M2[idx] ^ (M2[idx] ^ M[idx])) / (Mmag.j*Mmag.j) + Anh.j * delsq_M_A) / ((cuBReal)MU0*Ms_AFM.i*Ms_AFM.j);

			if (do_reduction) {

				int non_empty_cells = M.get_nonempty_cells();
				if (non_empty_cells) energy_ = -(cuBReal)MU0 * (M[idx] * Hexch  + M2[idx] * Hexch2) / (4 * non_empty_cells);
			}

			if (do_reduction && cuModule.pModule_Heff->linear_size()) (*cuModule.pModule_Heff)[idx] = Hexch;
			if (do_reduction && cuModule.pModule_Heff2->linear_size()) (*cuModule.pModule_Heff2)[idx] = Hexch2;
			if (do_reduction && cuModule.pModule_energy->linear_size()) (*cuModule.pModule_energy)[idx] = -MU0 * (M[idx] * Hexch) / 2;
			if (do_reduction && cuModule.pModule_energy2->linear_size()) (*cuModule.pModule_energy2)[idx] = -MU0 * (M2[idx] * Hexch2) / 2;
		}

		Heff[idx] += Hexch;
		Heff2[idx] += Hexch2;
	}

	if (do_reduction) reduction_sum(0, 1, &energy_, *cuModule.penergy);
}

//----------------------- UpdateField LAUNCHER

void Exch_6ngbr_NeuCUDA::UpdateField(void)
{
	if (pMeshCUDA->GetMeshType() == MESH_ANTIFERROMAGNETIC) {

		//anti-ferromagnetic mesh

		pMeshCUDA->M.exchange_halos();
		pMeshCUDA->M2.exchange_halos();

		if (pMeshCUDA->CurrentTimeStepSolved()) {

			ZeroEnergy();

			for (mGPU.device_begin(); mGPU != mGPU.device_end(); mGPU++) {

				ExchangeCUDA_AFM_UpdateField <<< (pMeshCUDA->M.device_size(mGPU) + CUDATHREADS) / CUDATHREADS, CUDATHREADS >>> 
					(pMeshCUDA->cuMesh.get_deviceobject(mGPU), cuModule.get_deviceobject(mGPU), true);
			}
		}
		else {

			for (mGPU.device_begin(); mGPU != mGPU.device_end(); mGPU++) {

				ExchangeCUDA_AFM_UpdateField <<< (pMeshCUDA->M.device_size(mGPU) + CUDATHREADS) / CUDATHREADS, CUDATHREADS >>> 
					(pMeshCUDA->cuMesh.get_deviceobject(mGPU), cuModule.get_deviceobject(mGPU), false);
			}
		}
	}
	else {

		//ferromagnetic mesh

		pMeshCUDA->M.exchange_halos();
		
		if (pMeshCUDA->CurrentTimeStepSolved()) {

			ZeroEnergy();
			
			for (mGPU.device_begin(); mGPU != mGPU.device_end(); mGPU++) {

				ExchangeCUDA_FM_UpdateField <<< (pMeshCUDA->M.device_size(mGPU) + CUDATHREADS) / CUDATHREADS, CUDATHREADS >>>
					(pMeshCUDA->cuMesh.get_deviceobject(mGPU), cuModule.get_deviceobject(mGPU), true);
			}
		}
		else {

			for (mGPU.device_begin(); mGPU != mGPU.device_end(); mGPU++) {

				ExchangeCUDA_FM_UpdateField <<< (pMeshCUDA->M.device_size(mGPU) + CUDATHREADS) / CUDATHREADS, CUDATHREADS >>>
					(pMeshCUDA->cuMesh.get_deviceobject(mGPU), cuModule.get_deviceobject(mGPU), false);
			}
		}
	}

	//if using UVA to compute differential operators then synchronization needed now
	//otherwise a kernel on a device could finish and continue on to diff eq update (which will update M on device), whilst neighboring devices are still accessing these data - data race!
	//if using halo exchanges instead this is not a problem
	mGPU.synchronize_if_uva();

	if (pMeshCUDA->GetMeshExchangeCoupling()) CalculateExchangeCoupling(energy);
}

#endif

#endif