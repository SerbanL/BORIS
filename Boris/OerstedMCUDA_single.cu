#include "hip/hip_runtime.h"
#include "OerstedMCUDA_single.h"

#if COMPILECUDA == 1

#ifdef MODULE_COMPILATION_OERSTED

#include "BorisCUDALib.cuh"

#include "OerstedCUDA.h"

///////////////////////////////////////////////////////////////////////////////////////////////////////////////
///////////////////////////////////////////////////////////////////////////////////////////////////////////////
///////////////////////////////////////////////////////////////////////////////////////////////////////////////

//used to copy data from xRegion_R * ny * nz outside of yRegion to appropriate component
//this gets component_idx, idx_out (output index in component), and idx_in (input index)
__device__ bool from_notyRegion_in_xRegion_R_Oersted(
	int& idx, int& component_idx, int& idx_out, int& idx_in,
	cuSZ3& n, cuSZ3& N, cuINT2& xRegion_R, cuINT2& yRegion, int& num_devices)
{
	int nxRegion_R = xRegion_R.j - xRegion_R.i;
	int nyRegion = yRegion.j - yRegion.i;
	int nCy = n.y - nyRegion;

	if (idx < nxRegion_R * nCy * n.z) {

		//more than 2 components, use the general routine
		if (num_devices > 2) {

			//form ijk in complement space, noting that kernel was launched with (ny - nyRegion) cells along y, since nyRegion data does not need to be transferred
			cuINT3 ijk = cuINT3(idx % nxRegion_R, (idx / nxRegion_R) % nCy, idx / (nxRegion_R * nCy));

			//find which component we need to use
			//if yRegion is not the last region, then yRegion.j - yRegion.i is the n.y value common to all components apart from the last one
			//In this case the last component has n.y value of n.y - nyRegion * (num_devices - 1)
			//If yRegion is the last region, then the common n.y value of components before it is yRegion.i / (num_devices - 1)

			//yRegion is the last region
			if (yRegion.j == n.y) {

				//y dimension of components (they are all the same in this case as only component in last region can be different)
				int component_ny = yRegion.i / (num_devices - 1);
				component_idx = ijk.j / component_ny;
				int component_j = ijk.j % component_ny;

				idx_out = ijk.i + component_j * nxRegion_R + ijk.k * nxRegion_R * component_ny;
				idx_in = ijk.i + ijk.j * nxRegion_R + ijk.k * nxRegion_R * n.y;
			}
			//yRegion is not the last region
			else {

				//component is below yRegion, hence itself not the last region
				if (ijk.j < yRegion.i) {

					int component_ny = nyRegion;
					component_idx = ijk.j / component_ny;
					int component_j = ijk.j % component_ny;

					idx_out = ijk.i + component_j * nxRegion_R + ijk.k * nxRegion_R * component_ny;
					idx_in = ijk.i + ijk.j * nxRegion_R + ijk.k * nxRegion_R * n.y;
				}
				//component is above yRegion, and could be the last region
				else {

					//y index is : ijk.j + yRegion.j - yRegion.i (i.e. skip over yRegion)
					component_idx = (ijk.j + nyRegion) / nyRegion;
					if (component_idx == num_devices) component_idx--;
					int component_j = (ijk.j + nyRegion) - nyRegion * component_idx;
					int component_ny;

					//if this is the last component, it is also the last region, hence find its ny value as :
					if (component_idx == num_devices - 1) component_ny = n.y - nyRegion * (num_devices - 1);
					//otherwise we have its ny value already as:
					else component_ny = nyRegion;

					idx_out = ijk.i + component_j * nxRegion_R + ijk.k * nxRegion_R * component_ny;
					idx_in = ijk.i + (ijk.j + nyRegion) * nxRegion_R + ijk.k * nxRegion_R * n.y;
				}
			}
		}
		//if number of components is 2 use a simpler routine
		else if (num_devices == 2) {

			//ijk in complement space (nxRegion_R, nCy, nz)
			cuINT3 ijk = cuINT3(idx % nxRegion_R, (idx / nxRegion_R) % nCy, idx / (nxRegion_R * nCy));

			if (yRegion.j == n.y) {

				component_idx = 0;
				idx_in = ijk.i + ijk.j * nxRegion_R + ijk.k * nxRegion_R * n.y;
			}
			else {

				component_idx = 1;
				idx_in = ijk.i + (ijk.j + yRegion.j) * nxRegion_R + ijk.k * nxRegion_R * n.y;
			}

			idx_out = idx;
		}

		return true;
	}
	else return false;
}

template <typename VECType>
__global__ void Copy_J_Input_xRegion_Oersted_kernel(
	VECType& J, cuReal3** J_Input_xRegion,
	cuSZ3& n, cuSZ3& N, cuINT2& xRegion_R, cuINT2& yRegion, int& num_devices)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;

	//the component index to use
	int component_idx;
	//form output index in component spaces, which are of dimension nxRegion * (respective y region size) * nz
	int idx_out;
	//form linear index in input data, which is of dimensions (nxRegion * ny * nz)
	int idx_in;

	if (from_notyRegion_in_xRegion_R_Oersted(idx, component_idx, idx_out, idx_in, n, N, xRegion_R, yRegion, num_devices))
		J_Input_xRegion[component_idx][idx_out] = J[idx_in];
}

template <typename VECType>
__global__ void Copy_J_Input_xRegion_Oersted_halfprecision_kernel(
	VECType& J, cuBHalf** J_Input_xRegion_half,
	cuSZ3& n, cuSZ3& N, cuINT2& xRegion_R, cuINT2& yRegion, int& num_devices, cuBReal& normalization)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;

	//the component index to use
	int component_idx;
	//form output index in component spaces, which are of dimension nxRegion * (respective y region size) * nz
	int idx_out;
	//form linear index in input data, which is of dimensions (nxRegion * ny * nz)
	int idx_in;

	if (from_notyRegion_in_xRegion_R_Oersted(idx, component_idx, idx_out, idx_in, n, N, xRegion_R, yRegion, num_devices)) {

		cuReal3 value_in = J[idx_in] / normalization;

#if SINGLEPRECISION
		J_Input_xRegion_half[component_idx][3 * idx_out + 0] = float2half_as_uint16(value_in.x);
		J_Input_xRegion_half[component_idx][3 * idx_out + 1] = float2half_as_uint16(value_in.y);
		J_Input_xRegion_half[component_idx][3 * idx_out + 2] = float2half_as_uint16(value_in.z);
#else
		J_Input_xRegion_half[component_idx][3 * idx_out + 0] = value_in.x;
		J_Input_xRegion_half[component_idx][3 * idx_out + 1] = value_in.y;
		J_Input_xRegion_half[component_idx][3 * idx_out + 2] = value_in.z;
#endif
	}
}

//Copy J data on this device to linear regions so we can transfer
void OerstedMCUDA_single::Copy_J_Input_xRegion(bool half_precision)
{
	if (!half_precision) {

		Copy_J_Input_xRegion_Oersted_kernel <<< (nxRegion_R * (n.y - nyRegion) * n.z + CUDATHREADS) / CUDATHREADS, CUDATHREADS >>>
			(pOerstedCUDA->sm_Vals.get_deviceobject(device_index), Real_xRegion_arr, cun, cuN, cuxRegion_R, cuyRegion, cunum_devices);
	}
	else {

		Copy_J_Input_xRegion_Oersted_halfprecision_kernel <<< (nxRegion_R * (n.y - nyRegion) * n.z + CUDATHREADS) / CUDATHREADS, CUDATHREADS >>>
			(pOerstedCUDA->sm_Vals.get_deviceobject(device_index), Real_xRegion_half_arr, cun, cuN, cuxRegion_R, cuyRegion, cunum_devices, normalization_J);
	}
}

#endif
#endif