#include "hip/hip_runtime.h"
#include "Atom_DiffEqCubicCUDA.h"

#if COMPILECUDA == 1
#ifdef ODE_EVAL_COMPILATION_AHEUN
#ifdef MESH_COMPILATION_ATOM_CUBIC

#include "Atom_MeshParamsControlCUDA.h"

#include "Reduction.cuh"

//defines evaluation methods kernel launchers

//----------------------------------------- EVALUATIONS : Trapezoidal Euler

__global__ void RunAHeun_Step0_withReductions_Kernel(ManagedAtom_DiffEqCubicCUDA& cuaDiffEq, ManagedAtom_MeshCUDA& cuaMesh)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;

	cuBReal dT = *cuaDiffEq.pdT;

	cuReal3 mxh = cuReal3();
	bool include_in_average = false;

	//multiplicative conversion factor from atomic moment (units of muB) to A/m
	cuBReal conversion = (cuBReal)MUB / cuaMesh.pM1->h.dim();

	if (idx < cuaMesh.pM1->linear_size()) {

		if (cuaMesh.pM1->is_not_empty(idx)) {

			//obtain average normalized torque term
			cuBReal Mnorm = (*cuaMesh.pM1)[idx].norm();
			mxh = ((*cuaMesh.pM1)[idx] ^ (*cuaMesh.pHeff1)[idx]) / (conversion * Mnorm * Mnorm);
			include_in_average = true;

			//Save current moment for the next step
			(*cuaDiffEq.psM1)[idx] = (*cuaMesh.pM1)[idx];

			if (!cuaMesh.pM1->is_skipcell(idx)) {

				//First evaluate RHS of set equation at the current time step
				cuReal3 rhs = (cuaDiffEq.*(cuaDiffEq.pODEFunc))(idx);

				//Now estimate moment for the next time step
				(*cuaMesh.pM1)[idx] += rhs * dT;
			}
		}
	}

	if (cuaMesh.pgrel->get0()) reduction_avg(0, 1, &mxh, *cuaDiffEq.pmxh_av, *cuaDiffEq.pavpoints, include_in_average);
}

__global__ void RunAHeun_Step0_Kernel(ManagedAtom_DiffEqCubicCUDA& cuaDiffEq, ManagedAtom_MeshCUDA& cuaMesh)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;

	cuBReal dT = *cuaDiffEq.pdT;

	if (idx < cuaMesh.pM1->linear_size()) {

		if (cuaMesh.pM1->is_not_empty(idx)) {

			//Save current moment for the next step
			(*cuaDiffEq.psM1)[idx] = (*cuaMesh.pM1)[idx];

			if (!cuaMesh.pM1->is_skipcell(idx)) {

				//First evaluate RHS of set equation at the current time step
				cuReal3 rhs = (cuaDiffEq.*(cuaDiffEq.pODEFunc))(idx);

				//Now estimate moment for the next time step
				(*cuaMesh.pM1)[idx] += rhs * dT;
			}
		}
	}
}

__global__ void RunAHeun_Step1_withReductions_Kernel(ManagedAtom_DiffEqCubicCUDA& cuaDiffEq, ManagedAtom_MeshCUDA& cuaMesh)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;

	cuBReal dT = *cuaDiffEq.pdT;

	cuReal3 dmdt = cuReal3();
	cuBReal lte = 0.0;
	bool include_in_average = false;

	//multiplicative conversion factor from atomic moment (units of muB) to A/m
	cuBReal conversion = (cuBReal)MUB / cuaMesh.pM1->h.dim();

	if (idx < cuaMesh.pM1->linear_size()) {

		if (cuaMesh.pM1->is_not_empty(idx)) {

			if (!cuaMesh.pM1->is_skipcell(idx)) {

				//First evaluate RHS of set equation at the current time step
				cuReal3 rhs = (cuaDiffEq.*(cuaDiffEq.pODEFunc))(idx);

				//First save predicted moment for lte calculation
				cuReal3 saveM = (*cuaMesh.pM1)[idx];

				//Now estimate moment using the second trapezoidal Euler step equation
				(*cuaMesh.pM1)[idx] = ((*cuaDiffEq.psM1)[idx] + (*cuaMesh.pM1)[idx] + rhs * dT) / 2;

				if (*cuaDiffEq.prenormalize) {

					cuBReal mu_s = *cuaMesh.pmu_s;
					cuaMesh.update_parameters_mcoarse(idx, *cuaMesh.pmu_s, mu_s);
					(*cuaMesh.pM1)[idx].renormalize(mu_s);
				}

				//obtain maximum normalized dmdt term
				cuBReal Mnorm = (*cuaMesh.pM1)[idx].norm();
				dmdt = ((*cuaMesh.pM1)[idx] - (*cuaDiffEq.psM1)[idx]) / (dT * (cuBReal)GAMMA * conversion * Mnorm * Mnorm);
				include_in_average = true;

				//local truncation error (between predicted and corrected)
				lte = cu_GetMagnitude((*cuaMesh.pM1)[idx] - saveM) / (*cuaMesh.pM1)[idx].norm();
			}
		}
	}

	if (cuaMesh.pgrel->get0()) reduction_avg(0, 1, &dmdt, *cuaDiffEq.pdmdt_av, *cuaDiffEq.pavpoints2, include_in_average);
	reduction_max(0, 1, &lte, *cuaDiffEq.plte);
}

__global__ void RunAHeun_Step1_Kernel(ManagedAtom_DiffEqCubicCUDA& cuaDiffEq, ManagedAtom_MeshCUDA& cuaMesh)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;

	cuBReal dT = *cuaDiffEq.pdT;

	cuBReal lte = 0.0;

	if (idx < cuaMesh.pM1->linear_size()) {

		if (cuaMesh.pM1->is_not_empty(idx)) {

			if (!cuaMesh.pM1->is_skipcell(idx)) {

				//First evaluate RHS of set equation at the current time step
				cuReal3 rhs = (cuaDiffEq.*(cuaDiffEq.pODEFunc))(idx);

				//First save predicted moment for lte calculation
				cuReal3 saveM = (*cuaMesh.pM1)[idx];

				//Now estimate moment using the second trapezoidal Euler step equation
				(*cuaMesh.pM1)[idx] = ((*cuaDiffEq.psM1)[idx] + (*cuaMesh.pM1)[idx] + rhs * dT) / 2;

				if (*cuaDiffEq.prenormalize) {

					cuBReal mu_s = *cuaMesh.pmu_s;
					cuaMesh.update_parameters_mcoarse(idx, *cuaMesh.pmu_s, mu_s);
					(*cuaMesh.pM1)[idx].renormalize(mu_s);
				}

				//local truncation error (between predicted and corrected)
				lte = cu_GetMagnitude((*cuaMesh.pM1)[idx] - saveM) / (*cuaMesh.pM1)[idx].norm();
			}
		}
	}

	reduction_max(0, 1, &lte, *cuaDiffEq.plte);
}

//----------------------------------------- DifferentialEquationCUDA Launchers

//TRAPEZOIDAL EULER

void Atom_DifferentialEquationCubicCUDA::RunAHeun(int step, bool calculate_mxh, bool calculate_dmdt)
{
	if (step == 0) {

		if (calculate_mxh) {

			for (mGPU.device_begin(); mGPU != mGPU.device_end(); mGPU++) {

				RunAHeun_Step0_withReductions_Kernel <<< (paMeshCUDA->M1.device_size(mGPU) + CUDATHREADS) / CUDATHREADS, CUDATHREADS >>> 
					(cuaDiffEq.get_deviceobject(mGPU), paMeshCUDA->cuaMesh.get_deviceobject(mGPU));
			}
		}
		else {

			for (mGPU.device_begin(); mGPU != mGPU.device_end(); mGPU++) {

				RunAHeun_Step0_Kernel <<< (paMeshCUDA->M1.device_size(mGPU) + CUDATHREADS) / CUDATHREADS, CUDATHREADS >>> 
					(cuaDiffEq.get_deviceobject(mGPU), paMeshCUDA->cuaMesh.get_deviceobject(mGPU));
			}
		}
	}
	else {

		if (calculate_dmdt) {

			for (mGPU.device_begin(); mGPU != mGPU.device_end(); mGPU++) {

				RunAHeun_Step1_withReductions_Kernel <<< (paMeshCUDA->M1.device_size(mGPU) + CUDATHREADS) / CUDATHREADS, CUDATHREADS >>> 
					(cuaDiffEq.get_deviceobject(mGPU), paMeshCUDA->cuaMesh.get_deviceobject(mGPU));
			}
		}
		else {

			for (mGPU.device_begin(); mGPU != mGPU.device_end(); mGPU++) {

				RunAHeun_Step1_Kernel <<< (paMeshCUDA->M1.device_size(mGPU) + CUDATHREADS) / CUDATHREADS, CUDATHREADS >>> 
					(cuaDiffEq.get_deviceobject(mGPU), paMeshCUDA->cuaMesh.get_deviceobject(mGPU));
			}
		}
	}
}

#endif
#endif
#endif