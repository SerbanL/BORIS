#include "hip/hip_runtime.h"
#include "Atom_DiffEqCubicCUDA.h"

#if COMPILECUDA == 1
#ifdef ODE_EVAL_COMPILATION_RKF45
#ifdef MESH_COMPILATION_ATOM_CUBIC

#include "Atom_MeshParamsControlCUDA.h"

#include "Reduction.cuh"

//defines evaluation methods kernel launchers

//----------------------------------------- EVALUATIONS : RKF45

__global__ void RunRKF45_Step0_withReductions_Kernel(ManagedAtom_DiffEqCubicCUDA& cuaDiffEq, ManagedAtom_MeshCUDA& cuaMesh)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;

	cuBReal dT = *cuaDiffEq.pdT;

	cuBReal mxh = 0.0;

	//multiplicative conversion factor from atomic moment (units of muB) to A/m
	cuBReal conversion = (cuBReal)MUB / cuaMesh.pM1->h.dim();

	if (idx < cuaMesh.pM1->linear_size()) {

		if (cuaMesh.pM1->is_not_empty(idx)) {

			//Save current moment for later use
			(*cuaDiffEq.psM1)[idx] = (*cuaMesh.pM1)[idx];

			if (!cuaMesh.pM1->is_skipcell(idx)) {

				//obtain maximum normalized torque term
				cuBReal Mnorm = (*cuaMesh.pM1)[idx].norm();
				mxh = cu_GetMagnitude((*cuaMesh.pM1)[idx] ^ (*cuaMesh.pHeff1)[idx]) / (conversion * Mnorm * Mnorm);

				//First evaluate RHS of set equation at the current time step
				(*cuaDiffEq.psEval0)[idx] = (cuaDiffEq.*(cuaDiffEq.pODEFunc))(idx);

				//Now estimate moment using RKF first step
				(*cuaMesh.pM1)[idx] += (*cuaDiffEq.psEval0)[idx] * (2 * dT / 9);
			}
		}
	}

	if (cuaMesh.pgrel->get0()) reduction_max(0, 1, &mxh, *cuaDiffEq.pmxh);
}

__global__ void RunRKF45_Step0_Kernel(ManagedAtom_DiffEqCubicCUDA& cuaDiffEq, ManagedAtom_MeshCUDA& cuaMesh)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;

	cuBReal dT = *cuaDiffEq.pdT;

	if (idx < cuaMesh.pM1->linear_size()) {

		if (cuaMesh.pM1->is_not_empty(idx)) {

			//Save current moment for later use
			(*cuaDiffEq.psM1)[idx] = (*cuaMesh.pM1)[idx];

			if (!cuaMesh.pM1->is_skipcell(idx)) {

				//First evaluate RHS of set equation at the current time step
				(*cuaDiffEq.psEval0)[idx] = (cuaDiffEq.*(cuaDiffEq.pODEFunc))(idx);

				//Now estimate moment using RKF first step
				(*cuaMesh.pM1)[idx] += (*cuaDiffEq.psEval0)[idx] * (2 * dT / 9);
			}
		}
	}
}

__global__ void RunRKF45_Step1_Kernel(ManagedAtom_DiffEqCubicCUDA& cuaDiffEq, ManagedAtom_MeshCUDA& cuaMesh)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;

	cuBReal dT = *cuaDiffEq.pdT;

	if (idx < cuaMesh.pM1->linear_size()) {

		if (cuaMesh.pM1->is_not_empty(idx) && !cuaMesh.pM1->is_skipcell(idx)) {

			//First evaluate RHS of set equation at the current time step
			(*cuaDiffEq.psEval1)[idx] = (cuaDiffEq.*(cuaDiffEq.pODEFunc))(idx);

			//Now estimate moment using RKF midle step 1
			(*cuaMesh.pM1)[idx] = (*cuaDiffEq.psM1)[idx] + ((*cuaDiffEq.psEval0)[idx] / 12 + (*cuaDiffEq.psEval1)[idx] / 4) * dT;
		}
	}
}

__global__ void RunRKF45_Step2_Kernel(ManagedAtom_DiffEqCubicCUDA& cuaDiffEq, ManagedAtom_MeshCUDA& cuaMesh)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;

	cuBReal dT = *cuaDiffEq.pdT;

	if (idx < cuaMesh.pM1->linear_size()) {

		if (cuaMesh.pM1->is_not_empty(idx) && !cuaMesh.pM1->is_skipcell(idx)) {

			//First evaluate RHS of set equation at the current time step
			(*cuaDiffEq.psEval2)[idx] = (cuaDiffEq.*(cuaDiffEq.pODEFunc))(idx);

			//Now estimate moment using RKF midle step 2
			(*cuaMesh.pM1)[idx] = (*cuaDiffEq.psM1)[idx] + (69 * (*cuaDiffEq.psEval0)[idx] / 128 - 243 * (*cuaDiffEq.psEval1)[idx] / 128 + 135 * (*cuaDiffEq.psEval2)[idx] / 64) * dT;
		}
	}
}

__global__ void RunRKF45_Step3_Kernel(ManagedAtom_DiffEqCubicCUDA& cuaDiffEq, ManagedAtom_MeshCUDA& cuaMesh)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;

	cuBReal dT = *cuaDiffEq.pdT;

	if (idx < cuaMesh.pM1->linear_size()) {

		if (cuaMesh.pM1->is_not_empty(idx) && !cuaMesh.pM1->is_skipcell(idx)) {

			//First evaluate RHS of set equation at the current time step
			(*cuaDiffEq.psEval3)[idx] = (cuaDiffEq.*(cuaDiffEq.pODEFunc))(idx);

			//Now estimate moment using RKF midle step 3
			(*cuaMesh.pM1)[idx] = (*cuaDiffEq.psM1)[idx] + (-17 * (*cuaDiffEq.psEval0)[idx] / 12 + 27 * (*cuaDiffEq.psEval1)[idx] / 4 - 27 * (*cuaDiffEq.psEval2)[idx] / 5 + 16 * (*cuaDiffEq.psEval3)[idx] / 15) * dT;
		}
	}
}

__global__ void RunRKF45_Step4_Kernel(ManagedAtom_DiffEqCubicCUDA& cuaDiffEq, ManagedAtom_MeshCUDA& cuaMesh)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;

	cuBReal dT = *cuaDiffEq.pdT;

	if (idx < cuaMesh.pM1->linear_size()) {

		if (cuaMesh.pM1->is_not_empty(idx) && !cuaMesh.pM1->is_skipcell(idx)) {

			//First evaluate RHS of set equation at the current time step
			(*cuaDiffEq.psEval4)[idx] = (cuaDiffEq.*(cuaDiffEq.pODEFunc))(idx);

			//Now estimate moment using RKF midle step 4
			(*cuaMesh.pM1)[idx] = (*cuaDiffEq.psM1)[idx] + (65 * (*cuaDiffEq.psEval0)[idx] / 432 - 5 * (*cuaDiffEq.psEval1)[idx] / 16 + 13 * (*cuaDiffEq.psEval2)[idx] / 16 + 4 * (*cuaDiffEq.psEval3)[idx] / 27 + 5 * (*cuaDiffEq.psEval4)[idx] / 144) * dT;
		}
	}
}

__global__ void RunRKF45_Step5_withReductions_Kernel(ManagedAtom_DiffEqCubicCUDA& cuaDiffEq, ManagedAtom_MeshCUDA& cuaMesh)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;

	cuBReal dT = *cuaDiffEq.pdT;

	cuBReal dmdt = 0.0;
	cuBReal lte = 0.0;

	//multiplicative conversion factor from atomic moment (units of muB) to A/m
	cuBReal conversion = (cuBReal)MUB / cuaMesh.pM1->h.dim();

	if (idx < cuaMesh.pM1->linear_size()) {

		if (cuaMesh.pM1->is_not_empty(idx)) {

			if (!cuaMesh.pM1->is_skipcell(idx)) {

				//First evaluate RHS of set equation at the current time step
				cuReal3 rhs = (cuaDiffEq.*(cuaDiffEq.pODEFunc))(idx);

				//4th order evaluation
				(*cuaMesh.pM1)[idx] = (*cuaDiffEq.psM1)[idx] + ((*cuaDiffEq.psEval0)[idx] / 9 + 9 * (*cuaDiffEq.psEval2)[idx] / 20 + 16 * (*cuaDiffEq.psEval3)[idx] / 45 + (*cuaDiffEq.psEval4)[idx] / 12) * dT;

				//5th order evaluation
				cuReal3 prediction = (*cuaDiffEq.psM1)[idx] + (47 * (*cuaDiffEq.psEval0)[idx] / 450 + 12 * (*cuaDiffEq.psEval2)[idx] / 25 + 32 * (*cuaDiffEq.psEval3)[idx] / 225 + 1 * (*cuaDiffEq.psEval4)[idx] / 30 + 6 * rhs / 25) * dT;

				if (*cuaDiffEq.prenormalize) {

					cuBReal mu_s = *cuaMesh.pmu_s;
					cuaMesh.update_parameters_mcoarse(idx, *cuaMesh.pmu_s, mu_s);
					(*cuaMesh.pM1)[idx].renormalize(mu_s);
				}

				//obtain maximum normalized dmdt term
				cuBReal Mnorm = (*cuaMesh.pM1)[idx].norm();
				dmdt = cu_GetMagnitude((*cuaMesh.pM1)[idx] - (*cuaDiffEq.psM1)[idx]) / (dT * (cuBReal)GAMMA * conversion * Mnorm * Mnorm);

				//local truncation error (between predicted and corrected)
				lte = cu_GetMagnitude((*cuaMesh.pM1)[idx] - prediction) / (*cuaMesh.pM1)[idx].norm();
			}
		}
	}

	if (cuaMesh.pgrel->get0()) reduction_max(0, 1, &dmdt, *cuaDiffEq.pdmdt);
	reduction_max(0, 1, &lte, *cuaDiffEq.plte);
}

__global__ void RunRKF45_Step5_Kernel(ManagedAtom_DiffEqCubicCUDA& cuaDiffEq, ManagedAtom_MeshCUDA& cuaMesh)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;

	cuBReal dT = *cuaDiffEq.pdT;

	cuBReal lte = 0.0;

	if (idx < cuaMesh.pM1->linear_size()) {

		if (cuaMesh.pM1->is_not_empty(idx)) {

			if (!cuaMesh.pM1->is_skipcell(idx)) {

				//First evaluate RHS of set equation at the current time step
				cuReal3 rhs = (cuaDiffEq.*(cuaDiffEq.pODEFunc))(idx);

				//4th order evaluation
				(*cuaMesh.pM1)[idx] = (*cuaDiffEq.psM1)[idx] + ((*cuaDiffEq.psEval0)[idx] / 9 + 9 * (*cuaDiffEq.psEval2)[idx] / 20 + 16 * (*cuaDiffEq.psEval3)[idx] / 45 + (*cuaDiffEq.psEval4)[idx] / 12) * dT;

				//5th order evaluation
				cuReal3 prediction = (*cuaDiffEq.psM1)[idx] + (47 * (*cuaDiffEq.psEval0)[idx] / 450 + 12 * (*cuaDiffEq.psEval2)[idx] / 25 + 32 * (*cuaDiffEq.psEval3)[idx] / 225 + 1 * (*cuaDiffEq.psEval4)[idx] / 30 + 6 * rhs / 25) * dT;
				
				if (*cuaDiffEq.prenormalize) {

					cuBReal mu_s = *cuaMesh.pmu_s;
					cuaMesh.update_parameters_mcoarse(idx, *cuaMesh.pmu_s, mu_s);
					(*cuaMesh.pM1)[idx].renormalize(mu_s);
				}

				//local truncation error (between predicted and corrected)
				lte = cu_GetMagnitude((*cuaMesh.pM1)[idx] - prediction) / (*cuaMesh.pM1)[idx].norm();
			}
		}
	}

	reduction_max(0, 1, &lte, *cuaDiffEq.plte);
}

//----------------------------------------- DifferentialEquationCUDA Launchers

//RUNGE KUTTA FEHLBERG

void Atom_DifferentialEquationCubicCUDA::RunRKF45(int step, bool calculate_mxh, bool calculate_dmdt)
{
	switch (step) {

	case 0:

		if (calculate_mxh) {

			for (mGPU.device_begin(); mGPU != mGPU.device_end(); mGPU++) {

				RunRKF45_Step0_withReductions_Kernel <<< (paMeshCUDA->M1.device_size(mGPU) + CUDATHREADS) / CUDATHREADS, CUDATHREADS >>> 
					(cuaDiffEq.get_deviceobject(mGPU), paMeshCUDA->cuaMesh.get_deviceobject(mGPU));
			}
		}
		else {

			for (mGPU.device_begin(); mGPU != mGPU.device_end(); mGPU++) {

				RunRKF45_Step0_Kernel <<< (paMeshCUDA->M1.device_size(mGPU) + CUDATHREADS) / CUDATHREADS, CUDATHREADS >>> 
					(cuaDiffEq.get_deviceobject(mGPU), paMeshCUDA->cuaMesh.get_deviceobject(mGPU));
			}
		}

		break;

	case 1:

		for (mGPU.device_begin(); mGPU != mGPU.device_end(); mGPU++) {

			RunRKF45_Step1_Kernel <<< (paMeshCUDA->M1.device_size(mGPU) + CUDATHREADS) / CUDATHREADS, CUDATHREADS >>> 
				(cuaDiffEq.get_deviceobject(mGPU), paMeshCUDA->cuaMesh.get_deviceobject(mGPU));
		}

		break;

	case 2:

		for (mGPU.device_begin(); mGPU != mGPU.device_end(); mGPU++) {

			RunRKF45_Step2_Kernel <<< (paMeshCUDA->M1.device_size(mGPU) + CUDATHREADS) / CUDATHREADS, CUDATHREADS >>> 
				(cuaDiffEq.get_deviceobject(mGPU), paMeshCUDA->cuaMesh.get_deviceobject(mGPU));
		}

		break;

	case 3:

		for (mGPU.device_begin(); mGPU != mGPU.device_end(); mGPU++) {

			RunRKF45_Step3_Kernel <<< (paMeshCUDA->M1.device_size(mGPU) + CUDATHREADS) / CUDATHREADS, CUDATHREADS >>> 
				(cuaDiffEq.get_deviceobject(mGPU), paMeshCUDA->cuaMesh.get_deviceobject(mGPU));
		}

		break;

	case 4:

		for (mGPU.device_begin(); mGPU != mGPU.device_end(); mGPU++) {

			RunRKF45_Step4_Kernel <<< (paMeshCUDA->M1.device_size(mGPU) + CUDATHREADS) / CUDATHREADS, CUDATHREADS >>>
				(cuaDiffEq.get_deviceobject(mGPU), paMeshCUDA->cuaMesh.get_deviceobject(mGPU));
		}

		break;

	case 5:

		if (calculate_dmdt) {

			for (mGPU.device_begin(); mGPU != mGPU.device_end(); mGPU++) {

				RunRKF45_Step5_withReductions_Kernel <<< (paMeshCUDA->M1.device_size(mGPU) + CUDATHREADS) / CUDATHREADS, CUDATHREADS >>> 
					(cuaDiffEq.get_deviceobject(mGPU), paMeshCUDA->cuaMesh.get_deviceobject(mGPU));
			}
		}
		else {

			for (mGPU.device_begin(); mGPU != mGPU.device_end(); mGPU++) {

				RunRKF45_Step5_Kernel <<< (paMeshCUDA->M1.device_size(mGPU) + CUDATHREADS) / CUDATHREADS, CUDATHREADS >>> 
					(cuaDiffEq.get_deviceobject(mGPU), paMeshCUDA->cuaMesh.get_deviceobject(mGPU));
			}
		}

		break;
	}
}

#endif
#endif
#endif