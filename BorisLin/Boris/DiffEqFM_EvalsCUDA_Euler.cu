#include "hip/hip_runtime.h"
#include "DiffEqFMCUDA.h"

#if COMPILECUDA == 1
#ifdef ODE_EVAL_COMPILATION_EULER
#ifdef MESH_COMPILATION_FERROMAGNETIC

#include "MeshParamsControlCUDA.h"

#include "Reduction.cuh"

//defines evaluation methods kernel launchers

//----------------------------------------- EVALUATIONS: Euler

__global__ void RunEuler_Kernel_withReductions(ManagedDiffEqFMCUDA& cuDiffEq, ManagedMeshCUDA& cuMesh)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;

	cuBReal dT = *cuDiffEq.pdT;

	cuReal3 mxh = cuReal3();
	cuReal3 dmdt = cuReal3();
	bool include_in_average = false;

	if (idx < cuMesh.pM->linear_size()) {

		if (cuMesh.pM->is_not_empty(idx)) {

			//Save current magnetization
			(*cuDiffEq.psM1)[idx] = (*cuMesh.pM)[idx];

			if (!cuMesh.pM->is_skipcell(idx)) {

				//obtain average normalized torque term
				cuBReal Mnorm = (*cuMesh.pM)[idx].norm();
				mxh = ((*cuMesh.pM)[idx] ^ (*cuMesh.pHeff)[idx]) / (Mnorm * Mnorm);
				include_in_average = true;

				//First evaluate RHS of set equation at the current time step
				cuReal3 rhs = (cuDiffEq.*(cuDiffEq.pODEFunc))(idx);

				//Now estimate magnetization for the next time step
				(*cuMesh.pM)[idx] += rhs * dT;

				if (*cuDiffEq.prenormalize) {

					cuBReal Ms = *cuMesh.pMs;
					cuMesh.update_parameters_mcoarse(idx, *cuMesh.pMs, Ms);
					(*cuMesh.pM)[idx].renormalize(Ms);
				}

				//obtain maximum normalized dmdt term
				dmdt = ((*cuMesh.pM)[idx] - (*cuDiffEq.psM1)[idx]) / (dT * (cuBReal)GAMMA * Mnorm * Mnorm);
			}
			else {

				cuBReal Ms = *cuMesh.pMs;
				cuMesh.update_parameters_mcoarse(idx, *cuMesh.pMs, Ms);
				(*cuMesh.pM)[idx].renormalize(Ms);		//re-normalize the skipped cells no matter what - temperature can change
			}
		}
	}

	//only reduce for dmdt (and mxh) if grel is not zero (if it's zero this means magnetization dynamics is disabled in this mesh)
	if (cuMesh.pgrel->get0()) {

		reduction_avg(0, 1, &mxh, *cuDiffEq.pmxh_av, *cuDiffEq.pavpoints, include_in_average);
		reduction_avg(0, 1, &dmdt, *cuDiffEq.pdmdt_av, *cuDiffEq.pavpoints2, include_in_average);
	}
}

__global__ void RunEuler_Kernel(ManagedDiffEqFMCUDA& cuDiffEq, ManagedMeshCUDA& cuMesh)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;

	cuBReal dT = *cuDiffEq.pdT;

	if (idx < cuMesh.pM->linear_size()) {

		if (cuMesh.pM->is_not_empty(idx)) {

			//Save current magnetization
			(*cuDiffEq.psM1)[idx] = (*cuMesh.pM)[idx];

			if (!cuMesh.pM->is_skipcell(idx)) {

				//First evaluate RHS of set equation at the current time step
				cuReal3 rhs = (cuDiffEq.*(cuDiffEq.pODEFunc))(idx);

				//Now estimate magnetization for the next time step
				(*cuMesh.pM)[idx] += rhs * dT;

				if (*cuDiffEq.prenormalize) {

					cuBReal Ms = *cuMesh.pMs;
					cuMesh.update_parameters_mcoarse(idx, *cuMesh.pMs, Ms);
					(*cuMesh.pM)[idx].renormalize(Ms);
				}
			}
			else {

				cuBReal Ms = *cuMesh.pMs;
				cuMesh.update_parameters_mcoarse(idx, *cuMesh.pMs, Ms);
				(*cuMesh.pM)[idx].renormalize(Ms);		//re-normalize the skipped cells no matter what - temperature can change
			}
		}
	}
}

//----------------------------------------- DifferentialEquationCUDA Launchers

//EULER

void DifferentialEquationFMCUDA::RunEuler(bool calculate_mxh, bool calculate_dmdt)
{
	for (mGPU.device_begin(); mGPU != mGPU.device_end(); mGPU++) {

		if (calculate_mxh || calculate_dmdt) {

			RunEuler_Kernel_withReductions <<< (pMeshCUDA->M.device_size(mGPU) + CUDATHREADS) / CUDATHREADS, CUDATHREADS >>>
				(cuDiffEq.get_deviceobject(mGPU), pMeshCUDA->cuMesh.get_deviceobject(mGPU));
		}
		else {

			RunEuler_Kernel <<< (pMeshCUDA->M.device_size(mGPU) + CUDATHREADS) / CUDATHREADS, CUDATHREADS >>>
				(cuDiffEq.get_deviceobject(mGPU), pMeshCUDA->cuMesh.get_deviceobject(mGPU));
		}
	}
}

#endif
#endif
#endif