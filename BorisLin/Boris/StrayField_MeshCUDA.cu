#include "hip/hip_runtime.h"
#include "StrayField_MeshCUDA.h"

#if COMPILECUDA == 1

#ifdef MODULE_COMPILATION_STRAYFIELD

#include "Reduction.cuh"

#include "MeshCUDA.h"
#include "MeshDefs.h"

//----------------------- Initialization

__global__ void set_StrayField_MeshCUDA_pointers_kernel(
	ManagedMeshCUDA& cuMesh, cuVEC<cuReal3>& strayField)
{
	if (threadIdx.x == 0) cuMesh.pstrayField = &strayField;
}

void StrayField_MeshCUDA::set_StrayField_MeshCUDA_pointers(void)
{
	for (mGPU.device_begin(); mGPU != mGPU.device_end(); mGPU++) {

		set_StrayField_MeshCUDA_pointers_kernel <<< 1, CUDATHREADS >>>
			(pMeshCUDA->cuMesh.get_deviceobject(mGPU), strayField.get_deviceobject(mGPU));
	}
}

//----------------------- Computation

__global__ void UpdateStrayField_FM_kernel(ManagedMeshCUDA& cuMesh, ManagedModulesCUDA& cuModule, cuVEC<cuReal3>& strayField, bool do_reduction)
{
	cuVEC_VC<cuReal3>& M = *cuMesh.pM;
	cuVEC<cuReal3>& Heff = *cuMesh.pHeff;

	int idx = blockIdx.x * blockDim.x + threadIdx.x;

	cuBReal energy_ = 0.0;

	if (idx < Heff.linear_size()) {

		cuReal3 Hstray = strayField[idx];

		Heff[idx] += Hstray;

		if (do_reduction) {

			int non_empty_cells = M.get_nonempty_cells();
			if (non_empty_cells) energy_ = -(cuBReal)MU0 * M[idx] * Hstray / non_empty_cells;
		}

		if (do_reduction && cuModule.pModule_Heff->linear_size()) (*cuModule.pModule_Heff)[idx] = Hstray;
		if (do_reduction && cuModule.pModule_energy->linear_size()) (*cuModule.pModule_energy)[idx] = -(cuBReal)MU0 * M[idx] * Hstray;
	}

	if (do_reduction) reduction_sum(0, 1, &energy_, *cuModule.penergy);
}

__global__ void UpdateStrayField_AFM_kernel(ManagedMeshCUDA& cuMesh, ManagedModulesCUDA& cuModule, cuVEC<cuReal3>& strayField, bool do_reduction)
{
	cuVEC_VC<cuReal3>& M = *cuMesh.pM;
	cuVEC_VC<cuReal3>& M2 = *cuMesh.pM2;
	cuVEC<cuReal3>& Heff = *cuMesh.pHeff;
	cuVEC<cuReal3>& Heff2 = *cuMesh.pHeff2;

	int idx = blockIdx.x * blockDim.x + threadIdx.x;

	cuBReal energy_ = 0.0;

	if (idx < Heff.linear_size()) {

		cuReal3 Hstray = strayField[idx];

		Heff[idx] += Hstray;
		Heff2[idx] += Hstray;

		if (do_reduction) {

			int non_empty_cells = M.get_nonempty_cells();
			if (non_empty_cells) energy_ = -(cuBReal)MU0 * (M[idx] + M2[idx]) * Hstray / (2 * non_empty_cells);
		}

		if (do_reduction && cuModule.pModule_Heff->linear_size()) (*cuModule.pModule_Heff)[idx] = Hstray;
		if (do_reduction && cuModule.pModule_Heff2->linear_size()) (*cuModule.pModule_Heff2)[idx] = Hstray;
		if (do_reduction && cuModule.pModule_energy->linear_size()) (*cuModule.pModule_energy)[idx] = -MU0 * M[idx] * Hstray;
		if (do_reduction && cuModule.pModule_energy2->linear_size()) (*cuModule.pModule_energy2)[idx] = -MU0 * M2[idx] * Hstray;
	}

	if (do_reduction) reduction_sum(0, 1, &energy_, *cuModule.penergy);
}

void StrayField_MeshCUDA::UpdateFieldCUDA(void)
{
	if (pMeshCUDA->GetMeshType() == MESH_ANTIFERROMAGNETIC) {

		//anti-ferromagnetic mesh

		if (pMeshCUDA->CurrentTimeStepSolved()) {

			ZeroEnergy();

			for (mGPU.device_begin(); mGPU != mGPU.device_end(); mGPU++) {

				UpdateStrayField_AFM_kernel <<< (pMeshCUDA->M.device_size(mGPU) + CUDATHREADS) / CUDATHREADS, CUDATHREADS >> >
					(pMeshCUDA->cuMesh.get_deviceobject(mGPU), cuModule.get_deviceobject(mGPU), strayField.get_deviceobject(mGPU), true);
			}
		}
		else {

			for (mGPU.device_begin(); mGPU != mGPU.device_end(); mGPU++) {

				UpdateStrayField_AFM_kernel <<< (pMeshCUDA->M.device_size(mGPU) + CUDATHREADS) / CUDATHREADS, CUDATHREADS >>>
					(pMeshCUDA->cuMesh.get_deviceobject(mGPU), cuModule.get_deviceobject(mGPU), strayField.get_deviceobject(mGPU), false);
			}
		}
	}
	else {

		//ferromagnetic mesh

		if (pMeshCUDA->CurrentTimeStepSolved()) {

			ZeroEnergy();

			for (mGPU.device_begin(); mGPU != mGPU.device_end(); mGPU++) {

				UpdateStrayField_FM_kernel <<< (pMeshCUDA->M.device_size(mGPU) + CUDATHREADS) / CUDATHREADS, CUDATHREADS >>>
					(pMeshCUDA->cuMesh.get_deviceobject(mGPU), cuModule.get_deviceobject(mGPU), strayField.get_deviceobject(mGPU), true);
			}
		}
		else {

			for (mGPU.device_begin(); mGPU != mGPU.device_end(); mGPU++) {

				UpdateStrayField_FM_kernel <<< (pMeshCUDA->M.device_size(mGPU) + CUDATHREADS) / CUDATHREADS, CUDATHREADS >>>
					(pMeshCUDA->cuMesh.get_deviceobject(mGPU), cuModule.get_deviceobject(mGPU), strayField.get_deviceobject(mGPU), false);
			}
		}
	}
}

#endif

#endif

//----------------------------------- MONTE-CARLO METHODS FOR ENERGY COMPUTATION

#if COMPILECUDA == 1 && MONTE_CARLO == 1

//Ferromagnetic
__device__ cuBReal ManagedMeshCUDA::Get_EnergyChange_FM_StrayField_MeshCUDA(int spin_index, cuReal3 Mnew)
{
	cuVEC_VC<cuReal3>& M = *pM;

	cuReal3 Hstray = cuReal3();

	if (pstrayField && pstrayField->linear_size()) {

		Hstray = (*pstrayField)[spin_index];
	}

	if (Mnew != cuReal3()) return -M.h.dim() * (Mnew - M[spin_index]) * (cuBReal)MU0 * Hstray;
	else return -M.h.dim() * M[spin_index] * (cuBReal)MU0 * Hstray;
}

//Antiferromagnetic
__device__ cuReal2 ManagedMeshCUDA::Get_EnergyChange_AFM_StrayField_MeshCUDA(int spin_index, cuReal3 Mnew_A, cuReal3 Mnew_B)
{
	cuVEC_VC<cuReal3>& M = *pM;
	cuVEC_VC<cuReal3>& M2 = *pM2;

	cuReal3 Hstray = cuReal3();

	if (pstrayField && pstrayField->linear_size()) {

		Hstray = (*pstrayField)[spin_index];
	}

	if (Mnew_A != cuReal3() && Mnew_B != cuReal3()) return -M.h.dim() * cuReal2((Mnew_A - M[spin_index]) * (cuBReal)MU0 * Hstray, (Mnew_B - M2[spin_index]) * (cuBReal)MU0 * Hstray);
	else return -M.h.dim() * cuReal2(M[spin_index] * (cuBReal)MU0 * Hstray, M2[spin_index] * (cuBReal)MU0 * Hstray);
}

#endif