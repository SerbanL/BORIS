#include "hip/hip_runtime.h"
#include "DiffEq_CommonBaseCUDA.h"

#if COMPILECUDA == 1

//defines evaluation methods kernel launchers

//----------------------------------------- AUXILIARY

__global__ void Zerovalues_kernel(cuBReal& mxh, cuReal3& mxh_av, size_t& avpoints, cuBReal& dmdt, cuReal3& dmdt_av, size_t& avpoints2, cuBReal& lte)
{
	if (threadIdx.x == 0) mxh = 0.0;
	else if (threadIdx.x == 1) mxh_av = cuReal3(0.0);
	else if (threadIdx.x == 2) avpoints = 0;
	else if (threadIdx.x == 3) dmdt = 0.0;
	else if (threadIdx.x == 4) dmdt_av = cuReal3(0.0);
	else if (threadIdx.x == 5) avpoints2 = 0;
	else if (threadIdx.x == 6) lte = 0.0;
}

void ODECommon_BaseCUDA::Zero_reduction_values(void)
{
	for (mGPU.device_begin(); mGPU != mGPU.device_end(); mGPU++) {

		Zerovalues_kernel <<< 1, CUDATHREADS >>> 
			((*pmxh)(mGPU), (*pmxh_av)(mGPU), (*pavpoints)(mGPU), (*pdmdt)(mGPU), (*pdmdt_av)(mGPU), (*pavpoints2)(mGPU), (*plte)(mGPU));
	}
}

//-----------------------------------------

__global__ void Zeromxh_kernel(cuBReal& mxh, cuReal3& mxh_av, size_t& avpoints, cuBReal& lte)
{
	if (threadIdx.x == 0) mxh = 0.0;
	else if (threadIdx.x == 1) mxh_av = cuReal3(0.0);
	else if (threadIdx.x == 2) avpoints = 0;
	else if (threadIdx.x == 3) lte = 0.0;
}

void ODECommon_BaseCUDA::Zero_mxh_lte_values(void)
{
	for (mGPU.device_begin(); mGPU != mGPU.device_end(); mGPU++) {

		Zeromxh_kernel <<< 1, CUDATHREADS >>> 
			((*pmxh)(mGPU), (*pmxh_av)(mGPU), (*pavpoints)(mGPU), (*plte)(mGPU));
	}
}

//-----------------------------------------

__global__ void Zerodmdt_kernel(cuBReal& dmdt, cuReal3& dmdt_av, size_t& avpoints2, cuBReal& lte)
{
	if (threadIdx.x == 0) dmdt = 0.0;
	else if (threadIdx.x == 1) dmdt_av = cuReal3(0.0);
	else if (threadIdx.x == 2) avpoints2 = 0;
	else if (threadIdx.x == 3) lte = 0.0;
}

void ODECommon_BaseCUDA::Zero_dmdt_lte_values(void)
{
	for (mGPU.device_begin(); mGPU != mGPU.device_end(); mGPU++) {

		Zerodmdt_kernel <<< 1, CUDATHREADS >>> 
			((*pdmdt)(mGPU), (*pdmdt_av)(mGPU), (*pavpoints2)(mGPU), (*plte)(mGPU));
	}
}

//-----------------------------------------

__global__ void Zerolte_kernel(cuBReal& lte)
{
	if (threadIdx.x == 0) lte = 0.0;
}

void ODECommon_BaseCUDA::Zero_lte_value(void)
{
	for (mGPU.device_begin(); mGPU != mGPU.device_end(); mGPU++) {

		Zerolte_kernel <<< 1, CUDATHREADS >>> ((*plte)(mGPU));
	}
}

//-----------------------------------------

__global__ void mxhav_to_mxh_kernel(cuBReal& mxh, cuReal3& mxh_av, size_t& avpoints)
{
	if (threadIdx.x == 0) {

		if (avpoints) {

			mxh = cu_GetMagnitude(mxh_av) / avpoints;
		}
		else {

			mxh = 0.0;
		}
	}
}

void ODECommon_BaseCUDA::mxhav_to_mxh(void)
{
	for (mGPU.device_begin(); mGPU != mGPU.device_end(); mGPU++) {

		mxhav_to_mxh_kernel <<< 1, CUDATHREADS >>>
			((*pmxh)(mGPU), (*pmxh_av)(mGPU), (*pavpoints)(mGPU));
	}
}

//-----------------------------------------

__global__ void dmdtav_to_dmdt_kernel(cuBReal& dmdt, cuReal3& dmdt_av, size_t& avpoints2)
{
	if (threadIdx.x == 0) {

		if (avpoints2) {

			dmdt = cu_GetMagnitude(dmdt_av) / avpoints2;
		}
		else {

			dmdt = 0.0;
		}
	}
}

void ODECommon_BaseCUDA::dmdtav_to_dmdt(void)
{
	for (mGPU.device_begin(); mGPU != mGPU.device_end(); mGPU++) {

		dmdtav_to_dmdt_kernel <<< 1, CUDATHREADS >>> 
			((*pdmdt)(mGPU), (*pdmdt_av)(mGPU), (*pavpoints2)(mGPU));
	}
}

#endif