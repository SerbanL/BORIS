#include "hip/hip_runtime.h"
#include "MElasticCUDA.h"

#if COMPILECUDA == 1

#ifdef MODULE_COMPILATION_MELASTIC

#include "MeshCUDA.h"

__global__ void Set_Strain_From_Formula_Sd_Sod_Kernel(
	ManagedMeshCUDA& cuMesh,
	ManagedFunctionCUDA<cuBReal, cuBReal, cuBReal, cuBReal>& Sd_equation_x,
	ManagedFunctionCUDA<cuBReal, cuBReal, cuBReal, cuBReal>& Sd_equation_y,
	ManagedFunctionCUDA<cuBReal, cuBReal, cuBReal, cuBReal>& Sd_equation_z,
	ManagedFunctionCUDA<cuBReal, cuBReal, cuBReal, cuBReal>& Sod_equation_x,
	ManagedFunctionCUDA<cuBReal, cuBReal, cuBReal, cuBReal>& Sod_equation_y,
	ManagedFunctionCUDA<cuBReal, cuBReal, cuBReal, cuBReal>& Sod_equation_z,
	cuBReal time)
{
	cuVEC_VC<cuReal3>& strain_diag = *cuMesh.pstrain_diag;
	cuVEC_VC<cuReal3>& strain_odiag = *cuMesh.pstrain_odiag;

	int idx = blockIdx.x * blockDim.x + threadIdx.x;

	if (idx < strain_diag.linear_size()) {

		if (strain_diag.is_not_empty(idx)) {

			cuReal3 relpos = strain_diag.cellidx_to_position(idx);
			strain_diag[idx] = cuReal3(
				Sd_equation_x.evaluate(relpos.x, relpos.y, relpos.z, time),
				Sd_equation_y.evaluate(relpos.x, relpos.y, relpos.z, time),
				Sd_equation_z.evaluate(relpos.x, relpos.y, relpos.z, time));

			strain_odiag[idx] = cuReal3(
				Sod_equation_x.evaluate(relpos.x, relpos.y, relpos.z, time),
				Sod_equation_y.evaluate(relpos.x, relpos.y, relpos.z, time),
				Sod_equation_z.evaluate(relpos.x, relpos.y, relpos.z, time));
		}
	}
}

__global__ void Set_Strain_From_Formula_Sd_Kernel(
	ManagedMeshCUDA& cuMesh,
	ManagedFunctionCUDA<cuBReal, cuBReal, cuBReal, cuBReal>& Sd_equation_x,
	ManagedFunctionCUDA<cuBReal, cuBReal, cuBReal, cuBReal>& Sd_equation_y,
	ManagedFunctionCUDA<cuBReal, cuBReal, cuBReal, cuBReal>& Sd_equation_z,
	cuBReal time)
{
	cuVEC_VC<cuReal3>& strain_diag = *cuMesh.pstrain_diag;
	cuVEC_VC<cuReal3>& strain_odiag = *cuMesh.pstrain_odiag;

	int idx = blockIdx.x * blockDim.x + threadIdx.x;

	if (idx < strain_diag.linear_size()) {

		if (strain_diag.is_not_empty(idx)) {

			cuReal3 relpos = strain_diag.cellidx_to_position(idx);
			strain_diag[idx] = cuReal3(
				Sd_equation_x.evaluate(relpos.x, relpos.y, relpos.z, time),
				Sd_equation_y.evaluate(relpos.x, relpos.y, relpos.z, time),
				Sd_equation_z.evaluate(relpos.x, relpos.y, relpos.z, time));

			strain_odiag[idx] = cuReal3();
		}
	}
}

__global__ void Set_Strain_From_Formula_Sod_Kernel(
	ManagedMeshCUDA& cuMesh,
	ManagedFunctionCUDA<cuBReal, cuBReal, cuBReal, cuBReal>& Sod_equation_x,
	ManagedFunctionCUDA<cuBReal, cuBReal, cuBReal, cuBReal>& Sod_equation_y,
	ManagedFunctionCUDA<cuBReal, cuBReal, cuBReal, cuBReal>& Sod_equation_z,
	cuBReal time)
{
	cuVEC_VC<cuReal3>& strain_diag = *cuMesh.pstrain_diag;
	cuVEC_VC<cuReal3>& strain_odiag = *cuMesh.pstrain_odiag;

	int idx = blockIdx.x * blockDim.x + threadIdx.x;

	if (idx < strain_diag.linear_size()) {

		if (strain_diag.is_not_empty(idx)) {

			cuReal3 relpos = strain_diag.cellidx_to_position(idx);
			strain_diag[idx] = cuReal3();

			strain_odiag[idx] = cuReal3(
				Sod_equation_x.evaluate(relpos.x, relpos.y, relpos.z, time),
				Sod_equation_y.evaluate(relpos.x, relpos.y, relpos.z, time),
				Sod_equation_z.evaluate(relpos.x, relpos.y, relpos.z, time));
		}
	}
}

//----------------------------------------------- Auxiliary

//Run-time auxiliary to set strain directly from user supplied text formulas
void MElasticCUDA::Set_Strain_From_Formula(void)
{
	if (Sd_equation.is_set() && Sod_equation.is_set()) {

		for (mGPU.device_begin(); mGPU != mGPU.device_end(); mGPU++) {

			Set_Strain_From_Formula_Sd_Sod_Kernel <<< (pMeshCUDA->u_disp.device_size(mGPU) + CUDATHREADS) / CUDATHREADS, CUDATHREADS >>>
				(pMeshCUDA->cuMesh.get_deviceobject(mGPU),
				Sd_equation.get_x(mGPU), Sd_equation.get_y(mGPU), Sd_equation.get_z(mGPU),
				Sod_equation.get_x(mGPU), Sod_equation.get_y(mGPU), Sod_equation.get_z(mGPU),
				pMeshCUDA->GetStageTime());
		}
	}
	else if (Sd_equation.is_set()) {

		for (mGPU.device_begin(); mGPU != mGPU.device_end(); mGPU++) {

			Set_Strain_From_Formula_Sd_Kernel <<< (pMeshCUDA->u_disp.device_size(mGPU) + CUDATHREADS) / CUDATHREADS, CUDATHREADS >>>
				(pMeshCUDA->cuMesh.get_deviceobject(mGPU),
				Sd_equation.get_x(mGPU), Sd_equation.get_y(mGPU), Sd_equation.get_z(mGPU),
				pMeshCUDA->GetStageTime());
		}
	}
	else if (Sod_equation.is_set()) {

		for (mGPU.device_begin(); mGPU != mGPU.device_end(); mGPU++) {

			Set_Strain_From_Formula_Sod_Kernel <<< (pMeshCUDA->u_disp.device_size(mGPU) + CUDATHREADS) / CUDATHREADS, CUDATHREADS >>>
				(pMeshCUDA->cuMesh.get_deviceobject(mGPU),
				Sod_equation.get_x(mGPU), Sod_equation.get_y(mGPU), Sod_equation.get_z(mGPU),
				pMeshCUDA->GetStageTime());
		}
	}
}

//----------------------- UpdateField LAUNCHER

void MElasticCUDA::UpdateField(void)
{
	if (Sd_equation.is_set() || Sod_equation.is_set()) {

		//strain specified using a formula
		Set_Strain_From_Formula();
	}
}

//----------------------------------------------- Computational Helpers

__global__ void Calculate_Strain_Kernel(
	ManagedMeshCUDA& cuMesh,
	cuVEC_VC<cuReal3>& sdd,
	cuVEC_VC<cuBReal>& sxy, cuVEC_VC<cuBReal>& sxz, cuVEC_VC<cuBReal>& syz)
{
	cuVEC_VC<cuReal3>& u_disp = *cuMesh.pu_disp;
	cuVEC_VC<cuReal3>& strain_diag = *cuMesh.pstrain_diag;
	cuVEC_VC<cuReal3>& strain_odiag = *cuMesh.pstrain_odiag;

	cuReal3& h_m = u_disp.h;
	cuSZ3& n_m = u_disp.n;

	//kernel launch with size n_m.i * n_m.j * n_m.k 
	int idx = blockIdx.x * blockDim.x + threadIdx.x;

	if (idx < u_disp.linear_size()) {

		if (u_disp.is_not_empty(idx)) {

			//get all 9 first-order differentials of u
			cuReal33 grad_u = u_disp.grad_sided(idx);

			//diagonal components
			strain_diag[idx] = cuReal3(grad_u.x.x, grad_u.y.y, grad_u.z.z);

			//off-diagonal components (yz, xz, xy)
			strain_odiag[idx] = 0.5 * cuReal3(grad_u.y.z + grad_u.z.y, grad_u.x.z + grad_u.z.x, grad_u.x.y + grad_u.y.x);
		}
		else {

			strain_diag[idx] = cuReal3();
			strain_odiag[idx] = cuReal3();
		}
	}
}

void MElasticCUDA::Calculate_Strain(void)
{
	for (mGPU.device_begin(); mGPU != mGPU.device_end(); mGPU++) {

		Calculate_Strain_Kernel <<< (pMeshCUDA->u_disp.device_size(mGPU) + CUDATHREADS) / CUDATHREADS, CUDATHREADS >>>
			(pMeshCUDA->cuMesh.get_deviceobject(mGPU), sdd.get_deviceobject(mGPU), sxy.get_deviceobject(mGPU), sxz.get_deviceobject(mGPU), syz.get_deviceobject(mGPU));
	}
}

#endif

#endif

//----------------------------------- MONTE-CARLO METHODS FOR ENERGY COMPUTATION

#if COMPILECUDA == 1 && MONTE_CARLO == 1

#include "MeshParamsControlCUDA.h"

//Ferromagnetic
__device__ cuBReal ManagedMeshCUDA::Get_EnergyChange_FM_MElasticCUDA(int spin_index, cuReal3 Mnew)
{
	cuVEC_VC<cuReal3>& M = *pM;
	cuVEC_VC<cuReal3>& strain_diag = *pstrain_diag;
	cuVEC_VC<cuReal3>& strain_odiag = *pstrain_odiag;

	cuBReal Ms = *pMs;
	cuReal3 mcanis_ea1 = *pmcanis_ea1;
	cuReal3 mcanis_ea2 = *pmcanis_ea2;
	cuReal3 mcanis_ea3 = *pmcanis_ea3;
	cuReal2 MEc = *pMEc;
	update_parameters_mcoarse(spin_index, *pMs, Ms, *pMEc, MEc, *pmcanis_ea1, mcanis_ea1, *pmcanis_ea2, mcanis_ea2, *pmcanis_ea3, mcanis_ea3);

	cuReal3 position = M.cellidx_to_position(spin_index);
	//xx, yy, zz
	cuReal3 Sd = strain_diag[position];
	//yz, xz, xy
	cuReal3 Sod = strain_odiag[position];

	//normalised magnetization
	//Magneto-elastic term here applicable for a cubic crystal. We use the mcanis_ea1 and mcanis_ea2 axes to fix the cubic lattice orientation, thus rotate the m, Sd and Sod vectors.

	Sd = cuReal3(Sd * mcanis_ea1, Sd * mcanis_ea2, Sd * mcanis_ea3);
	Sod = cuReal3(Sod * mcanis_ea1, Sod * mcanis_ea2, Sod * mcanis_ea3);

	auto Get_Energy = [&](cuReal3 M) -> cuBReal
	{
		cuReal3 m = cuReal3(M * mcanis_ea1, M * mcanis_ea2, M * mcanis_ea3) / Ms;

		cuReal3 Hmel_1 = (-2.0 * MEc.i / ((cuBReal)MU0 * Ms)) * cuReal3(
			m.x * Sd.x * mcanis_ea1.x + m.y * Sd.y * mcanis_ea2.x + m.z * Sd.z * mcanis_ea3.x,
			m.x * Sd.x * mcanis_ea1.y + m.y * Sd.y * mcanis_ea2.y + m.z * Sd.z * mcanis_ea3.y,
			m.x * Sd.x * mcanis_ea1.z + m.y * Sd.y * mcanis_ea2.z + m.z * Sd.z * mcanis_ea3.z);

		cuReal3 Hmel_2 = (-2.0 * MEc.j / ((cuBReal)MU0 * Ms)) * cuReal3(
			Sod.z * (mcanis_ea1.x * m.y + mcanis_ea2.x * m.x) + Sod.y * (mcanis_ea1.x * m.z + mcanis_ea3.x * m.x) + Sod.x * (mcanis_ea2.x * m.z + mcanis_ea3.x * m.y),
			Sod.z * (mcanis_ea1.y * m.y + mcanis_ea2.y * m.x) + Sod.y * (mcanis_ea1.y * m.z + mcanis_ea3.y * m.x) + Sod.x * (mcanis_ea2.y * m.z + mcanis_ea3.y * m.y),
			Sod.z * (mcanis_ea1.z * m.y + mcanis_ea2.z * m.x) + Sod.y * (mcanis_ea1.z * m.z + mcanis_ea3.z * m.x) + Sod.x * (mcanis_ea2.z * m.z + mcanis_ea3.z * m.y));

		return -(cuBReal)MU0 * M * (Hmel_1 + Hmel_2) / 2;
	};

	if (Mnew != cuReal3()) return M.h.dim() * (Get_Energy(Mnew) - Get_Energy(M[spin_index]));
	else return M.h.dim() * Get_Energy(M[spin_index]);
}

//Antiferromagnetic
//N/A

#endif