#include "hip/hip_runtime.h"
#include "Atom_DiffEqCubicCUDA.h"

#if COMPILECUDA == 1
#ifdef MESH_COMPILATION_ATOM_CUBIC

//defines evaluation methods kernel launchers

#include "Atom_MeshParamsControlCUDA.h"

//-----------------------------------------

__global__ void RestoreMoments_Cubic_kernel(cuVEC_VC<cuReal3>& M1, cuVEC<cuReal3>& sM1)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;

	if (idx < M1.linear_size()) {

		M1[idx] = sM1[idx];
	}
}

//Restore magnetization after a failed step for adaptive time-step methods
void Atom_DifferentialEquationCubicCUDA::RestoreMoments(void)
{
	for (mGPU.device_begin(); mGPU != mGPU.device_end(); mGPU++) {

		RestoreMoments_Cubic_kernel <<< (sM1.device_size(mGPU) + CUDATHREADS) / CUDATHREADS, CUDATHREADS >>>
			(paMeshCUDA->M1.get_deviceobject(mGPU), sM1.get_deviceobject(mGPU));
	}
}

//Save current moments in sM VECs (e.g. useful to reset dM / dt calculation)
__global__ void SaveMoments_Cubic_kernel(cuVEC_VC<cuReal3>& M1, cuVEC<cuReal3>& sM1)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;

	if (idx < M1.linear_size()) {

		sM1[idx] = M1[idx];
	}
}

//Restore magnetization after a failed step for adaptive time-step methods
void Atom_DifferentialEquationCubicCUDA::SaveMoments(void)
{
	for (mGPU.device_begin(); mGPU != mGPU.device_end(); mGPU++) {

		SaveMoments_Cubic_kernel <<< (sM1.device_size(mGPU) + CUDATHREADS) / CUDATHREADS, CUDATHREADS >>>
			(paMeshCUDA->M1.get_deviceobject(mGPU), sM1.get_deviceobject(mGPU));
	}
}

//-----------------------------------------

__global__ void RenormalizeMoments_Cubic_kernel(ManagedAtom_MeshCUDA& cuaMesh)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;

	if (idx < cuaMesh.pM1->linear_size()) {

		if (cuaMesh.pM1->is_not_empty(idx)) {

			cuBReal mu_s = *cuaMesh.pmu_s;
			cuaMesh.update_parameters_mcoarse(idx, *cuaMesh.pmu_s, mu_s);

			if (mu_s) (*cuaMesh.pM1)[idx].renormalize(mu_s);
		}
	}
}

//Restore magnetization after a failed step for adaptive time-step methods
void Atom_DifferentialEquationCubicCUDA::RenormalizeMoments(void)
{
	for (mGPU.device_begin(); mGPU != mGPU.device_end(); mGPU++) {

		RenormalizeMoments_Cubic_kernel <<< (paMeshCUDA->M1.device_size(mGPU) + CUDATHREADS) / CUDATHREADS, CUDATHREADS >>> 
			(paMeshCUDA->cuaMesh.get_deviceobject(mGPU));
	}
}

//-----------------------------------------

#endif
#endif