#include "hip/hip_runtime.h"
#include "TransportCUDA.h"

#if COMPILECUDA == 1

#ifdef MODULE_COMPILATION_TRANSPORT

#include "mcuVEC_halo.cuh"

#include "MeshCUDA.h"
#include "SuperMeshCUDA.h"
#include "MeshParamsControlCUDA.h"

//-------------------Display Calculation Methods

//--------------------------------------------------------------- Current Density

__global__ void CalculateFixedCurrentDensity_Charge_Kernel(cuVEC_VC<cuReal3>& Jc, cuVEC_VC<cuReal3>& E, cuVEC_VC<cuBReal>& elC)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;

	if (idx < Jc.linear_size()) {

		//only calculate current on non-empty cells - empty cells have already been assigned 0 at UpdateConfiguration
		if (elC.is_not_empty(idx)) {

			Jc[idx] = elC[idx] * E[idx];
		}
		else Jc[idx] = cuReal3(0.0);
	}
}

__global__ void CalculateCurrentDensity_Spin_Kernel(cuVEC_VC<cuReal3>& Jc, ManagedMeshCUDA& cuMesh, TransportCUDA_Spin_V_Funcs& poisson_Spin_V, cuVEC_VC<cuReal3>& dM_dt)
{
	cuVEC<cuReal3>& E = *cuMesh.pE;
	cuVEC_VC<cuBReal>& V = *cuMesh.pV;
	cuVEC_VC<cuBReal>& elC = *cuMesh.pelC;
	cuVEC_VC<cuReal3>& S = *cuMesh.pS;
	cuVEC_VC<cuReal3>& M = *cuMesh.pM;

	int idx = blockIdx.x * blockDim.x + threadIdx.x;

	if (idx < Jc.linear_size()) {

		//only calculate current on non-empty cells - empty cells have already been assigned 0 at UpdateConfiguration
		if (V.is_not_empty(idx)) {

			bool cppgmr_enabled = cuIsNZ(cuMesh.pbetaD->get0());
			bool cpump_enabled = cuIsNZ(cuMesh.pcpump_eff->get0());
			bool the_enabled = cuIsNZ(cuMesh.pthe_eff->get0());

			if (poisson_Spin_V.stsolve == STSOLVE_NORMALMETAL) {

				//non-magnetic mesh

				if (cuIsZ(cuMesh.piSHA->get0())) {

					//no iSHE contribution.
					Jc[idx] = -elC[idx] * V.grad_diri(idx);
				}
				else {

					cuBReal SHA = *cuMesh.pSHA;
					cuBReal iSHA = *cuMesh.piSHA;
					cuBReal De = *cuMesh.pDe;
					cuMesh.update_parameters_ecoarse(idx, *cuMesh.pSHA, SHA, *cuMesh.piSHA, iSHA, *cuMesh.pDe, De);

					//iSHE enabled, must use non-homogeneous Neumann boundary condition for grad V -> Note homogeneous Neumann boundary conditions apply when calculating S differentials here (due to Jc.n = 0 at boundaries)
					Jc[idx] = -elC[idx] * V.grad_diri_nneu(idx, (iSHA * De / ((cuBReal)MUB_E * elC[idx])) * S.curl_neu(idx));

					//must also add iSHE contribution -> here we must use non-homogeneous Neumann boundary conditions when calculating S differentials
					Jc[idx] += (iSHA * De / (cuBReal)MUB_E) * S.curl_nneu(idx, cu_epsilon3(E[idx]) * (SHA * elC[idx] * (cuBReal)MUB_E / De));
				}
			}
			else {

				//magnetic mesh

				cuReal3 grad_V = V.grad_diri(idx);

				//1. principal term : always present
				Jc[idx] = -elC[idx] * grad_V;

				//additional contributions if enabled
				if (cppgmr_enabled || cpump_enabled || the_enabled) {

					int idx_M = M.position_to_cellidx(S.cellidx_to_position(idx));

					cuReal3 m = cu_normalize(M[idx_M]);
					cuReal33 grad_S = S.grad_neu(idx);		//homogeneous Neumann since SHA = 0 in magnetic meshes

					//2. CPP-GMR contribution
					if (cppgmr_enabled) {

						cuBReal betaD = *cuMesh.pbetaD;
						cuBReal De = *cuMesh.pDe;
						cuMesh.update_parameters_ecoarse(idx, *cuMesh.pbetaD, betaD, *cuMesh.pDe, De);

						Jc[idx] += (grad_S * m) * betaD * De / (cuBReal)MUB_E;
					}

					//3. topological Hall effect contribution
					//4. charge pumping contribution
					if (cpump_enabled || the_enabled) {

						cuBReal P = *cuMesh.pP;
						cuBReal n_density = *cuMesh.pn_density;
						cuMesh.update_parameters_ecoarse(idx, *cuMesh.pP, P, *cuMesh.pn_density, n_density);

						cuReal33 grad_M = M.grad_neu(idx_M);
						cuReal3 dx_m = cu_normalize(grad_M.x, M[idx_M]);
						cuReal3 dy_m = cu_normalize(grad_M.y, M[idx_M]);

						//topological Hall effect contribution
						if (the_enabled) {

							cuBReal Bz_the = (dx_m ^ dy_m) * m;
							Jc[idx] += cuMesh.pthe_eff->get0() * (P * elC[idx] * (cuBReal)HBAR_E / ((cuBReal)ECHARGE * n_density)) * elC[idx] * cuReal3(grad_V.y * Bz_the, -grad_V.x *Bz_the, 0.0);
						}

						//charge pumping contribution
						if (cpump_enabled) {

							cuReal3 dm_dt = cu_normalize(dM_dt[idx_M], M[idx_M]);
							Jc[idx] += cuMesh.pcpump_eff->get0() * (P * elC[idx] * (cuBReal)HBAR_E / 2) * cuReal3((dm_dt ^ dx_m) * m, (dm_dt ^ dy_m) * m, 0.0);
						}
					}
				}
			}
		}
		else Jc[idx] = cuReal3(0);
	}
}

//-------------------Calculation Methods : Charge Current Density

//calculate charge current density over the mesh
mcu_VEC_VC(cuReal3)& TransportCUDA::GetChargeCurrent(void)
{
	if (!PrepareDisplayVEC_VC(pMeshCUDA->h_e)) return displayVEC_VC;

	if (!pSMeshCUDA->DisabledTransportSolver()) {

		if (stsolve == STSOLVE_NONE) {

			for (mGPU.device_begin(); mGPU != mGPU.device_end(); mGPU++) {

				CalculateFixedCurrentDensity_Charge_Kernel <<< (pMeshCUDA->elC.device_size(mGPU) + CUDATHREADS) / CUDATHREADS, CUDATHREADS >>>
					(displayVEC_VC.get_deviceobject(mGPU), pMeshCUDA->E.get_deviceobject(mGPU), pMeshCUDA->elC.get_deviceobject(mGPU));
			}
		}
		else {
			
			pMeshCUDA->V.exchange_halos();
			pMeshCUDA->S.exchange_halos();
			pMeshCUDA->M.exchange_halos();

			for (mGPU.device_begin(); mGPU != mGPU.device_end(); mGPU++) {

				CalculateCurrentDensity_Spin_Kernel <<< (pMeshCUDA->elC.device_size(mGPU) + CUDATHREADS) / CUDATHREADS, CUDATHREADS >>>
					(displayVEC_VC.get_deviceobject(mGPU), pMeshCUDA->cuMesh.get_deviceobject(mGPU), poisson_Spin_V.get_deviceobject(mGPU), dM_dt.get_deviceobject(mGPU));
			}
		}
	}
	else {

		//if transport solver disabled we need to set displayVEC_VC directly from E and elC as Jc = elC * E
		for (mGPU.device_begin(); mGPU != mGPU.device_end(); mGPU++) {

			CalculateFixedCurrentDensity_Charge_Kernel <<< (pMeshCUDA->elC.device_size(mGPU) + CUDATHREADS) / CUDATHREADS, CUDATHREADS >>>
				(displayVEC_VC.get_deviceobject(mGPU), pMeshCUDA->E.get_deviceobject(mGPU), pMeshCUDA->elC.get_deviceobject(mGPU));
		}
	}

	return displayVEC_VC;
}

#endif

#endif