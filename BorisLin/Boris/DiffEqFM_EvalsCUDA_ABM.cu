#include "hip/hip_runtime.h"
#include "DiffEqFMCUDA.h"

#if COMPILECUDA == 1
#ifdef ODE_EVAL_COMPILATION_ABM
#ifdef MESH_COMPILATION_FERROMAGNETIC

#include "MeshParamsControlCUDA.h"

#include "Reduction.cuh"

//defines evaluation methods kernel launchers

//----------------------------------------- EVALUATIONS : ABM

__global__ void RunABM_Predictor_withReductions_Kernel(ManagedDiffEqFMCUDA& cuDiffEq, ManagedMeshCUDA& cuMesh)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;

	cuBReal dT = *cuDiffEq.pdT;

	cuBReal mxh = 0.0;

	if (idx < cuMesh.pM->linear_size()) {

		if (cuMesh.pM->is_not_empty(idx)) {

			//Save current magnetization for the next step
			(*cuDiffEq.psM1)[idx] = (*cuMesh.pM)[idx];

			if (!cuMesh.pM->is_skipcell(idx)) {

				//obtain maximum normalized torque term
				cuBReal Mnorm = (*cuMesh.pM)[idx].norm();
				mxh = cu_GetMagnitude((*cuMesh.pM)[idx] ^ (*cuMesh.pHeff)[idx]) / (Mnorm * Mnorm);

				//First evaluate RHS of set equation at the current time step
				cuReal3 rhs = (cuDiffEq.*(cuDiffEq.pODEFunc))(idx);

				//ABM predictor : pk+1 = mk + (dt/2) * (3*fk - fk-1)
				if (*cuDiffEq.palternator) {

					(*cuMesh.pM)[idx] += dT * (3 * rhs - (*cuDiffEq.psEval0)[idx]) / 2;
					(*cuDiffEq.psEval1)[idx] = rhs;
				}
				else {

					(*cuMesh.pM)[idx] += dT * (3 * rhs - (*cuDiffEq.psEval1)[idx]) / 2;
					(*cuDiffEq.psEval0)[idx] = rhs;
				}
			}
		}
	}

	//only reduce for mxh if grel is not zero (if it's zero this means magnetization dynamics is disabled in this mesh)
	if (cuMesh.pgrel->get0()) {

		reduction_max(0, 1, &mxh, *cuDiffEq.pmxh);
	}
}

__global__ void RunABM_Predictor_Kernel(ManagedDiffEqFMCUDA& cuDiffEq, ManagedMeshCUDA& cuMesh)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;

	cuBReal dT = *cuDiffEq.pdT;

	if (idx < cuMesh.pM->linear_size()) {

		if (cuMesh.pM->is_not_empty(idx)) {

			//Save current magnetization for the next step
			(*cuDiffEq.psM1)[idx] = (*cuMesh.pM)[idx];

			if (!cuMesh.pM->is_skipcell(idx)) {

				//First evaluate RHS of set equation at the current time step
				cuReal3 rhs = (cuDiffEq.*(cuDiffEq.pODEFunc))(idx);

				//ABM predictor : pk+1 = mk + (dt/2) * (3*fk - fk-1)
				if (*cuDiffEq.palternator) {

					(*cuMesh.pM)[idx] += dT * (3 * rhs - (*cuDiffEq.psEval0)[idx]) / 2;
					(*cuDiffEq.psEval1)[idx] = rhs;
				}
				else {

					(*cuMesh.pM)[idx] += dT * (3 * rhs - (*cuDiffEq.psEval1)[idx]) / 2;
					(*cuDiffEq.psEval0)[idx] = rhs;
				}
			}
		}
	}
}

__global__ void RunABM_Corrector_withReductions_Kernel(ManagedDiffEqFMCUDA& cuDiffEq, ManagedMeshCUDA& cuMesh)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;

	cuBReal dT = *cuDiffEq.pdT;

	cuBReal dmdt = 0.0;
	cuBReal lte = 0.0;

	if (idx < cuMesh.pM->linear_size()) {

		if (cuMesh.pM->is_not_empty(idx)) {

			if (!cuMesh.pM->is_skipcell(idx)) {

				//First evaluate RHS of set equation at the current time step
				cuReal3 rhs = (cuDiffEq.*(cuDiffEq.pODEFunc))(idx);

				//First save predicted magnetization for lte calculation
				cuReal3 saveM = (*cuMesh.pM)[idx];

				//ABM corrector : mk+1 = mk + (dt/2) * (fk+1 + fk)
				if (*cuDiffEq.palternator) {

					(*cuMesh.pM)[idx] = (*cuDiffEq.psM1)[idx] + dT * (rhs + (*cuDiffEq.psEval1)[idx]) / 2;
				}
				else {

					(*cuMesh.pM)[idx] = (*cuDiffEq.psM1)[idx] + dT * (rhs + (*cuDiffEq.psEval0)[idx]) / 2;
				}

				if (*cuDiffEq.prenormalize) {

					cuBReal Ms = *cuMesh.pMs;
					cuMesh.update_parameters_mcoarse(idx, *cuMesh.pMs, Ms);
					(*cuMesh.pM)[idx].renormalize(Ms);
				}

				//obtain maximum normalized dmdt term
				cuBReal Mnorm = (*cuMesh.pM)[idx].norm();
				dmdt = cu_GetMagnitude((*cuMesh.pM)[idx] - (*cuDiffEq.psM1)[idx]) / (dT * (cuBReal)GAMMA * Mnorm * Mnorm);

				//local truncation error (between predicted and corrected)
				lte = cu_GetMagnitude((*cuMesh.pM)[idx] - saveM) / (*cuMesh.pM)[idx].norm();
			}
			else {

				cuBReal Ms = *cuMesh.pMs;
				cuMesh.update_parameters_mcoarse(idx, *cuMesh.pMs, Ms);
				(*cuMesh.pM)[idx].renormalize(Ms);		//re-normalize the skipped cells no matter what - temperature can change
			}
		}
	}

	//only reduce for dmdt if grel is not zero (if it's zero this means magnetization dynamics is disabled in this mesh)
	if (cuMesh.pgrel->get0()) {

		reduction_max(0, 1, &dmdt, *cuDiffEq.pdmdt);
	}

	reduction_max(0, 1, &lte, *cuDiffEq.plte);
}

__global__ void RunABM_Corrector_Kernel(ManagedDiffEqFMCUDA& cuDiffEq, ManagedMeshCUDA& cuMesh)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;

	cuBReal dT = *cuDiffEq.pdT;

	cuBReal lte = 0.0;

	if (idx < cuMesh.pM->linear_size()) {

		if (cuMesh.pM->is_not_empty(idx)) {

			if (!cuMesh.pM->is_skipcell(idx)) {

				//First evaluate RHS of set equation at the current time step
				cuReal3 rhs = (cuDiffEq.*(cuDiffEq.pODEFunc))(idx);

				//First save predicted magnetization for lte calculation
				cuReal3 saveM = (*cuMesh.pM)[idx];

				//ABM corrector : mk+1 = mk + (dt/2) * (fk+1 + fk)
				if (*cuDiffEq.palternator) {

					(*cuMesh.pM)[idx] = (*cuDiffEq.psM1)[idx] + dT * (rhs + (*cuDiffEq.psEval1)[idx]) / 2;
				}
				else {

					(*cuMesh.pM)[idx] = (*cuDiffEq.psM1)[idx] + dT * (rhs + (*cuDiffEq.psEval0)[idx]) / 2;
				}

				if (*cuDiffEq.prenormalize) {

					cuBReal Ms = *cuMesh.pMs;
					cuMesh.update_parameters_mcoarse(idx, *cuMesh.pMs, Ms);
					(*cuMesh.pM)[idx].renormalize(Ms);
				}

				//local truncation error (between predicted and corrected)
				lte = cu_GetMagnitude((*cuMesh.pM)[idx] - saveM) / (*cuMesh.pM)[idx].norm();
			}
			else {

				cuBReal Ms = *cuMesh.pMs;
				cuMesh.update_parameters_mcoarse(idx, *cuMesh.pMs, Ms);
				(*cuMesh.pM)[idx].renormalize(Ms);		//re-normalize the skipped cells no matter what - temperature can change
			}
		}
	}

	reduction_max(0, 1, &lte, *cuDiffEq.plte);
}

__global__ void RunABMTEuler_Step0_Kernel(ManagedDiffEqFMCUDA& cuDiffEq, ManagedMeshCUDA& cuMesh)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;

	cuBReal dT = *cuDiffEq.pdT;

	if (idx < cuMesh.pM->linear_size()) {

		if (cuMesh.pM->is_not_empty(idx)) {

			//Save current magnetization for the next step
			(*cuDiffEq.psM1)[idx] = (*cuMesh.pM)[idx];

			if (!cuMesh.pM->is_skipcell(idx)) {

				//First evaluate RHS of set equation at the current time step
				(*cuDiffEq.psEval0)[idx] = (cuDiffEq.*(cuDiffEq.pODEFunc))(idx);

				//Now estimate magnetization for the next time step
				(*cuMesh.pM)[idx] += (*cuDiffEq.psEval0)[idx] * dT;
			}
		}
	}
}

__global__ void RunABMTEuler_Step1_Kernel(ManagedDiffEqFMCUDA& cuDiffEq, ManagedMeshCUDA& cuMesh)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;

	cuBReal dT = *cuDiffEq.pdT;

	if (idx < cuMesh.pM->linear_size()) {

		if (cuMesh.pM->is_not_empty(idx)) {

			if (!cuMesh.pM->is_skipcell(idx)) {

				//First evaluate RHS of set equation at the current time step
				cuReal3 rhs = (cuDiffEq.*(cuDiffEq.pODEFunc))(idx);

				//Now estimate magnetization using the second trapezoidal Euler step equation
				(*cuMesh.pM)[idx] = ((*cuDiffEq.psM1)[idx] + (*cuMesh.pM)[idx] + rhs * dT) / 2;

				if (*cuDiffEq.prenormalize) {

					cuBReal Ms = *cuMesh.pMs;
					cuMesh.update_parameters_mcoarse(idx, *cuMesh.pMs, Ms);
					(*cuMesh.pM)[idx].renormalize(Ms);
				}
			}
			else {

				cuBReal Ms = *cuMesh.pMs;
				cuMesh.update_parameters_mcoarse(idx, *cuMesh.pMs, Ms);
				(*cuMesh.pM)[idx].renormalize(Ms);		//re-normalize the skipped cells no matter what - temperature can change
			}
		}
	}
}

//----------------------------------------- DifferentialEquationCUDA Launchers

//Adams-Bashforth-Moulton 2nd order

void DifferentialEquationFMCUDA::RunABM(int step, bool calculate_mxh, bool calculate_dmdt)
{
	if (step == 0) {

		for (mGPU.device_begin(); mGPU != mGPU.device_end(); mGPU++) {

			if (calculate_mxh) {

				RunABM_Predictor_withReductions_Kernel <<< (pMeshCUDA->M.device_size(mGPU) + CUDATHREADS) / CUDATHREADS, CUDATHREADS >>>
					(cuDiffEq.get_deviceobject(mGPU), pMeshCUDA->cuMesh.get_deviceobject(mGPU));
			}
			else {

				RunABM_Predictor_Kernel <<< (pMeshCUDA->M.device_size(mGPU) + CUDATHREADS) / CUDATHREADS, CUDATHREADS >>>
					(cuDiffEq.get_deviceobject(mGPU), pMeshCUDA->cuMesh.get_deviceobject(mGPU));
			}
		}
	}
	else {

		for (mGPU.device_begin(); mGPU != mGPU.device_end(); mGPU++) {

			if (calculate_dmdt) {

				RunABM_Corrector_withReductions_Kernel <<< (pMeshCUDA->M.device_size(mGPU) + CUDATHREADS) / CUDATHREADS, CUDATHREADS >>>
					(cuDiffEq.get_deviceobject(mGPU), pMeshCUDA->cuMesh.get_deviceobject(mGPU));
			}
			else {

				RunABM_Corrector_Kernel <<< (pMeshCUDA->M.device_size(mGPU) + CUDATHREADS) / CUDATHREADS, CUDATHREADS >>>
					(cuDiffEq.get_deviceobject(mGPU), pMeshCUDA->cuMesh.get_deviceobject(mGPU));
			}
		}
	}
}

//Adams-Bashforth-Moulton 2nd order priming using Trapezoidal Euler

void DifferentialEquationFMCUDA::RunABMTEuler(int step)
{
	if (step == 0) {

		for (mGPU.device_begin(); mGPU != mGPU.device_end(); mGPU++) {

			RunABMTEuler_Step0_Kernel <<< (pMeshCUDA->M.device_size(mGPU) + CUDATHREADS) / CUDATHREADS, CUDATHREADS >>> 
				(cuDiffEq.get_deviceobject(mGPU), pMeshCUDA->cuMesh.get_deviceobject(mGPU));
		}
	}
	else {

		for (mGPU.device_begin(); mGPU != mGPU.device_end(); mGPU++) {

			RunABMTEuler_Step1_Kernel <<< (pMeshCUDA->M.device_size(mGPU) + CUDATHREADS) / CUDATHREADS, CUDATHREADS >>> 
				(cuDiffEq.get_deviceobject(mGPU), pMeshCUDA->cuMesh.get_deviceobject(mGPU));
		}
	}
}

#endif
#endif
#endif