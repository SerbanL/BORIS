#include "hip/hip_runtime.h"
#include "TransportCUDA.h"

#if COMPILECUDA == 1

#ifdef MODULE_COMPILATION_TRANSPORT

#include "mcuVEC_halo.cuh"

#include "MeshCUDA.h"
#include "SuperMeshCUDA.h"
#include "MeshParamsControlCUDA.h"

//-------------------Display Calculation Methods

//SPIN CURRENT

__global__ void GetSpinCurrent_Kernel(int component, cuVEC<cuReal3>& displayVEC, ManagedMeshCUDA& cuMesh, TransportCUDA_Spin_S_Funcs& poisson_Spin_S, cuVEC_VC<cuReal3>& dM_dt)
{
	cuVEC_VC<cuReal3>& M = *cuMesh.pM;
	cuVEC_VC<cuReal3>& S = *cuMesh.pS;
	cuVEC_VC<cuReal3>& E = *cuMesh.pE;
	cuVEC_VC<cuBReal>& elC = *cuMesh.pelC;

	int idx = blockIdx.x * blockDim.x + threadIdx.x;

	if (idx < S.linear_size()) {

		bool cpump_enabled = cuIsNZ(cuMesh.pcpump_eff->get0());
		bool the_enabled = cuIsNZ(cuMesh.pthe_eff->get0());

		cuReal33 Js = cuReal33();

		if (S.is_not_empty(idx)) {
			
			if (poisson_Spin_S.stsolve == STSOLVE_FERROMAGNETIC) {

				//magnetic mesh terms

				cuBReal P = *cuMesh.pP;
				cuBReal De = *cuMesh.pDe;
				cuMesh.update_parameters_ecoarse(idx, *cuMesh.pP, P, *cuMesh.pDe, De);

				//1. drift
				int idx_M = M.position_to_cellidx(S.cellidx_to_position(idx));

				cuReal3 mval = cu_normalize(M[idx_M]);
				cuReal33 grad_S = S.grad_neu(idx);

				Js = (E[idx] | mval) * (P * elC[idx]) * (-(cuBReal)MUB_E);

				//2. diffusion with homogeneous Neumann boundary condition
				Js -= grad_S * De;

				//3. charge pumping
				//4. topological Hall effect

				if (component != 2 && (cpump_enabled || the_enabled)) {

					cuReal33 grad_m = cu_normalize(M.grad_neu(idx_M), M[idx_M]);

					//topological Hall effect contribution
					if (the_enabled) {

						cuBReal n_density = *cuMesh.pn_density;
						cuMesh.update_parameters_ecoarse(idx, *cuMesh.pn_density, n_density);

						cuReal3 B = (grad_m.x ^ grad_m.y);
						Js += cuMesh.pthe_eff->get0() * ((cuBReal)HBAR_E * (cuBReal)MUB_E * elC[idx] * elC[idx] / ((cuBReal)ECHARGE * n_density)) * cuReal33(-E[idx].y * B, E[idx].x * B, cuReal3());
					}

					//charge pumping contribution
					if (cpump_enabled) {

						//value a1
						cuReal3 dm_dt = cu_normalize(dM_dt[idx_M], M[idx_M]);
						Js += cuMesh.pcpump_eff->get0() * ((cuBReal)HBAR_E * (cuBReal)MUB_E * elC[idx] / 2) * cuReal33(dm_dt ^ grad_m.x, dm_dt ^ grad_m.y, cuReal3());
					}
				}
			}
			else {

				//non-magnetic mesh terms

				cuBReal De = *cuMesh.pDe;
				cuBReal SHA = *cuMesh.pSHA;
				cuMesh.update_parameters_ecoarse(idx, *cuMesh.pDe, De, *cuMesh.pSHA, SHA);

				//1. SHE contribution
				Js = cu_epsilon3(E[idx]) * SHA * elC[idx] * (cuBReal)MUB_E;

				//2. diffusion with non-homogeneous Neumann boundary condition
				Js -= S.grad_nneu(idx, cu_epsilon3(E[idx]) * (SHA * elC[idx] * (cuBReal)MUB_E / De)) * De;
			}
		}

		switch (component) {

		case 0:
			displayVEC[idx] = Js.x;
			break;
		case 1:
			displayVEC[idx] = Js.y;
			break;
		case 2:
			displayVEC[idx] = Js.z;
			break;
		}
	}
}

//SPIN TORQUE

__global__ void GetSpinTorque_Kernel(cuVEC<cuReal3>& displayVEC, ManagedMeshCUDA& cuMesh)
{
	cuVEC_VC<cuReal3>& M = *cuMesh.pM;
	cuVEC_VC<cuReal3>& S = *cuMesh.pS;

	int idx = blockIdx.x * blockDim.x + threadIdx.x;

	if (idx < M.linear_size()) {

		if (M.is_empty(idx)) {

			displayVEC[idx] = cuReal3();
			return;
		}

		cuBReal De = *cuMesh.pDe;
		cuBReal ts_eff = *cuMesh.pts_eff;
		cuBReal l_ex = *cuMesh.pl_ex;
		cuBReal l_ph = *cuMesh.pl_ph;
		cuMesh.update_parameters_mcoarse(idx, *cuMesh.pDe, De, *cuMesh.pts_eff, ts_eff, *cuMesh.pl_ex, l_ex, *cuMesh.pl_ph, l_ph);

		cuReal3 Sav = S.weighted_average(M.cellidx_to_position(idx), M.h);
		cuReal3 m = cu_normalize(M[idx]);

		displayVEC[idx] = ts_eff * ((Sav ^ m) * De / (l_ex * l_ex) + (m ^ (Sav ^ m)) * De / (l_ph * l_ph));
	}
}

//SPIN INTERFACE TORQUE

__global__ void CalculateDisplaySAInterfaceTorque_Kernel(
	CMBNDInfoCUDA& contact, 
	TransportCUDA_Spin_S_CMBND_Sec& cmbndFuncs_sec, TransportCUDA_Spin_S_CMBND_Pri& cmbndFuncs_pri, 
	cuVEC<cuReal3>& displayVEC)
{
	cuVEC_VC<cuReal3>& M = *cmbndFuncs_pri.pcuMesh->pM;
	cuVEC_VC<cuReal3>& S_pri = *cmbndFuncs_pri.pcuMesh->pS;
	//access S on first device, which contains origin of entire mcuVEC
	cuVEC_VC<cuReal3>& S0_sec = *cmbndFuncs_sec.ppcuMesh[0]->pS;

	int box_idx = blockIdx.x * blockDim.x + threadIdx.x;

	//interface conductance method with F being the primary mesh : calculate and set spin torque

	//convert the cells box from S mesh to M mesh
	cuINT3 mbox_start = M.cellidx_from_position(S_pri.cellidx_to_position(contact.cells_box.s) + M.rect.s);
	cuINT3 mbox_end = M.cellidx_from_position(S_pri.cellidx_to_position(contact.cells_box.e - cuINT3(1)) + M.rect.s) + cuINT3(1);

	if ((mbox_end.i - mbox_start.i) == 0) mbox_end.i = mbox_start.i + 1;
	if ((mbox_end.j - mbox_start.j) == 0) mbox_end.j = mbox_start.j + 1;
	if ((mbox_end.k - mbox_start.k) == 0) mbox_end.k = mbox_start.k + 1;

	cuINT3 box_sizes = mbox_end - mbox_start;

	if (box_idx < box_sizes.dim()) {

		//the cellsize perpendicular to the contact (in the M mesh)
		cuBReal dh = (cuReal3(contact.cell_shift) & M.h).norm();

		int i = (box_idx % box_sizes.x) + mbox_start.i;
		int j = ((box_idx / box_sizes.x) % box_sizes.y) + mbox_start.j;
		int k = (box_idx / (box_sizes.x * box_sizes.y)) + mbox_start.k;

		//index of magnetic cell 1
		int mcell1_idx = i + j * M.n.x + k * M.n.x*M.n.y;

		if (M.is_empty(mcell1_idx)) return;

		cuBReal tsi_eff = *cmbndFuncs_pri.pcuMesh->ptsi_eff;
		cmbndFuncs_pri.pcuMesh->update_parameters_mcoarse(mcell1_idx, *cmbndFuncs_pri.pcuMesh->ptsi_eff, tsi_eff);

		//position at interface relative to primary mesh
		
		cuReal3 mhshift_primary = contact.hshift_primary.normalized() & M.h;
		cuReal3 relpos_interf = ((cuReal3(i, j, k) + cuReal3(0.5)) & M.h) + mhshift_primary / 2;

		cuReal3 relpos_1 = relpos_interf - contact.hshift_primary / 2;

		//relpos_m1 is relative to entire mcuVEC (sec)
		cuReal3 relpos_m1 = S_pri.rect.s - S0_sec.rect.s + relpos_interf + contact.hshift_secondary / 2;
		//get device and device-relative position on secondary side
		int device = 0;
		cuReal3 devrelpos_m1 = cmbndFuncs_sec.global_relpos_to_device_relpos(relpos_m1, device);
		cuVEC_VC<cuReal3>& S_sec = *cmbndFuncs_sec.ppcuMesh[device]->pS;

		cuReal3 stencil_sec = M.h - cu_mod(mhshift_primary) + cu_mod(contact.hshift_secondary);
		cuReal3 stencil_pri = M.h - cu_mod(mhshift_primary) + cu_mod(contact.hshift_primary);

		//S values
		cuReal3 S_1 = S_pri.weighted_average(relpos_1, stencil_pri);
		cuReal3 S_2 = S_pri.weighted_average(relpos_1 - contact.hshift_primary, stencil_pri);
		cuReal3 S_m1 = S_sec.weighted_average(devrelpos_m1, stencil_sec);
		cuReal3 S_m2 = S_sec.weighted_average(devrelpos_m1 + contact.hshift_secondary, stencil_sec);

		//c values
		cuBReal c_1 = cmbndFuncs_pri.c_func_sec(relpos_1, stencil_pri);
		cuBReal c_2 = cmbndFuncs_pri.c_func_sec(relpos_1 - contact.hshift_primary, stencil_pri);
		cuBReal c_m1 = cmbndFuncs_sec.c_func_sec(relpos_m1, stencil_sec);
		cuBReal c_m2 = cmbndFuncs_sec.c_func_sec(relpos_m1 + contact.hshift_secondary, stencil_sec);

		//Calculate S drop at the interface
		cuReal3 Vs_F = 1.5 * c_1 * S_1 - 0.5 * c_2 * S_2;
		cuReal3 Vs_N = 1.5 * c_m1 * S_m1 - 0.5 * c_m2 * S_m2;
		cuReal3 dVs = Vs_F - Vs_N;

		//Get G values from top contacting mesh
		cuReal2 Gmix;
		if (contact.IsPrimaryTop()) {

			Gmix = *cmbndFuncs_pri.pcuMesh->pGmix;
			cmbndFuncs_pri.pcuMesh->update_parameters_mcoarse(mcell1_idx, *cmbndFuncs_pri.pcuMesh->pGmix, Gmix);
		}
		else {

			Gmix = *cmbndFuncs_sec.ppcuMesh[device]->pGmix;
			cmbndFuncs_sec.ppcuMesh[device]->update_parameters_atposition(relpos_m1, *cmbndFuncs_sec.ppcuMesh[device]->pGmix, Gmix);
		}

		cuBReal Mnorm = M[mcell1_idx].norm();
		if (Mnorm > 0.0) {

			cuBReal gI = (2.0 * (cuBReal)GMUB_2E / dh) * Gmix.j / Mnorm;
			cuBReal gR = (2.0 * (cuBReal)GMUB_2E / dh) * Gmix.i / Mnorm;

			displayVEC[mcell1_idx] += tsi_eff * (gI * (M[mcell1_idx] ^ dVs) + gR * (M[mcell1_idx] ^ (M[mcell1_idx] ^ dVs)) / Mnorm);
		}
	}
}

//Launchers

//return x, y, or z component of spin current (component = 0, 1, or 2)
mcu_VEC(cuReal3)& TransportCUDA::GetSpinCurrent(int component)
{
	if (!PrepareDisplayVEC(pMeshCUDA->h_e)) return displayVEC;

	if (stsolve != STSOLVE_NONE) {

		pMeshCUDA->S.exchange_halos();
		pMeshCUDA->M.exchange_halos();

		for (mGPU.device_begin(); mGPU != mGPU.device_end(); mGPU++) {

			GetSpinCurrent_Kernel <<< (pMeshCUDA->elC.device_size(mGPU) + CUDATHREADS) / CUDATHREADS, CUDATHREADS >>>
				(component, displayVEC.get_deviceobject(mGPU), pMeshCUDA->cuMesh.get_deviceobject(mGPU), poisson_Spin_S.get_deviceobject(mGPU), dM_dt.get_deviceobject(mGPU));
		}
	}

	return displayVEC;
}

//return spin torque computed from spin accumulation
mcu_VEC(cuReal3)& TransportCUDA::GetSpinTorque(void)
{
	if (!PrepareDisplayVEC(pMeshCUDA->h)) return displayVEC;
	
	if (stsolve == STSOLVE_FERROMAGNETIC) {

		for (mGPU.device_begin(); mGPU != mGPU.device_end(); mGPU++) {

			GetSpinTorque_Kernel <<< (pMeshCUDA->M.device_size(mGPU) + CUDATHREADS) / CUDATHREADS, CUDATHREADS >>>
				(displayVEC.get_deviceobject(mGPU), pMeshCUDA->cuMesh.get_deviceobject(mGPU));
		}
	}

	return displayVEC;
}

//Calculate the interface spin accumulation torque for a given contact (in magnetic meshes for NF interfaces with G interface conductance set), accumulating result in displayVEC
void TransportCUDA::CalculateDisplaySAInterfaceTorque(TransportBaseCUDA* ptrans_sec, mCMBNDInfoCUDA& contactCUDA, bool primary_top)
{
	//the top contacting mesh sets G values
	bool isGInterface_Enabled = ((primary_top && GInterface_Enabled()) || (!primary_top && ptrans_sec->GInterface_Enabled()));

	if (isGInterface_Enabled && stsolve == STSOLVE_FERROMAGNETIC && (ptrans_sec->Get_STSolveType() == STSOLVE_NORMALMETAL || ptrans_sec->Get_STSolveType() == STSOLVE_TUNNELING)) {

		for (mGPU.device_begin(); mGPU != mGPU.device_end(); mGPU++) {

			CalculateDisplaySAInterfaceTorque_Kernel <<< (pMeshCUDA->M.device_size(mGPU) + CUDATHREADS) / CUDATHREADS, CUDATHREADS >>>
				(contactCUDA.get_deviceobject(mGPU), ptrans_sec->spin_S_cmbnd_funcs_sec.get_deviceobject(mGPU), spin_S_cmbnd_funcs_pri.get_deviceobject(mGPU), displayVEC.get_deviceobject(mGPU));
		}
	}
}

#endif

#endif