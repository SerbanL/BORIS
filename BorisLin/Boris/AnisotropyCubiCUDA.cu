#include "hip/hip_runtime.h"
#include "AnisotropyCubiCUDA.h"

#if COMPILECUDA == 1

#ifdef MODULE_COMPILATION_ANICUBI

#include "Reduction.cuh"

#include "MeshCUDA.h"
#include "MeshParamsControlCUDA.h"
#include "MeshDefs.h"

__global__ void Anisotropy_CubicCUDA_FM_UpdateField(ManagedMeshCUDA& cuMesh, ManagedModulesCUDA& cuModule, bool do_reduction)
{
	cuVEC_VC<cuReal3>& M = *cuMesh.pM;
	cuVEC<cuReal3>& Heff = *cuMesh.pHeff;

	int idx = blockIdx.x * blockDim.x + threadIdx.x;

	cuBReal energy_ = 0.0;

	if (idx < Heff.linear_size()) {

		cuReal3 Heff_value = cuReal3();

		if (M.is_not_empty(idx)) {

			cuBReal Ms = *cuMesh.pMs;
			cuBReal K1 = *cuMesh.pK1;
			cuBReal K2 = *cuMesh.pK2;
			cuReal3 mcanis_ea1 = *cuMesh.pmcanis_ea1;
			cuReal3 mcanis_ea2 = *cuMesh.pmcanis_ea2;
			cuReal3 mcanis_ea3 = *cuMesh.pmcanis_ea3;
			cuMesh.update_parameters_mcoarse(idx, *cuMesh.pMs, Ms, *cuMesh.pK1, K1, *cuMesh.pK2, K2, *cuMesh.pmcanis_ea1, mcanis_ea1, *cuMesh.pmcanis_ea2, mcanis_ea2, *cuMesh.pmcanis_ea3, mcanis_ea3);

			//calculate m.ea1, m.ea2 and m.ea3 dot products
			cuBReal d1 = (M[idx] * mcanis_ea1) / Ms;
			cuBReal d2 = (M[idx] * mcanis_ea2) / Ms;
			cuBReal d3 = (M[idx] * mcanis_ea3) / Ms;

			//terms for K1 contribution
			cuBReal a1 = d1 * (d2*d2 + d3 * d3);
			cuBReal a2 = d2 * (d1*d1 + d3 * d3);
			cuBReal a3 = d3 * (d1*d1 + d2 * d2);

			//terms for K2 contribution
			cuBReal d123 = d1 * d2*d3;

			cuBReal b1 = d123 * d2*d3;
			cuBReal b2 = d123 * d1*d3;
			cuBReal b3 = d123 * d1*d2;

			//update effective field with the anisotropy field
			Heff_value = cuReal3(
				(-2 * K1 / ((cuBReal)MU0*Ms)) * (mcanis_ea1.i * a1 + mcanis_ea2.i * a2 + mcanis_ea3.i * a3)
				+ (-2 * K2 / ((cuBReal)MU0*Ms)) * (mcanis_ea1.i * b1 + mcanis_ea2.i * b2 + mcanis_ea3.i * b3),

				(-2 * K1 / ((cuBReal)MU0*Ms)) * (mcanis_ea1.j * a1 + mcanis_ea2.j * a2 + mcanis_ea3.j * a3)
				+ (-2 * K2 / ((cuBReal)MU0*Ms)) * (mcanis_ea1.j * b1 + mcanis_ea2.j * b2 + mcanis_ea3.j * b3),

				(-2 * K1 / ((cuBReal)MU0*Ms)) * (mcanis_ea1.k * a1 + mcanis_ea2.k * a2 + mcanis_ea3.k * a3)
				+ (-2 * K2 / ((cuBReal)MU0*Ms)) * (mcanis_ea1.k * b1 + mcanis_ea2.k * b2 + mcanis_ea3.k * b3)
			);

			if (do_reduction) {

				//update energy (E/V)		
				int non_empty_cells = M.get_nonempty_cells();
				if (non_empty_cells) energy_ = (K1 * (d1*d1*d2*d2 + d1 * d1*d3*d3 + d2 * d2*d3*d3) + K2 * d123*d123) / non_empty_cells;
			}

			if (do_reduction && cuModule.pModule_Heff->linear_size()) (*cuModule.pModule_Heff)[idx] = Heff_value;
			if (do_reduction && cuModule.pModule_energy->linear_size()) (*cuModule.pModule_energy)[idx] = (K1 * (d1*d1*d2*d2 + d1 * d1*d3*d3 + d2 * d2*d3*d3) + K2 * d123*d123);
		}

		Heff[idx] += Heff_value;
	}

	if (do_reduction) reduction_sum(0, 1, &energy_, *cuModule.penergy);
}

__global__ void Anisotropy_CubicCUDA_AFM_UpdateField(ManagedMeshCUDA& cuMesh, ManagedModulesCUDA& cuModule, bool do_reduction)
{
	cuVEC_VC<cuReal3>& M = *cuMesh.pM;
	cuVEC<cuReal3>& Heff = *cuMesh.pHeff;

	cuVEC_VC<cuReal3>& M2 = *cuMesh.pM2;
	cuVEC<cuReal3>& Heff2 = *cuMesh.pHeff2;

	int idx = blockIdx.x * blockDim.x + threadIdx.x;

	cuBReal energy_ = 0.0;

	if (idx < Heff.linear_size()) {

		cuReal3 Heff_value = cuReal3();
		cuReal3 Heff2_value = cuReal3();

		if (M.is_not_empty(idx)) {

			cuReal2 Ms_AFM = *cuMesh.pMs_AFM;
			cuReal2 K1_AFM = *cuMesh.pK1_AFM;
			cuReal2 K2_AFM = *cuMesh.pK2_AFM;
			cuReal3 mcanis_ea1 = *cuMesh.pmcanis_ea1;
			cuReal3 mcanis_ea2 = *cuMesh.pmcanis_ea2;
			cuReal3 mcanis_ea3 = *cuMesh.pmcanis_ea3;
			cuMesh.update_parameters_mcoarse(idx, *cuMesh.pMs_AFM, Ms_AFM, *cuMesh.pK1_AFM, K1_AFM, *cuMesh.pK2_AFM, K2_AFM, *cuMesh.pmcanis_ea1, mcanis_ea1, *cuMesh.pmcanis_ea2, mcanis_ea2, *cuMesh.pmcanis_ea3, mcanis_ea3);

			//calculate m.ea1, m.ea2 and m.ea3 dot products
			cuBReal d1 = (M[idx] * mcanis_ea1) / Ms_AFM.i;
			cuBReal d2 = (M[idx] * mcanis_ea2) / Ms_AFM.i;
			cuBReal d3 = (M[idx] * mcanis_ea3) / Ms_AFM.i;

			//terms for K1 contribution
			cuBReal a1 = d1 * (d2*d2 + d3 * d3);
			cuBReal a2 = d2 * (d1*d1 + d3 * d3);
			cuBReal a3 = d3 * (d1*d1 + d2 * d2);

			//terms for K2 contribution
			cuBReal d123 = d1*d2*d3;

			cuBReal b1 = d123 * d2*d3;
			cuBReal b2 = d123 * d1*d3;
			cuBReal b3 = d123 * d1*d2;

			//update effective field with the anisotropy field
			Heff_value = cuReal3(
				(-2 * K1_AFM.i / ((cuBReal)MU0*Ms_AFM.i)) * (mcanis_ea1.i * a1 + mcanis_ea2.i * a2 + mcanis_ea3.i * a3)
				+ (-2 * K2_AFM.i / ((cuBReal)MU0*Ms_AFM.i)) * (mcanis_ea1.i * b1 + mcanis_ea2.i * b2 + mcanis_ea3.i * b3),

				(-2 * K1_AFM.i / ((cuBReal)MU0*Ms_AFM.i)) * (mcanis_ea1.j * a1 + mcanis_ea2.j * a2 + mcanis_ea3.j * a3)
				+ (-2 * K2_AFM.i / ((cuBReal)MU0*Ms_AFM.i)) * (mcanis_ea1.j * b1 + mcanis_ea2.j * b2 + mcanis_ea3.j * b3),

				(-2 * K1_AFM.i / ((cuBReal)MU0*Ms_AFM.i)) * (mcanis_ea1.k * a1 + mcanis_ea2.k * a2 + mcanis_ea3.k * a3)
				+ (-2 * K2_AFM.i / ((cuBReal)MU0*Ms_AFM.i)) * (mcanis_ea1.k * b1 + mcanis_ea2.k * b2 + mcanis_ea3.k * b3)
			);

			//same thing for sub-lattice B

			cuBReal d1B = (M2[idx] * mcanis_ea1) / Ms_AFM.j;
			cuBReal d2B = (M2[idx] * mcanis_ea2) / Ms_AFM.j;
			cuBReal d3B = (M2[idx] * mcanis_ea3) / Ms_AFM.j;

			cuBReal a1B = d1B * (d2B*d2B + d3B*d3B);
			cuBReal a2B = d2B * (d1B*d1B + d3B*d3B);
			cuBReal a3B = d3B * (d1B*d1B + d2B*d2B);

			cuBReal d123B = d1B*d2B*d3B;

			cuBReal b1B = d123B*d2B*d3B;
			cuBReal b2B = d123B*d1B*d3B;
			cuBReal b3B = d123B*d1B*d2B;

			Heff2_value = cuReal3(
				(-2 * K1_AFM.j / ((cuBReal)MU0*Ms_AFM.j)) * (mcanis_ea1.i * a1B + mcanis_ea2.i * a2B + mcanis_ea3.i * a3B)
				+ (-2 * K2_AFM.j / ((cuBReal)MU0*Ms_AFM.j)) * (mcanis_ea1.i * b1B + mcanis_ea2.i * b2B + mcanis_ea3.i * b3B),

				(-2 * K1_AFM.j / ((cuBReal)MU0*Ms_AFM.j)) * (mcanis_ea1.j * a1B + mcanis_ea2.j * a2B + mcanis_ea3.j * a3B)
				+ (-2 * K2_AFM.j / ((cuBReal)MU0*Ms_AFM.j)) * (mcanis_ea1.j * b1B + mcanis_ea2.j * b2B + mcanis_ea3.j * b3B),

				(-2 * K1_AFM.j / ((cuBReal)MU0*Ms_AFM.j)) * (mcanis_ea1.k * a1B + mcanis_ea2.k * a2B + mcanis_ea3.k * a3B)
				+ (-2 * K2_AFM.j / ((cuBReal)MU0*Ms_AFM.j)) * (mcanis_ea1.k * b1B + mcanis_ea2.k * b2B + mcanis_ea3.k * b3B)
			);

			if (do_reduction) {

				//update energy (E/V)		
				int non_empty_cells = M.get_nonempty_cells();
				if (non_empty_cells) energy_ = ((K1_AFM.i * (d1*d1*d2*d2 + d1*d1*d3*d3 + d2*d2*d3*d3) + K2_AFM.i * d123*d123) + (K1_AFM.j * (d1B*d1B*d2B*d2B + d1B*d1B*d3B*d3B + d2B*d2B*d3B*d3B) + K2_AFM.j * d123B*d123B)) / (2*non_empty_cells);
			}

			if (do_reduction && cuModule.pModule_Heff->linear_size()) (*cuModule.pModule_Heff)[idx] = Heff_value;
			if (do_reduction && cuModule.pModule_Heff2->linear_size()) (*cuModule.pModule_Heff2)[idx] = Heff2_value;
			if (do_reduction && cuModule.pModule_energy->linear_size()) (*cuModule.pModule_energy)[idx] = K1_AFM.i * (d1*d1*d2*d2 + d1*d1*d3*d3 + d2*d2*d3*d3) + K2_AFM.i * d123*d123;
			if (do_reduction && cuModule.pModule_energy2->linear_size()) (*cuModule.pModule_energy2)[idx] = K1_AFM.j * (d1B*d1B*d2B*d2B + d1B*d1B*d3B*d3B + d2B*d2B*d3B*d3B) + K2_AFM.j * d123B*d123B;
		}

		Heff[idx] += Heff_value;
		Heff2[idx] += Heff2_value;
	}

	if (do_reduction) reduction_sum(0, 1, &energy_, *cuModule.penergy);
}

//----------------------- UpdateField LAUNCHER

void Anisotropy_CubicCUDA::UpdateField(void)
{
	if (pMeshCUDA->GetMeshType() == MESH_ANTIFERROMAGNETIC) {

		//anti-ferromagnetic mesh

		if (pMeshCUDA->CurrentTimeStepSolved()) {

			ZeroEnergy();

			for (mGPU.device_begin(); mGPU != mGPU.device_end(); mGPU++) {

				Anisotropy_CubicCUDA_AFM_UpdateField <<< (pMeshCUDA->M.device_size(mGPU) + CUDATHREADS) / CUDATHREADS, CUDATHREADS >>> 
					(pMeshCUDA->cuMesh.get_deviceobject(mGPU), cuModule.get_deviceobject(mGPU), true);
			}
		}
		else {

			for (mGPU.device_begin(); mGPU != mGPU.device_end(); mGPU++) {

				Anisotropy_CubicCUDA_AFM_UpdateField <<< (pMeshCUDA->M.device_size(mGPU) + CUDATHREADS) / CUDATHREADS, CUDATHREADS >>> 
					(pMeshCUDA->cuMesh.get_deviceobject(mGPU), cuModule.get_deviceobject(mGPU), false);
			}
		}
	}
	else {

		//ferromagnetic mesh

		if (pMeshCUDA->CurrentTimeStepSolved()) {

			ZeroEnergy();

			for (mGPU.device_begin(); mGPU != mGPU.device_end(); mGPU++) {

				Anisotropy_CubicCUDA_FM_UpdateField <<< (pMeshCUDA->M.device_size(mGPU) + CUDATHREADS) / CUDATHREADS, CUDATHREADS >>> 
					(pMeshCUDA->cuMesh.get_deviceobject(mGPU), cuModule.get_deviceobject(mGPU), true);
			}
		}
		else {

			for (mGPU.device_begin(); mGPU != mGPU.device_end(); mGPU++) {

				Anisotropy_CubicCUDA_FM_UpdateField <<< (pMeshCUDA->M.device_size(mGPU) + CUDATHREADS) / CUDATHREADS, CUDATHREADS >>>
					(pMeshCUDA->cuMesh.get_deviceobject(mGPU), cuModule.get_deviceobject(mGPU), false);
			}
		}
	}
}

#endif

#endif

//----------------------------------- MONTE-CARLO METHODS FOR ENERGY COMPUTATION

#if COMPILECUDA == 1 && MONTE_CARLO == 1

//Ferromagnetic
__device__ cuBReal ManagedMeshCUDA::Get_EnergyChange_FM_AnisotropyCubiCUDA(int spin_index, cuReal3 Mnew)
{
	cuVEC_VC<cuReal3>& M = *pM;

	cuBReal K1 = *pK1;
	cuBReal K2 = *pK2;
	cuBReal Ms = *pMs;
	cuReal3 mcanis_ea1 = *pmcanis_ea1;
	cuReal3 mcanis_ea2 = *pmcanis_ea2;
	cuReal3 mcanis_ea3 = *pmcanis_ea3;
	update_parameters_mcoarse(spin_index, *pMs, Ms, *pK1, K1, *pK2, K2, *pmcanis_ea1, mcanis_ea1, *pmcanis_ea2, mcanis_ea2, *pmcanis_ea3, mcanis_ea3);

	cuReal3 S = M[spin_index] / Ms;
	cuReal3 S_new = Mnew / Ms;

	//calculate m.ea1, m.ea2 and m.ea3 dot products
	cuBReal d1 = S * mcanis_ea1;
	cuBReal d2 = S * mcanis_ea2;
	cuBReal d3 = S * mcanis_ea3;
	cuBReal d123 = d1 * d2 * d3;

	if (Mnew != cuReal3()) {

		cuBReal d1_new = S_new * mcanis_ea1;
		cuBReal d2_new = S_new * mcanis_ea2;
		cuBReal d3_new = S_new * mcanis_ea3;
		cuBReal d123_new = d1_new * d2_new * d3_new;

		return M.h.dim() * (
			(K1 * (d1_new * d1_new * d2_new * d2_new + d1_new * d1_new * d3_new * d3_new + d2_new * d2_new * d3_new * d3_new) + K2 * d123_new * d123_new)
			- (K1 * (d1 * d1 * d2 * d2 + d1 * d1 * d3 * d3 + d2 * d2 * d3 * d3) + K2 * d123 * d123));
	}
	else return M.h.dim() * (K1 * (d1 * d1 * d2 * d2 + d1 * d1 * d3 * d3 + d2 * d2 * d3 * d3) + K2 * d123 * d123);
}

//Antiferromagnetic
__device__ cuReal2 ManagedMeshCUDA::Get_EnergyChange_AFM_AnisotropyCubiCUDA(int spin_index, cuReal3 Mnew_A, cuReal3 Mnew_B)
{
	cuVEC_VC<cuReal3>& M = *pM;
	cuVEC_VC<cuReal3>& M2 = *pM2;

	cuReal2 Ms_AFM = *pMs_AFM;
	cuReal2 K1_AFM = *pK1_AFM;
	cuReal2 K2_AFM = *pK2_AFM;
	cuReal3 mcanis_ea1 = *pmcanis_ea1;
	cuReal3 mcanis_ea2 = *pmcanis_ea2;
	cuReal3 mcanis_ea3 = *pmcanis_ea3;
	update_parameters_mcoarse(spin_index, *pMs_AFM, Ms_AFM, *pK1_AFM, K1_AFM, *pK2_AFM, K2_AFM, *pmcanis_ea1, mcanis_ea1, *pmcanis_ea2, mcanis_ea2, *pmcanis_ea3, mcanis_ea3);

	//calculate m.ea1, m.ea2 and m.ea3 dot products
	cuBReal d1 = (M[spin_index] * mcanis_ea1) / Ms_AFM.i;
	cuBReal d2 = (M[spin_index] * mcanis_ea2) / Ms_AFM.i;
	cuBReal d3 = (M[spin_index] * mcanis_ea3) / Ms_AFM.i;
	cuBReal d123 = d1 * d2 * d3;

	//same thing for sub-lattice B

	cuBReal d1B = (M2[spin_index] * mcanis_ea1) / Ms_AFM.j;
	cuBReal d2B = (M2[spin_index] * mcanis_ea2) / Ms_AFM.j;
	cuBReal d3B = (M2[spin_index] * mcanis_ea3) / Ms_AFM.j;
	cuBReal d123B = d1B * d2B * d3B;

	cuBReal energyA = K1_AFM.i * (d1 * d1 * d2 * d2 + d1 * d1 * d3 * d3 + d2 * d2 * d3 * d3) + K2_AFM.i * d123 * d123;
	cuBReal energyB = K1_AFM.j * (d1B * d1B * d2B * d2B + d1B * d1B * d3B * d3B + d2B * d2B * d3B * d3B) + K2_AFM.j * d123B * d123B;

	if (Mnew_A != cuReal3() && Mnew_B != cuReal3()) {

		//calculate m.ea1, m.ea2 and m.ea3 dot products
		cuBReal d1new = (M[spin_index] * mcanis_ea1) / Ms_AFM.i;
		cuBReal d2new = (M[spin_index] * mcanis_ea2) / Ms_AFM.i;
		cuBReal d3new = (M[spin_index] * mcanis_ea3) / Ms_AFM.i;
		cuBReal d123new = d1new * d2new * d3new;

		//same thing for sub-lattice B

		cuBReal d1Bnew = (M2[spin_index] * mcanis_ea1) / Ms_AFM.j;
		cuBReal d2Bnew = (M2[spin_index] * mcanis_ea2) / Ms_AFM.j;
		cuBReal d3Bnew = (M2[spin_index] * mcanis_ea3) / Ms_AFM.j;
		cuBReal d123Bnew = d1Bnew * d2Bnew * d3Bnew;

		cuBReal energyAnew = K1_AFM.i * (d1new * d1new * d2new * d2new + d1new * d1new * d3new * d3new + d2new * d2new * d3new * d3new) + K2_AFM.i * d123new * d123new;
		cuBReal energyBnew = K1_AFM.j * (d1Bnew * d1Bnew * d2Bnew * d2Bnew + d1Bnew * d1Bnew * d3Bnew * d3Bnew + d2Bnew * d2Bnew * d3Bnew * d3Bnew) + K2_AFM.j * d123Bnew * d123Bnew;

		return M.h.dim() * cuReal2(energyAnew - energyA, energyBnew - energyB);
	}
	else return M.h.dim() * cuReal2(energyA, energyB);
}

#endif