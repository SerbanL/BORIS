#include "hip/hip_runtime.h"
#include "MElasticCUDA.h"

#if COMPILECUDA == 1

#ifdef MODULE_COMPILATION_MELASTIC

#include "Reduction.cuh"
#include "mcuVEC_halo.cuh"

#include "MeshDefs.h"

#include "ManagedDiffEqPolicyFMCUDA.h"
#include "ManagedDiffEqPolicyAFMCUDA.h"
#include "MeshParamsControlCUDA.h"

#include "MElastic_PolicyBoundariesCUDA.h"

//----------------------- Calculate_MElastic_Field KERNELS

__global__ void MElasticCUDA_Trigonal_UpdateField_FM(ManagedMeshCUDA& cuMesh, ManagedModulesCUDA& cuModule, bool do_reduction)
{
	cuVEC_VC<cuReal3>& M = *cuMesh.pM;
	cuVEC<cuReal3>& Heff = *cuMesh.pHeff;

	cuVEC_VC<cuReal3>& strain_diag = *cuMesh.pstrain_diag;
	cuVEC_VC<cuReal3>& strain_odiag = *cuMesh.pstrain_odiag;

	int idx = blockIdx.x * blockDim.x + threadIdx.x;

	cuBReal energy_ = 0.0;

	if (idx < Heff.linear_size()) {

		if (M.is_not_empty(idx)) {

			cuBReal Ms = *cuMesh.pMs;
			cuReal3 mcanis_ea1 = *cuMesh.pmcanis_ea1;
			cuReal3 mcanis_ea2 = *cuMesh.pmcanis_ea2;
			cuReal3 mcanis_ea3 = *cuMesh.pmcanis_ea3;
			cuReal2 MEc = *cuMesh.pMEc;
			cuReal2 MEc2 = *cuMesh.pMEc2;
			cuReal2 MEc3 = *cuMesh.pMEc3;
			cuMesh.update_parameters_mcoarse(idx, *cuMesh.pMs, Ms, *cuMesh.pMEc, MEc, *cuMesh.pMEc2, MEc2, *cuMesh.pMEc3, MEc3, *cuMesh.pmcanis_ea1, mcanis_ea1, *cuMesh.pmcanis_ea2, mcanis_ea2, *cuMesh.pmcanis_ea3, mcanis_ea3);
			
			cuBReal& B21 = MEc.i;
			cuBReal& B22 = MEc.j;
			cuBReal& B3 = MEc2.i;
			cuBReal& B4 = MEc2.j;
			cuBReal& B14 = MEc3.i;
			cuBReal& B34 = MEc3.j;
			
			cuReal3 position = M.cellidx_to_position(idx);
			//xx, yy, zz
			cuReal3 Sd = strain_diag[position];
			//yz, xz, xy
			cuReal3 Sod = strain_odiag[position];

			//normalised magnetization
			//Magneto-elastic term here applicable for a cubic crystal. We use the mcanis_ea1 and mcanis_ea2 axes to fix the cubic lattice orientation, thus rotate the m, Sd and Sod vectors.

			cuReal3 m = cuReal3(M[idx] * mcanis_ea1, M[idx] * mcanis_ea2, M[idx] * mcanis_ea3) / Ms;
			Sd = cuReal3(Sd * mcanis_ea1, Sd * mcanis_ea2, Sd * mcanis_ea3);
			Sod = cuReal3(Sod * mcanis_ea1, Sod * mcanis_ea2, Sod * mcanis_ea3);

			cuReal3 Hmel = cuReal3();
			Hmel += 2 * B21*(Sd.x + Sd.y)*m.z*mcanis_ea3 + 2 * B22*Sd.z*m.z*mcanis_ea3;
			Hmel += B3 * (Sd.x - Sd.y) * (m.x*mcanis_ea1 - m.y*mcanis_ea2) + 2 * B3 * Sod.z * (m.x*mcanis_ea2 + m.y*mcanis_ea1);
			Hmel += 2 * B4*Sod.y*(m.x*mcanis_ea3 + m.z*mcanis_ea1) + 2 * B4*Sod.x * (m.y*mcanis_ea3 + m.z*mcanis_ea2);
			Hmel += 2 * B14*Sod.x*(m.x*mcanis_ea1 - m.y*mcanis_ea2) + 2 * B14*Sod.y*(m.x*mcanis_ea2 + m.y*mcanis_ea1);
			Hmel += B34 * (Sd.x - Sd.y) * (m.y*mcanis_ea3 + m.z*mcanis_ea2) + 2 * B34*Sod.z*(m.x*mcanis_ea3 + m.z*mcanis_ea1);
			Hmel *= -1 / ((cuBReal)MU0 * Ms);

			Heff[idx] += Hmel;

			if (do_reduction) {

				int non_empty_cells = M.get_nonempty_cells();
				if (non_empty_cells) energy_ = -(cuBReal)MU0 * M[idx] * Hmel / (2 * non_empty_cells);
			}

			if (do_reduction && cuModule.pModule_Heff->linear_size()) (*cuModule.pModule_Heff)[idx] = Hmel;
			if (do_reduction && cuModule.pModule_energy->linear_size()) (*cuModule.pModule_energy)[idx] = -(cuBReal)MU0 * M[idx] * Hmel / 2;
		}
	}

	if (do_reduction) reduction_sum(0, 1, &energy_, *cuModule.penergy);
}

__global__ void MElasticCUDA_Trigonal_UpdateField_AFM(ManagedMeshCUDA& cuMesh, ManagedModulesCUDA& cuModule, bool do_reduction)
{
	cuVEC_VC<cuReal3>& M = *cuMesh.pM;
	cuVEC_VC<cuReal3>& M2 = *cuMesh.pM2;
	cuVEC<cuReal3>& Heff = *cuMesh.pHeff;
	cuVEC<cuReal3>& Heff2 = *cuMesh.pHeff2;

	cuVEC_VC<cuReal3>& strain_diag = *cuMesh.pstrain_diag;
	cuVEC_VC<cuReal3>& strain_odiag = *cuMesh.pstrain_odiag;

	int idx = blockIdx.x * blockDim.x + threadIdx.x;

	cuBReal energy_ = 0.0;

	if (idx < Heff.linear_size()) {

		if (M.is_not_empty(idx)) {

			cuReal2 Ms_AFM = *cuMesh.pMs_AFM;
			cuReal3 mcanis_ea1 = *cuMesh.pmcanis_ea1;
			cuReal3 mcanis_ea2 = *cuMesh.pmcanis_ea2;
			cuReal3 mcanis_ea3 = *cuMesh.pmcanis_ea3;
			cuReal2 MEc = *cuMesh.pMEc;
			cuReal2 MEc2 = *cuMesh.pMEc2;
			cuReal2 MEc3 = *cuMesh.pMEc3;
			cuMesh.update_parameters_mcoarse(idx, *cuMesh.pMs_AFM, Ms_AFM, *cuMesh.pMEc, MEc, *cuMesh.pMEc2, MEc2, *cuMesh.pMEc3, MEc3, *cuMesh.pmcanis_ea1, mcanis_ea1, *cuMesh.pmcanis_ea2, mcanis_ea2, *cuMesh.pmcanis_ea3, mcanis_ea3);

			cuBReal& B21 = MEc.i;
			cuBReal& B22 = MEc.j;
			cuBReal& B3 = MEc2.i;
			cuBReal& B4 = MEc2.j;
			cuBReal& B14 = MEc3.i;
			cuBReal& B34 = MEc3.j;

			cuReal3 position = M.cellidx_to_position(idx);
			//xx, yy, zz
			cuReal3 Sd = strain_diag[position];
			//yz, xz, xy
			cuReal3 Sod = strain_odiag[position];

			//normalised magnetization
			//Magneto-elastic term here applicable for a cubic crystal. We use the mcanis_ea1 and mcanis_ea2 axes to fix the cubic lattice orientation, thus rotate the m, Sd and Sod vectors.

			cuReal3 mA = cuReal3(M[idx] * mcanis_ea1, M[idx] * mcanis_ea2, M[idx] * mcanis_ea3) / Ms_AFM.i;
			cuReal3 mB = cuReal3(M2[idx] * mcanis_ea1, M2[idx] * mcanis_ea2, M2[idx] * mcanis_ea3) / Ms_AFM.j;

			Sd = cuReal3(Sd * mcanis_ea1, Sd * mcanis_ea2, Sd * mcanis_ea3);
			Sod = cuReal3(Sod * mcanis_ea1, Sod * mcanis_ea2, Sod * mcanis_ea3);

			cuReal3 Hmel_A = cuReal3();
			Hmel_A += 2 * B21*(Sd.x + Sd.y)*mA.z*mcanis_ea3 + 2 * B22*Sd.z*mA.z*mcanis_ea3;
			Hmel_A += B3 * (Sd.x - Sd.y) * (mA.x*mcanis_ea1 - mA.y*mcanis_ea2) + 2 * B3 * Sod.z * (mA.x*mcanis_ea2 + mA.y*mcanis_ea1);
			Hmel_A += 2 * B4*Sod.y*(mA.x*mcanis_ea3 + mA.z*mcanis_ea1) + 2 * B4*Sod.x * (mA.y*mcanis_ea3 + mA.z*mcanis_ea2);
			Hmel_A += 2 * B14*Sod.x*(mA.x*mcanis_ea1 - mA.y*mcanis_ea2) + 2 * B14*Sod.y*(mA.x*mcanis_ea2 + mA.y*mcanis_ea1);
			Hmel_A += B34 * (Sd.x - Sd.y) * (mA.y*mcanis_ea3 + mA.z*mcanis_ea2) + 2 * B34*Sod.z*(mA.x*mcanis_ea3 + mA.z*mcanis_ea1);
			Hmel_A *= -1 / ((cuBReal)MU0 * Ms_AFM.i);

			cuReal3 Hmel_B = cuReal3();
			Hmel_B += 2 * B21*(Sd.x + Sd.y)*mB.z*mcanis_ea3 + 2 * B22*Sd.z*mB.z*mcanis_ea3;
			Hmel_B += B3 * (Sd.x - Sd.y) * (mB.x*mcanis_ea1 - mB.y*mcanis_ea2) + 2 * B3 * Sod.z * (mB.x*mcanis_ea2 + mB.y*mcanis_ea1);
			Hmel_B += 2 * B4*Sod.y*(mB.x*mcanis_ea3 + mB.z*mcanis_ea1) + 2 * B4*Sod.x * (mB.y*mcanis_ea3 + mB.z*mcanis_ea2);
			Hmel_B += 2 * B14*Sod.x*(mB.x*mcanis_ea1 - mB.y*mcanis_ea2) + 2 * B14*Sod.y*(mB.x*mcanis_ea2 + mB.y*mcanis_ea1);
			Hmel_B += B34 * (Sd.x - Sd.y) * (mB.y*mcanis_ea3 + mB.z*mcanis_ea2) + 2 * B34*Sod.z*(mB.x*mcanis_ea3 + mB.z*mcanis_ea1);
			Hmel_B *= -1 / ((cuBReal)MU0 * Ms_AFM.j);

			Heff[idx] += Hmel_A;
			Heff2[idx] += Hmel_B;

			if (do_reduction) {

				int non_empty_cells = M.get_nonempty_cells();
				if (non_empty_cells) energy_ = -(cuBReal)MU0 * (M[idx] * Hmel_A + M2[idx] * Hmel_B) / (2 * non_empty_cells);
			}

			if (do_reduction && cuModule.pModule_Heff->linear_size()) (*cuModule.pModule_Heff)[idx] = Hmel_A;
			if (do_reduction && cuModule.pModule_Heff2->linear_size()) (*cuModule.pModule_Heff2)[idx] = Hmel_B;
			if (do_reduction && cuModule.pModule_energy->linear_size()) (*cuModule.pModule_energy)[idx] = -(cuBReal)MU0 * M[idx] * Hmel_A / 2;
			if (do_reduction && cuModule.pModule_energy2->linear_size()) (*cuModule.pModule_energy2)[idx] = -(cuBReal)MU0 * M2[idx] * Hmel_B / 2;
		}
	}

	if (do_reduction) reduction_sum(0, 1, &energy_, *cuModule.penergy);
}

//----------------------- Calculate_MElastic_Field LAUNCHER

//compute magnetoelastic effective field to use in magnetization equation.
void MElasticCUDA::Calculate_MElastic_Field_Trigonal(void)
{
	//disabled by setting magnetoelastic coefficient to zero (also disabled in non-magnetic meshes)
	if (melastic_field_disabled) return;

	ZeroEnergy();

	if (pMeshCUDA->GetMeshType() == MESH_ANTIFERROMAGNETIC) {

		//anti-ferromagnetic mesh

		if (pMeshCUDA->CurrentTimeStepSolved()) {

			for (mGPU.device_begin(); mGPU != mGPU.device_end(); mGPU++) {

				MElasticCUDA_Trigonal_UpdateField_AFM <<< (pMeshCUDA->M.device_size(mGPU) + CUDATHREADS) / CUDATHREADS, CUDATHREADS >>>
					(pMeshCUDA->cuMesh.get_deviceobject(mGPU), cuModule.get_deviceobject(mGPU), true);
			}
		}
		else {

			for (mGPU.device_begin(); mGPU != mGPU.device_end(); mGPU++) {

				MElasticCUDA_Trigonal_UpdateField_AFM <<< (pMeshCUDA->M.device_size(mGPU) + CUDATHREADS) / CUDATHREADS, CUDATHREADS >>>
					(pMeshCUDA->cuMesh.get_deviceobject(mGPU), cuModule.get_deviceobject(mGPU), false);
			}
		}
	}
	else if (pMeshCUDA->GetMeshType() == MESH_FERROMAGNETIC) {

		//ferromagnetic mesh

		if (pMeshCUDA->CurrentTimeStepSolved()) {

			for (mGPU.device_begin(); mGPU != mGPU.device_end(); mGPU++) {

				MElasticCUDA_Trigonal_UpdateField_FM <<< (pMeshCUDA->M.device_size(mGPU) + CUDATHREADS) / CUDATHREADS, CUDATHREADS >>>
					(pMeshCUDA->cuMesh.get_deviceobject(mGPU), cuModule.get_deviceobject(mGPU), true);
			}
		}
		else {

			for (mGPU.device_begin(); mGPU != mGPU.device_end(); mGPU++) {

				MElasticCUDA_Trigonal_UpdateField_FM <<< (pMeshCUDA->M.device_size(mGPU) + CUDATHREADS) / CUDATHREADS, CUDATHREADS >>>
					(pMeshCUDA->cuMesh.get_deviceobject(mGPU), cuModule.get_deviceobject(mGPU), false);
			}
		}
	}
}

//----------------------------------------------- Computational Helpers

//----- Velocity

__global__ void Iterate_Elastic_Solver_Velocity2_Kernel(
	ManagedMeshCUDA& cuMesh,
	MElastic_BoundaryCUDA** pexternal_stress_surfaces, size_t num_surfaces,
	cuVEC_VC<cuBReal>& vx2, cuVEC_VC<cuBReal>& vy2, cuVEC_VC<cuBReal>& vz2,
	cuVEC_VC<cuReal3>& sdd2,
	cuVEC_VC<cuBReal>& sxy2, cuVEC_VC<cuBReal>& sxz2, cuVEC_VC<cuBReal>& syz2,
	cuBReal time, cuBReal dT)
{
	cuVEC_VC<cuReal3>& u_disp = *cuMesh.pu_disp;
	cuVEC_VC<cuReal3>& strain_diag = *cuMesh.pstrain_diag;

	cuReal3& h_m = u_disp.h;
	cuSZ3& n_m = u_disp.n;

	int idx = blockIdx.x * blockDim.x + threadIdx.x;

	//kernel launched with size sdd.device_size(mGPU). For a single GPU this has (n_m.i + 1) * (n_m.j + 1) * (n_m.k + 1) cells.
	//for multiple GPUs only last one has an extra cell along partition dimension, the other ones have same dimension as u_disp along partition dimension
	//this means the ijk index can always be used for reading and writing, but with +/-1 along partition direction need to use the () operator to read values
	int i = idx % syz2.n.i;
	int j = (idx / syz2.n.i) % syz2.n.j;
	int k = idx / (syz2.n.i * syz2.n.j);

	if (idx < syz2.n.dim()) {

		//convert vertex index to cell-center index by capping maximum index size (use this to index u_disp)
		cuINT3 ijk_u = cuINT3(i < n_m.i ? i : n_m.i - 1, j < n_m.j ? j : n_m.j - 1, k < n_m.k ? k : n_m.k - 1);
		int idx_u = ijk_u.i + ijk_u.j * n_m.x + ijk_u.k * n_m.x * n_m.y;

		cuBReal density = *cuMesh.pdensity;
		cuBReal mdamping = *cuMesh.pmdamping;
		cuMesh.update_parameters_scoarse(idx_u, *cuMesh.pdensity, density, *cuMesh.pmdamping, mdamping);

		cuINT3 ijk = cuINT3(i, j, k);

		//external forces on different faces (keep track separately in case an edge cell is excited simultaneously by 2 or more external forces
		cuReal3 Fext_xface = cuReal3(), Fext_yface = cuReal3(), Fext_zface = cuReal3();

		//is there an external force? If so, get it, otherwise it will be zero
		if (
			((i == 0 || i == n_m.i) && strain_diag.is_dirichlet_x(idx_u)) ||
			((j == 0 || j == n_m.j) && strain_diag.is_dirichlet_y(idx_u)) ||
			((k == 0 || k == n_m.k) && strain_diag.is_dirichlet_z(idx_u))) {

			//search through all available surfaces to get external force
			for (int sidx = 0; sidx < num_surfaces; sidx++) {

				int orientation = pexternal_stress_surfaces[sidx]->contains(ijk_u);
				if (orientation) {

					switch (abs(orientation)) {

						//x face
					case 1:
						Fext_xface = pexternal_stress_surfaces[sidx]->get_ext_force_edges(ijk, time);
						break;

						//y face
					case 2:
						Fext_yface = pexternal_stress_surfaces[sidx]->get_ext_force_edges(ijk, time);
						break;

						//z face
					case 3:
						Fext_zface = pexternal_stress_surfaces[sidx]->get_ext_force_edges(ijk, time);
						break;
					};
				}
			}
		}

		//update vx2
		if (i < n_m.i && j < n_m.j && k < n_m.k) {

			if (u_disp.is_not_empty(idx_u)) {

				cuBReal dsxx2_dx = (sdd2(cuINT3(i + 1, j, k)).x - sdd2[ijk].x) / h_m.x;
				cuBReal dsxy2_dy = (sxy2(cuINT3(i, j + 1, k)) - sxy2[ijk]) / h_m.y;
				cuBReal dsxz2_dz = (sxz2(cuINT3(i, j, k + 1)) - sxz2[ijk]) / h_m.z;

				vx2[ijk] += dT * (dsxx2_dx + dsxy2_dy + dsxz2_dz - mdamping * vx2[ijk]) / density;
			}
			else vx2[ijk] = 0.0;
		}

		//update vy2
		if (k < n_m.k) {

			//set zero at fixed faces (for vy2 only x and y faces are applicable)
			if (((i == 0 || i == n_m.i) && u_disp.is_dirichlet_x(idx_u)) || ((j == 0 || j == n_m.j) && u_disp.is_dirichlet_y(idx_u))) {

				vy2[ijk] = 0.0;
			}
			else {

				int niend = (i < n_m.i);
				int njend = (j < n_m.j);

				//check for required axis normal faces being present
				bool yface_u = i < n_m.i && u_disp.is_face_y(idx_u);
				bool yface_l = (i > 0 && u_disp.is_face_y(idx_u - niend)) || (i == 0 && (u_disp.is_halo_nx(idx_u) || (j > 0 && u_disp.is_halo_nx(idx_u - njend * n_m.x))));
				bool xface_u = j < n_m.j && u_disp.is_face_x(idx_u);
				bool xface_l = j > 0 && u_disp.is_face_x(idx_u - njend * n_m.x);

				//at least one face is required, otherwise velocity must be zero
				if (yface_u || yface_l || xface_u || xface_l) {

					cuBReal dsxy2_dx = 0.0, dsyy2_dy = 0.0, dsyz2_dz = 0.0;

					//always interior
					dsyz2_dz = (syz2(cuINT3(i, j, k + 1)) - syz2[ijk]) / h_m.z;

					//interior
					if (yface_u && yface_l) dsxy2_dx = (sxy2[ijk] - sxy2(cuINT3(i - 1, j, k))) / h_m.x;
					else if (yface_l) dsxy2_dx = (Fext_xface.y - sxy2(cuINT3(i - 1, j, k))) / (h_m.x / 2);
					else if (yface_u) dsxy2_dx = (sxy2[ijk] - Fext_xface.y) / (h_m.x / 2);

					//interior
					if (xface_u && xface_l) dsyy2_dy = (sdd2[ijk].y - sdd2(cuINT3(i, j - 1, k)).y) / h_m.y;
					else if (xface_l) dsyy2_dy = (Fext_yface.y - sdd2(cuINT3(i, j - 1, k)).y) / (h_m.y / 2);
					else if (xface_u) dsyy2_dy = (sdd2[ijk].y - Fext_yface.y) / (h_m.y / 2);

					vy2[ijk] += dT * (dsxy2_dx + dsyy2_dy + dsyz2_dz - mdamping * vy2[ijk]) / density;
				}
				else vy2[ijk] = 0.0;
			}
		}

		//update vz2
		if (j < n_m.j) {

			//set zero at fixed faces (for vz2 only x and z faces are applicable)
			if (((i == 0 || i == n_m.i) && u_disp.is_dirichlet_x(idx_u)) || ((k == 0 || k == n_m.k) && u_disp.is_dirichlet_z(idx_u))) {

				vz2[ijk] = 0.0;
			}
			else {

				int niend = (i < n_m.i);
				int nkend = (k < n_m.k);

				//check for required axis normal faces being present
				bool zface_u = i < n_m.i && u_disp.is_face_z(idx_u);
				bool zface_l = (i > 0 && u_disp.is_face_z(idx_u - niend)) || (i == 0 && (u_disp.is_halo_nx(idx_u) || (k > 0 && u_disp.is_halo_nx(idx_u - nkend * n_m.x * n_m.y))));
				bool xface_u = k < n_m.k && u_disp.is_face_x(idx_u);
				bool xface_l = k > 0 && u_disp.is_face_x(idx_u - nkend * n_m.x * n_m.y);

				//at least one face is required, otherwise velocity must be zero
				if (zface_u || zface_l || xface_u || xface_l) {

					cuBReal dsxz2_dx = 0.0, dsyz2_dy = 0.0, dszz2_dz = 0.0;

					//always interior
					dsyz2_dy = (syz2(cuINT3(i, j + 1, k)) - syz2[ijk]) / h_m.y;

					//interior
					if (zface_u && zface_l) dsxz2_dx = (sxz2[ijk] - sxz2(cuINT3(i - 1, j, k))) / h_m.x;
					else if (zface_l) dsxz2_dx = (Fext_xface.z - sxz2(cuINT3(i - 1, j, k))) / (h_m.x / 2);
					else if (zface_u) dsxz2_dx = (sxz2[ijk] - Fext_xface.z) / (h_m.x / 2);

					//interior
					if (xface_u && xface_l) dszz2_dz = (sdd2[ijk].z - sdd2(cuINT3(i, j, k - 1)).z) / h_m.z;
					else if (xface_l) dszz2_dz = (Fext_zface.z - sdd2(cuINT3(i, j, k - 1)).z) / (h_m.z / 2);
					else if (xface_u) dszz2_dz = (sdd2[ijk].z - Fext_zface.z) / (h_m.z / 2);

					vz2[ijk] += dT * (dsxz2_dx + dsyz2_dy + dszz2_dz - mdamping * vz2[ijk]) / density;
				}
				else vz2[ijk] = 0.0;
			}
		}
	}
}

//----------------------- Iterate_Elastic_Solver LAUNCHERS

//update velocity for dT time increment (also updating displacement)
void MElasticCUDA::Iterate_Elastic_Solver_Velocity2(double dT)
{
	//use sdd device dimensions, since this has total size (pMeshCUDA->n_m.i + 1) * (pMeshCUDA->n_m.j + 1) * (pMeshCUDA->n_m.k + 1)

	sdd2.exchange_halos();
	sxy2.exchange_halos();
	sxz2.exchange_halos();
	syz2.exchange_halos();

	//1a. Update velocity
	for (mGPU.device_begin(); mGPU != mGPU.device_end(); mGPU++) {

		Iterate_Elastic_Solver_Velocity2_Kernel <<< (sdd.device_size(mGPU) + CUDATHREADS) / CUDATHREADS, CUDATHREADS >>>
			(pMeshCUDA->cuMesh.get_deviceobject(mGPU), 
			external_stress_surfaces_arr(mGPU), external_stress_surfaces.size(),
			vx2.get_deviceobject(mGPU), vy2.get_deviceobject(mGPU), vz2.get_deviceobject(mGPU), 
			sdd2.get_deviceobject(mGPU), sxy2.get_deviceobject(mGPU), sxz2.get_deviceobject(mGPU), syz2.get_deviceobject(mGPU),
			pMeshCUDA->GetStageTime(), dT);
	}
}

//----------------------- Iterate_Elastic_Solver KERNELS

__device__ void Iterate_Elastic_Solver_Stress_Trigonal_CUDA(
	cuINT3 ijk, cuINT3 ijk_u, int idx_u,
	ManagedMeshCUDA& cuMesh,
	MElastic_BoundaryCUDA** pexternal_stress_surfaces, size_t num_surfaces,
	cuVEC_VC<cuBReal>& vx, cuVEC_VC<cuBReal>& vy, cuVEC_VC<cuBReal>& vz,
	cuVEC_VC<cuReal3>& sdd, cuVEC_VC<cuBReal>& sxy, cuVEC_VC<cuBReal>& sxz, cuVEC_VC<cuBReal>& syz,
	cuVEC_VC<cuBReal>& vx2, cuVEC_VC<cuBReal>& vy2, cuVEC_VC<cuBReal>& vz2,
	cuVEC_VC<cuReal3>& sdd2, cuVEC_VC<cuBReal>& sxy2, cuVEC_VC<cuBReal>& sxz2, cuVEC_VC<cuBReal>& syz2,
	cuBReal time, cuBReal dT,
	bool thermoelasticity_enabled,
	cuVEC<cuBReal>& Temp_previous, cuBReal magnetic_dT,
	cuReal3 dsdd_dt_ms, cuReal3 dsod_dt_ms)
{
	cuVEC_VC<cuReal3>& u_disp = *cuMesh.pu_disp;
	cuVEC_VC<cuReal3>& strain_diag = *cuMesh.pstrain_diag;

	cuReal3& h_m = u_disp.h;
	cuSZ3& n_m = u_disp.n;

	cuReal3 cC = *cuMesh.pcC;
	cuReal3 cC3 = *cuMesh.pcC3;
	cuReal3 cCs = *cuMesh.pcCs;
	cuMesh.update_parameters_scoarse(idx_u, *cuMesh.pcC, cC, *cuMesh.pcC3, cC3, *cuMesh.pcCs, cCs);

	//get named coefficients for clarity to avoid typing errors
	cuBReal& c11 = cC.i;
	cuBReal& c12 = cC.j;
	cuBReal& c44 = cC.k;
	cuBReal& c33 = cC3.i;
	cuBReal& c13 = cC3.j;
	cuBReal& c66 = cC3.k;
	cuBReal& c14 = cCs.i;

	cuBReal r12_11 = c12 / c11;
	cuBReal r13_11 = c13 / c11;
	cuBReal r14_11 = c14 / c11;
	cuBReal r13_33 = c13 / c33;
	cuBReal r14_44 = c14 / c44;
	cuBReal r14_66 = c14 / c66;
	cuBReal r11_44 = c11 / c44;
	cuBReal r12_44 = c12 / c44;
	cuBReal r13_44 = c13 / c44;

	int& i = ijk.i;
	int& j = ijk.j;
	int& k = ijk.k;

	///////////// THERMOELASTICITY CONTRIBUTION

	//needed for thermoelasticity (includes time derivative of temperature)
	cuBReal dsxx_yy_dt_te = 0.0;
	cuBReal dszz_dt_te = 0.0;
	if (thermoelasticity_enabled) {

		cuVEC_VC<cuBReal>& Temp = *cuMesh.pTemp;
		cuVEC_VC<cuBReal>& Temp_l = *cuMesh.pTemp_l;

		int idx_T = Temp.position_to_cellidx(u_disp.cellidx_to_position(idx_u));

		if (Temp.is_not_empty(idx_T)) {

			cuBReal thalpha = *cuMesh.pthalpha;
			cuMesh.update_parameters_scoarse(idx_u, *cuMesh.pthalpha, thalpha);

			cuBReal Temperature = 0.0;
			//for 2TM we need to use the lattice temperature
			if (Temp_l.linear_size()) Temperature = Temp_l[idx_T];
			else Temperature = Temp[idx_T];

			dsxx_yy_dt_te = (c11 + c12 + c13) * thalpha * (Temperature - Temp_previous[idx_T]) / magnetic_dT;
			dszz_dt_te = (c33 + 2 * c13) * thalpha * (Temperature - Temp_previous[idx_T]) / magnetic_dT;
		}
	}

	///////////// EXTERNAL FORCES

	//external forces on different faces (keep track separately in case an edge cell is excited simultaneously by 2 or more external forces
	cuReal3 Fext_xface = cuReal3(), Fext_yface = cuReal3(), Fext_zface = cuReal3();

	//time derivatives of forces on the different faces
	//first index is face normal, second index is force component. e.g. dFyz is for y-face, force z component.
	cuBReal dFxx = -dsdd_dt_ms.x + dsxx_yy_dt_te;
	cuBReal dFyy = -dsdd_dt_ms.y + dsxx_yy_dt_te;
	cuBReal dFzz = -dsdd_dt_ms.z + dszz_dt_te;
	cuBReal dFyz = -dsod_dt_ms.x;
	cuBReal dFzx = -dsod_dt_ms.y;
	cuBReal dFyx = -dsod_dt_ms.z;

	//is there an external force? If so, get it, otherwise it will be zero
	if (
		((i == 0 || i == n_m.i) && strain_diag.is_dirichlet_x(idx_u)) ||
		((j == 0 || j == n_m.j) && strain_diag.is_dirichlet_y(idx_u)) ||
		((k == 0 || k == n_m.k) && strain_diag.is_dirichlet_z(idx_u))) {

		//search through all available surfaces to get external force
		for (int sidx = 0; sidx < num_surfaces; sidx++) {

			int orientation = pexternal_stress_surfaces[sidx]->contains(ijk_u);
			if (orientation) {

				switch (abs(orientation)) {

					//x face
				case 1:
					Fext_xface = pexternal_stress_surfaces[sidx]->get_ext_force_vertices(ijk, time);
					break;

					//y face
				case 2:
					Fext_yface = pexternal_stress_surfaces[sidx]->get_ext_force_vertices(ijk, time);
					break;

					//z face
				case 3:
					Fext_zface = pexternal_stress_surfaces[sidx]->get_ext_force_vertices(ijk, time);
					break;
				};
			}
		}
	}

	//update sxx, syy, szz
	int niend = (i < n_m.i);
	int njend = (j < n_m.j);
	int nkend = (k < n_m.k);

	//check if required edges are present
	bool xedge_u = ijk.i < n_m.i && u_disp.is_edge_x(idx_u);
	bool xedge_l =
		(ijk.i > 0 && u_disp.is_edge_x(idx_u - niend)) ||
		(ijk.i == 0 &&
		(u_disp.is_halo_nx(idx_u) ||
		(ijk.k > 0 && u_disp.is_halo_nx(idx_u - nkend * n_m.x * n_m.y)) ||
		(ijk.j > 0 && u_disp.is_halo_nx(idx_u - njend * n_m.x)) ||
		(ijk.j > 0 && ijk.k > 0 && u_disp.is_halo_nx(idx_u - nkend * n_m.x * n_m.y - njend * n_m.x))));
	bool yedge_u = ijk.j < n_m.j && u_disp.is_edge_y(idx_u);
	bool yedge_l = ijk.j > 0 && u_disp.is_edge_y(idx_u - njend * n_m.x);
	bool zedge_u = ijk.k < n_m.k && u_disp.is_edge_z(idx_u);
	bool zedge_l = ijk.k > 0 && u_disp.is_edge_z(idx_u - nkend * n_m.x * n_m.y);

	//check for fixed faces at ends
	bool xfixed_l = (ijk.i == 0 && u_disp.is_dirichlet_px(idx_u));
	bool xfixed_u = (ijk.i == n_m.i && u_disp.is_dirichlet_nx(idx_u));
	bool yfixed_l = (ijk.j == 0 && u_disp.is_dirichlet_py(idx_u));
	bool yfixed_u = (ijk.j == n_m.j && u_disp.is_dirichlet_ny(idx_u));
	bool zfixed_l = (ijk.k == 0 && u_disp.is_dirichlet_pz(idx_u));
	bool zfixed_u = (ijk.k == n_m.k && u_disp.is_dirichlet_nz(idx_u));

	//dvx/dx at vertex
	cuBReal dvx_dx = 0.0;

	//interior
	if (xedge_u && xedge_l) dvx_dx = (vx[ijk] - vx(cuINT3(i - 1, j, k))) / h_m.x;
	//fixed face : Dirichlet value of zero for velocity derivative
	else if (xedge_l && xfixed_u) {

		dvx_dx = -vx(cuINT3(i - 1, j, k)) / (h_m.x / 2);
	}
	else if (xedge_u && xfixed_l) {

		dvx_dx = vx[ijk] / (h_m.x / 2);
	}
	//free x face
	else {

		//both side derivatives
		if (yedge_l && yedge_u && zedge_l && zedge_u) {

			//VERIFIED
			dvx_dx = (dFxx / c11 - r12_11 * (vy[ijk] - vy(cuINT3(i, j - 1, k))) / h_m.y - r13_11 * (vz[ijk] - vz(cuINT3(i, j, k - 1))) / h_m.z - r14_11 * ((vz2[ijk] - vz2(cuINT3(i, j - 1, k))) / h_m.y + (vy2[ijk] - vy2(cuINT3(i, j, k - 1))) / h_m.z));
		}
		//only z derivative
		else if (zedge_l && zedge_u) {

			//VERIFIED
			cuBReal rdiv = 1 - r12_11 * r12_11 - 4 * r14_11 * r14_44 * (1 + r12_11);
			dvx_dx = ((dFxx / c11) * (1 - 2 * r14_11 * r14_44) - (dFyy / c11) * (r12_11 + 2 * r14_11 * r14_44) - (dFyz / c44) * r14_11 * (1 + r12_11) - ((vz[ijk] - vz(cuINT3(i, j, k - 1))) / h_m.z) * r13_11 * (1 - r12_11 - 4 * r14_11 * r14_44)) / rdiv;
		}
		//only y derivative
		else if (yedge_l && yedge_u) {

			//VERIFIED
			dvx_dx = ((dFxx / c11) - (dFzz / c33) * r13_11 - (dFyz / c44) * r14_11 - ((vy[ijk] - vy(cuINT3(i, j - 1, k))) / h_m.y) * (r12_11 - r13_11 * r13_33 + 2 * r14_11 * r14_44)) / (1 - r13_11 * r13_33 - 2 * r14_11 * r14_44);
		}
		//no side derivatives : corner point. In this case all diagonal stress components set from external conditions, so derivatives not needed (set zero)
		else dvx_dx = 0.0;
	}

	//dvy/dy and dvz2_dy at vertex
	cuBReal dvy_dy = 0.0;
	cuBReal dvz2_dy = 0.0;

	//interior
	if (yedge_u && yedge_l) {

		dvy_dy = (vy[ijk] - vy(cuINT3(i, j - 1, k))) / h_m.y;
		dvz2_dy = (vz2[ijk] - vz2(cuINT3(i, j - 1, k))) / h_m.y;
	}
	//fixed face : Dirichlet value of zero for velocity derivative
	else if (yedge_l && yfixed_u) {

		dvy_dy = -vy(cuINT3(i, j - 1, k)) / (h_m.y / 2);
		dvz2_dy = -vz2(cuINT3(i, j - 1, k)) / (h_m.y / 2);
	}
	else if (yedge_u && yfixed_l) {

		dvy_dy = vy[ijk] / (h_m.y / 2);
		dvz2_dy = vz2[ijk] / (h_m.y / 2);
	}
	//free face
	else {

		//z derivative present (if x derivative not present, then dvx_dx is calculated above when only z derivative present, so no need to consider this again here)
		if (zedge_l && zedge_u) {

			//VERIFIED
			dvy_dy = ((dFyy / c11) + r14_11 * (dFyz / c44) - dvx_dx * (r12_11 + 2 * r14_11 * r14_44) - r13_11 * (vz[ijk] - vz(cuINT3(i, j, k - 1))) / h_m.z) / (1 - 2 * r14_11 * r14_44);
			//VERIFIED
			//now that we have dvy_dy. this can be used (together with dvx_dx) in formula for dvz2_dy, even if x derivative not present
			dvz2_dy = (dFyz / c44) - 2 * r14_44 * (dvx_dx - dvy_dy) - (vy2[ijk] - vy2(cuINT3(i, j, k - 1))) / h_m.z;
		}
		//only x derivative
		else if (xedge_l && xedge_u) {

			//VERIFIED
			dvy_dy = ((dFyy / c11) - r13_11 * (dFzz / c33) + r14_11 * (dFyz / c44) - dvx_dx * (r12_11 - r13_11 * r13_33 + 2 * r14_11 * r14_44)) / (1 - r13_11 * r13_33 - 2 * r14_11 * r14_44);

			//VERIFIED
			//need dvz2_dy also. In this case it's not possible to obtain it separately, but we only need (dvz2_dy + dvy2_dz), which is possible to obtain
			//this is : (dvz2_dy + dvy2_dz) = ((dFyy / c11) * 2 * r14_44 + (dFyz / c44) * (1 - r13_11 * r13_33) - (dFzz / c33) * 2 * r13_11*r14_44 - dvx_dx * 2 * r14_44 * (1 + r12_11 - 2 * r13_11*r13_33)) / (1 - r13_11*r13_33 - 2*r14_11*r14_44);
			//thus here set dvz2_dy as this value, and later when dvy2_dz is calculated below, set dvy2_dz to zero if only x derivative available
			dvz2_dy = ((dFyy / c11) * 2 * r14_44 + (dFyz / c44) * (1 - r13_11 * r13_33) - (dFzz / c33) * 2 * r13_11 * r14_44 - dvx_dx * 2 * r14_44 * (1 + r12_11 - 2 * r13_11 * r13_33)) / (1 - r13_11 * r13_33 - 2 * r14_11 * r14_44);

			//there are exceptions to this : if fixed z surface then dvy2_dz will be calculated using Dirichlet boundary condition, so use it here also
			if (zedge_u && zedge_l) {

				dvz2_dy -= (vy2[ijk] - vy2(cuINT3(i, j, k - 1))) / h_m.z;
			}
			else if (zedge_l && zfixed_u) {

				dvz2_dy -= -vy2(cuINT3(i, j, k - 1)) / (h_m.z / 2);
			}
		}
		//no side derivatives : corner point. In this case all diagonal stress components set from external conditions, so derivatives not needed (set zero)
		//similarly dvz2_dy set to zero since this is also used to obtain sig_yz, but at corner point this is set from external conditions
		else {

			dvy_dy = 0.0;
			dvz2_dy = 0.0;
		}
	}

	//dvz/dz and dyz2_dz at vertex
	cuBReal dvz_dz = 0.0;
	cuBReal dvy2_dz = 0.0;

	//interior
	if (zedge_u && zedge_l) {

		dvz_dz = (vz[ijk] - vz(cuINT3(i, j, k - 1))) / h_m.z;
		dvy2_dz = (vy2[ijk] - vy2(cuINT3(i, j, k - 1))) / h_m.z;
	}
	//fixed face : Dirichlet value of zero for velocity derivative
	else if (zedge_l && zfixed_u) {

		dvz_dz = -vz(cuINT3(i, j, k - 1)) / (h_m.z / 2);
		dvy2_dz = -vy2(cuINT3(i, j, k - 1)) / (h_m.z / 2);
	}
	//fixed face : Dirichlet value of zero for velocity derivative
	else if (zedge_u && zfixed_l) {

		dvz_dz = vz[ijk] / (h_m.z / 2);
		dvy2_dz = vy2[ijk] / (h_m.z / 2);
	}
	//free face
	else {

		//VERIFIED
		//don't need to check if x and y derivatives present, as these are calculated above when obtaining dvy/dy, dvx/dx and dvz/dy
		//if this is a corner point then derivative values won't matter as values at corner points set from external conditions
		dvz_dz = (dFzz / c33 - r13_33 * (dvx_dx + dvy_dy));

		//VERIFIED
		//y derivative present
		if (yedge_l && yedge_u) dvy2_dz = (dFyz / c44 - 2 * r14_44 * (dvx_dx - dvy_dy)) - dvz2_dy;
		//if no y derivative present, then either only x is present (in which case keep dvy2_dz as zero - see comments for dvz2_dy calculated above when only x derivative is present), 
		//or else this is a corner point and dvy2_dz is not required, so still zero.
	}

	//update sdd and syz2 if not empty
	if ((xedge_u || xedge_l) && (yedge_u || yedge_l) && (zedge_u || zedge_l)) {

		//update sdd
		if ((!xedge_u && !xfixed_u) || (!xedge_l && !xfixed_l)) sdd[ijk].x = Fext_xface.x;
		else sdd[ijk].x += dT * (c11 * dvx_dx + c12 * dvy_dy + c13 * dvz_dz + c14 * (dvz2_dy + dvy2_dz) - dsxx_yy_dt_te + dsdd_dt_ms.x);

		bool yfree = (!yedge_u && !yfixed_u) || (!yedge_l && !yfixed_l);
		bool zfree = (!zedge_u && !zfixed_u) || (!zedge_l && !zfixed_l);

		if (yfree) {

			sdd[ijk].y = Fext_yface.y;
			syz2[ijk] = Fext_yface.z;
		}
		else sdd[ijk].y += dT * (c11 * dvy_dy + c12 * dvx_dx + c13 * dvz_dz - c14 * (dvz2_dy + dvy2_dz) - dsxx_yy_dt_te + dsdd_dt_ms.y);

		if (zfree) {

			sdd[ijk].z = Fext_zface.z;
			syz2[ijk] = Fext_zface.y;
		}
		else sdd[ijk].z += dT * (c33 * dvz_dz + c13 * (dvx_dx + dvy_dy) - dszz_dt_te + dsdd_dt_ms.z);

		//update syz2 if we don't have free y or z faces (otherwise it is set from external conditions above)
		if (!yfree && !zfree) syz2[ijk] += dT * (c44 * (dvz2_dy + dvy2_dz) + 2 * c14 * (dvx_dx - dvy_dy) + dsod_dt_ms.x);
	}
	else {

		sdd[ijk] = cuReal3();
		syz2[ijk] = 0.0;
	}

	//update sxy and sxz2
	if (i < n_m.i && j < n_m.j) {

		bool zface = u_disp.is_face_z(idx_u);

		//both cells (distinct) present either side of the z face
		bool zstencil = k < n_m.z && u_disp.is_not_empty(idx_u) && k > 0 && u_disp.is_not_empty(idx_u - nkend * n_m.x * n_m.y);

		if (zface) {

			cuBReal dvx_dy = (vx(cuINT3(i, j + 1, k)) - vx[ijk]) / h_m.y;
			cuBReal dvy_dx = (vy(cuINT3(i + 1, j, k)) - vy[ijk]) / h_m.x;

			cuBReal dvz2_dx = (vz2(cuINT3(i + 1, j, k)) - vz2[ijk]) / h_m.x;
			cuBReal dvx2_dz = 0.0;

			//interior
			if (zstencil) dvx2_dz = (vx2[ijk] - vx2(cuINT3(i, j, k - 1))) / h_m.z;
			else {

				//fixed surfaces use Dirichlet
				if (zfixed_l) dvx2_dz = vx2[ijk] / (h_m.z / 2);
				else if (zfixed_u) dvx2_dz = -vx2(cuINT3(i, j, k - 1)) / (h_m.z / 2);
				//free surfaces
				else {

					//VERIFIED
					dvx2_dz = (dFzx / c44 - r14_44 * (dvy_dx + dvx_dy)) - dvz2_dx;
				}
			}

			sxy[ijk] += dT * (c66 * (dvx_dy + dvy_dx) + c14 * (dvz2_dx + dvx2_dz) + dsod_dt_ms.z);

			if (zstencil || zfixed_l || zfixed_u) sxz2[ijk] += dT * (c44 * (dvz2_dx + dvx2_dz) + c14 * (dvy_dx + dvx_dy) + dsod_dt_ms.y);
			//for free surface (z normal), set sxz2 directly from external condition
			else sxz2[ijk] = Fext_zface.x;
		}
		else {

			sxy[ijk] = 0.0;
			sxz2[ijk] = 0.0;
		}
	}

	//update sxz and sxy2
	if (i < n_m.i && k < n_m.k) {

		bool yface = u_disp.is_face_y(idx_u);

		//both cells (distinct) present either side of the y face
		bool ystencil = j < n_m.y && u_disp.is_not_empty(idx_u) && j > 0 && u_disp.is_not_empty(idx_u - njend * n_m.x);

		if (yface) {

			cuBReal dvx_dz = (vx(cuINT3(i, j, k + 1)) - vx[ijk]) / h_m.z;
			cuBReal dvz_dx = (vz(cuINT3(i + 1, j, k)) - vz[ijk]) / h_m.x;

			cuBReal dvy2_dx = (vy2(cuINT3(i + 1, j, k)) - vy2[ijk]) / h_m.x;
			cuBReal dvx2_dy = 0.0;

			//interior
			if (ystencil) dvx2_dy = (vx2[ijk] - vx2(cuINT3(i, j - 1, k))) / h_m.y;
			else {

				//fixed surfaces use Dirichlet
				if (yfixed_l) dvx2_dy = vx2[ijk] / (h_m.y / 2);
				else if (yfixed_u) dvx2_dy = -vx2(cuINT3(i, j - 1, k)) / (h_m.y / 2);
				//free surfaces
				else {

					//VERIFIED
					dvx2_dy = (dFyx / c66 - r14_66 * (dvz_dx + dvx_dz)) - dvy2_dx;
				}
			}

			sxz[ijk] += dT * (c44 * (dvx_dz + dvz_dx) + c14 * (dvy2_dx + dvx2_dy) + dsod_dt_ms.y);

			if (ystencil || yfixed_l || yfixed_u)  sxy2[ijk] += dT * (c66 * (dvy2_dx + dvx2_dy) + c14 * (dvz_dx + dvx_dz) + dsod_dt_ms.z);
			//for free surface (y normal), set sxy2 directly from external condition
			else sxy2[ijk] = Fext_yface.x;
		}
		else {

			sxz[ijk] = 0.0;
			sxy2[ijk] = 0.0;
		}
	}

	//update syz and sdd2
	if (j < n_m.j && k < n_m.k) {

		bool xface = u_disp.is_face_x(idx_u);

		//both cells (distinct) present either side of the x face
		bool xstencil = i < n_m.x && u_disp.is_not_empty(idx_u) && i > 0 && u_disp.is_not_empty(idx_u - niend);

		if (xface) {

			cuBReal dvy_dz = (vy(cuINT3(i, j, k + 1)) - vy[ijk]) / h_m.z;
			cuBReal dvz_dy = (vz(cuINT3(i, j + 1, k)) - vz[ijk]) / h_m.y;
			cuBReal dvz2_dz = (vz2(cuINT3(i, j, k + 1)) - vz2[ijk]) / h_m.z;

			cuBReal dvy2_dy = (vy2(cuINT3(i, j + 1, k)) - vy2[ijk]) / h_m.y;
			cuBReal dvx2_dx = 0.0;

			//interior
			if (xstencil) dvx2_dx = (vx2[ijk] - vx2(cuINT3(i - 1, j, k))) / h_m.x;
			else {

				//fixed surfaces use Dirichlet
				if (xfixed_l) dvx2_dx = vx2[ijk] / (h_m.x / 2);
				else if (xfixed_u) dvx2_dx = -vx2(cuINT3(i - 1, j, k)) / (h_m.x / 2);
				//free surfaces
				else {

					//VERIFIED
					dvx2_dx = (dFxx / c11 - r12_11 * dvy2_dy - r13_11 * dvz2_dz - r14_11 * (dvz_dy + dvy_dz));
				}
			}

			syz[ijk] += dT * (c44 * (dvy_dz + dvz_dy) + 2 * c14 * (dvx2_dx - dvy2_dy) + dsod_dt_ms.x);

			if (xstencil || xfixed_l || xfixed_u) sdd2[ijk].x += dT * (c11 * dvx2_dx + c12 * dvy2_dy + c13 * dvz2_dz + c14 * (dvz_dy + dvy_dz) - dsxx_yy_dt_te + dsdd_dt_ms.x);
			//for free surface (x normal), set sdd2.x directly from external condition
			else sdd2[ijk].x = Fext_xface.x;

			//the y and z components cannot be set from external conditions here
			sdd2[ijk].y += dT * (c11 * dvy2_dy + c12 * dvx2_dx + c13 * dvz2_dz - c14 * (dvz_dy + dvy_dz) - dsxx_yy_dt_te + dsdd_dt_ms.y);
			sdd2[ijk].z += dT * (c33 * dvz2_dz + c13 * (dvx2_dx + dvy2_dy) - dszz_dt_te + dsdd_dt_ms.z);
		}
		else {

			syz[ijk] = 0.0;
			sdd2[ijk] = cuReal3();
		}
	}

	///////////// MECHANICAL DISPLACEMENT

	//update mechanical displacement using velocity (remember u is cell-centred)
	if (i < n_m.i && j < n_m.j && k < n_m.k) {

		if (u_disp.is_not_empty(idx_u)) {

			//find velocity values cell-centred
			cuBReal vx_cc = (vx[ijk] + vx(ijk + cuINT3(0, 1, 0)) + vx(ijk + cuINT3(0, 0, 1)) + vx(ijk + cuINT3(0, 1, 1))) / 4;
			cuBReal vy_cc = (vy[ijk] + vy(ijk + cuINT3(1, 0, 0)) + vy(ijk + cuINT3(0, 0, 1)) + vy(ijk + cuINT3(1, 0, 1))) / 4;
			cuBReal vz_cc = (vz[ijk] + vz(ijk + cuINT3(1, 0, 0)) + vz(ijk + cuINT3(0, 1, 0)) + vz(ijk + cuINT3(1, 1, 0))) / 4;

			u_disp[idx_u] += dT * cuReal3(vx_cc, vy_cc, vz_cc);
		}
		else u_disp[idx_u] = cuReal3();
	}
}

__global__ void Iterate_Elastic_Solver_Stress_FM_Trigonal_Kernel(
	ManagedMeshCUDA& cuMesh,
	MElastic_BoundaryCUDA** pexternal_stress_surfaces, size_t num_surfaces,
	cuVEC_VC<cuBReal>& vx, cuVEC_VC<cuBReal>& vy, cuVEC_VC<cuBReal>& vz,
	cuVEC_VC<cuReal3>& sdd, cuVEC_VC<cuBReal>& sxy, cuVEC_VC<cuBReal>& sxz, cuVEC_VC<cuBReal>& syz,
	cuVEC_VC<cuBReal>& vx2, cuVEC_VC<cuBReal>& vy2, cuVEC_VC<cuBReal>& vz2,
	cuVEC_VC<cuReal3>& sdd2, cuVEC_VC<cuBReal>& sxy2, cuVEC_VC<cuBReal>& sxz2, cuVEC_VC<cuBReal>& syz2,
	cuBReal time, cuBReal dT,
	bool magnetostriction_enabled, bool thermoelasticity_enabled,
	cuVEC<cuBReal>& Temp_previous, cuBReal magnetic_dT,
	ManagedDiffEqFMCUDA& cuDiffEq_FM)
{
	cuVEC_VC<cuReal3>& u_disp = *cuMesh.pu_disp;

	cuSZ3& n_m = u_disp.n;

	int idx = blockIdx.x * blockDim.x + threadIdx.x;

	//kernel launched with size sdd.device_size(mGPU). For a single GPU this has (n_m.i + 1) * (n_m.j + 1) * (n_m.k + 1) cells.
	//for multiple GPUs only last one has an extra cell along partition dimension, the other ones have same dimension as u_disp along partition dimension
	//this means the ijk index can always be used for reading and writing, but with +/-1 along partition direction need to use the () operator to read values
	int i = idx % sdd.n.i;
	int j = (idx / sdd.n.i) % sdd.n.j;
	int k = idx / (sdd.n.i * sdd.n.j);

	if (idx < sdd.n.dim()) {

		cuINT3 ijk = cuINT3(i, j, k);

		//convert vertex index to cell-center index by capping maximum index size (use this to index u_disp)
		cuINT3 ijk_u = cuINT3(i < n_m.i ? i : n_m.i - 1, j < n_m.j ? j : n_m.j - 1, k < n_m.k ? k : n_m.k - 1);
		int idx_u = ijk_u.i + ijk_u.j * n_m.x + ijk_u.k * n_m.x * n_m.y;

		//needed for magnetostriction (time derivatives of stress due to magnetostriction)
		//xx, yy, zz
		cuReal3 dsdd_dt_ms = cuReal3();
		//yz, xz, xy
		cuReal3 dsod_dt_ms = cuReal3();
		if (magnetostriction_enabled) {

			cuVEC_VC<cuReal3>& u_disp = *cuMesh.pu_disp;
			cuVEC_VC<cuReal3>& M = *cuMesh.pM;

			int idx_M = M.position_to_cellidx(u_disp.cellidx_to_position(idx_u));

			if (M.is_not_empty(idx_M)) {

				cuBReal Ms = *cuMesh.pMs;
				cuReal3 mcanis_ea1 = *cuMesh.pmcanis_ea1;
				cuReal3 mcanis_ea2 = *cuMesh.pmcanis_ea2;
				cuReal3 mcanis_ea3 = *cuMesh.pmcanis_ea3;
				cuReal2 mMEc = *cuMesh.pmMEc;
				cuReal2 mMEc2 = *cuMesh.pmMEc2;
				cuReal2 mMEc3 = *cuMesh.pmMEc3;
				cuMesh.update_parameters_mcoarse(idx_M, *cuMesh.pMs, Ms, *cuMesh.pmMEc, mMEc, *cuMesh.pmMEc2, mMEc2, *cuMesh.pmMEc3, mMEc3, *cuMesh.pmcanis_ea1, mcanis_ea1, *cuMesh.pmcanis_ea2, mcanis_ea2, *cuMesh.pmcanis_ea3, mcanis_ea3);

				cuBReal& B21 = mMEc.i;
				cuBReal& B22 = mMEc.j;
				cuBReal& B3 = mMEc2.i;
				cuBReal& B4 = mMEc2.j;
				cuBReal& B14 = mMEc3.i;
				cuBReal& B34 = mMEc3.j;

				cuReal3 m = cuReal3(M[idx_M] * mcanis_ea1, M[idx_M] * mcanis_ea2, M[idx_M] * mcanis_ea3) / Ms;
				cuReal3 dM_dt = (M[idx_M] - (*cuDiffEq_FM.psM1)[idx_M]) / magnetic_dT;
				cuReal3 dm_dt = cuReal3(dM_dt * mcanis_ea1, dM_dt * mcanis_ea2, dM_dt * mcanis_ea3) / Ms;

				dsdd_dt_ms += 2 * B21 * m.z*dm_dt.z * (mcanis_ea1 + mcanis_ea2) + 2 * B22 * m.z*dm_dt.z * mcanis_ea3;
				dsdd_dt_ms += B3 * (m.x*dm_dt.x - m.y*dm_dt.y)*(mcanis_ea1 - mcanis_ea2);
				dsdd_dt_ms += B34 * (m.y*dm_dt.z + dm_dt.y*m.z)*(mcanis_ea1 - mcanis_ea2);

				dsod_dt_ms += B3 * (m.x*dm_dt.y + dm_dt.x*m.y)*mcanis_ea3;
				dsod_dt_ms += B4 * (m.x*dm_dt.z + dm_dt.x*m.z)*mcanis_ea2 + B4 * (m.y*dm_dt.z + dm_dt.y*m.z)*mcanis_ea1;
				dsod_dt_ms += B14 * (m.x*dm_dt.x - m.y*dm_dt.y) * mcanis_ea1 + B14 * (m.x*dm_dt.y + dm_dt.x*m.y)*mcanis_ea2;
				dsod_dt_ms += B34 * (m.x*dm_dt.z + dm_dt.x*m.z) * mcanis_ea3;
			}
		}

		//now solve the main part, with the possible addition of magnetostriction contribution
		Iterate_Elastic_Solver_Stress_Trigonal_CUDA(
			ijk, ijk_u, idx_u,
			cuMesh,
			pexternal_stress_surfaces, num_surfaces,
			vx, vy, vz,
			sdd, sxy, sxz, syz,
			vx2, vy2, vz2,
			sdd2, sxy2, sxz2, syz2,
			time, dT,
			thermoelasticity_enabled,
			Temp_previous, magnetic_dT,
			dsdd_dt_ms, dsod_dt_ms);
	}
}

__global__ void Iterate_Elastic_Solver_Stress_AFM_Trigonal_Kernel(
	ManagedMeshCUDA& cuMesh,
	MElastic_BoundaryCUDA** pexternal_stress_surfaces, size_t num_surfaces,
	cuVEC_VC<cuBReal>& vx, cuVEC_VC<cuBReal>& vy, cuVEC_VC<cuBReal>& vz,
	cuVEC_VC<cuReal3>& sdd, cuVEC_VC<cuBReal>& sxy, cuVEC_VC<cuBReal>& sxz, cuVEC_VC<cuBReal>& syz,
	cuVEC_VC<cuBReal>& vx2, cuVEC_VC<cuBReal>& vy2, cuVEC_VC<cuBReal>& vz2,
	cuVEC_VC<cuReal3>& sdd2, cuVEC_VC<cuBReal>& sxy2, cuVEC_VC<cuBReal>& sxz2, cuVEC_VC<cuBReal>& syz2,
	cuBReal time, cuBReal dT,
	bool magnetostriction_enabled, bool thermoelasticity_enabled,
	cuVEC<cuBReal>& Temp_previous, cuBReal magnetic_dT,
	ManagedDiffEqAFMCUDA& cuDiffEq_AFM)
{
	cuVEC_VC<cuReal3>& u_disp = *cuMesh.pu_disp;

	cuSZ3& n_m = u_disp.n;

	int idx = blockIdx.x * blockDim.x + threadIdx.x;

	//kernel launched with size sdd.device_size(mGPU). For a single GPU this has (n_m.i + 1) * (n_m.j + 1) * (n_m.k + 1) cells.
	//for multiple GPUs only last one has an extra cell along partition dimension, the other ones have same dimension as u_disp along partition dimension
	//this means the ijk index can always be used for reading and writing, but with +/-1 along partition direction need to use the () operator to read values
	int i = idx % sdd.n.i;
	int j = (idx / sdd.n.i) % sdd.n.j;
	int k = idx / (sdd.n.i * sdd.n.j);

	if (idx < sdd.n.dim()) {

		cuINT3 ijk = cuINT3(i, j, k);

		//convert vertex index to cell-center index by capping maximum index size (use this to index u_disp)
		cuINT3 ijk_u = cuINT3(i < n_m.i ? i : n_m.i - 1, j < n_m.j ? j : n_m.j - 1, k < n_m.k ? k : n_m.k - 1);
		int idx_u = ijk_u.i + ijk_u.j * n_m.x + ijk_u.k * n_m.x * n_m.y;

		//needed for magnetostriction (time derivatives of stress due to magnetostriction)
		//xx, yy, zz
		cuReal3 dsdd_dt_ms = cuReal3();
		//yz, xz, xy
		cuReal3 dsod_dt_ms = cuReal3();
		if (magnetostriction_enabled) {

			cuVEC_VC<cuReal3>& u_disp = *cuMesh.pu_disp;
			cuVEC_VC<cuReal3>& M = *cuMesh.pM;
			cuVEC_VC<cuReal3>& M2 = *cuMesh.pM2;

			int idx_M = M.position_to_cellidx(u_disp.cellidx_to_position(idx_u));

			if (M.is_not_empty(idx_M)) {

				cuReal2 Ms_AFM = *cuMesh.pMs_AFM;
				cuReal3 mcanis_ea1 = *cuMesh.pmcanis_ea1;
				cuReal3 mcanis_ea2 = *cuMesh.pmcanis_ea2;
				cuReal3 mcanis_ea3 = *cuMesh.pmcanis_ea3;
				cuReal2 mMEc = *cuMesh.pmMEc;
				cuReal2 mMEc2 = *cuMesh.pmMEc2;
				cuReal2 mMEc3 = *cuMesh.pmMEc3;
				cuMesh.update_parameters_mcoarse(idx_M, *cuMesh.pMs_AFM, Ms_AFM, *cuMesh.pmMEc, mMEc, *cuMesh.pmMEc2, mMEc2, *cuMesh.pmMEc3, mMEc3, *cuMesh.pmcanis_ea1, mcanis_ea1, *cuMesh.pmcanis_ea2, mcanis_ea2, *cuMesh.pmcanis_ea3, mcanis_ea3);

				cuBReal& B21 = mMEc.i;
				cuBReal& B22 = mMEc.j;
				cuBReal& B3 = mMEc2.i;
				cuBReal& B4 = mMEc2.j;
				cuBReal& B14 = mMEc3.i;
				cuBReal& B34 = mMEc3.j;

				cuReal3 mA = cuReal3(M[idx_M] * mcanis_ea1, M[idx_M] * mcanis_ea2, M[idx_M] * mcanis_ea3) / Ms_AFM.i;
				cuReal3 mB = cuReal3(M2[idx_M] * mcanis_ea1, M2[idx_M] * mcanis_ea2, M2[idx_M] * mcanis_ea3) / Ms_AFM.j;
				cuReal3 dM_dtA = (M[idx_M] - (*cuDiffEq_AFM.psM1)[idx_M]) / magnetic_dT;
				cuReal3 dm_dtA = cuReal3(dM_dtA * mcanis_ea1, dM_dtA * mcanis_ea2, dM_dtA * mcanis_ea3) / Ms_AFM.i;
				cuReal3 dM_dtB = (M2[idx_M] - (*cuDiffEq_AFM.psM1_2)[idx_M]) / magnetic_dT;
				cuReal3 dm_dtB = cuReal3(dM_dtB * mcanis_ea1, dM_dtB * mcanis_ea2, dM_dtB * mcanis_ea3) / Ms_AFM.j;

				dsdd_dt_ms += B21 * (mA.z*dm_dtA.z + mB.z*dm_dtB.z) * (mcanis_ea1 + mcanis_ea2) + B22 * (mA.z*dm_dtA.z + mB.z*dm_dtB.z) * mcanis_ea3;
				dsdd_dt_ms += 0.5 * B3 * (mA.x*dm_dtA.x + mB.x*dm_dtB.x - mA.y*dm_dtA.y - mB.y*dm_dtB.y)*(mcanis_ea1 - mcanis_ea2);
				dsdd_dt_ms += 0.5 * B34 * (mA.y*dm_dtA.z + dm_dtA.y*mA.z + mB.y*dm_dtB.z + dm_dtB.y*mB.z)*(mcanis_ea1 - mcanis_ea2);

				dsod_dt_ms += B3 * (mA.x*dm_dtA.y + mB.x*dm_dtB.y)*mcanis_ea3;
				dsod_dt_ms += 0.5 * B4 * (mA.x*dm_dtA.z + dm_dtA.x*mA.z + mB.x*dm_dtB.z + dm_dtB.x*mB.z)*mcanis_ea2 + 0.5 * B4 * (mA.y*dm_dtA.z + dm_dtA.y*mA.z + mB.y*dm_dtB.z + dm_dtB.y*mB.z)*mcanis_ea1;
				dsod_dt_ms += 0.5 * B14 * (mA.x*dm_dtA.x + mB.x*dm_dtB.x - mA.y*dm_dtA.y - mB.y*dm_dtB.y) * mcanis_ea1 + 0.5 * B14 * (mA.x*dm_dtA.y + dm_dtA.x*mA.y + mB.x*dm_dtB.y + dm_dtB.x*mB.y)*mcanis_ea2;
				dsod_dt_ms += 0.5 * B34 * (mA.x*dm_dtA.z + dm_dtA.x*mA.z + mB.x*dm_dtB.z + dm_dtB.x*mB.z) * mcanis_ea3;
			}
		}

		//now solve the main part, with the possible addition of magnetostriction contribution
		Iterate_Elastic_Solver_Stress_Trigonal_CUDA(
			ijk, ijk_u, idx_u,
			cuMesh,
			pexternal_stress_surfaces, num_surfaces,
			vx, vy, vz,
			sdd, sxy, sxz, syz,
			vx2, vy2, vz2,
			sdd2, sxy2, sxz2, syz2,
			time, dT,
			thermoelasticity_enabled,
			Temp_previous, magnetic_dT,
			dsdd_dt_ms, dsod_dt_ms);
	}
}

__global__ void Iterate_Elastic_Solver_Stress_NoMS_Trigonal_Kernel(
	ManagedMeshCUDA& cuMesh,
	MElastic_BoundaryCUDA** pexternal_stress_surfaces, size_t num_surfaces,
	cuVEC_VC<cuBReal>& vx, cuVEC_VC<cuBReal>& vy, cuVEC_VC<cuBReal>& vz,
	cuVEC_VC<cuReal3>& sdd, cuVEC_VC<cuBReal>& sxy, cuVEC_VC<cuBReal>& sxz, cuVEC_VC<cuBReal>& syz,
	cuVEC_VC<cuBReal>& vx2, cuVEC_VC<cuBReal>& vy2, cuVEC_VC<cuBReal>& vz2,
	cuVEC_VC<cuReal3>& sdd2, cuVEC_VC<cuBReal>& sxy2, cuVEC_VC<cuBReal>& sxz2, cuVEC_VC<cuBReal>& syz2,
	cuBReal time, cuBReal dT,
	bool thermoelasticity_enabled,
	cuVEC<cuBReal>& Temp_previous, cuBReal magnetic_dT)
{
	cuVEC_VC<cuReal3>& u_disp = *cuMesh.pu_disp;

	cuSZ3& n_m = u_disp.n;

	int idx = blockIdx.x * blockDim.x + threadIdx.x;

	//kernel launched with size sdd.device_size(mGPU). For a single GPU this has (n_m.i + 1) * (n_m.j + 1) * (n_m.k + 1) cells.
	//for multiple GPUs only last one has an extra cell along partition dimension, the other ones have same dimension as u_disp along partition dimension
	//this means the ijk index can always be used for reading and writing, but with +/-1 along partition direction need to use the () operator to read values
	int i = idx % sdd.n.i;
	int j = (idx / sdd.n.i) % sdd.n.j;
	int k = idx / (sdd.n.i * sdd.n.j);

	if (idx < sdd.n.dim()) {

		cuINT3 ijk = cuINT3(i, j, k);

		//convert vertex index to cell-center index by capping maximum index size (use this to index u_disp)
		cuINT3 ijk_u = cuINT3(i < n_m.i ? i : n_m.i - 1, j < n_m.j ? j : n_m.j - 1, k < n_m.k ? k : n_m.k - 1);
		int idx_u = ijk_u.i + ijk_u.j * n_m.x + ijk_u.k * n_m.x * n_m.y;

		//now solve the main part without magnetostriction
		Iterate_Elastic_Solver_Stress_Trigonal_CUDA(
			ijk, ijk_u, idx_u,
			cuMesh,
			pexternal_stress_surfaces, num_surfaces,
			vx, vy, vz,
			sdd, sxy, sxz, syz,
			vx2, vy2, vz2,
			sdd2, sxy2, sxz2, syz2,
			time, dT,
			thermoelasticity_enabled,
			Temp_previous, magnetic_dT,
			cuReal3(), cuReal3());
	}
}

//----------------------- Iterate_Elastic_Solver LAUNCHERS

//update stress for dT time increment
void MElasticCUDA::Iterate_Elastic_Solver_Stress_Trigonal(double dT, double magnetic_dT)
{
	//use sdd device dimensions, since this has total size (pMeshCUDA->n_m.i + 1) * (pMeshCUDA->n_m.j + 1) * (pMeshCUDA->n_m.k + 1)

	vx.exchange_halos();
	vy.exchange_halos();
	vz.exchange_halos();
	vx2.exchange_halos();
	vy2.exchange_halos();
	vz2.exchange_halos();

	//1b. Update stress
	if (magnetostriction_enabled) {

		if (pMeshCUDA->GetMeshType() == MESH_ANTIFERROMAGNETIC) {

			for (mGPU.device_begin(); mGPU != mGPU.device_end(); mGPU++) {

				Iterate_Elastic_Solver_Stress_AFM_Trigonal_Kernel <<< (sdd.device_size(mGPU) + CUDATHREADS) / CUDATHREADS, CUDATHREADS >>>
					(pMeshCUDA->cuMesh.get_deviceobject(mGPU), 
					external_stress_surfaces_arr(mGPU), external_stress_surfaces.size(),
					vx.get_deviceobject(mGPU), vy.get_deviceobject(mGPU), vz.get_deviceobject(mGPU), 
					sdd.get_deviceobject(mGPU), sxy.get_deviceobject(mGPU), sxz.get_deviceobject(mGPU), syz.get_deviceobject(mGPU),
					vx2.get_deviceobject(mGPU), vy2.get_deviceobject(mGPU), vz2.get_deviceobject(mGPU), 
					sdd2.get_deviceobject(mGPU), sxy2.get_deviceobject(mGPU), sxz2.get_deviceobject(mGPU), syz2.get_deviceobject(mGPU),
					pMeshCUDA->GetStageTime(), dT,
					magnetostriction_enabled, thermoelasticity_enabled,
					Temp_previous.get_deviceobject(mGPU), magnetic_dT,
					reinterpret_cast<AFMeshCUDA*>(pMeshCUDA)->Get_ManagedDiffEqCUDA().get_deviceobject(mGPU));
			}
		}
		else if (pMeshCUDA->GetMeshType() == MESH_FERROMAGNETIC) {

			for (mGPU.device_begin(); mGPU != mGPU.device_end(); mGPU++) {

				Iterate_Elastic_Solver_Stress_FM_Trigonal_Kernel <<< (sdd.device_size(mGPU) + CUDATHREADS) / CUDATHREADS, CUDATHREADS >>>
					(pMeshCUDA->cuMesh.get_deviceobject(mGPU), 
					external_stress_surfaces_arr(mGPU), external_stress_surfaces.size(),
					vx.get_deviceobject(mGPU), vy.get_deviceobject(mGPU), vz.get_deviceobject(mGPU),
					sdd.get_deviceobject(mGPU), sxy.get_deviceobject(mGPU), sxz.get_deviceobject(mGPU), syz.get_deviceobject(mGPU),
					vx2.get_deviceobject(mGPU), vy2.get_deviceobject(mGPU), vz2.get_deviceobject(mGPU),
					sdd2.get_deviceobject(mGPU), sxy2.get_deviceobject(mGPU), sxz2.get_deviceobject(mGPU), syz2.get_deviceobject(mGPU),
					pMeshCUDA->GetStageTime(), dT,
					magnetostriction_enabled, thermoelasticity_enabled,
					Temp_previous.get_deviceobject(mGPU), magnetic_dT,
					reinterpret_cast<FMeshCUDA*>(pMeshCUDA)->Get_ManagedDiffEqCUDA().get_deviceobject(mGPU));
			}
		}
	}
	else {

		for (mGPU.device_begin(); mGPU != mGPU.device_end(); mGPU++) {

			Iterate_Elastic_Solver_Stress_NoMS_Trigonal_Kernel <<< (sdd.device_size(mGPU) + CUDATHREADS) / CUDATHREADS, CUDATHREADS >>>
				(pMeshCUDA->cuMesh.get_deviceobject(mGPU), 
				external_stress_surfaces_arr(mGPU), external_stress_surfaces.size(),
				vx.get_deviceobject(mGPU), vy.get_deviceobject(mGPU), vz.get_deviceobject(mGPU),
				sdd.get_deviceobject(mGPU), sxy.get_deviceobject(mGPU), sxz.get_deviceobject(mGPU), syz.get_deviceobject(mGPU),
				vx2.get_deviceobject(mGPU), vy2.get_deviceobject(mGPU), vz2.get_deviceobject(mGPU),
				sdd2.get_deviceobject(mGPU), sxy2.get_deviceobject(mGPU), sxz2.get_deviceobject(mGPU), syz2.get_deviceobject(mGPU),
				pMeshCUDA->GetStageTime(), dT,
				thermoelasticity_enabled,
				Temp_previous.get_deviceobject(mGPU), magnetic_dT);
		}
	}
}

//---------------------------------------------- Initial Conditions Launchers and Kernels

__global__ void Set_Initial_Stress_Trigonal_Kernel(
	ManagedMeshCUDA& cuMesh,
	cuVEC_VC<cuReal3>& sdd, cuVEC_VC<cuReal3>& sdd2,
	cuVEC_VC<cuBReal>& sxy, cuVEC_VC<cuBReal>& sxz, cuVEC_VC<cuBReal>& syz,
	cuVEC_VC<cuBReal>& sxy2, cuVEC_VC<cuBReal>& sxz2, cuVEC_VC<cuBReal>& syz2,
	bool magnetostriction_enabled, bool thermoelasticity_enabled, cuBReal& T_ambient)
{
	cuVEC_VC<cuReal3>& u_disp = *cuMesh.pu_disp;
	cuVEC_VC<cuReal3>& strain_diag = *cuMesh.pstrain_diag;
	cuVEC_VC<cuReal3>& strain_odiag = *cuMesh.pstrain_odiag;

	cuReal3& h_m = u_disp.h;
	cuSZ3& n_m = u_disp.n;

	int idx = blockIdx.x * blockDim.x + threadIdx.x;

	//kernel launched with size sdd.device_size(mGPU). For a single GPU this has (n_m.i + 1) * (n_m.j + 1) * (n_m.k + 1) cells.
	//for multiple GPUs only last one has an extra cell along partition dimension, the other ones have same dimension as u_disp along partition dimension
	//this means the ijk index can always be used for reading and writing, but when adding +1 along partition direction need to use the () operator to read values
	int i = idx % sdd.n.i;
	int j = (idx / sdd.n.i) % sdd.n.j;
	int k = idx / (sdd.n.i * sdd.n.j);

	if (idx < sdd.n.dim()) {

		//convert vertex index to cell-center index by capping maximum index size (use this to index u_disp)
		cuINT3 ijk_u = cuINT3(i < n_m.i ? i : n_m.i - 1, j < n_m.j ? j : n_m.j - 1, k < n_m.k ? k : n_m.k - 1);
		int idx_u = ijk_u.i + ijk_u.j * n_m.x + ijk_u.k * n_m.x * n_m.y;

		cuINT3 ijk = cuINT3(i, j, k);

		//update sxx, syy, szz
		int niend = (ijk.i < n_m.i);
		int njend = (ijk.j < n_m.j);
		int nkend = (ijk.k < n_m.k);

		//check if required edges are present
		bool xedge_u = ijk.i < n_m.i && u_disp.is_edge_x(idx_u);
		bool xedge_l =
			(ijk.i > 0 && u_disp.is_edge_x(idx_u - niend)) ||
			(ijk.i == 0 &&
			(u_disp.is_halo_nx(idx_u) ||
			(ijk.k > 0 && u_disp.is_halo_nx(idx_u - nkend * n_m.x * n_m.y)) ||
			(ijk.j > 0 && u_disp.is_halo_nx(idx_u - njend * n_m.x)) ||
			(ijk.j > 0 && ijk.k > 0 && u_disp.is_halo_nx(idx_u - nkend * n_m.x * n_m.y - njend * n_m.x))));
		bool yedge_u = ijk.j < n_m.j && u_disp.is_edge_y(idx_u);
		bool yedge_l = ijk.j > 0 && u_disp.is_edge_y(idx_u - njend * n_m.x);
		bool zedge_u = ijk.k < n_m.k && u_disp.is_edge_z(idx_u);
		bool zedge_l = ijk.k > 0 && u_disp.is_edge_z(idx_u - nkend * n_m.x * n_m.y);

		//check for fixed faces at ends
		bool xfixed_l = (ijk.i == 0 && u_disp.is_dirichlet_px(idx_u));
		bool xfixed_u = (ijk.i == n_m.i && u_disp.is_dirichlet_nx(idx_u));
		bool yfixed_l = (ijk.j == 0 && u_disp.is_dirichlet_py(idx_u));
		bool yfixed_u = (ijk.j == n_m.j && u_disp.is_dirichlet_ny(idx_u));
		bool zfixed_l = (ijk.k == 0 && u_disp.is_dirichlet_pz(idx_u));
		bool zfixed_u = (ijk.k == n_m.k && u_disp.is_dirichlet_nz(idx_u));

		//xx, yy, zz
		cuReal3 Stress_MS_dd = cuReal3();
		//yz, xz, xy
		cuReal3 Stress_MS_od = cuReal3();
		if (magnetostriction_enabled) {

			cuVEC_VC<cuReal3>& M = *cuMesh.pM;
			cuVEC_VC<cuReal3>& M2 = *cuMesh.pM2;

			int idx_M = M.position_to_cellidx(u_disp.cellidx_to_position(idx_u));

			if (M.is_not_empty(idx_M)) {

				//MESH_ANTIFERROMAGNETIC
				if (M2.linear_size()) {

					cuReal2 Ms_AFM = *cuMesh.pMs_AFM;
					cuReal3 mcanis_ea1 = *cuMesh.pmcanis_ea1;
					cuReal3 mcanis_ea2 = *cuMesh.pmcanis_ea2;
					cuReal3 mcanis_ea3 = *cuMesh.pmcanis_ea3;
					cuReal2 mMEc = *cuMesh.pmMEc;
					cuReal2 mMEc2 = *cuMesh.pmMEc2;
					cuReal2 mMEc3 = *cuMesh.pmMEc3;
					cuMesh.update_parameters_mcoarse(idx_M, *cuMesh.pMs_AFM, Ms_AFM, *cuMesh.pmMEc, mMEc, *cuMesh.pmMEc2, mMEc2, *cuMesh.pmMEc3, mMEc3, *cuMesh.pmcanis_ea1, mcanis_ea1, *cuMesh.pmcanis_ea2, mcanis_ea2, *cuMesh.pmcanis_ea3, mcanis_ea3);

					cuBReal& B21 = mMEc.i;
					cuBReal& B22 = mMEc.j;
					cuBReal& B3 = mMEc2.i;
					cuBReal& B4 = mMEc2.j;
					cuBReal& B14 = mMEc3.i;
					cuBReal& B34 = mMEc3.j;

					cuReal3 mA = cuReal3(M[idx_M] * mcanis_ea1, M[idx_M] * mcanis_ea2, M[idx_M] * mcanis_ea3) / Ms_AFM.i;
					cuReal3 mB = cuReal3(M2[idx_M] * mcanis_ea1, M2[idx_M] * mcanis_ea2, M2[idx_M] * mcanis_ea3) / Ms_AFM.j;

					Stress_MS_dd += 0.5 * B21 * (mA.z*mA.z + mB.z*mB.z) * (mcanis_ea1 + mcanis_ea2) + 0.5 * B22 * (mA.z*mA.z + mB.z*mB.z) * mcanis_ea3;
					Stress_MS_dd += 0.25 * B3 * (mA.x*mA.x + mB.x*mB.x - mA.y*mA.y - mB.y*mB.y)*(mcanis_ea1 - mcanis_ea2);
					Stress_MS_dd += 0.5 * B34 * (mA.y*mA.z + mB.y*mB.z)*(mcanis_ea1 - mcanis_ea2);

					Stress_MS_od += 0.5 * B3 * (mA.x*mA.y + mB.x*mB.y)*mcanis_ea3;
					Stress_MS_od += 0.5 * B4 * (mA.x*mA.z + mB.x*mB.z)*mcanis_ea2 + 0.5 * B4 * (mA.y*mA.z + mB.y*mB.z)*mcanis_ea1;
					Stress_MS_od += 0.25 * B14 * (mA.x*mA.x + mB.x*mB.x - mA.y*mA.y - mB.y*mB.y) * mcanis_ea1 + 0.5 * B14 * (mA.x*mA.y + mB.x*mB.y)*mcanis_ea2;
					Stress_MS_od += 0.5 * B34 * (mA.x*mA.z + mB.x*mB.z) * mcanis_ea3;
				}
				//MESH_FERROMAGNETIC
				else {

					cuBReal Ms = *cuMesh.pMs;
					cuReal3 mcanis_ea1 = *cuMesh.pmcanis_ea1;
					cuReal3 mcanis_ea2 = *cuMesh.pmcanis_ea2;
					cuReal3 mcanis_ea3 = *cuMesh.pmcanis_ea3;
					cuReal2 mMEc = *cuMesh.pmMEc;
					cuReal2 mMEc2 = *cuMesh.pmMEc2;
					cuReal2 mMEc3 = *cuMesh.pmMEc3;
					cuMesh.update_parameters_mcoarse(idx_M, *cuMesh.pMs, Ms, *cuMesh.pmMEc, mMEc, *cuMesh.pmMEc2, mMEc2, *cuMesh.pmMEc3, mMEc3, *cuMesh.pmcanis_ea1, mcanis_ea1, *cuMesh.pmcanis_ea2, mcanis_ea2, *cuMesh.pmcanis_ea3, mcanis_ea3);

					cuBReal& B21 = mMEc.i;
					cuBReal& B22 = mMEc.j;
					cuBReal& B3 = mMEc2.i;
					cuBReal& B4 = mMEc2.j;
					cuBReal& B14 = mMEc3.i;
					cuBReal& B34 = mMEc3.j;

					cuReal3 m = cuReal3(M[idx_M] * mcanis_ea1, M[idx_M] * mcanis_ea2, M[idx_M] * mcanis_ea3) / Ms;

					Stress_MS_dd += B21 * m.z*m.z * (mcanis_ea1 + mcanis_ea2) + B22 * m.z*m.z * mcanis_ea3;
					Stress_MS_dd += 0.5 * B3 * (m.x*m.x - m.y*m.y)*(mcanis_ea1 - mcanis_ea2);
					Stress_MS_dd += B34 * m.y*m.z*(mcanis_ea1 - mcanis_ea2);

					Stress_MS_od += B3 * m.x*m.y*mcanis_ea3;
					Stress_MS_od += B4 * m.x*m.z*mcanis_ea2 + B4 * m.y*m.z*mcanis_ea1;
					Stress_MS_od += 0.5 * B14 * (m.x*m.x - m.y*m.y) * mcanis_ea1 + B14 * m.x*m.y*mcanis_ea2;
					Stress_MS_od += B34 * m.x*m.z * mcanis_ea3;
				}
			}
		}

		cuBReal Stress_Temp_xx_yy = 0.0;
		cuBReal Stress_Temp_zz = 0.0;
		if (thermoelasticity_enabled) {

			cuVEC_VC<cuBReal>& Temp = *cuMesh.pTemp;
			cuVEC_VC<cuBReal>& Temp_l = *cuMesh.pTemp_l;

			int idx_T = Temp.position_to_cellidx(u_disp.cellidx_to_position(idx_u));

			if (Temp.is_not_empty(idx_T)) {

				cuBReal thalpha = *cuMesh.pthalpha;
				cuReal3 cC = *cuMesh.pcC;
				cuReal3 cC3 = *cuMesh.pcC3;
				cuMesh.update_parameters_scoarse(idx_u, *cuMesh.pcC, cC, *cuMesh.pcC3, cC3, *cuMesh.pthalpha, thalpha);

				cuBReal Temperature = 0.0;
				//for 2TM we need to use the lattice temperature
				if (Temp_l.linear_size()) Temperature = Temp_l[idx_T];
				else Temperature = Temp[idx_T];

				Stress_Temp_xx_yy = (cC.i + cC.j + cC3.j) * thalpha * (Temperature - T_ambient);
				Stress_Temp_zz = (cC3.i + 2 * cC3.j) * thalpha * (Temperature - T_ambient);
			}
		}

		//update sdd if not empty
		if ((xedge_u || xedge_l) && (yedge_u || yedge_l) && (zedge_u || zedge_l)) {

			sdd[ijk].x = -Stress_Temp_xx_yy + Stress_MS_dd.x;
			sdd[ijk].y = -Stress_Temp_xx_yy + Stress_MS_dd.y;
			sdd[ijk].z = -Stress_Temp_zz + Stress_MS_dd.z;
		}
		else sdd[ijk] = cuReal3();

		//update sdd and syz2
		if ((xedge_u || xedge_l) && (yedge_u || yedge_l) && (zedge_u || zedge_l)) {

			sdd[ijk].x = -Stress_Temp_xx_yy + Stress_MS_dd.x;
			sdd[ijk].y = -Stress_Temp_xx_yy + Stress_MS_dd.y;
			sdd[ijk].z = -Stress_Temp_zz + Stress_MS_dd.z;

			syz2[ijk] = Stress_MS_od.x;
		}
		else {

			sdd[ijk] = cuReal3();
			syz2[ijk] = 0.0;
		}

		//update sxy and sxz2
		if (i < n_m.i && j < n_m.j) {

			bool zface = u_disp.is_face_z(idx_u);

			if (zface) {

				sxy[ijk] = Stress_MS_od.z;
				sxz2[ijk] = Stress_MS_od.y;
			}
			else {

				sxy[ijk] = 0.0;
				sxz2[ijk] = 0.0;
			}
		}

		//update sxz and sxy2
		if (i < n_m.i && k < n_m.k) {

			bool yface = u_disp.is_face_y(idx_u);

			if (yface) {

				sxz[ijk] = Stress_MS_od.y;
				sxy2[ijk] = Stress_MS_od.z;
			}
			else {

				sxz[ijk] = 0.0;
				sxy2[ijk] = 0.0;
			}
		}

		//update syz and sdd2
		if (j < n_m.j && k < n_m.k) {

			bool xface = u_disp.is_face_x(idx_u);

			if (xface) {

				syz[ijk] = Stress_MS_od.x;
				sdd2[ijk].x = -Stress_Temp_xx_yy + Stress_MS_dd.x;
				sdd2[ijk].x = -Stress_Temp_xx_yy + Stress_MS_dd.y;
				sdd2[ijk].x = -Stress_Temp_zz + Stress_MS_dd.z;
			}
			else {

				syz[ijk] = 0.0;
				sdd2[ijk] = cuReal3();
			}
		}
	}
}

//if thermoelasticity or magnetostriction is enabled, then initial stress must be set correctly
void MElasticCUDA::Set_Initial_Stress_Trigonal(void)
{
	if (!magnetostriction_enabled && !thermoelasticity_enabled) {

		sdd.set(cuReal3());
		sdd2.set(cuReal3());
		sxy.set(0.0); sxz.set(0.0); syz.set(0.0);
		sxy2.set(0.0); sxz2.set(0.0); syz2.set(0.0);
	}
	else {

		//reset for dT / dt computation
		if (thermoelasticity_enabled) {

			if (Temp_previous.resize(pMeshCUDA->n_t.dim())) Save_Current_Temperature();
		}

		//reset for dm / dt computation
		if (magnetostriction_enabled) pMeshCUDA->SaveMagnetization();

		//use sdd device dimensions, since this has total size (pMeshCUDA->n_m.i + 1) * (pMeshCUDA->n_m.j + 1) * (pMeshCUDA->n_m.k + 1)

		for (mGPU.device_begin(); mGPU != mGPU.device_end(); mGPU++) {

			Set_Initial_Stress_Trigonal_Kernel <<< (sdd.device_size(mGPU) + CUDATHREADS) / CUDATHREADS, CUDATHREADS >>>
				(pMeshCUDA->cuMesh.get_deviceobject(mGPU), 
				sdd.get_deviceobject(mGPU), sdd2.get_deviceobject(mGPU), sxy.get_deviceobject(mGPU), sxz.get_deviceobject(mGPU), syz.get_deviceobject(mGPU), 
				sxy2.get_deviceobject(mGPU), sxz2.get_deviceobject(mGPU), syz2.get_deviceobject(mGPU), 
				magnetostriction_enabled, thermoelasticity_enabled, T_ambient(mGPU));
		}
	}
}

#endif

#endif
