#include "hip/hip_runtime.h"
#include "RoughnessCUDA.h"

#if COMPILECUDA == 1

#ifdef MODULE_COMPILATION_ROUGHNESS

#include "Reduction.cuh"

#include "Mesh_FerromagneticCUDA.h"
#include "MeshDefs.h"

//----------------------- Initialization

__global__ void set_RoughnessCUDA_pointers_kernel(
	ManagedMeshCUDA& cuMesh, cuVEC<cuReal3>& Fmul_rough, cuVEC<cuReal3>& Fomul_rough)
{
	if (threadIdx.x == 0) cuMesh.pFmul_rough = &Fmul_rough;
	if (threadIdx.x == 1) cuMesh.pFomul_rough = &Fomul_rough;
}

void RoughnessCUDA::set_RoughnessCUDA_pointers(void)
{
	for (mGPU.device_begin(); mGPU != mGPU.device_end(); mGPU++) {

		set_RoughnessCUDA_pointers_kernel <<< 1, CUDATHREADS >>>
			(pMeshCUDA->cuMesh.get_deviceobject(mGPU), Fmul_rough.get_deviceobject(mGPU), Fomul_rough.get_deviceobject(mGPU));
	}
}

__global__ void RoughnessCUDA_FM_UpdateField_Kernel(ManagedMeshCUDA& cuMesh, cuVEC<cuReal3>& Fmul_rough, cuVEC<cuReal3>& Fomul_rough, ManagedModulesCUDA& cuModule, bool do_reduction)
{
	cuVEC_VC<cuReal3>& M = *cuMesh.pM;
	cuVEC<cuReal3>& Heff = *cuMesh.pHeff;

	int idx = blockIdx.x * blockDim.x + threadIdx.x;

	cuBReal energy_ = 0.0;

	if (idx < Heff.linear_size()) {

		cuReal3 Hrough = cuReal3();

		if (M.is_not_empty(idx)) {

			Hrough = cuReal33(
				cuReal3(Fmul_rough[idx].x, Fomul_rough[idx].x, Fomul_rough[idx].y),
				cuReal3(Fomul_rough[idx].x, Fmul_rough[idx].y, Fomul_rough[idx].z),
				cuReal3(Fomul_rough[idx].y, Fomul_rough[idx].z, Fmul_rough[idx].z)) * M[idx];

			if (do_reduction) {

				int non_empty_cells = M.get_nonempty_cells();
				if (non_empty_cells) energy_ = -(cuBReal)MU0 * M[idx] * Hrough / (2 * non_empty_cells);
			}

			if (do_reduction && cuModule.pModule_Heff->linear_size()) (*cuModule.pModule_Heff)[idx] = Hrough;
			if (do_reduction && cuModule.pModule_energy->linear_size()) (*cuModule.pModule_energy)[idx] = -(cuBReal)MU0 * M[idx] * Hrough / 2;
		}

		Heff[idx] += Hrough;
	}

	if (do_reduction) reduction_sum(0, 1, &energy_, *cuModule.penergy);
}

__global__ void RoughnessCUDA_AFM_UpdateField_Kernel(ManagedMeshCUDA& cuMesh, cuVEC<cuReal3>& Fmul_rough, cuVEC<cuReal3>& Fomul_rough, ManagedModulesCUDA& cuModule, bool do_reduction)
{
	cuVEC_VC<cuReal3>& M = *cuMesh.pM;
	cuVEC<cuReal3>& Heff = *cuMesh.pHeff;

	cuVEC_VC<cuReal3>& M2 = *cuMesh.pM2;
	cuVEC<cuReal3>& Heff2 = *cuMesh.pHeff2;

	int idx = blockIdx.x * blockDim.x + threadIdx.x;

	cuBReal energy_ = 0.0;

	if (idx < Heff.linear_size()) {

		cuReal3 Hrough = cuReal3();

		if (M.is_not_empty(idx)) {

			Hrough = cuReal33(
				cuReal3(Fmul_rough[idx].x, Fomul_rough[idx].x, Fomul_rough[idx].y),
				cuReal3(Fomul_rough[idx].x, Fmul_rough[idx].y, Fomul_rough[idx].z),
				cuReal3(Fomul_rough[idx].y, Fomul_rough[idx].z, Fmul_rough[idx].z)) * (M[idx] + M2[idx]) / 2;

			if (do_reduction) {

				int non_empty_cells = M.get_nonempty_cells();
				if (non_empty_cells) energy_ = -(cuBReal)MU0 * (M[idx] + M2[idx]) * Hrough / (4 * non_empty_cells);
			}

			if (do_reduction && cuModule.pModule_Heff->linear_size()) (*cuModule.pModule_Heff)[idx] = Hrough;
			if (do_reduction && cuModule.pModule_Heff2->linear_size()) (*cuModule.pModule_Heff2)[idx] = Hrough;
			if (do_reduction && cuModule.pModule_energy->linear_size()) (*cuModule.pModule_energy)[idx] = -MU0 * M[idx] * Hrough / 2;
			if (do_reduction && cuModule.pModule_energy2->linear_size()) (*cuModule.pModule_energy2)[idx] = -MU0 * M2[idx] * Hrough / 2;
		}

		Heff[idx] += Hrough;
		Heff2[idx] += Hrough;
	}

	if (do_reduction) reduction_sum(0, 1, &energy_, *cuModule.penergy);
}

void RoughnessCUDA::UpdateField(void)
{
	if (pMeshCUDA->GetMeshType() == MESH_FERROMAGNETIC) {

		if (pMeshCUDA->CurrentTimeStepSolved()) {

			ZeroEnergy();

			for (mGPU.device_begin(); mGPU != mGPU.device_end(); mGPU++) {

				RoughnessCUDA_FM_UpdateField_Kernel <<< (pMeshCUDA->M.device_size(mGPU) + CUDATHREADS) / CUDATHREADS, CUDATHREADS >>> 
					(pMeshCUDA->cuMesh.get_deviceobject(mGPU), Fmul_rough.get_deviceobject(mGPU), Fomul_rough.get_deviceobject(mGPU), cuModule.get_deviceobject(mGPU), true);
			}
		}
		else {

			for (mGPU.device_begin(); mGPU != mGPU.device_end(); mGPU++) {

				RoughnessCUDA_FM_UpdateField_Kernel <<< (pMeshCUDA->M.device_size(mGPU) + CUDATHREADS) / CUDATHREADS, CUDATHREADS >>> 
					(pMeshCUDA->cuMesh.get_deviceobject(mGPU), Fmul_rough.get_deviceobject(mGPU), Fomul_rough.get_deviceobject(mGPU), cuModule.get_deviceobject(mGPU), false);
			}
		}
	}

	else if (pMeshCUDA->GetMeshType() == MESH_ANTIFERROMAGNETIC) {

		if (pMeshCUDA->CurrentTimeStepSolved()) {

			ZeroEnergy();

			for (mGPU.device_begin(); mGPU != mGPU.device_end(); mGPU++) {

				RoughnessCUDA_AFM_UpdateField_Kernel <<< (pMeshCUDA->M.device_size(mGPU) + CUDATHREADS) / CUDATHREADS, CUDATHREADS >>> 
					(pMeshCUDA->cuMesh.get_deviceobject(mGPU), Fmul_rough.get_deviceobject(mGPU), Fomul_rough.get_deviceobject(mGPU), cuModule.get_deviceobject(mGPU), true);
			}
		}
		else {

			for (mGPU.device_begin(); mGPU != mGPU.device_end(); mGPU++) {

				RoughnessCUDA_AFM_UpdateField_Kernel <<< (pMeshCUDA->M.device_size(mGPU) + CUDATHREADS) / CUDATHREADS, CUDATHREADS >>> 
					(pMeshCUDA->cuMesh.get_deviceobject(mGPU), Fmul_rough.get_deviceobject(mGPU), Fomul_rough.get_deviceobject(mGPU), cuModule.get_deviceobject(mGPU), false);
			}
		}
	}
}

#endif

#endif

//----------------------------------- MONTE-CARLO METHODS FOR ENERGY COMPUTATION

#if COMPILECUDA == 1 && MONTE_CARLO == 1

//Ferromagnetic
__device__ cuBReal ManagedMeshCUDA::Get_EnergyChange_FM_RoughnessCUDA(int spin_index, cuReal3 Mnew)
{
	if (pFmul_rough && pFmul_rough->linear_size()) {

		cuVEC_VC<cuReal3>& M = *pM;

		cuReal33 Fmat = cuReal33(
			cuReal3((*pFmul_rough)[spin_index].x, (*pFomul_rough)[spin_index].x, (*pFomul_rough)[spin_index].y),
			cuReal3((*pFomul_rough)[spin_index].x, (*pFmul_rough)[spin_index].y, (*pFomul_rough)[spin_index].z),
			cuReal3((*pFomul_rough)[spin_index].y, (*pFomul_rough)[spin_index].z, (*pFmul_rough)[spin_index].z));

		cuReal3 Hrough = Fmat * M[spin_index];

		if (Mnew != cuReal3()) {

			cuReal3 Hrough_new = Fmat * Mnew;

			return -M.h.dim() * (cuBReal)MU0 * (Hrough_new * Mnew - Hrough * M[spin_index]);
		}
		else return -M.h.dim() * (cuBReal)MU0 * Hrough * M[spin_index];
	}
	else return 0.0;
}

//Antiferromagnetic
__device__ cuReal2 ManagedMeshCUDA::Get_EnergyChange_AFM_RoughnessCUDA(int spin_index, cuReal3 Mnew_A, cuReal3 Mnew_B)
{
	if (pFmul_rough && pFmul_rough->linear_size()) {

		cuVEC_VC<cuReal3>& M = *pM;
		cuVEC_VC<cuReal3>& M2 = *pM2;

		cuReal33 Fmat = cuReal33(
			cuReal3((*pFmul_rough)[spin_index].x, (*pFomul_rough)[spin_index].x, (*pFomul_rough)[spin_index].y),
			cuReal3((*pFomul_rough)[spin_index].x, (*pFmul_rough)[spin_index].y, (*pFomul_rough)[spin_index].z),
			cuReal3((*pFomul_rough)[spin_index].y, (*pFomul_rough)[spin_index].z, (*pFmul_rough)[spin_index].z));

		cuBReal energy_ = 0.0;

		cuReal3 Mval = (M[spin_index] + M2[spin_index]) / 2;
		cuReal3 Hrough = Fmat * Mval;

		if (Mnew_A != cuReal3() && Mnew_B != cuReal3()) {

			cuReal3 Mvalnew = (Mnew_A + Mnew_B) / 2;
			cuReal3 Hrough_new = Fmat * Mvalnew;

			energy_ = -M.h.dim() * MU0 * (Hrough_new * Mvalnew - Hrough * Mval);
		}
		else energy_ = -M.h.dim() * MU0 * Hrough * Mval;

		return cuReal2(energy_, energy_);
	}
	else return cuReal2();
}

#endif