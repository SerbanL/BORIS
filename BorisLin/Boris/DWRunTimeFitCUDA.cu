#include "hip/hip_runtime.h"
#include "DWRunTimeFitCUDA.h"

#if COMPILECUDA == 1

#include "Reduction.cuh"

///////////////////////////////////////////////////////////////
// AUXILIARY

__global__ void Zero_DWRunTimeFitCUDA_Values(cuBReal& As, cuBReal& Ae, cuBReal& x0, cuBReal& dw, size_t& av_points_x0, cuBReal& weight)
{
	if (threadIdx.x == 0) As = 0.0;
	else if (threadIdx.x == 1) Ae = 0.0;
	else if (threadIdx.x == 2) x0 = 0.0;
	else if (threadIdx.x == 3) dw = 0.0;
	else if (threadIdx.x == 4) av_points_x0 = 0;
	else if (threadIdx.x == 5) weight = 0.0;
}

///////////////////////////////////////////////////////////////
// PARALLEL MONTE-CARLO METROPOLIS - WITH REDUCTION

//reduce start and end values : finish average at the end by dividing by av_points_s and av_points_e
__global__ void DWRunTimeFitCUDA_endpoints_kernel(size_t size, cuReal2* pxy_data, cuBReal& As, cuBReal& Ae)
{
	//kernel launched with at least 2*num_end_points : lower for As, upper for Ae. Better than launching 2 separate kernels as these are usually very small.
	int idx = blockIdx.x * blockDim.x + threadIdx.x;

	cuBReal value_s = 0.0, value_e = 0.0;

	int num_end_points = size * (cuBReal)DWPOS_ENDSTENCIL;

	if (idx < num_end_points) {

		value_s = pxy_data[idx].j / num_end_points;
	}
	else if (idx < 2 * num_end_points) {

		value_e = pxy_data[size - 2 * num_end_points + idx].j / num_end_points;
	}

	reduction_sum(0, 1, &value_s, As);
	reduction_sum(0, 1, &value_e, Ae);
}

__global__ void DWRunTimeFitCUDA_smoothing_kernel(cuReal2* pxy_data, size_t size_smoothed, int num_stencil_points, cuReal2* pxy_data_smoothed)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;

	if (idx < size_smoothed) {

		idx += num_stencil_points;

		cuBReal value = 0.0;
		for (int sidx = idx - num_stencil_points; sidx < idx + num_stencil_points; sidx++) {

			value += pxy_data[sidx].j / (2 * num_stencil_points + 1);
		}

		pxy_data_smoothed[idx - num_stencil_points].i = pxy_data[idx].i;
		pxy_data_smoothed[idx - num_stencil_points].j = value;
	}
}

__global__ void DWRunTimeFitCUDA_x0_kernel(size_t size_smoothed, cuReal2* pxy_data_smoothed, cuBReal& As, cuBReal& Ae, cuBReal& x0, size_t& av_points_x0)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;

	cuBReal x0value = 0.0;
	bool include_in_average = false;

	if (idx < size_smoothed - 1) {

		cuBReal ycentre = (As + Ae) / 2;

		if ((pxy_data_smoothed[idx].j - ycentre) * (pxy_data_smoothed[idx + 1].j - ycentre) < 0) {

			x0value = cu_interpolate(
				cuReal2(pxy_data_smoothed[idx].j, pxy_data_smoothed[idx].i),
				cuReal2(pxy_data_smoothed[idx + 1].j, pxy_data_smoothed[idx + 1].i),
				ycentre);

			include_in_average = true;
		}
	}

	reduction_avg(0, 1, &x0value, x0, av_points_x0, include_in_average);
}

__global__ void DWRunTimeFitCUDA_dw_kernel(size_t size, cuReal2* pxy_data, cuBReal& As, cuBReal& Ae, cuBReal& x0, size_t& av_points_x0, cuBReal& dw, cuBReal& weight)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;

	//domain wall width value times weight
	cuBReal w_i_dw_i = 0.0;
	//weight only (need to reduce to find total weight)
	cuBReal w_i = 0.0;
	bool include_in_average = false;

	int num_end_points = size * (cuBReal)DWPOS_ENDSTENCIL;

	if (idx < size - num_end_points && idx >= num_end_points) {
		
		cuBReal c = (As + Ae) / 2;
		cuBReal m = (As - Ae) / 2;

		//function is f(x) = [ (As - Ae) * tanh(-PI * (x - x0) / dw) + (As + Ae) ] / 2 = m * tanh(-PI*(x - x0) / dw) + c
		//at each point find f(x), then solve for a dw value and an attached weight obtained from least squares equation. Obtain final domain wall width using weighted average.
		cuBReal nval = (pxy_data[idx].j - c) / m;
		if (abs(nval) < (cuBReal)DWPOS_YTHRESHOLD_MAX && abs(nval) > (cuBReal)DWPOS_YTHRESHOLD_MIN) {

			//domain wall width for this point
			cuBReal dw_i = fabs(PI * (pxy_data[idx].i - x0) / atanh(nval));

			if (dw_i) {

				//function evaluated for dw_i
				cuBReal f_i = m * tanh(-PI * (pxy_data[idx].i - x0) / dw_i) + c;
				//weight for dw_i (obtained from least squares equation)
				w_i = fabs((m*m - f_i * f_i) * (pxy_data[idx].i - x0));

				//total domain wall as weighted sum
				w_i_dw_i = w_i * dw_i;
				include_in_average = true;
			}
		}
	}

	//reduce numerator and denominator for weighted sum
	reduction_sum(0, 1, &w_i_dw_i, dw, include_in_average);
	reduction_sum(0, 1, &w_i, weight, include_in_average);
}

//Fit the extracted profile for position and width, assuming the magnetization component follows f(x) = [ (As - Ae) * tanh(-PI * (x - x0) / dw) + (As + Ae) ] / 2
//Here As, Ae are the start and end values - profile must be long enough to include at least DWPOS_ENDSTENCIL (length ratio) flat parts of the tanh profile
//x0 is the centre position relative to start of profile
//dw is the domain wall width
//xy_data contains profile as x coordinates and corresponding y values
//return x0, dw
cuReal2 DWPosWidthCUDA::FitDomainWallCUDA(double length)
{	
	size_t size = xy_data.size();
	if (size < DWPOS_MINPROFILEPOINTS) return cuReal2();

	//0. profile preparation if needed

	//select base device for processing
	mGPU.select_device(0);

	//which xy_data array should we work with? If only one GPU then just use xy_data; if multiple GPUs, then we'll need to transfer profile to xy_data_base to use
	//either way xy_data_ref will be a cu_arr on the base device containing the extracted profile to process
	cu_arr<cuReal2>& xy_data_ref = (mGPU.get_num_devices() == 1 ? xy_data.get_cu_arr(0) : xy_data_base);

	//for multiple GPUs, now make sure profile is actually in xy_data_base
	if (mGPU.get_num_devices() > 1) {

		//xy_data_base must have correct size to hold full profile
		if (xy_data_base.size() != size) xy_data_base.resize(size);

		//refresh transfer object if needed
		setup_xy_data_transf();

		//now do the transfers
		size_t profile_offset = 0;
		for (int idx = 0; idx < mGPU.get_num_devices(); idx++) {

			//transfer to handle 1 on base device (xy_data_base) from xy_data on device idx its respective number of profile points
			xy_data_transf.transfer(0, 1, idx, 0, xy_data.size(idx), profile_offset, 0);
			profile_offset += xy_data.size(idx);
		}

		//this synchronization is necessary (always synchronize after asynch transfers)
		mGPU.synchronize_if_multi();
	}

	//1. Find end values

	//zero all reduction values

	Zero_DWRunTimeFitCUDA_Values <<< 1, CUDATHREADS >>> (As, Ae, x0, dw, av_points_x0, weight);

	int num_end_points = size * DWPOS_ENDSTENCIL;
	DWRunTimeFitCUDA_endpoints_kernel <<< (2 * num_end_points + CUDATHREADS) / CUDATHREADS, CUDATHREADS >>>
		(size, xy_data_ref, As, Ae);

	//2. Find centre position using calculated ycentre and amplitude values

	int num_stencil_points = size * DWPOS_STENCIL;

	//produce smoothed xy data using nearest neighbor average
	if (xy_data_smoothed_size != size - 2 * num_stencil_points) {

		if (!xy_data_smoothed.resize(size - 2 * num_stencil_points)) return cuReal2();
		else xy_data_smoothed_size = size - 2 * num_stencil_points;
	}

	//smooth
	DWRunTimeFitCUDA_smoothing_kernel <<< (xy_data_smoothed_size + CUDATHREADS) / CUDATHREADS, CUDATHREADS >>>
		(xy_data_ref, xy_data_smoothed_size, num_stencil_points, xy_data_smoothed);

	//find x0
	DWRunTimeFitCUDA_x0_kernel <<< (xy_data_smoothed_size + CUDATHREADS) / CUDATHREADS, CUDATHREADS >> >
		(xy_data_smoothed_size, xy_data_smoothed, As, Ae, x0, av_points_x0);

	//If x0 is not within reasonable bounds then fail
	cuBReal x0_cpu = x0.to_cpu();
	if (x0_cpu < length * DWPOS_ENDSTENCIL || x0_cpu > length * (1.0 - DWPOS_ENDSTENCIL)) return cuReal2();

	//3. Find DW width using x0, As, Ae values

	//find dw
	DWRunTimeFitCUDA_dw_kernel <<< (size + CUDATHREADS) / CUDATHREADS, CUDATHREADS >>>
		(size, xy_data_ref, As, Ae, x0, av_points_x0, dw, weight);

	cuBReal weight_cpu = weight.to_cpu();
	cuBReal dw_cpu = dw.to_cpu();

	if (weight_cpu) dw_cpu /= weight_cpu;
	else return cuReal2();

	//if dw width is not within reasonable bounds then fail
	if (dw_cpu > length * (1.0 - 2 * DWPOS_ENDSTENCIL) || dw_cpu < 0) return cuReal2();

	//remember x0 is relative to start of profile, so caller will have to adjust for this to make it relative to start of mesh
	return cuReal2(x0_cpu, dw_cpu);
}

#endif