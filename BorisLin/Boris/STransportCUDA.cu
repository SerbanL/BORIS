#include "hip/hip_runtime.h"
#include "STransportCUDA.h"

#if COMPILECUDA == 1

#ifdef MODULE_COMPILATION_TRANSPORT

__global__ void Zero_Errors_kernel(cuBReal& max_error, cuBReal& max_value)
{
	if (threadIdx.x == 0) max_error = 0.0;
	else if (threadIdx.x == 1) max_value = 0.0;
}

void STransportCUDA::Zero_Errors(void)
{
	for (mGPU.device_begin(); mGPU != mGPU.device_end(); mGPU++) {

		Zero_Errors_kernel <<< 1, CUDATHREADS >>>
			(max_error(mGPU), max_value(mGPU));
	}
}

#endif

#endif