#include "hip/hip_runtime.h"
#include "Atom_DiffEqCubicCUDA.h"

#if COMPILECUDA == 1
#ifdef ODE_EVAL_COMPILATION_RKCK
#ifdef MESH_COMPILATION_ATOM_CUBIC

#include "Atom_MeshParamsControlCUDA.h"

#include "Reduction.cuh"

//defines evaluation methods kernel launchers

//----------------------------------------- EVALUATIONS : RKCK45

__global__ void RunRKCK45_Step0_withReductions_Kernel(ManagedAtom_DiffEqCubicCUDA& cuaDiffEq, ManagedAtom_MeshCUDA& cuaMesh)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;

	cuBReal dT = *cuaDiffEq.pdT;

	cuBReal mxh = 0.0;

	//multiplicative conversion factor from atomic moment (units of muB) to A/m
	cuBReal conversion = (cuBReal)MUB / cuaMesh.pM1->h.dim();

	if (idx < cuaMesh.pM1->linear_size()) {

		if (cuaMesh.pM1->is_not_empty(idx)) {

			//Save current moment for later use
			(*cuaDiffEq.psM1)[idx] = (*cuaMesh.pM1)[idx];

			if (!cuaMesh.pM1->is_skipcell(idx)) {

				//obtain maximum normalized torque term
				cuBReal Mnorm = (*cuaMesh.pM1)[idx].norm();
				mxh = cu_GetMagnitude((*cuaMesh.pM1)[idx] ^ (*cuaMesh.pHeff1)[idx]) / (conversion * Mnorm * Mnorm);

				//First evaluate RHS of set equation at the current time step
				(*cuaDiffEq.psEval0)[idx] = (cuaDiffEq.*(cuaDiffEq.pODEFunc))(idx);

				//Now estimate moment using RKCK first step
				(*cuaMesh.pM1)[idx] += (*cuaDiffEq.psEval0)[idx] * (dT / 5);
			}
		}
	}

	if (cuaMesh.pgrel->get0()) reduction_max(0, 1, &mxh, *cuaDiffEq.pmxh);
}

__global__ void RunRKCK45_Step0_Kernel(ManagedAtom_DiffEqCubicCUDA& cuaDiffEq, ManagedAtom_MeshCUDA& cuaMesh)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;

	cuBReal dT = *cuaDiffEq.pdT;

	if (idx < cuaMesh.pM1->linear_size()) {

		if (cuaMesh.pM1->is_not_empty(idx)) {

			//Save current moment for later use
			(*cuaDiffEq.psM1)[idx] = (*cuaMesh.pM1)[idx];

			if (!cuaMesh.pM1->is_skipcell(idx)) {

				//First evaluate RHS of set equation at the current time step
				(*cuaDiffEq.psEval0)[idx] = (cuaDiffEq.*(cuaDiffEq.pODEFunc))(idx);

				//Now estimate moment using RKCK first step
				(*cuaMesh.pM1)[idx] += (*cuaDiffEq.psEval0)[idx] * (dT / 5);
			}
		}
	}
}

__global__ void RunRKCK45_Step1_Kernel(ManagedAtom_DiffEqCubicCUDA& cuaDiffEq, ManagedAtom_MeshCUDA& cuaMesh)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;

	cuBReal dT = *cuaDiffEq.pdT;

	if (idx < cuaMesh.pM1->linear_size()) {

		if (cuaMesh.pM1->is_not_empty(idx) && !cuaMesh.pM1->is_skipcell(idx)) {

			//First evaluate RHS of set equation at the current time step
			(*cuaDiffEq.psEval1)[idx] = (cuaDiffEq.*(cuaDiffEq.pODEFunc))(idx);

			//Now estimate moment using RKCK midle step 1
			(*cuaMesh.pM1)[idx] = (*cuaDiffEq.psM1)[idx] + (3 * (*cuaDiffEq.psEval0)[idx] / 40 + 9 * (*cuaDiffEq.psEval1)[idx] / 40) * dT;
		}
	}
}

__global__ void RunRKCK45_Step2_Kernel(ManagedAtom_DiffEqCubicCUDA& cuaDiffEq, ManagedAtom_MeshCUDA& cuaMesh)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;

	cuBReal dT = *cuaDiffEq.pdT;

	if (idx < cuaMesh.pM1->linear_size()) {

		if (cuaMesh.pM1->is_not_empty(idx) && !cuaMesh.pM1->is_skipcell(idx)) {

			//First evaluate RHS of set equation at the current time step
			(*cuaDiffEq.psEval2)[idx] = (cuaDiffEq.*(cuaDiffEq.pODEFunc))(idx);

			//Now estimate moment using RKCK midle step 2
			(*cuaMesh.pM1)[idx] = (*cuaDiffEq.psM1)[idx] + (3 * (*cuaDiffEq.psEval0)[idx] / 10 - 9 * (*cuaDiffEq.psEval1)[idx] / 10 + 6 * (*cuaDiffEq.psEval2)[idx] / 5) * dT;
		}
	}
}

__global__ void RunRKCK45_Step3_Kernel(ManagedAtom_DiffEqCubicCUDA& cuaDiffEq, ManagedAtom_MeshCUDA& cuaMesh)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;

	cuBReal dT = *cuaDiffEq.pdT;

	if (idx < cuaMesh.pM1->linear_size()) {

		if (cuaMesh.pM1->is_not_empty(idx) && !cuaMesh.pM1->is_skipcell(idx)) {

			//First evaluate RHS of set equation at the current time step
			(*cuaDiffEq.psEval3)[idx] = (cuaDiffEq.*(cuaDiffEq.pODEFunc))(idx);

			//Now estimate moment using RKCK midle step 3
			(*cuaMesh.pM1)[idx] = (*cuaDiffEq.psM1)[idx] + (-11 * (*cuaDiffEq.psEval0)[idx] / 54 + 5 * (*cuaDiffEq.psEval1)[idx] / 2 - 70 * (*cuaDiffEq.psEval2)[idx] / 27 + 35 * (*cuaDiffEq.psEval3)[idx] / 27) * dT;
		}
	}
}

__global__ void RunRKCK45_Step4_Kernel(ManagedAtom_DiffEqCubicCUDA& cuaDiffEq, ManagedAtom_MeshCUDA& cuaMesh)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;

	cuBReal dT = *cuaDiffEq.pdT;

	if (idx < cuaMesh.pM1->linear_size()) {

		if (cuaMesh.pM1->is_not_empty(idx) && !cuaMesh.pM1->is_skipcell(idx)) {

			//First evaluate RHS of set equation at the current time step
			(*cuaDiffEq.psEval4)[idx] = (cuaDiffEq.*(cuaDiffEq.pODEFunc))(idx);

			//Now estimate moment using RKF midle step 4
			(*cuaMesh.pM1)[idx] = (*cuaDiffEq.psM1)[idx] + (1631 * (*cuaDiffEq.psEval0)[idx] / 55296 + 175 * (*cuaDiffEq.psEval1)[idx] / 512 + 575 * (*cuaDiffEq.psEval2)[idx] / 13824 + 44275 * (*cuaDiffEq.psEval3)[idx] / 110592 + 253 * (*cuaDiffEq.psEval4)[idx] / 4096) * dT;
		}
	}
}

__global__ void RunRKCK45_Step5_withReductions_Kernel(ManagedAtom_DiffEqCubicCUDA& cuaDiffEq, ManagedAtom_MeshCUDA& cuaMesh)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;

	cuBReal dT = *cuaDiffEq.pdT;

	cuBReal dmdt = 0.0;
	cuBReal lte = 0.0;

	//multiplicative conversion factor from atomic moment (units of muB) to A/m
	cuBReal conversion = (cuBReal)MUB / cuaMesh.pM1->h.dim();

	if (idx < cuaMesh.pM1->linear_size()) {

		if (cuaMesh.pM1->is_not_empty(idx)) {

			if (!cuaMesh.pM1->is_skipcell(idx)) {

				//First evaluate RHS of set equation at the current time step
				cuReal3 rhs = (cuaDiffEq.*(cuaDiffEq.pODEFunc))(idx);

				//RKCK45 : 4th order evaluation
				(*cuaMesh.pM1)[idx] = (*cuaDiffEq.psM1)[idx] + (2825 * (*cuaDiffEq.psEval0)[idx] / 27648 + 18575 * (*cuaDiffEq.psEval2)[idx] / 48384 + 13525 * (*cuaDiffEq.psEval3)[idx] / 55296 + 277 * (*cuaDiffEq.psEval4)[idx] / 14336 + rhs / 4) * dT;

				//Now calculate 5th order evaluation for adaptive time step
				cuReal3 prediction = (*cuaDiffEq.psM1)[idx] + (37 * (*cuaDiffEq.psEval0)[idx] / 378 + 250 * (*cuaDiffEq.psEval2)[idx] / 621 + 125 * (*cuaDiffEq.psEval3)[idx] / 594 + 512 * rhs / 1771) * dT;

				if (*cuaDiffEq.prenormalize) {

					cuBReal mu_s = *cuaMesh.pmu_s;
					cuaMesh.update_parameters_mcoarse(idx, *cuaMesh.pmu_s, mu_s);
					(*cuaMesh.pM1)[idx].renormalize(mu_s);
				}

				//obtain maximum normalized dmdt term
				cuBReal Mnorm = (*cuaMesh.pM1)[idx].norm();
				dmdt = cu_GetMagnitude((*cuaMesh.pM1)[idx] - (*cuaDiffEq.psM1)[idx]) / (dT * (cuBReal)GAMMA * conversion * Mnorm * Mnorm);

				//local truncation error (between predicted and corrected)
				lte = cu_GetMagnitude((*cuaMesh.pM1)[idx] - prediction) / (*cuaMesh.pM1)[idx].norm();
			}
		}
	}

	if (cuaMesh.pgrel->get0()) reduction_max(0, 1, &dmdt, *cuaDiffEq.pdmdt);
	reduction_max(0, 1, &lte, *cuaDiffEq.plte);
}

__global__ void RunRKCK45_Step5_Kernel(ManagedAtom_DiffEqCubicCUDA& cuaDiffEq, ManagedAtom_MeshCUDA& cuaMesh)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;

	cuBReal dT = *cuaDiffEq.pdT;

	cuBReal lte = 0.0;

	if (idx < cuaMesh.pM1->linear_size()) {

		if (cuaMesh.pM1->is_not_empty(idx)) {

			if (!cuaMesh.pM1->is_skipcell(idx)) {

				//First evaluate RHS of set equation at the current time step
				cuReal3 rhs = (cuaDiffEq.*(cuaDiffEq.pODEFunc))(idx);

				//RKCK45 : 4th order evaluation
				(*cuaMesh.pM1)[idx] = (*cuaDiffEq.psM1)[idx] + (2825 * (*cuaDiffEq.psEval0)[idx] / 27648 + 18575 * (*cuaDiffEq.psEval2)[idx] / 48384 + 13525 * (*cuaDiffEq.psEval3)[idx] / 55296 + 277 * (*cuaDiffEq.psEval4)[idx] / 14336 + rhs / 4) * dT;

				//Now calculate 5th order evaluation for adaptive time step
				cuReal3 prediction = (*cuaDiffEq.psM1)[idx] + (37 * (*cuaDiffEq.psEval0)[idx] / 378 + 250 * (*cuaDiffEq.psEval2)[idx] / 621 + 125 * (*cuaDiffEq.psEval3)[idx] / 594 + 512 * rhs / 1771) * dT;

				if (*cuaDiffEq.prenormalize) {

					cuBReal mu_s = *cuaMesh.pmu_s;
					cuaMesh.update_parameters_mcoarse(idx, *cuaMesh.pmu_s, mu_s);
					(*cuaMesh.pM1)[idx].renormalize(mu_s);
				}

				//local truncation error (between predicted and corrected)
				lte = cu_GetMagnitude((*cuaMesh.pM1)[idx] - prediction) / (*cuaMesh.pM1)[idx].norm();
			}
		}
	}

	reduction_max(0, 1, &lte, *cuaDiffEq.plte);
}

//----------------------------------------- DifferentialEquationCUDA Launchers

//RUNGE KUTTA CASH-KARP

void Atom_DifferentialEquationCubicCUDA::RunRKCK45(int step, bool calculate_mxh, bool calculate_dmdt)
{
	switch (step) {

	case 0:

		if (calculate_mxh) {

			for (mGPU.device_begin(); mGPU != mGPU.device_end(); mGPU++) {

				RunRKCK45_Step0_withReductions_Kernel <<< (paMeshCUDA->M1.device_size(mGPU) + CUDATHREADS) / CUDATHREADS, CUDATHREADS >>> 
					(cuaDiffEq.get_deviceobject(mGPU), paMeshCUDA->cuaMesh.get_deviceobject(mGPU));
			}
		}
		else {

			for (mGPU.device_begin(); mGPU != mGPU.device_end(); mGPU++) {

				RunRKCK45_Step0_Kernel <<< (paMeshCUDA->M1.device_size(mGPU) + CUDATHREADS) / CUDATHREADS, CUDATHREADS >>> 
					(cuaDiffEq.get_deviceobject(mGPU), paMeshCUDA->cuaMesh.get_deviceobject(mGPU));
			}
		}

		break;

	case 1:

		for (mGPU.device_begin(); mGPU != mGPU.device_end(); mGPU++) {

			RunRKCK45_Step1_Kernel <<< (paMeshCUDA->M1.device_size(mGPU) + CUDATHREADS) / CUDATHREADS, CUDATHREADS >>> 
				(cuaDiffEq.get_deviceobject(mGPU), paMeshCUDA->cuaMesh.get_deviceobject(mGPU));
		}

		break;

	case 2:

		for (mGPU.device_begin(); mGPU != mGPU.device_end(); mGPU++) {

			RunRKCK45_Step2_Kernel <<< (paMeshCUDA->M1.device_size(mGPU) + CUDATHREADS) / CUDATHREADS, CUDATHREADS >>> 
				(cuaDiffEq.get_deviceobject(mGPU), paMeshCUDA->cuaMesh.get_deviceobject(mGPU));
		}

		break;

	case 3:

		for (mGPU.device_begin(); mGPU != mGPU.device_end(); mGPU++) {

			RunRKCK45_Step3_Kernel <<< (paMeshCUDA->M1.device_size(mGPU) + CUDATHREADS) / CUDATHREADS, CUDATHREADS >>> 
				(cuaDiffEq.get_deviceobject(mGPU), paMeshCUDA->cuaMesh.get_deviceobject(mGPU));
		}

		break;

	case 4:

		for (mGPU.device_begin(); mGPU != mGPU.device_end(); mGPU++) {

			RunRKCK45_Step4_Kernel <<< (paMeshCUDA->M1.device_size(mGPU) + CUDATHREADS) / CUDATHREADS, CUDATHREADS >>> 
				(cuaDiffEq.get_deviceobject(mGPU), paMeshCUDA->cuaMesh.get_deviceobject(mGPU));
		}

		break;

	case 5:

		if (calculate_dmdt) {

			for (mGPU.device_begin(); mGPU != mGPU.device_end(); mGPU++) {

				RunRKCK45_Step5_withReductions_Kernel <<< (paMeshCUDA->M1.device_size(mGPU) + CUDATHREADS) / CUDATHREADS, CUDATHREADS >>> 
					(cuaDiffEq.get_deviceobject(mGPU), paMeshCUDA->cuaMesh.get_deviceobject(mGPU));
			}
		}
		else {

			for (mGPU.device_begin(); mGPU != mGPU.device_end(); mGPU++) {

				RunRKCK45_Step5_Kernel <<< (paMeshCUDA->M1.device_size(mGPU) + CUDATHREADS) / CUDATHREADS, CUDATHREADS >>> 
					(cuaDiffEq.get_deviceobject(mGPU), paMeshCUDA->cuaMesh.get_deviceobject(mGPU));
			}
		}

		break;
	}
}

#endif
#endif
#endif