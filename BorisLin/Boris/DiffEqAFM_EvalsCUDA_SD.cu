#include "hip/hip_runtime.h"
#include "DiffEqAFMCUDA.h"

#if COMPILECUDA == 1
#ifdef ODE_EVAL_COMPILATION_SD
#ifdef MESH_COMPILATION_ANTIFERROMAGNETIC

#include "MeshParamsControlCUDA.h"

#include "Reduction.cuh"

//defines evaluation methods kernel launchers

//----------------------------------------- EVALUATIONS: SD Solver

__global__ void RunSD_Start_Kernel(ManagedDiffEqAFMCUDA& cuDiffEq, ManagedMeshCUDA& cuMesh)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;

	cuBReal dT = *cuDiffEq.pdT;

	if (idx < cuMesh.pM->linear_size()) {

		if (cuMesh.pM->is_not_empty(idx)) {

			if (!cuMesh.pM->is_skipcell(idx)) {

				/////////////////////////

				cuReal2 Ms_AFM = *cuMesh.pMs_AFM;
				cuReal2 grel_AFM = *cuMesh.pgrel_AFM;
				cuMesh.update_parameters_mcoarse(idx, *cuMesh.pMs_AFM, Ms_AFM, *cuMesh.pgrel_AFM, grel_AFM);

				/////////////////////////

				cuReal3 m = (*cuMesh.pM)[idx] / Ms_AFM.i;
				cuReal3 H = (*cuMesh.pHeff)[idx];

				cuReal3 m2 = (*cuMesh.pM2)[idx] / Ms_AFM.j;
				cuReal3 H2 = (*cuMesh.pHeff2)[idx];

				//calculate m cross Heff (multiplication by GAMMA/2 not necessary as this could be absorbed in the stepsize, but keep it for a more natural step size value from the user point of view - i.e. a time step).
				cuReal3 mxHeff = ((cuBReal)GAMMA / 2) * (m ^ H);
				cuReal3 mxHeff2 = ((cuBReal)GAMMA / 2) * (m2 ^ H2);

				/////////////////////////

				//current torque value G = m x (M x H)
				cuReal3 G = m ^ mxHeff;
				cuReal3 G2 = m2 ^ mxHeff2;

				//save calculated torque for next time
				(*cuDiffEq.psEval0)[idx] = G;
				(*cuDiffEq.psEval0_2)[idx] = G2;

				//save current m for next time
				(*cuDiffEq.psM1)[idx] = m;
				(*cuDiffEq.psM1_2)[idx] = m2;

				/////////////////////////

				//The updating equation is (see https://doi.org/10.1063/1.4862839):

				//m_next = m - (dT/2) * (m_next + m) x ((gamma/2)m x Heff)
				//Here gamma = mu0 * |gamma_e| as usual., m is the current normalized M value, Heff is the current effective field, and we need to find m_next.
				//This is applicable to the LLGStatic approach, i.e. no precession term and damping set to 1.
				//M_next = m_next * Ms

				//The above equation can be solved for m_next explicitly.

				cuBReal s = dT * (cuBReal)GAMMA * grel_AFM.i / 4.0;

				cuReal3 s_mxH = m ^ (s*H);
				m = ((1 - (s_mxH * s_mxH)) * m - 2 * (m ^ s_mxH)) / (1 + (s_mxH * s_mxH));

				cuReal3 s_mxH2 = m2 ^ (s*H2);
				m2 = ((1 - (s_mxH2 * s_mxH2)) * m2 - 2 * (m2 ^ s_mxH2)) / (1 + (s_mxH2 * s_mxH2));

				//set new M
				(*cuMesh.pM)[idx] = m * Ms_AFM.i;
				(*cuMesh.pM2)[idx] = m2 * Ms_AFM.j;

				//renormalize - method is supposed to conserve norm, but best to renormalize anyway.
				(*cuMesh.pM)[idx].renormalize(Ms_AFM.i);
				(*cuMesh.pM2)[idx].renormalize(Ms_AFM.j);

				/////////////////////////
			}
			else {

				cuReal2 Ms_AFM = *cuMesh.pMs_AFM;
				cuMesh.update_parameters_mcoarse(idx, *cuMesh.pMs_AFM, Ms_AFM);
				(*cuMesh.pM)[idx].renormalize(Ms_AFM.i);
				(*cuMesh.pM2)[idx].renormalize(Ms_AFM.j);
			}
		}
	}
}

__global__ void RunSD_BB_Kernel(ManagedDiffEqAFMCUDA& cuDiffEq, ManagedMeshCUDA& cuMesh)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;

	cuBReal _delta_m_sq = 0.0;
	cuBReal _delta_G_sq = 0.0;
	cuBReal _delta_m_dot_delta_G = 0.0;

	cuBReal _delta_m2_sq = 0.0;
	cuBReal _delta_G2_sq = 0.0;
	cuBReal _delta_m2_dot_delta_G2 = 0.0;

	if (idx < cuMesh.pM->linear_size()) {

		if (cuMesh.pM->is_not_empty(idx)) {

			if (!cuMesh.pM->is_skipcell(idx)) {

				/////////////////////////

				cuReal2 Ms_AFM = *cuMesh.pMs_AFM;
				cuMesh.update_parameters_mcoarse(idx, *cuMesh.pMs_AFM, Ms_AFM);

				cuReal3 m = (*cuMesh.pM)[idx] / Ms_AFM.i;
				cuReal3 H = (*cuMesh.pHeff)[idx];

				cuReal3 m2 = (*cuMesh.pM2)[idx] / Ms_AFM.j;
				cuReal3 H2 = (*cuMesh.pHeff2)[idx];

				//calculate m cross Heff (multiplication by GAMMA/2 not necessary as this could be absorbed in the stepsize, but keep it for a more natural step size value from the user point of view - i.e. a time step).
				cuReal3 mxHeff = ((cuBReal)GAMMA / 2) * (m ^ H);
				cuReal3 mxHeff2 = ((cuBReal)GAMMA / 2) * (m2 ^ H2);

				/////////////////////////

				//current torque value G = m x (M x H)
				cuReal3 G = m ^ mxHeff;
				cuReal3 G2 = m2 ^ mxHeff2;

				//change in torque
				//divide by 1e6 to stop the accumulated value having a large exponent -> both num and denom are divided by same value; if exponent too large when dividing num by denom significant loss of precision can occur.
				cuReal3 delta_G = (G - (*cuDiffEq.psEval0)[idx]) / 1e6;
				cuReal3 delta_G2 = (G2 - (*cuDiffEq.psEval0_2)[idx]) / 1e6;

				//save calculated torque for next time
				(*cuDiffEq.psEval0)[idx] = G;
				(*cuDiffEq.psEval0_2)[idx] = G2;

				/////////////////////////

				//change in m
				//divide by 1e6 to stop the accumulated value having a large exponent -> both num and denom are divided by same value; if exponent too large when dividing num by denom significant loss of precision can occur.
				cuReal3 delta_m = (m - (*cuDiffEq.psM1)[idx]) / 1e6;
				cuReal3 delta_m2 = (m2 - (*cuDiffEq.psM1_2)[idx]) / 1e6;

				//save current m for next time
				(*cuDiffEq.psM1)[idx] = m;
				(*cuDiffEq.psM1_2)[idx] = m2;

				/////////////////////////

				//calculate num and denom for the two Barzilai-Borwein stepsize solutions (see Journal of Numerical Analysis (1988) 8, 141-148) so we can find new stepsize
				_delta_m_sq = delta_m * delta_m;
				_delta_G_sq = delta_G * delta_G;
				_delta_m_dot_delta_G = delta_m * delta_G;

				_delta_m2_sq = delta_m2 * delta_m2;
				_delta_G2_sq = delta_G2 * delta_G2;
				_delta_m2_dot_delta_G2 = delta_m2 * delta_G2;
			}
		}
	}

	//the delta_... quantities in which we accumulate are cuBReal (thus likely single precision depending on compilation flag SINGLEPRECISION)
	//Bear in mind this could potentially cause catastrophic loss of precision for large simulations even though we normalized to 1e6*1e6 - unlikely however so leave them like this for now.
	//e.g. when I used these quantities as not normalized the stepsize calculation was all wrong even for medium sized meshes.
	//If you ever have problems then this is the likely culprit
	reduction_sum(0, 1, &_delta_m_sq, *cuDiffEq.pdelta_M_sq);
	reduction_sum(0, 1, &_delta_G_sq, *cuDiffEq.pdelta_G_sq);
	reduction_sum(0, 1, &_delta_m_dot_delta_G, *cuDiffEq.pdelta_M_dot_delta_G);

	reduction_sum(0, 1, &_delta_m2_sq, *cuDiffEq.pdelta_M2_sq);
	reduction_sum(0, 1, &_delta_G2_sq, *cuDiffEq.pdelta_G2_sq);
	reduction_sum(0, 1, &_delta_m2_dot_delta_G2, *cuDiffEq.pdelta_M2_dot_delta_G2);
}

__global__ void RunSD_Advance_withReductions_Kernel(ManagedDiffEqAFMCUDA& cuDiffEq, ManagedMeshCUDA& cuMesh)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;

	cuBReal dT = *cuDiffEq.pdT;

	cuBReal mxh = 0.0;
	cuBReal dmdt = 0.0;

	if (idx < cuMesh.pM->linear_size()) {

		if (cuMesh.pM->is_not_empty(idx)) {

			if (!cuMesh.pM->is_skipcell(idx)) {

				/////////////////////////

				cuReal2 Ms_AFM = *cuMesh.pMs_AFM;
				cuReal2 grel_AFM = *cuMesh.pgrel_AFM;
				cuMesh.update_parameters_mcoarse(idx, *cuMesh.pMs_AFM, Ms_AFM, *cuMesh.pgrel_AFM, grel_AFM);

				cuReal3 m = (*cuMesh.pM)[idx] / Ms_AFM.i;
				cuReal3 H = (*cuMesh.pHeff)[idx];

				cuReal3 m2 = (*cuMesh.pM2)[idx] / Ms_AFM.j;
				cuReal3 H2 = (*cuMesh.pHeff2)[idx];

				//obtained maximum normalized torque term
				cuBReal Mnorm = (*cuMesh.pM)[idx].norm();
				if (cuIsNZ(grel_AFM.i)) mxh = cu_GetMagnitude(m ^ H) / Mnorm;

				//The updating equation is (see https://doi.org/10.1063/1.4862839):

				//m_next = m - (dT/2) * (m_next + m) x ((gamma/2)m x Heff)
				//Here gamma = mu0 * |gamma_e| as usual., m is the current normalized M value, Heff is the current effective field, and we need to find m_next.
				//This is applicable to the LLGStatic approach, i.e. no precession term and damping set to 1.
				//M_next = m_next * Ms

				//The above equation can be solved for m_next explicitly.

				cuBReal s = dT * (cuBReal)GAMMA * grel_AFM.i / 4.0;

				cuReal3 s_mxH = m ^ (s*H);
				m = ((1 - (s_mxH * s_mxH)) * m - 2 * (m ^ s_mxH)) / (1 + (s_mxH * s_mxH));

				cuReal3 s_mxH2 = m2 ^ (s*H2);
				m2 = ((1 - (s_mxH2 * s_mxH2)) * m2 - 2 * (m2 ^ s_mxH2)) / (1 + (s_mxH2 * s_mxH2));

				//set new M
				(*cuMesh.pM)[idx] = m * Ms_AFM.i;
				(*cuMesh.pM2)[idx] = m2 * Ms_AFM.j;

				//renormalize - method is supposed to conserve norm, but best to renormalize anyway.
				(*cuMesh.pM)[idx].renormalize(Ms_AFM.i);
				(*cuMesh.pM2)[idx].renormalize(Ms_AFM.j);

				//obtain maximum normalized dmdt term
				if (cuIsNZ(grel_AFM.i)) dmdt = cu_GetMagnitude((*cuMesh.pM)[idx] - (*cuDiffEq.psM1)[idx]) / (dT * (cuBReal)GAMMA * grel_AFM.i * Mnorm * Mnorm);
			}
			else {

				cuReal2 Ms_AFM = *cuMesh.pMs_AFM;
				cuMesh.update_parameters_mcoarse(idx, *cuMesh.pMs_AFM, Ms_AFM);
				(*cuMesh.pM)[idx].renormalize(Ms_AFM.i);
				(*cuMesh.pM2)[idx].renormalize(Ms_AFM.j);
			}
		}
	}

	reduction_max(0, 1, &mxh, *cuDiffEq.pmxh);
	reduction_max(0, 1, &dmdt, *cuDiffEq.pdmdt);
}

__global__ void RunSD_Advance_withReduction_mxh_Kernel(ManagedDiffEqAFMCUDA& cuDiffEq, ManagedMeshCUDA& cuMesh)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;

	cuBReal dT = *cuDiffEq.pdT;

	cuBReal mxh = 0.0;

	if (idx < cuMesh.pM->linear_size()) {

		if (cuMesh.pM->is_not_empty(idx)) {

			if (!cuMesh.pM->is_skipcell(idx)) {

				/////////////////////////

				cuReal2 Ms_AFM = *cuMesh.pMs_AFM;
				cuReal2 grel_AFM = *cuMesh.pgrel_AFM;
				cuMesh.update_parameters_mcoarse(idx, *cuMesh.pMs_AFM, Ms_AFM, *cuMesh.pgrel_AFM, grel_AFM);

				cuReal3 m = (*cuMesh.pM)[idx] / Ms_AFM.i;
				cuReal3 H = (*cuMesh.pHeff)[idx];

				cuReal3 m2 = (*cuMesh.pM2)[idx] / Ms_AFM.j;
				cuReal3 H2 = (*cuMesh.pHeff2)[idx];

				//obtained maximum normalized torque term
				if (cuIsNZ(grel_AFM.i)) mxh = cu_GetMagnitude(m ^ H) / (*cuMesh.pM)[idx].norm();

				//The updating equation is (see https://doi.org/10.1063/1.4862839):

				//m_next = m - (dT/2) * (m_next + m) x ((gamma/2)m x Heff)
				//Here gamma = mu0 * |gamma_e| as usual., m is the current normalized M value, Heff is the current effective field, and we need to find m_next.
				//This is applicable to the LLGStatic approach, i.e. no precession term and damping set to 1.
				//M_next = m_next * Ms

				//The above equation can be solved for m_next explicitly.

				cuBReal s = dT * (cuBReal)GAMMA * grel_AFM.i / 4.0;

				cuReal3 s_mxH = m ^ (s*H);
				m = ((1 - (s_mxH * s_mxH)) * m - 2 * (m ^ s_mxH)) / (1 + (s_mxH * s_mxH));

				cuReal3 s_mxH2 = m2 ^ (s*H2);
				m2 = ((1 - (s_mxH2 * s_mxH2)) * m2 - 2 * (m2 ^ s_mxH2)) / (1 + (s_mxH2 * s_mxH2));

				//set new M
				(*cuMesh.pM)[idx] = m * Ms_AFM.i;
				(*cuMesh.pM2)[idx] = m2 * Ms_AFM.j;

				//renormalize - method is supposed to conserve norm, but best to renormalize anyway.
				(*cuMesh.pM)[idx].renormalize(Ms_AFM.i);
				(*cuMesh.pM2)[idx].renormalize(Ms_AFM.j);
			}
			else {

				cuReal2 Ms_AFM = *cuMesh.pMs_AFM;
				cuMesh.update_parameters_mcoarse(idx, *cuMesh.pMs_AFM, Ms_AFM);
				(*cuMesh.pM)[idx].renormalize(Ms_AFM.i);
				(*cuMesh.pM2)[idx].renormalize(Ms_AFM.j);
			}
		}
	}

	reduction_max(0, 1, &mxh, *cuDiffEq.pmxh);
}

__global__ void RunSD_Advance_withReduction_dmdt_Kernel(ManagedDiffEqAFMCUDA& cuDiffEq, ManagedMeshCUDA& cuMesh)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;

	cuBReal dT = *cuDiffEq.pdT;

	cuBReal dmdt = 0.0;

	if (idx < cuMesh.pM->linear_size()) {

		if (cuMesh.pM->is_not_empty(idx)) {

			if (!cuMesh.pM->is_skipcell(idx)) {

				/////////////////////////

				cuReal2 Ms_AFM = *cuMesh.pMs_AFM;
				cuReal2 grel_AFM = *cuMesh.pgrel_AFM;
				cuMesh.update_parameters_mcoarse(idx, *cuMesh.pMs_AFM, Ms_AFM, *cuMesh.pgrel_AFM, grel_AFM);

				cuReal3 m = (*cuMesh.pM)[idx] / Ms_AFM.i;
				cuReal3 H = (*cuMesh.pHeff)[idx];

				cuReal3 m2 = (*cuMesh.pM2)[idx] / Ms_AFM.j;
				cuReal3 H2 = (*cuMesh.pHeff2)[idx];

				//The updating equation is (see https://doi.org/10.1063/1.4862839):

				//m_next = m - (dT/2) * (m_next + m) x ((gamma/2)m x Heff)
				//Here gamma = mu0 * |gamma_e| as usual., m is the current normalized M value, Heff is the current effective field, and we need to find m_next.
				//This is applicable to the LLGStatic approach, i.e. no precession term and damping set to 1.
				//M_next = m_next * Ms

				//The above equation can be solved for m_next explicitly.

				cuBReal s = dT * (cuBReal)GAMMA * grel_AFM.i / 4.0;

				cuReal3 s_mxH = m ^ (s*H);
				m = ((1 - (s_mxH * s_mxH)) * m - 2 * (m ^ s_mxH)) / (1 + (s_mxH * s_mxH));

				cuReal3 s_mxH2 = m2 ^ (s*H2);
				m2 = ((1 - (s_mxH2 * s_mxH2)) * m2 - 2 * (m2 ^ s_mxH2)) / (1 + (s_mxH2 * s_mxH2));

				//set new M
				(*cuMesh.pM)[idx] = m * Ms_AFM.i;
				(*cuMesh.pM2)[idx] = m2 * Ms_AFM.j;

				//renormalize - method is supposed to conserve norm, but best to renormalize anyway.
				(*cuMesh.pM)[idx].renormalize(Ms_AFM.i);
				(*cuMesh.pM2)[idx].renormalize(Ms_AFM.j);

				//obtain maximum normalized dmdt term
				if (cuIsNZ(grel_AFM.i)) {

					cuBReal Mnorm = (*cuMesh.pM)[idx].norm();
					dmdt = cu_GetMagnitude((*cuMesh.pM)[idx] - (*cuDiffEq.psM1)[idx]) / (dT * (cuBReal)GAMMA * grel_AFM.i * Mnorm * Mnorm);
				}
			}
			else {

				cuReal2 Ms_AFM = *cuMesh.pMs_AFM;
				cuMesh.update_parameters_mcoarse(idx, *cuMesh.pMs_AFM, Ms_AFM);
				(*cuMesh.pM)[idx].renormalize(Ms_AFM.i);
				(*cuMesh.pM2)[idx].renormalize(Ms_AFM.j);
			}
		}
	}

	reduction_max(0, 1, &dmdt, *cuDiffEq.pdmdt);
}

__global__ void RunSD_Advance_Kernel(ManagedDiffEqAFMCUDA& cuDiffEq, ManagedMeshCUDA& cuMesh)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;

	cuBReal dT = *cuDiffEq.pdT;

	if (idx < cuMesh.pM->linear_size()) {

		if (cuMesh.pM->is_not_empty(idx)) {

			if (!cuMesh.pM->is_skipcell(idx)) {

				/////////////////////////

				cuReal2 Ms_AFM = *cuMesh.pMs_AFM;
				cuReal2 grel_AFM = *cuMesh.pgrel_AFM;
				cuMesh.update_parameters_mcoarse(idx, *cuMesh.pMs_AFM, Ms_AFM, *cuMesh.pgrel_AFM, grel_AFM);

				cuReal3 m = (*cuMesh.pM)[idx] / Ms_AFM.i;
				cuReal3 H = (*cuMesh.pHeff)[idx];

				cuReal3 m2 = (*cuMesh.pM2)[idx] / Ms_AFM.j;
				cuReal3 H2 = (*cuMesh.pHeff2)[idx];

				//The updating equation is (see https://doi.org/10.1063/1.4862839):

				//m_next = m - (dT/2) * (m_next + m) x ((gamma/2)m x Heff)
				//Here gamma = mu0 * |gamma_e| as usual., m is the current normalized M value, Heff is the current effective field, and we need to find m_next.
				//This is applicable to the LLGStatic approach, i.e. no precession term and damping set to 1.
				//M_next = m_next * Ms

				//The above equation can be solved for m_next explicitly.

				cuBReal s = dT * (cuBReal)GAMMA * grel_AFM.i / 4.0;

				cuReal3 s_mxH = m ^ (s*H);
				m = ((1 - (s_mxH * s_mxH)) * m - 2 * (m ^ s_mxH)) / (1 + (s_mxH * s_mxH));

				cuReal3 s_mxH2 = m2 ^ (s*H2);
				m2 = ((1 - (s_mxH2 * s_mxH2)) * m2 - 2 * (m2 ^ s_mxH2)) / (1 + (s_mxH2 * s_mxH2));

				//set new M
				(*cuMesh.pM)[idx] = m * Ms_AFM.i;
				(*cuMesh.pM2)[idx] = m2 * Ms_AFM.j;

				//renormalize - method is supposed to conserve norm, but best to renormalize anyway.
				(*cuMesh.pM)[idx].renormalize(Ms_AFM.i);
				(*cuMesh.pM2)[idx].renormalize(Ms_AFM.j);
			}
			else {

				cuReal2 Ms_AFM = *cuMesh.pMs_AFM;
				cuMesh.update_parameters_mcoarse(idx, *cuMesh.pMs_AFM, Ms_AFM);
				(*cuMesh.pM)[idx].renormalize(Ms_AFM.i);
				(*cuMesh.pM2)[idx].renormalize(Ms_AFM.j);
			}
		}
	}
}

//----------------------------------------- DifferentialEquationCUDA Launchers

//SD Solver

void DifferentialEquationAFMCUDA::RunSD_Start(void)
{
	for (mGPU.device_begin(); mGPU != mGPU.device_end(); mGPU++) {

		RunSD_Start_Kernel <<< (pMeshCUDA->M.device_size(mGPU) + CUDATHREADS) / CUDATHREADS, CUDATHREADS >>> 
			(cuDiffEq.get_deviceobject(mGPU), pMeshCUDA->cuMesh.get_deviceobject(mGPU));
	}
}

void DifferentialEquationAFMCUDA::RunSD_BB(void)
{
	for (mGPU.device_begin(); mGPU != mGPU.device_end(); mGPU++) {

		RunSD_BB_Kernel <<< (pMeshCUDA->M.device_size(mGPU) + CUDATHREADS) / CUDATHREADS, CUDATHREADS >>> 
			(cuDiffEq.get_deviceobject(mGPU), pMeshCUDA->cuMesh.get_deviceobject(mGPU));
	}
}

void DifferentialEquationAFMCUDA::RunSD_Advance(bool calculate_mxh, bool calculate_dmdt)
{
	if (calculate_mxh && calculate_dmdt) {

		for (mGPU.device_begin(); mGPU != mGPU.device_end(); mGPU++) {

			RunSD_Advance_withReductions_Kernel <<< (pMeshCUDA->M.device_size(mGPU) + CUDATHREADS) / CUDATHREADS, CUDATHREADS >>> 
				(cuDiffEq.get_deviceobject(mGPU), pMeshCUDA->cuMesh.get_deviceobject(mGPU));
		}
	}
	else if (calculate_mxh) {

		for (mGPU.device_begin(); mGPU != mGPU.device_end(); mGPU++) {

			RunSD_Advance_withReduction_mxh_Kernel <<< (pMeshCUDA->M.device_size(mGPU) + CUDATHREADS) / CUDATHREADS, CUDATHREADS >>> 
				(cuDiffEq.get_deviceobject(mGPU), pMeshCUDA->cuMesh.get_deviceobject(mGPU));
		}
	}
	else if (calculate_dmdt) {

		for (mGPU.device_begin(); mGPU != mGPU.device_end(); mGPU++) {

			RunSD_Advance_withReduction_dmdt_Kernel <<< (pMeshCUDA->M.device_size(mGPU) + CUDATHREADS) / CUDATHREADS, CUDATHREADS >>> 
				(cuDiffEq.get_deviceobject(mGPU), pMeshCUDA->cuMesh.get_deviceobject(mGPU));
		}
	}
	else {

		for (mGPU.device_begin(); mGPU != mGPU.device_end(); mGPU++) {

			RunSD_Advance_Kernel <<< (pMeshCUDA->M.device_size(mGPU) + CUDATHREADS) / CUDATHREADS, CUDATHREADS >>> 
				(cuDiffEq.get_deviceobject(mGPU), pMeshCUDA->cuMesh.get_deviceobject(mGPU));
		}
	}
}

#endif
#endif
#endif