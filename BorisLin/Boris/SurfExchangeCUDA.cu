#include "hip/hip_runtime.h"
#include "SurfExchangeCUDA.h"

#if COMPILECUDA == 1

#ifdef MODULE_COMPILATION_SURFEXCHANGE

#include "Reduction.cuh"
#include "cuVEC_VC_mcuVEC.cuh"

#include "Mesh_FerromagneticCUDA.h"
#include "MeshParamsControlCUDA.h"

#include "ManagedAtom_MeshCUDA.h"

#include "MeshDefs.h"

///////////////////////////////////////////////////////////////////////////////////////////////////////////////
///////////////////////////////////////////////////////////////////////////////////////////////////////////////
//------------------ SURFACE COUPLING Z STACKING

//Top mesh is ferromagnetic
__global__ void SurfExchangeCUDA_TopFM_UpdateField(ManagedMeshCUDA& cuMesh, ManagedMeshCUDA** ppMesh_Top, size_t coupled_meshes, ManagedModulesCUDA& cuModule, bool do_reduction)
{
	cuVEC_VC<cuReal3>& M = *cuMesh.pM;
	cuVEC<cuReal3>& Heff = *cuMesh.pHeff;

	int idx = blockIdx.x * blockDim.x + threadIdx.x;

	cuSZ3 n = M.n;
	cuReal3 h = M.h;

	cuBReal energy_ = 0.0;

	if (idx < n.x * n.y) {

		int i = idx % n.x;
		int j = idx / n.x;
		int cell_idx = i + j * n.x + (n.z - 1) * n.x * n.y;

		//skip empty cells
		if (M.is_not_empty(cell_idx)) {

			cuBReal Ms = *cuMesh.pMs;
			cuMesh.update_parameters_mcoarse(cell_idx, *cuMesh.pMs, Ms);

			//check all meshes for coupling
			for (int mesh_idx = 0; mesh_idx < coupled_meshes; mesh_idx++) {

				mcuVEC_Managed<cuVEC_VC<cuReal3>, cuReal3>& M_Top = ppMesh_Top[mesh_idx]->pM->mcuvec();

				//relative coordinates to read value from top mesh (the one we're coupling to here) - relative to top mesh
				cuReal3 cell_rel_pos = cuReal3(
					(i + 0.5) * h.x + M.rect.s.x - M_Top.rect.s.x,
					(j + 0.5) * h.y + M.rect.s.y - M_Top.rect.s.y,
					M_Top.h.z / 2);

				//can't couple to an empty cell
				if (!M_Top.rect.contains(cell_rel_pos + M_Top.rect.s) || M_Top.is_empty(cell_rel_pos)) continue;

				cuBReal J1 = *(ppMesh_Top[mesh_idx]->pJ1);
				cuBReal J2 = *(ppMesh_Top[mesh_idx]->pJ2);
				ppMesh_Top[mesh_idx]->update_parameters_atposition(cell_rel_pos, *(ppMesh_Top[mesh_idx]->pJ1), J1, *(ppMesh_Top[mesh_idx]->pJ2), J2);

				//get magnetization value in top mesh cell to couple with
				cuReal3 m_j = cu_normalize(M_Top[cell_rel_pos]);
				cuReal3 m_i = cu_normalize(M[cell_idx]);

				cuBReal dot_prod = m_i * m_j;

				//total surface exchange field in coupling cells, including bilinear and biquadratic terms
				cuReal3 Hsurfexch = (m_j / ((cuBReal)MU0 * Ms * h.z)) * (J1 + 2 * J2 * dot_prod);

				if (do_reduction) {

					energy_ = (-1 * J1 - 2 * J2 * dot_prod) * dot_prod / (h.z * M.get_nonempty_cells());
				}

				Heff[cell_idx] += Hsurfexch;

				//NOTE : we must add into the module display VECs, since there could be 2 contributions for some cells (top and bottom). This is why we had to zero the VECs before calling this kernel.
				if (do_reduction && cuModule.pModule_Heff->linear_size()) (*cuModule.pModule_Heff)[cell_idx] += Hsurfexch;
				if (do_reduction && cuModule.pModule_energy->linear_size()) (*cuModule.pModule_energy)[cell_idx] += energy_ * M.get_nonempty_cells();

				//for each cell, either it's not coupled to any other mesh cell (so we never get here), or else it's coupled to exactly one cell on this surface (thus can stop looping over meshes now)
				break;
			}
		}
	}

	if (do_reduction) reduction_sum(0, 1, &energy_, *cuModule.penergy);
}

//Top mesh is antiferromagnetic
__global__ void SurfExchangeCUDA_TopAFM_UpdateField(ManagedMeshCUDA& cuMesh, ManagedMeshCUDA** ppMesh_Top, size_t coupled_meshes, ManagedModulesCUDA& cuModule, bool do_reduction)
{
	cuVEC_VC<cuReal3>& M = *cuMesh.pM;
	cuVEC<cuReal3>& Heff = *cuMesh.pHeff;

	int idx = blockIdx.x * blockDim.x + threadIdx.x;

	cuSZ3 n = M.n;
	cuReal3 h = M.h;

	cuBReal energy_ = 0.0;

	if (idx < n.x * n.y) {

		int i = idx % n.x;
		int j = idx / n.x;
		int k = idx / (n.x * n.y);
		int cell_idx = i + j * n.x + (n.z - 1) * n.x * n.y;

		//skip empty cells
		if (M.is_not_empty(cell_idx)) {

			cuBReal Ms = *cuMesh.pMs;
			cuMesh.update_parameters_mcoarse(cell_idx, *cuMesh.pMs, Ms);

			//check all meshes for coupling
			for (int mesh_idx = 0; mesh_idx < coupled_meshes; mesh_idx++) {

				mcuVEC_Managed<cuVEC_VC<cuReal3>, cuReal3>& M_Top = ppMesh_Top[mesh_idx]->pM->mcuvec();
				mcuVEC_Managed<cuVEC_VC<cuReal3>, cuReal3>& M2_Top = ppMesh_Top[mesh_idx]->pM2->mcuvec();

				//relative coordinates to read value from top mesh (the one we're coupling to here) - relative to top mesh
				cuReal3 cell_rel_pos = cuReal3(
					(i + 0.5) * h.x + M.rect.s.x - M_Top.rect.s.x,
					(j + 0.5) * h.y + M.rect.s.y - M_Top.rect.s.y,
					M_Top.h.z / 2);

				//can't couple to an empty cell
				if (!M_Top.rect.contains(cell_rel_pos + M_Top.rect.s) || M_Top.is_empty(cell_rel_pos)) continue;

				cuBReal J1 = *(ppMesh_Top[mesh_idx]->pJ1);
				cuBReal J2 = *(ppMesh_Top[mesh_idx]->pJ2);
				ppMesh_Top[mesh_idx]->update_parameters_atposition(cell_rel_pos, *(ppMesh_Top[mesh_idx]->pJ1), J1, *(ppMesh_Top[mesh_idx]->pJ2), J2);

				//get magnetization value in top mesh cell to couple with
				cuReal3 m_j1 = cu_normalize(M_Top[cell_rel_pos]);
				cuReal3 m_j2 = cu_normalize(M2_Top[cell_rel_pos]);
				cuReal3 m_i = cu_normalize(M[cell_idx]);

				//total surface exchange field in coupling cells, including contributions from both sub-lattices
				cuReal3 Hsurfexch = (m_j1 / ((cuBReal)MU0 * Ms * h.z)) * J1 + (m_j2 / ((cuBReal)MU0 * Ms * h.z)) * J2;

				if (do_reduction) {

					energy_ = (-J1 * (m_i * m_j1) - J2 * (m_i * m_j2)) / (h.z * M.get_nonempty_cells());
				}

				Heff[cell_idx] += Hsurfexch;

				if (do_reduction && cuModule.pModule_Heff->linear_size()) (*cuModule.pModule_Heff)[cell_idx] += Hsurfexch;
				if (do_reduction && cuModule.pModule_energy->linear_size()) (*cuModule.pModule_energy)[cell_idx] += energy_ * M.get_nonempty_cells();

				//for each cell, either it's not coupled to any other mesh cell (so we never get here), or else it's coupled to exactly one cell on this surface (thus can stop looping over meshes now)
				break;
			}
		}
	}

	if (do_reduction) reduction_sum(0, 1, &energy_, *cuModule.penergy);
}

//Top mesh is atomistic
__global__ void SurfExchangeCUDA_TopAtom_UpdateField(ManagedMeshCUDA& cuMesh, ManagedAtom_MeshCUDA** ppMesh_Top, size_t coupled_meshes, ManagedModulesCUDA& cuModule, bool do_reduction)
{
	cuVEC_VC<cuReal3>& M = *cuMesh.pM;
	cuVEC<cuReal3>& Heff = *cuMesh.pHeff;

	int idx = blockIdx.x * blockDim.x + threadIdx.x;

	cuSZ3 n = M.n;
	cuReal3 h = M.h;

	cuBReal energy_ = 0.0;

	if (idx < n.x * n.y) {

		int i = idx % n.x;
		int j = idx / n.x;
		int cell_idx = i + j * n.x + (n.z - 1) * n.x * n.y;

		//skip empty cells
		if (M.is_not_empty(cell_idx)) {

			cuBReal Ms = *cuMesh.pMs;
			cuBReal J1 = *cuMesh.pJ1;
			cuBReal J2 = *cuMesh.pJ2;
			cuMesh.update_parameters_mcoarse(cell_idx, *cuMesh.pMs, Ms, *cuMesh.pJ1, J1, *cuMesh.pJ2, J2);

			//check all meshes for coupling
			for (int mesh_idx = 0; mesh_idx < coupled_meshes; mesh_idx++) {

				mcuVEC_Managed<cuVEC_VC<cuReal3>, cuReal3>& M1 = ppMesh_Top[mesh_idx]->pM1->mcuvec();

				//coupling rectangle in atomistic mesh in absolute coordinates
				cuRect rect_c = cuRect(
					cuReal3(i * h.x, j * h.y, M.rect.e.z),
					cuReal3((i + 1) * h.x, (j + 1) * h.y, M1.h.z + M.rect.e.z));
				rect_c += cuReal3(M.rect.s.x, M.rect.s.y, 0.0);

				//cells box in atomistic mesh
				cuBox acells = M1.box_from_rect_min(rect_c);

				cuReal3 m_j = cuReal3();
				for (int ai = acells.s.i; ai < acells.e.i; ai++) {
					for (int aj = acells.s.j; aj < acells.e.j; aj++) {

						cuReal3 rel_pos = cuReal3((ai + 0.5) * M1.h.x, (aj + 0.5) * M1.h.y, M1.h.z / 2);

						if (M1.is_empty(rel_pos)) continue;

						m_j += M1[rel_pos];
					}
				}

				m_j = cu_normalize(m_j);
				cuReal3 m_i = cu_normalize(M[cell_idx]);

				cuBReal dot_prod = m_i * m_j;

				//total surface exchange field in coupling cells, including bilinear and biquadratic terms
				cuReal3 Hsurfexch = (m_j / ((cuBReal)MU0 * Ms * h.z)) * (J1 + 2 * J2 * dot_prod);

				if (do_reduction) {

					energy_ = (-1 * J1 - 2 * J2 * dot_prod) * dot_prod / (h.z * M.get_nonempty_cells());
				}

				Heff[cell_idx] += Hsurfexch;

				//NOTE : we must add into the module display VECs, since there could be 2 contributions for some cells (top and bottom). This is why we had to zero the VECs before calling this kernel.
				if (do_reduction && cuModule.pModule_Heff->linear_size()) (*cuModule.pModule_Heff)[cell_idx] += Hsurfexch;
				if (do_reduction && cuModule.pModule_energy->linear_size()) (*cuModule.pModule_energy)[cell_idx] += energy_ * M.get_nonempty_cells();

				//for each cell, either it's not coupled to any other mesh cell (so we never get here), or else it's coupled to exactly one cell on this surface (thus can stop looping over meshes now)
				break;
			}
		}
	}

	if (do_reduction) reduction_sum(0, 1, &energy_, *cuModule.penergy);
}

//Bottom mesh is ferromagnetic
__global__ void SurfExchangeCUDA_BotFM_UpdateField(ManagedMeshCUDA& cuMesh, ManagedMeshCUDA** ppMesh_Bot, size_t coupled_meshes, ManagedModulesCUDA& cuModule, bool do_reduction)
{
	cuVEC_VC<cuReal3>& M = *cuMesh.pM;
	cuVEC<cuReal3>& Heff = *cuMesh.pHeff;

	int idx = blockIdx.x * blockDim.x + threadIdx.x;

	cuSZ3 n = M.n;
	cuReal3 h = M.h;

	cuBReal energy_ = 0.0;

	if (idx < n.x * n.y) {

		int i = idx % n.x;
		int j = idx / n.x;
		int cell_idx = i + j * n.x;

		//skip empty cells
		if (M.is_not_empty(cell_idx)) {

			cuBReal Ms = *cuMesh.pMs;
			cuBReal J1 = *cuMesh.pJ1;
			cuBReal J2 = *cuMesh.pJ2;
			cuMesh.update_parameters_mcoarse(cell_idx, *cuMesh.pMs, Ms, *cuMesh.pJ1, J1, *cuMesh.pJ2, J2);
			
			//check all meshes for coupling
			for (int mesh_idx = 0; mesh_idx < coupled_meshes; mesh_idx++) {

				mcuVEC_Managed<cuVEC_VC<cuReal3>, cuReal3>& M_Bot = ppMesh_Bot[mesh_idx]->pM->mcuvec();

				//relative coordinates to read value from bottom mesh (the one we're coupling to here) - relative to bottom mesh
				cuReal3 cell_rel_pos = cuReal3(
					(i + 0.5) * h.x + M.rect.s.x - M_Bot.rect.s.x,
					(j + 0.5) * h.y + M.rect.s.y - M_Bot.rect.s.y,
					M_Bot.rect.height() - M_Bot.h.z / 2);

				//can't couple to an empty cell
				if (!M_Bot.rect.contains(cell_rel_pos + M_Bot.rect.s) || M_Bot.is_empty(cell_rel_pos)) continue;

				//get value of magnetization used in coupling with current cell at cell_idx
				cuReal3 m_j = cu_normalize(M_Bot[cell_rel_pos]);
				cuReal3 m_i = cu_normalize(M[cell_idx]);

				cuBReal dot_prod = m_i * m_j;

				//total surface exchange field in coupling cells, including bilinear and biquadratic terms
				cuReal3 Hsurfexch = (m_j / ((cuBReal)MU0 * Ms * h.z)) * (J1 + 2 * J2 * dot_prod);

				if (do_reduction) {

					energy_ = (-1 * J1 - 2 * J2 * dot_prod) * dot_prod / (h.z * M.get_nonempty_cells());
				}

				Heff[cell_idx] += Hsurfexch;

				if (do_reduction && cuModule.pModule_Heff->linear_size()) (*cuModule.pModule_Heff)[cell_idx] += Hsurfexch;
				if (do_reduction && cuModule.pModule_energy->linear_size()) (*cuModule.pModule_energy)[cell_idx] += energy_ * M.get_nonempty_cells();

				//for each cell, either it's not coupled to any other mesh cell (so we never get here), or else it's coupled to exactly one cell on this surface (thus can stop looping over meshes now)
				break;
			}
		}
	}

	if (do_reduction) reduction_sum(0, 1, &energy_, *cuModule.penergy);
}

//Bottom mesh is antiferromagnetic
__global__ void SurfExchangeCUDA_BotAFM_UpdateField(ManagedMeshCUDA& cuMesh, ManagedMeshCUDA** ppMesh_Bot, size_t coupled_meshes, ManagedModulesCUDA& cuModule, bool do_reduction)
{
	cuVEC_VC<cuReal3>& M = *cuMesh.pM;
	cuVEC<cuReal3>& Heff = *cuMesh.pHeff;

	int idx = blockIdx.x * blockDim.x + threadIdx.x;

	cuSZ3 n = M.n;
	cuReal3 h = M.h;

	cuBReal energy_ = 0.0;

	if (idx < n.x * n.y) {

		int i = idx % n.x;
		int j = idx / n.x;
		int k = idx / (n.x * n.y);
		int cell_idx = i + j * n.x;

		//skip empty cells
		if (M.is_not_empty(cell_idx)) {

			cuBReal Ms = *cuMesh.pMs;
			cuBReal J1 = *cuMesh.pJ1;
			cuBReal J2 = *cuMesh.pJ2;
			cuMesh.update_parameters_mcoarse(cell_idx, *cuMesh.pMs, Ms, *cuMesh.pJ1, J1, *cuMesh.pJ2, J2);

			//check all meshes for coupling
			for (int mesh_idx = 0; mesh_idx < coupled_meshes; mesh_idx++) {

				mcuVEC_Managed<cuVEC_VC<cuReal3>, cuReal3>& M_Bot = ppMesh_Bot[mesh_idx]->pM->mcuvec();
				mcuVEC_Managed<cuVEC_VC<cuReal3>, cuReal3>& M2_Bot = ppMesh_Bot[mesh_idx]->pM2->mcuvec();

				//relative coordinates to read value from bottom mesh (the one we're coupling to here) - relative to bottom mesh
				cuReal3 cell_rel_pos = cuReal3(
					(i + 0.5) * h.x + M.rect.s.x - M_Bot.rect.s.x,
					(j + 0.5) * h.y + M.rect.s.y - M_Bot.rect.s.y,
					M_Bot.rect.height() - M_Bot.h.z / 2);

				//can't couple to an empty cell
				if (!M_Bot.rect.contains(cell_rel_pos + M_Bot.rect.s) || M_Bot.is_empty(cell_rel_pos)) continue;

				//yes, then get value of magnetization used in coupling with current cell at cell_idx
				cuReal3 m_j1 = cu_normalize(M_Bot[cell_rel_pos]);
				cuReal3 m_j2 = cu_normalize(M2_Bot[cell_rel_pos]);
				cuReal3 m_i = cu_normalize(M[cell_idx]);

				//total surface exchange field in coupling cells, including bilinear and biquadratic terms
				cuReal3 Hsurfexch = (m_j1 / ((cuBReal)MU0 * Ms * h.z)) * J1 + (m_j2 / ((cuBReal)MU0 * Ms * h.z)) * J2;

				if (do_reduction) {

					energy_ = (-J1 * (m_i * m_j1) - J2 * (m_i * m_j2)) / (h.z * M.get_nonempty_cells());
				}

				Heff[cell_idx] += Hsurfexch;

				if (do_reduction && cuModule.pModule_Heff->linear_size()) (*cuModule.pModule_Heff)[cell_idx] += Hsurfexch;
				if (do_reduction && cuModule.pModule_energy->linear_size()) (*cuModule.pModule_energy)[cell_idx] += energy_ * M.get_nonempty_cells();

				//for each cell, either it's not coupled to any other mesh cell (so we never get here), or else it's coupled to exactly one cell on this surface (thus can stop looping over meshes now)
				break;
			}
		}
	}

	if (do_reduction) reduction_sum(0, 1, &energy_, *cuModule.penergy);
}

//Bottom mesh is atomistic
__global__ void SurfExchangeCUDA_BotAtom_UpdateField(ManagedMeshCUDA& cuMesh, ManagedAtom_MeshCUDA** ppMesh_Bot, size_t coupled_meshes, ManagedModulesCUDA& cuModule, bool do_reduction)
{
	cuVEC_VC<cuReal3>& M = *cuMesh.pM;
	cuVEC<cuReal3>& Heff = *cuMesh.pHeff;

	int idx = blockIdx.x * blockDim.x + threadIdx.x;

	cuSZ3 n = M.n;
	cuReal3 h = M.h;

	cuBReal energy_ = 0.0;

	if (idx < n.x * n.y) {

		int i = idx % n.x;
		int j = idx / n.x;
		int cell_idx = i + j * n.x;

		//skip empty cells
		if (M.is_not_empty(cell_idx)) {

			cuBReal Ms = *cuMesh.pMs;
			cuBReal J1 = *cuMesh.pJ1;
			cuBReal J2 = *cuMesh.pJ2;
			cuMesh.update_parameters_mcoarse(cell_idx, *cuMesh.pMs, Ms, *cuMesh.pJ1, J1, *cuMesh.pJ2, J2);

			//check all meshes for coupling
			for (int mesh_idx = 0; mesh_idx < coupled_meshes; mesh_idx++) {

				mcuVEC_Managed<cuVEC_VC<cuReal3>, cuReal3>& M1 = ppMesh_Bot[mesh_idx]->pM1->mcuvec();

				//coupling rectangle in atomistic mesh in absolute coordinates
				cuRect rect_c = cuRect(
					cuReal3(i * h.x, j * h.y, M1.rect.e.z - M1.h.z),
					cuReal3((i + 1) * h.x, (j + 1) * h.y, M1.rect.e.z));
				rect_c += cuReal3(M.rect.s.x, M.rect.s.y, 0.0);

				//cells box in atomistic mesh
				cuBox acells = M1.box_from_rect_min(rect_c);

				cuReal3 m_j = cuReal3();
				for (int ai = acells.s.i; ai < acells.e.i; ai++) {
					for (int aj = acells.s.j; aj < acells.e.j; aj++) {

						cuReal3 rel_pos = cuReal3((ai + 0.5) * M1.h.x, (aj + 0.5) * M1.h.y, M1.rect.e.z - M1.h.z / 2);

						if (M1.is_empty(rel_pos)) continue;

						m_j += M1[rel_pos];
					}
				}

				m_j = cu_normalize(m_j);
				cuReal3 m_i = cu_normalize(M[cell_idx]);

				cuBReal dot_prod = m_i * m_j;

				//total surface exchange field in coupling cells, including bilinear and biquadratic terms
				cuReal3 Hsurfexch = (m_j / ((cuBReal)MU0 * Ms * h.z)) * (J1 + 2 * J2 * dot_prod);

				if (do_reduction) {

					energy_ = (-1 * J1 - 2 * J2 * dot_prod) * dot_prod / (h.z * M.get_nonempty_cells());
				}

				Heff[cell_idx] += Hsurfexch;

				if (do_reduction && cuModule.pModule_Heff->linear_size()) (*cuModule.pModule_Heff)[cell_idx] += Hsurfexch;
				if (do_reduction && cuModule.pModule_energy->linear_size()) (*cuModule.pModule_energy)[cell_idx] += energy_ * M.get_nonempty_cells();

				//for each cell, either it's not coupled to any other mesh cell (so we never get here), or else it's coupled to exactly one cell on this surface (thus can stop looping over meshes now)
				break;
			}
		}
	}

	if (do_reduction) reduction_sum(0, 1, &energy_, *cuModule.penergy);
}

///////////////////////////////////////////////////////////////////////////////////////////////////////////////
///////////////////////////////////////////////////////////////////////////////////////////////////////////////
//------------------ SURFACE COUPLING FOR OVERLAPPING MESHES

__global__ void SurfExchangeFMCUDA_Bulk_UpdateField(
	ManagedMeshCUDA& cuMesh,
	cuVEC<cuINT3>& bulk_coupling_mask,
	ManagedMeshCUDA** ppMeshFM_Bulk, size_t coupledFM_meshes, 
	ManagedMeshCUDA** ppMeshAFM_Bulk, size_t coupledAFM_meshes,
	ManagedAtom_MeshCUDA** ppaMesh_Bulk, size_t coupled_ameshes,
	ManagedModulesCUDA& cuModule, bool do_reduction)
{
	//------------------ Coupling functions

	auto calculate_mm_FM_coupling = [](
		cuVEC_VC<cuReal3>& M, int cell_idx,
		ManagedMeshCUDA& MeshCoupled, cuReal3 cell_rel_pos,
		cuBReal Ms, cuBReal J1, cuBReal J2, cuBReal cell_size,
		cuReal3& Hsurfexch, cuBReal& cell_energy, bool do_reduction) -> void
	{
		//NOTE : no need to check here if pMeshCoupled->M contains cell_rel_pos, or if cell is not empty there.
		//This check is done before calling this function for z stacking, or when initializing bulk_coupling_mask for bulk coupling.

		//Surface exchange field from a ferromagnetic mesh (RKKY)

		mcuVEC_Managed<cuVEC_VC<cuReal3>, cuReal3>& M_Bulk = MeshCoupled.pM->mcuvec();

		cuReal3 m_j = cu_normalize(M_Bulk[cell_rel_pos]);
		cuReal3 m_i = cu_normalize(M[cell_idx]);

		cuBReal dot_prod = m_i * m_j;

		//total surface exchange field in coupling cells, including bilinear and biquadratic terms
		Hsurfexch += (m_j / ((cuBReal)MU0 * Ms * cell_size)) * (J1 + 2 * J2 * dot_prod);
		if (do_reduction) cell_energy += (-1 * J1 - 2 * J2 * dot_prod) * dot_prod / (cell_size * M.get_nonempty_cells());
	};

	auto calculate_mm_AFM_coupling = [](
		cuVEC_VC<cuReal3>& M, int cell_idx,
		ManagedMeshCUDA& MeshCoupled, cuReal3 cell_rel_pos,
		cuBReal Ms, cuBReal J1, cuBReal J2, cuBReal cell_size,
		cuReal3& Hsurfexch, cuBReal& cell_energy, bool do_reduction) -> void
	{
		//NOTE : no need to check here if pMeshCoupled->M contains cell_rel_pos, or if cell is not empty there.
		//This check is done before calling this function for z stacking, or when initializing bulk_coupling_mask for bulk coupling.

		//Surface exchange field from an antiferromagnetic mesh (exchange bias)

		mcuVEC_Managed<cuVEC_VC<cuReal3>, cuReal3>& M_Bulk = MeshCoupled.pM->mcuvec();
		mcuVEC_Managed<cuVEC_VC<cuReal3>, cuReal3>& M2_Bulk = MeshCoupled.pM2->mcuvec();

		cuReal3 m_j1 = cu_normalize(M_Bulk[cell_rel_pos]);
		cuReal3 m_j2 = cu_normalize(M2_Bulk[cell_rel_pos]);
		cuReal3 m_i = cu_normalize(M[cell_idx]);

		//total surface exchange field in coupling cells, including contributions from both sub-lattices
		Hsurfexch += (m_j1 / ((cuBReal)MU0 * Ms * cell_size)) * J1;
		Hsurfexch += (m_j2 / ((cuBReal)MU0 * Ms * cell_size)) * J2;
		if (do_reduction) cell_energy += (-J1 * (m_i * m_j1) - J2 * (m_i * m_j2)) / (cell_size * M.get_nonempty_cells());
	};

	auto calculate_atom_coupling = [](
		cuVEC_VC<cuReal3>& M, int cell_idx,
		ManagedAtom_MeshCUDA& aMeshCoupled, cuRect& rect_c,
		cuBReal Ms, cuBReal J1, cuBReal J2, cuBReal cell_size,
		cuReal3& Hsurfexch, cuBReal& cell_energy, bool do_reduction) -> void
	{
		mcuVEC_Managed<cuVEC_VC<cuReal3>, cuReal3>& M1 = aMeshCoupled.pM1->mcuvec();

		//cells box in atomistic mesh. NOTE : acells is capped to mesh dimensions, so we are guaranteed proper indexes inside the mesh.
		cuBox acells = M1.box_from_rect_min(rect_c);

		cuReal3 m_j = cuReal3();
		for (int ai = acells.s.i; ai < acells.e.i; ai++) {
			for (int aj = acells.s.j; aj < acells.e.j; aj++) {
				for (int ak = acells.s.k; ak < acells.e.k; ak++) {

					cuReal3 rel_pos = cuReal3((ai + 0.5) * M1.h.x, (aj + 0.5) * M1.h.y, (ak + 0.5) * M1.h.z);

					if (M1.is_empty(rel_pos)) continue;

					m_j += M1[rel_pos];
				}
			}
		}

		m_j = cu_normalize(m_j);
		cuReal3 m_i = cu_normalize(M[cell_idx]);

		cuBReal dot_prod = m_i * m_j;

		//total surface exchange field in coupling cells, including bilinear and biquadratic terms
		Hsurfexch += (m_j / ((cuBReal)MU0 * Ms * cell_size)) * (J1 + 2 * J2 * dot_prod);
		if (do_reduction) cell_energy += (-1 * J1 - 2 * J2 * dot_prod) * dot_prod / (cell_size * M.get_nonempty_cells());
	};

	//------------------

	cuVEC_VC<cuReal3>& M = *cuMesh.pM;
	cuVEC<cuReal3>& Heff = *cuMesh.pHeff;

	int idx = blockIdx.x * blockDim.x + threadIdx.x;

	cuSZ3 n = M.n;
	cuReal3 h = M.h;

	cuBReal energy_ = 0.0;

	if (idx < n.dim()) {

		int i = idx % n.x;
		int j = (idx / n.x) % n.y;
		int k = idx / (n.x * n.y);
		int idx = i + j * n.x + k * n.x * n.y;

		//skip empty cells
		if (M.is_not_empty(idx) && bulk_coupling_mask[idx] != cuINT3()) {
			
			cuBReal Ms = *cuMesh.pMs;
			cuBReal J1 = *cuMesh.pJ1;
			cuBReal J2 = *cuMesh.pJ2;
			cuMesh.update_parameters_mcoarse(idx, *cuMesh.pMs, Ms, *cuMesh.pJ1, J1, *cuMesh.pJ2, J2);

			//surface cell which needs to be exchange coupled
			cuReal3 Hsurfexch = cuReal3();
			int num_couplings = 0;

			cuReal3 abs_pos = M.cellidx_to_position(idx) + M.rect.s;

			//+x coupling direction
			if (bulk_coupling_mask[idx].x & 0x0000ffff) {

				int mesh_idx = (bulk_coupling_mask[idx].x & 0x0000ffff) - 1;
				num_couplings++;

				//coupling for micromagnetic FM mesh
				if (mesh_idx < coupledFM_meshes) {
					calculate_mm_FM_coupling(
						M, idx,
						*ppMeshFM_Bulk[mesh_idx], abs_pos - ppMeshFM_Bulk[mesh_idx]->pM->origin + cuReal3((M.h.x + ppMeshFM_Bulk[mesh_idx]->pM->h.x) / 2, 0, 0),
						Ms, J1, J2, M.h.x,
						Hsurfexch, energy_, do_reduction);
				}
				//coupling for micromagnetic AFM mesh
				else if (mesh_idx < coupledFM_meshes + coupledAFM_meshes) {
					calculate_mm_AFM_coupling(
						M, idx,
						*ppMeshAFM_Bulk[mesh_idx - coupledFM_meshes], abs_pos - ppMeshAFM_Bulk[mesh_idx - coupledFM_meshes]->pM->origin + cuReal3((M.h.x + ppMeshAFM_Bulk[mesh_idx - coupledFM_meshes]->pM->h.x) / 2, 0, 0),
						Ms, J1, J2, M.h.x,
						Hsurfexch, energy_, do_reduction);
				}
				//coupling for atomistic mesh
				else if (mesh_idx < coupledFM_meshes + coupledAFM_meshes + coupled_ameshes) {

					//coupling rectangle in atomistic mesh in absolute coordinates
					cuVEC_VC<cuReal3>& M1 = *ppaMesh_Bulk[mesh_idx - coupledFM_meshes - coupledAFM_meshes]->pM1;
					cuRect rect_c = cuRect(
						cuReal3(abs_pos.x + M.h.x / 2, abs_pos.y - M.h.y / 2, abs_pos.z - M.h.z / 2),
						cuReal3(abs_pos.x + M.h.x / 2 + M1.h.x, abs_pos.y + M.h.y / 2, abs_pos.z + M.h.z / 2));

					calculate_atom_coupling(
						M, idx,
						*ppaMesh_Bulk[mesh_idx - coupledFM_meshes - coupledAFM_meshes], rect_c,
						Ms, J1, J2, M.h.x,
						Hsurfexch, energy_, do_reduction);
				}
			}
			
			//-x coupling direction
			if (bulk_coupling_mask[idx].x & 0xffff0000) {

				int mesh_idx = (bulk_coupling_mask[idx].x >> 16) - 1;
				num_couplings++;

				if (mesh_idx < coupledFM_meshes) {
					calculate_mm_FM_coupling(
						M, idx,
						*ppMeshFM_Bulk[mesh_idx], abs_pos - ppMeshFM_Bulk[mesh_idx]->pM->origin + cuReal3(-(M.h.x + ppMeshFM_Bulk[mesh_idx]->pM->h.x) / 2, 0, 0),
						Ms, J1, J2, M.h.x,
						Hsurfexch, energy_, do_reduction);
				}
				//coupling for micromagnetic AFM mesh
				else if (mesh_idx < coupledFM_meshes + coupledAFM_meshes) {
					calculate_mm_AFM_coupling(
						M, idx,
						*ppMeshAFM_Bulk[mesh_idx - coupledFM_meshes], abs_pos - ppMeshAFM_Bulk[mesh_idx - coupledFM_meshes]->pM->origin + cuReal3(-(M.h.x + ppMeshAFM_Bulk[mesh_idx - coupledFM_meshes]->pM->h.x) / 2, 0, 0),
						Ms, J1, J2, M.h.x,
						Hsurfexch, energy_, do_reduction);
				}
				//coupling for atomistic mesh
				else if (mesh_idx < coupledFM_meshes + coupledAFM_meshes + coupled_ameshes) {

					//coupling rectangle in atomistic mesh in absolute coordinates
					cuVEC_VC<cuReal3>& M1 = *ppaMesh_Bulk[mesh_idx - coupledFM_meshes - coupledAFM_meshes]->pM1;
					cuRect rect_c = cuRect(
						cuReal3(abs_pos.x - M.h.x / 2 - M1.h.x, abs_pos.y - M.h.y / 2, abs_pos.z - M.h.z / 2),
						cuReal3(abs_pos.x - M.h.x / 2, abs_pos.y + M.h.y / 2, abs_pos.z + M.h.z / 2));

					calculate_atom_coupling(
						M, idx,
						*ppaMesh_Bulk[mesh_idx - coupledFM_meshes - coupledAFM_meshes], rect_c,
						Ms, J1, J2, M.h.x,
						Hsurfexch, energy_, do_reduction);
				}
			}

			//+y coupling direction
			if (bulk_coupling_mask[idx].y & 0x0000ffff) {

				int mesh_idx = (bulk_coupling_mask[idx].y & 0x0000ffff) - 1;
				num_couplings++;

				//coupling for micromagnetic mesh
				if (mesh_idx < coupledFM_meshes) {
					calculate_mm_FM_coupling(
						M, idx,
						*ppMeshFM_Bulk[mesh_idx], abs_pos - ppMeshFM_Bulk[mesh_idx]->pM->origin + cuReal3(0, (M.h.y + ppMeshFM_Bulk[mesh_idx]->pM->h.y) / 2, 0),
						Ms, J1, J2, M.h.y,
						Hsurfexch, energy_, do_reduction);
				}
				//coupling for micromagnetic AFM mesh
				else if (mesh_idx < coupledFM_meshes + coupledAFM_meshes) {
					calculate_mm_AFM_coupling(
						M, idx,
						*ppMeshAFM_Bulk[mesh_idx - coupledFM_meshes], abs_pos - ppMeshAFM_Bulk[mesh_idx - coupledFM_meshes]->pM->origin + cuReal3(0, (M.h.y + ppMeshAFM_Bulk[mesh_idx - coupledFM_meshes]->pM->h.y) / 2, 0),
						Ms, J1, J2, M.h.y,
						Hsurfexch, energy_, do_reduction);
				}
				//coupling for atomistic mesh
				else if (mesh_idx < coupledFM_meshes + coupledAFM_meshes + coupled_ameshes) {

					//coupling rectangle in atomistic mesh in absolute coordinates
					cuVEC_VC<cuReal3>& M1 = *ppaMesh_Bulk[mesh_idx - coupledFM_meshes - coupledAFM_meshes]->pM1;
					cuRect rect_c = cuRect(
						cuReal3(abs_pos.x - M.h.x / 2, abs_pos.y + M.h.y / 2, abs_pos.z - M.h.z / 2),
						cuReal3(abs_pos.x + M.h.x / 2, abs_pos.y + M.h.y / 2 + M1.h.y, abs_pos.z + M.h.z / 2));

					calculate_atom_coupling(
						M, idx,
						*ppaMesh_Bulk[mesh_idx - coupledFM_meshes - coupledAFM_meshes], rect_c,
						Ms, J1, J2, M.h.y,
						Hsurfexch, energy_, do_reduction);
				}
			}

			//-y coupling direction
			if (bulk_coupling_mask[idx].y & 0xffff0000) {

				int mesh_idx = (bulk_coupling_mask[idx].y >> 16) - 1;
				num_couplings++;

				//coupling for micromagnetic mesh
				if (mesh_idx < coupledFM_meshes) {
					calculate_mm_FM_coupling(
						M, idx,
						*ppMeshFM_Bulk[mesh_idx], abs_pos - ppMeshFM_Bulk[mesh_idx]->pM->origin + cuReal3(0, -(M.h.y + ppMeshFM_Bulk[mesh_idx]->pM->h.y) / 2, 0),
						Ms, J1, J2, M.h.y,
						Hsurfexch, energy_, do_reduction);
				}
				//coupling for micromagnetic AFM mesh
				else if (mesh_idx < coupledFM_meshes + coupledAFM_meshes) {
					calculate_mm_AFM_coupling(
						M, idx,
						*ppMeshAFM_Bulk[mesh_idx - coupledFM_meshes], abs_pos - ppMeshAFM_Bulk[mesh_idx - coupledFM_meshes]->pM->origin + cuReal3(0, -(M.h.y + ppMeshAFM_Bulk[mesh_idx - coupledFM_meshes]->pM->h.y) / 2, 0),
						Ms, J1, J2, M.h.y,
						Hsurfexch, energy_, do_reduction);
				}
				//coupling for atomistic mesh
				else if (mesh_idx < coupledFM_meshes + coupledAFM_meshes + coupled_ameshes) {

					//coupling rectangle in atomistic mesh in absolute coordinates
					cuVEC_VC<cuReal3>& M1 = *ppaMesh_Bulk[mesh_idx - coupledFM_meshes - coupledAFM_meshes]->pM1;
					cuRect rect_c = cuRect(
						cuReal3(abs_pos.x - M.h.x / 2, abs_pos.y - M.h.y / 2 - M1.h.y, abs_pos.z - M.h.z / 2),
						cuReal3(abs_pos.x + M.h.x / 2, abs_pos.y - M.h.y / 2, abs_pos.z + M.h.z / 2));

					calculate_atom_coupling(
						M, idx,
						*ppaMesh_Bulk[mesh_idx - coupledFM_meshes - coupledAFM_meshes], rect_c,
						Ms, J1, J2, M.h.y,
						Hsurfexch, energy_, do_reduction);
				}
			}

			//+z coupling direction
			if (bulk_coupling_mask[idx].z & 0x0000ffff) {

				int mesh_idx = (bulk_coupling_mask[idx].z & 0x0000ffff) - 1;
				num_couplings++;

				//coupling for micromagnetic mesh
				if (mesh_idx < coupledFM_meshes) {
					calculate_mm_FM_coupling(
						M, idx,
						*ppMeshFM_Bulk[mesh_idx], abs_pos - ppMeshFM_Bulk[mesh_idx]->pM->origin + cuReal3(0, 0, (M.h.z + ppMeshFM_Bulk[mesh_idx]->pM->h.z) / 2),
						Ms, J1, J2, M.h.z,
						Hsurfexch, energy_, do_reduction);
				}
				//coupling for micromagnetic AFM mesh
				else if (mesh_idx < coupledFM_meshes + coupledAFM_meshes) {
					calculate_mm_AFM_coupling(
						M, idx,
						*ppMeshAFM_Bulk[mesh_idx - coupledFM_meshes], abs_pos - ppMeshAFM_Bulk[mesh_idx - coupledFM_meshes]->pM->origin + cuReal3(0, 0, (M.h.z + ppMeshAFM_Bulk[mesh_idx - coupledFM_meshes]->pM->h.z) / 2),
						Ms, J1, J2, M.h.z,
						Hsurfexch, energy_, do_reduction);
				}
				//coupling for atomistic mesh
				else if (mesh_idx < coupledFM_meshes + coupledAFM_meshes + coupled_ameshes) {

					//coupling rectangle in atomistic mesh in absolute coordinates
					cuVEC_VC<cuReal3>& M1 = *ppaMesh_Bulk[mesh_idx - coupledFM_meshes - coupledAFM_meshes]->pM1;
					cuRect rect_c = cuRect(
						cuReal3(abs_pos.x - M.h.x / 2, abs_pos.y - M.h.y / 2, abs_pos.z + M.h.z / 2),
						cuReal3(abs_pos.x + M.h.x / 2, abs_pos.y + M.h.y / 2, abs_pos.z + M.h.z / 2 + M1.h.z));

					calculate_atom_coupling(
						M, idx,
						*ppaMesh_Bulk[mesh_idx - coupledFM_meshes - coupledAFM_meshes], rect_c,
						Ms, J1, J2, M.h.z,
						Hsurfexch, energy_, do_reduction);
				}
			}

			//-z coupling direction
			if (bulk_coupling_mask[idx].z & 0xffff0000) {

				int mesh_idx = (bulk_coupling_mask[idx].z >> 16) - 1;
				num_couplings++;

				//coupling for micromagnetic mesh
				if (mesh_idx < coupledFM_meshes) {
					calculate_mm_FM_coupling(
						M, idx,
						*ppMeshFM_Bulk[mesh_idx], abs_pos - ppMeshFM_Bulk[mesh_idx]->pM->origin + cuReal3(0, 0, -(M.h.z + ppMeshFM_Bulk[mesh_idx]->pM->h.z) / 2),
						Ms, J1, J2, M.h.z,
						Hsurfexch, energy_, do_reduction);
				}
				//coupling for micromagnetic AFM mesh
				else if (mesh_idx < coupledFM_meshes + coupledAFM_meshes) {
					calculate_mm_AFM_coupling(
						M, idx,
						*ppMeshAFM_Bulk[mesh_idx - coupledFM_meshes], abs_pos - ppMeshAFM_Bulk[mesh_idx - coupledFM_meshes]->pM->origin + cuReal3(0, 0, -(M.h.z + ppMeshAFM_Bulk[mesh_idx - coupledFM_meshes]->pM->h.z) / 2),
						Ms, J1, J2, M.h.z,
						Hsurfexch, energy_, do_reduction);
				}
				//coupling for atomistic mesh
				else if (mesh_idx < coupledFM_meshes + coupledAFM_meshes + coupled_ameshes) {

					//coupling rectangle in atomistic mesh in absolute coordinates
					cuVEC_VC<cuReal3>& M1 = *ppaMesh_Bulk[mesh_idx - coupledFM_meshes - coupledAFM_meshes]->pM1;
					cuRect rect_c = cuRect(
						cuReal3(abs_pos.x - M.h.x / 2, abs_pos.y - M.h.y / 2, abs_pos.z - M.h.z / 2 - M1.h.z),
						cuReal3(abs_pos.x + M.h.x / 2, abs_pos.y + M.h.y / 2, abs_pos.z - M.h.z / 2));

					calculate_atom_coupling(
						M, idx,
						*ppaMesh_Bulk[mesh_idx - coupledFM_meshes - coupledAFM_meshes], rect_c,
						Ms, J1, J2, M.h.z,
						Hsurfexch, energy_, do_reduction);
				}
			}
			
			if (num_couplings) {

				//need average if cell receives multiple coupling contributions
				Hsurfexch /= num_couplings;
				energy_ /= num_couplings;
			}

			Heff[idx] += Hsurfexch;

			//NOTE : we must add into the module display VECs, since there could be 2 contributions for some cells (top and bottom). This is why we had to zero the VECs before calling this kernel.
			if (do_reduction && cuModule.pModule_Heff->linear_size()) (*cuModule.pModule_Heff)[idx] += Hsurfexch;
			if (do_reduction && cuModule.pModule_energy->linear_size()) (*cuModule.pModule_energy)[idx] += energy_ * M.get_nonempty_cells();
			
		}
	}

	if (do_reduction) reduction_sum(0, 1, &energy_, *cuModule.penergy);
}

//----------------------- UpdateField LAUNCHER

void SurfExchangeCUDA::UpdateField(void)
{
	if (pMeshCUDA->CurrentTimeStepSolved()) {

		ZeroEnergy();
		ZeroModuleVECs();

		//------------------ SURFACE COUPLING Z STACKING

		//Coupling from ferromagnetic meshes

		//Top
		if (pMeshFM_Top.size()) {

			for (mGPU.device_begin(); mGPU != mGPU.device_end(); mGPU++) {

				cuSZ3 dn = pMeshCUDA->M.device_n(mGPU);
				SurfExchangeCUDA_TopFM_UpdateField <<< (dn.x * dn.y + CUDATHREADS) / CUDATHREADS, CUDATHREADS >>>
					(pMeshCUDA->cuMesh.get_deviceobject(mGPU), pMeshFM_Top.get_array(mGPU), pMeshFM_Top.size(mGPU), cuModule.get_deviceobject(mGPU), true);
			}
		}

		//Bottom
		if (pMeshFM_Bot.size()) {

			for (mGPU.device_begin(); mGPU != mGPU.device_end(); mGPU++) {

				cuSZ3 dn = pMeshCUDA->M.device_n(mGPU);
				SurfExchangeCUDA_BotFM_UpdateField <<< (dn.x * dn.y + CUDATHREADS) / CUDATHREADS, CUDATHREADS >>>
					(pMeshCUDA->cuMesh.get_deviceobject(mGPU), pMeshFM_Bot.get_array(mGPU), pMeshFM_Bot.size(mGPU), cuModule.get_deviceobject(mGPU), true);
			}
		}

		//Coupling from antiferromagnetic meshes

		//Top
		if (pMeshAFM_Top.size()) {

			for (mGPU.device_begin(); mGPU != mGPU.device_end(); mGPU++) {

				cuSZ3 dn = pMeshCUDA->M.device_n(mGPU);
				SurfExchangeCUDA_TopAFM_UpdateField <<< (dn.x * dn.y + CUDATHREADS) / CUDATHREADS, CUDATHREADS >>>
					(pMeshCUDA->cuMesh.get_deviceobject(mGPU), pMeshAFM_Top.get_array(mGPU), pMeshAFM_Top.size(mGPU), cuModule.get_deviceobject(mGPU), true);
			}
		}

		//Bottom
		if (pMeshAFM_Bot.size()) {

			for (mGPU.device_begin(); mGPU != mGPU.device_end(); mGPU++) {

				cuSZ3 dn = pMeshCUDA->M.device_n(mGPU);
				SurfExchangeCUDA_BotAFM_UpdateField <<< (dn.x * dn.y + CUDATHREADS) / CUDATHREADS, CUDATHREADS >>>
					(pMeshCUDA->cuMesh.get_deviceobject(mGPU), pMeshAFM_Bot.get_array(mGPU), pMeshAFM_Bot.size(mGPU), cuModule.get_deviceobject(mGPU), true);
			}
		}

		//Coupling from atomistic meshes

		//Top
		if (pMeshAtom_Top.size()) {

			for (mGPU.device_begin(); mGPU != mGPU.device_end(); mGPU++) {

				cuSZ3 dn = pMeshCUDA->M.device_n(mGPU);
				SurfExchangeCUDA_TopAtom_UpdateField <<< (dn.x * dn.y + CUDATHREADS) / CUDATHREADS, CUDATHREADS >>>
					(pMeshCUDA->cuMesh.get_deviceobject(mGPU), pMeshAtom_Top.get_array(mGPU), pMeshAtom_Top.size(mGPU), cuModule.get_deviceobject(mGPU), true);
			}
		}

		//Bottom
		if (pMeshAtom_Bot.size()) {

			for (mGPU.device_begin(); mGPU != mGPU.device_end(); mGPU++) {

				cuSZ3 dn = pMeshCUDA->M.device_n(mGPU);
				SurfExchangeCUDA_BotAtom_UpdateField << < (dn.x * dn.y + CUDATHREADS) / CUDATHREADS, CUDATHREADS >> >
					(pMeshCUDA->cuMesh.get_deviceobject(mGPU), pMeshAtom_Bot.get_array(mGPU), pMeshAtom_Bot.size(mGPU), cuModule.get_deviceobject(mGPU), true);
			}
		}

		//------------------ SURFACE COUPLING FOR OVERLAPPING MESHES

		if (pMeshFM_Bulk.size() + pMeshAFM_Bulk.size() + paMesh_Bulk.size()) {

			for (mGPU.device_begin(); mGPU != mGPU.device_end(); mGPU++) {

				SurfExchangeFMCUDA_Bulk_UpdateField <<< (pMeshCUDA->M.device_size(mGPU) + CUDATHREADS) / CUDATHREADS, CUDATHREADS >>>
					(pMeshCUDA->cuMesh.get_deviceobject(mGPU), 
					bulk_coupling_mask.get_deviceobject(mGPU),
					pMeshFM_Bulk.get_array(mGPU), pMeshFM_Bulk.size(mGPU), 
					pMeshAFM_Bulk.get_array(mGPU), pMeshAFM_Bulk.size(mGPU), 
					paMesh_Bulk.get_array(mGPU), paMesh_Bulk.size(mGPU),
					cuModule.get_deviceobject(mGPU), true);
			}
		}
	}
	else {

		//------------------ SURFACE COUPLING Z STACKING

		//Coupling from ferromagnetic meshes

		//Top
		if (pMeshFM_Top.size()) {

			for (mGPU.device_begin(); mGPU != mGPU.device_end(); mGPU++) {

				cuSZ3 dn = pMeshCUDA->M.device_n(mGPU);
				SurfExchangeCUDA_TopFM_UpdateField <<< (dn.x * dn.y + CUDATHREADS) / CUDATHREADS, CUDATHREADS >>>
					(pMeshCUDA->cuMesh.get_deviceobject(mGPU), pMeshFM_Top.get_array(mGPU), pMeshFM_Top.size(mGPU), cuModule.get_deviceobject(mGPU), false);
			}
		}

		//Bottom
		if (pMeshFM_Bot.size()) {

			for (mGPU.device_begin(); mGPU != mGPU.device_end(); mGPU++) {

				cuSZ3 dn = pMeshCUDA->M.device_n(mGPU);
				SurfExchangeCUDA_BotFM_UpdateField <<< (dn.x * dn.y + CUDATHREADS) / CUDATHREADS, CUDATHREADS >>>
					(pMeshCUDA->cuMesh.get_deviceobject(mGPU), pMeshFM_Bot.get_array(mGPU), pMeshFM_Bot.size(mGPU), cuModule.get_deviceobject(mGPU), false);
			}
		}

		//Coupling from antiferromagnetic meshes

		//Top
		if (pMeshAFM_Top.size()) {

			for (mGPU.device_begin(); mGPU != mGPU.device_end(); mGPU++) {

				cuSZ3 dn = pMeshCUDA->M.device_n(mGPU);
				SurfExchangeCUDA_TopAFM_UpdateField <<< (dn.x * dn.y + CUDATHREADS) / CUDATHREADS, CUDATHREADS >>>
					(pMeshCUDA->cuMesh.get_deviceobject(mGPU), pMeshAFM_Top.get_array(mGPU), pMeshAFM_Top.size(mGPU), cuModule.get_deviceobject(mGPU), false);
			}
		}

		//Bottom
		if (pMeshAFM_Bot.size()) {

			for (mGPU.device_begin(); mGPU != mGPU.device_end(); mGPU++) {

				cuSZ3 dn = pMeshCUDA->M.device_n(mGPU);
				SurfExchangeCUDA_BotAFM_UpdateField <<< (dn.x * dn.y + CUDATHREADS) / CUDATHREADS, CUDATHREADS >>>
					(pMeshCUDA->cuMesh.get_deviceobject(mGPU), pMeshAFM_Bot.get_array(mGPU), pMeshAFM_Bot.size(mGPU), cuModule.get_deviceobject(mGPU), false);
			}
		}

		//Coupling from atomistic meshes

		//Top
		if (pMeshAtom_Top.size()) {

			for (mGPU.device_begin(); mGPU != mGPU.device_end(); mGPU++) {

				cuSZ3 dn = pMeshCUDA->M.device_n(mGPU);
				SurfExchangeCUDA_TopAtom_UpdateField <<< (dn.x * dn.y + CUDATHREADS) / CUDATHREADS, CUDATHREADS >>>
					(pMeshCUDA->cuMesh.get_deviceobject(mGPU), pMeshAtom_Top.get_array(mGPU), pMeshAtom_Top.size(mGPU), cuModule.get_deviceobject(mGPU), false);
			}
		}

		//Bottom
		if (pMeshAtom_Bot.size()) {

			for (mGPU.device_begin(); mGPU != mGPU.device_end(); mGPU++) {

				cuSZ3 dn = pMeshCUDA->M.device_n(mGPU);
				SurfExchangeCUDA_BotAtom_UpdateField <<< (dn.x * dn.y + CUDATHREADS) / CUDATHREADS, CUDATHREADS >>>
					(pMeshCUDA->cuMesh.get_deviceobject(mGPU), pMeshAtom_Bot.get_array(mGPU), pMeshAtom_Bot.size(mGPU), cuModule.get_deviceobject(mGPU), false);
			}
		}

		//------------------ SURFACE COUPLING FOR OVERLAPPING MESHES

		if (pMeshFM_Bulk.size() + pMeshAFM_Bulk.size() + paMesh_Bulk.size()) {

			for (mGPU.device_begin(); mGPU != mGPU.device_end(); mGPU++) {

				SurfExchangeFMCUDA_Bulk_UpdateField <<< (pMeshCUDA->M.device_size(mGPU) + CUDATHREADS) / CUDATHREADS, CUDATHREADS >>>
					(pMeshCUDA->cuMesh.get_deviceobject(mGPU),
					bulk_coupling_mask.get_deviceobject(mGPU),
					pMeshFM_Bulk.get_array(mGPU), pMeshFM_Bulk.size(mGPU),
					pMeshAFM_Bulk.get_array(mGPU), pMeshAFM_Bulk.size(mGPU),
					paMesh_Bulk.get_array(mGPU), paMesh_Bulk.size(mGPU),
					cuModule.get_deviceobject(mGPU), false);
			}
		}
	}
}

//----------------------- Initialization

//Current mesh is ferromagnetic
__global__ void set_SurfExchangeCUDA_pointers_kernel(
	ManagedMeshCUDA& cuMesh,
	ManagedMeshCUDA** ppMeshFM_Bot, size_t coupledFM_bot_meshes,
	ManagedMeshCUDA** ppMeshFM_Top, size_t coupledFM_top_meshes,
	ManagedMeshCUDA** ppMeshAFM_Bot, size_t coupledAFM_bot_meshes,
	ManagedMeshCUDA** ppMeshAFM_Top, size_t coupledAFM_top_meshes,
	ManagedAtom_MeshCUDA** ppMeshAtom_Bot, size_t coupledAtom_bot_meshes,
	ManagedAtom_MeshCUDA** ppMeshAtom_Top, size_t coupledAtom_top_meshes,
	ManagedMeshCUDA** ppMeshFM_Bulk, size_t pMeshFM_Bulk_size,
	ManagedMeshCUDA** ppMeshAFM_Bulk, size_t pMeshAFM_Bulk_size,
	ManagedAtom_MeshCUDA** ppaMesh_Bulk, size_t paMesh_Bulk_size,
	cuVEC<cuINT3>& bulk_coupling_mask)
{
	if (threadIdx.x == 0) cuMesh.ppMeshFM_Bot = ppMeshFM_Bot;
	if (threadIdx.x == 1) cuMesh.pMeshFM_Bot_size = coupledFM_bot_meshes;
	if (threadIdx.x == 2) cuMesh.ppMeshFM_Top = ppMeshFM_Top;
	if (threadIdx.x == 3) cuMesh.pMeshFM_Top_size = coupledFM_top_meshes;

	if (threadIdx.x == 4) cuMesh.ppMeshAFM_Bot = ppMeshAFM_Bot;
	if (threadIdx.x == 5) cuMesh.pMeshAFM_Bot_size = coupledAFM_bot_meshes;
	if (threadIdx.x == 6) cuMesh.ppMeshAFM_Top = ppMeshAFM_Top;
	if (threadIdx.x == 7) cuMesh.pMeshAFM_Top_size = coupledAFM_top_meshes;

	if (threadIdx.x == 8) cuMesh.ppMeshAtom_Bot = ppMeshAtom_Bot;
	if (threadIdx.x == 9) cuMesh.pMeshAtom_Bot_size = coupledAtom_bot_meshes;
	if (threadIdx.x == 10) cuMesh.ppMeshAtom_Top = ppMeshAtom_Top;
	if (threadIdx.x == 11) cuMesh.pMeshAtom_Top_size = coupledAtom_top_meshes;

	if (threadIdx.x == 12) cuMesh.ppMeshFM_Bulk = ppMeshFM_Bulk;
	if (threadIdx.x == 13) cuMesh.pMeshFM_Bulk_size = pMeshFM_Bulk_size;
	if (threadIdx.x == 14) cuMesh.ppMeshAFM_Bulk = ppMeshAFM_Bulk;
	if (threadIdx.x == 15) cuMesh.pMeshAFM_Bulk_size = pMeshAFM_Bulk_size;
	if (threadIdx.x == 16) cuMesh.ppaMesh_Bulk = ppaMesh_Bulk;
	if (threadIdx.x == 17) cuMesh.paMesh_Bulk_size = paMesh_Bulk_size;
	if (threadIdx.x == 18) cuMesh.pbulk_coupling_mask = &bulk_coupling_mask;
}

//Called by SurfExchangeCUDA module
void SurfExchangeCUDA::set_SurfExchangeCUDA_pointers(void)
{   
	for (mGPU.device_begin(); mGPU != mGPU.device_end(); mGPU++) {

		set_SurfExchangeCUDA_pointers_kernel <<< 1, CUDATHREADS >>>
			(pMeshCUDA->cuMesh.get_deviceobject(mGPU),
				pMeshFM_Bot.get_array(mGPU), pMeshFM_Bot.size(mGPU), pMeshFM_Top.get_array(mGPU), pMeshFM_Top.size(mGPU),
				pMeshAFM_Bot.get_array(mGPU), pMeshAFM_Bot.size(mGPU), pMeshAFM_Top.get_array(mGPU), pMeshAFM_Top.size(mGPU),
				pMeshAtom_Bot.get_array(mGPU), pMeshAtom_Bot.size(mGPU), pMeshAtom_Top.get_array(mGPU), pMeshAtom_Top.size(mGPU),
				pMeshFM_Bulk.get_array(mGPU), pMeshFM_Bulk.size(mGPU),
				pMeshAFM_Bulk.get_array(mGPU), pMeshAFM_Bulk.size(mGPU),
				paMesh_Bulk.get_array(mGPU), paMesh_Bulk.size(mGPU),
				bulk_coupling_mask.get_deviceobject(mGPU));
	}
}

#endif

#endif

#if COMPILECUDA == 1 && MONTE_CARLO == 1

//Ferromagnetic
__device__ cuBReal ManagedMeshCUDA::Get_EnergyChange_FM_SurfExchangeCUDA(int spin_index, cuReal3 Mnew)
{
	cuBReal energy_new = 0, energy_old = 0;

	cuVEC_VC<cuReal3>& M = *pM;

	cuSZ3 n = M.n;
	cuReal3 h = M.h;

	//------------------ Coupling functions

	auto calculate_mm_FM_coupling = [](
		cuVEC_VC<cuReal3>& M, cuReal3 Mnew, int spin_index,
		ManagedMeshCUDA& MeshCoupled, cuReal3& cell_rel_pos,
		cuBReal J1, cuBReal J2, cuBReal cell_size,
		cuBReal& energy_old, cuBReal& energy_new) -> void
	{
		//Surface exchange field from a ferromagnetic mesh (RKKY)

		mcuVEC_Managed<cuVEC_VC<cuReal3>, cuReal3>& M_Bulk = MeshCoupled.pM->mcuvec();

		cuReal3 m_j = cu_normalize(M_Bulk[cell_rel_pos]);
		cuReal3 m_i = cu_normalize(M[spin_index]);

		//total surface exchange field in coupling cells, including bilinear and biquadratic terms
		cuBReal dot_prod = m_i * m_j;
		energy_old += (-1 * J1 - 2 * J2 * dot_prod) * dot_prod / cell_size;

		if (Mnew != cuReal3()) {

			cuReal3 mnew_i = cu_normalize(Mnew);
			cuBReal dot_prod_new = mnew_i * m_j;
			energy_new += (-1 * J1 - 2 * J2 * dot_prod_new) * dot_prod_new / cell_size;
		}
	};

	auto calculate_mm_AFM_coupling = [](
		cuVEC_VC<cuReal3>& M, cuReal3 Mnew, int spin_index,
		ManagedMeshCUDA& MeshCoupled, cuReal3& cell_rel_pos,
		cuBReal J1, cuBReal J2, cuBReal cell_size,
		cuBReal& energy_old, cuBReal& energy_new) -> void
	{
		//Surface exchange field from an antiferromagnetic mesh (exchange bias)

		mcuVEC_Managed<cuVEC_VC<cuReal3>, cuReal3>& M_Bulk = MeshCoupled.pM->mcuvec();
		mcuVEC_Managed<cuVEC_VC<cuReal3>, cuReal3>& M2_Bulk = MeshCoupled.pM2->mcuvec();

		cuReal3 m_j1 = cu_normalize(M_Bulk[cell_rel_pos]);
		cuReal3 m_j2 = cu_normalize(M2_Bulk[cell_rel_pos]);
		cuReal3 m_i = cu_normalize(M[spin_index]);

		//total surface exchange field in coupling cells, including contributions from both sub-lattices
		energy_old += (-J1 * (m_i * m_j1) - J2 * (m_i * m_j2)) / cell_size;

		if (Mnew != cuReal3()) {

			cuReal3 mnew_i = cu_normalize(Mnew);
			energy_new += (-J1 * (mnew_i * m_j1) - J2 * (mnew_i * m_j2)) / cell_size;
		}
	};

	auto calculate_atom_coupling = [](
		cuVEC_VC<cuReal3>& M, cuReal3 Mnew, int spin_index,
		ManagedAtom_MeshCUDA& aMeshCoupled, cuRect& rect_c,
		cuBReal J1, cuBReal J2, cuBReal cell_size,
		cuBReal& energy_old, cuBReal& energy_new) -> void
	{
		mcuVEC_Managed<cuVEC_VC<cuReal3>, cuReal3>& M1 = aMeshCoupled.pM1->mcuvec();

		//cells box in atomistic mesh
		cuBox acells = M1.box_from_rect_min(rect_c);

		cuReal3 m_j = cuReal3();
		for (int ai = acells.s.i; ai < acells.e.i; ai++) {
			for (int aj = acells.s.j; aj < acells.e.j; aj++) {
				for (int ak = acells.s.k; ak < acells.e.k; ak++) {

					cuReal3 rel_pos = cuReal3((ai + 0.5) * M1.h.x, (aj + 0.5) * M1.h.y, (ak + 0.5) * M1.h.z);

					if (M1.is_empty(rel_pos)) continue;

					m_j += M1[rel_pos];
				}
			}
		}

		//total surface exchange field in coupling cells, including bilinear and biquadratic terms
		m_j = cu_normalize(m_j);
		cuBReal dot_prod = cu_normalize(M[spin_index]) * m_j;
		energy_old += (-1 * J1 - 2 * J2 * dot_prod) * dot_prod / cell_size;

		if (Mnew != cuReal3()) {

			cuReal3 mnew_i = cu_normalize(Mnew);
			cuBReal dot_prod_new = mnew_i * m_j;
			energy_new += (-1 * J1 - 2 * J2 * dot_prod_new) * dot_prod_new / cell_size;
		}
	};

	//------------------ SURFACE COUPLING Z STACKING

	//if spin is on top surface then look at paMesh_Top
	if (spin_index / (n.x * n.y) == n.z - 1 && (pMeshFM_Top_size + pMeshAFM_Top_size + pMeshAtom_Top_size > 0)) {

		if (!M.is_empty(spin_index)) {

			int i = spin_index % n.x;
			int j = (spin_index / n.x) % n.y;

			bool cell_coupled = false;

			//check all meshes for coupling : FM meshes first
			for (int mesh_idx = 0; mesh_idx < (int)pMeshFM_Top_size; mesh_idx++) {

				mcuVEC_Managed<cuVEC_VC<cuReal3>, cuReal3>& M_Top = ppMeshFM_Top[mesh_idx]->pM->mcuvec();

				//relative coordinates to read value from top mesh (the one we're coupling to here) - relative to top mesh
				cuReal3 cell_rel_pos = cuReal3(
					(i + 0.5) * h.x + M.rect.s.x - M_Top.rect.s.x,
					(j + 0.5) * h.y + M.rect.s.y - M_Top.rect.s.y,
					M_Top.h.z / 2);

				//can't couple to an empty cell
				if (!M_Top.rect.contains(cell_rel_pos + M_Top.rect.s) || M_Top.is_empty(cell_rel_pos)) continue;

				//Surface exchange field from a ferromagnetic mesh (RKKY)

				//Top mesh sets J1 and J2 values
				cuBReal J1 = *(ppMeshFM_Top[mesh_idx]->pJ1);
				cuBReal J2 = *(ppMeshFM_Top[mesh_idx]->pJ2);
				ppMeshFM_Top[mesh_idx]->update_parameters_atposition(cell_rel_pos, *(ppMeshFM_Top[mesh_idx]->pJ1), J1, *(ppMeshFM_Top[mesh_idx]->pJ2), J2);

				calculate_mm_FM_coupling(
					M, Mnew, spin_index,
					*ppMeshFM_Top[mesh_idx], cell_rel_pos,
					J1, J2, h.z,
					energy_old, energy_new);

				//for each cell, either it's not coupled to any other mesh cell (so we never get here), or else it's coupled to exactly one cell on this surface (thus can stop looping over meshes now)
				cell_coupled = true;
				break;
			}

			if (!cell_coupled) {

				//next AFM meshes
				for (int mesh_idx = 0; mesh_idx < (int)pMeshAFM_Top_size; mesh_idx++) {

					mcuVEC_Managed<cuVEC_VC<cuReal3>, cuReal3>& M_Top = ppMeshAFM_Top[mesh_idx]->pM->mcuvec();
					mcuVEC_Managed<cuVEC_VC<cuReal3>, cuReal3>& M2_Top = ppMeshAFM_Top[mesh_idx]->pM2->mcuvec();

					//relative coordinates to read value from top mesh (the one we're coupling to here) - relative to top mesh
					cuReal3 cell_rel_pos = cuReal3(
						(i + 0.5) * h.x + M.rect.s.x - M_Top.rect.s.x,
						(j + 0.5) * h.y + M.rect.s.y - M_Top.rect.s.y,
						M_Top.h.z / 2);

					//can't couple to an empty cell
					if (!M_Top.rect.contains(cell_rel_pos + M_Top.rect.s) || M_Top.is_empty(cell_rel_pos)) continue;

					//Surface exchange field from an antiferromagnetic mesh (exchange bias)

					//Top mesh sets J1 and J2 values
					cuBReal J1 = *(ppMeshAFM_Top[mesh_idx]->pJ1);
					cuBReal J2 = *(ppMeshAFM_Top[mesh_idx]->pJ2);
					ppMeshAFM_Top[mesh_idx]->update_parameters_atposition(cell_rel_pos, *(ppMeshAFM_Top[mesh_idx]->pJ1), J1, *(ppMeshAFM_Top[mesh_idx]->pJ2), J2);

					calculate_mm_AFM_coupling(
						M, Mnew, spin_index,
						*ppMeshAFM_Top[mesh_idx], cell_rel_pos,
						J1, J2, h.z,
						energy_old, energy_new);

					//for each cell, either it's not coupled to any other mesh cell (so we never get here), or else it's coupled to exactly one cell on this surface (thus can stop looping over meshes now)
					cell_coupled = true;
					break;
				}
			}

			if (!cell_coupled) {

				//next atomistic meshes
				for (int mesh_idx = 0; mesh_idx < (int)pMeshAtom_Top_size; mesh_idx++) {

					//coupling rectangle in atomistic mesh in absolute coordinates
					mcuVEC_Managed<cuVEC_VC<cuReal3>, cuReal3>& M1 = ppMeshAtom_Top[mesh_idx]->pM1->mcuvec();
					cuRect rect_c = cuRect(
						cuReal3(i * h.x, j * h.y, M.rect.e.z),
						cuReal3((i + 1) * h.x, (j + 1) * h.y, M1.h.z + M.rect.e.z));
					rect_c += cuReal3(M.rect.s.x, M.rect.s.y, 0.0);

					//current mesh sets coupling in micromagnetic to atomistic meshes coupling
					cuBReal J1 = *pJ1;
					cuBReal J2 = *pJ2;
					update_parameters_mcoarse(spin_index, *pJ1, J1, *pJ2, J2);

					calculate_atom_coupling(
						M, Mnew, spin_index,
						*ppMeshAtom_Top[mesh_idx], rect_c,
						J1, J2, h.z,
						energy_old, energy_new);

					//for each cell, either it's not coupled to any other mesh cell (so we never get here), or else it's coupled to exactly one cell on this surface (thus can stop looping over meshes now)
					break;
				}
			}
		}
	}

	if (spin_index / (n.x * n.y) == 0 && (pMeshFM_Bot_size + pMeshAFM_Bot_size + pMeshAtom_Bot_size > 0)) {

		//surface exchange coupling at the bottom

		if (!M.is_empty(spin_index)) {

			int i = spin_index % n.x;
			int j = (spin_index / n.x) % n.y;

			cuBReal J1 = *pJ1;
			cuBReal J2 = *pJ2;
			update_parameters_mcoarse(spin_index, *pJ1, J1, *pJ2, J2);

			bool cell_coupled = false;

			//check all meshes for coupling : FM meshes first
			for (int mesh_idx = 0; mesh_idx < (int)pMeshFM_Bot_size; mesh_idx++) {

				mcuVEC_Managed<cuVEC_VC<cuReal3>, cuReal3>& M_Bot = ppMeshFM_Bot[mesh_idx]->pM->mcuvec();

				//relative coordinates to read value from bottom mesh (the one we're coupling to here) - relative to bottom mesh
				cuReal3 cell_rel_pos = cuReal3(
					(i + 0.5) * h.x + M.rect.s.x - M_Bot.rect.s.x,
					(j + 0.5) * h.y + M.rect.s.y - M_Bot.rect.s.y,
					M_Bot.rect.e.z - M_Bot.rect.s.z - M_Bot.h.z / 2);

				//can't couple to an empty cell
				if (!M_Bot.rect.contains(cell_rel_pos + M_Bot.rect.s) || M_Bot.is_empty(cell_rel_pos)) continue;

				//Surface exchange field from a ferromagnetic mesh (RKKY)

				calculate_mm_FM_coupling(
					M, Mnew, spin_index,
					*ppMeshFM_Bot[mesh_idx], cell_rel_pos,
					J1, J2, h.z,
					energy_old, energy_new);

				//for each cell, either it's not coupled to any other mesh cell (so we never get here), or else it's coupled to exactly one cell on this surface (thus can stop looping over meshes now)
				cell_coupled = true;
				break;
			}

			if (!cell_coupled) {

				//next AFM meshes
				for (int mesh_idx = 0; mesh_idx < (int)pMeshAFM_Bot_size; mesh_idx++) {

					mcuVEC_Managed<cuVEC_VC<cuReal3>, cuReal3>& M_Bot = ppMeshAFM_Bot[mesh_idx]->pM->mcuvec();
					mcuVEC_Managed<cuVEC_VC<cuReal3>, cuReal3>& M2_Bot = ppMeshAFM_Bot[mesh_idx]->pM2->mcuvec();

					//relative coordinates to read value from bottom mesh (the one we're coupling to here) - relative to bottom mesh
					cuReal3 cell_rel_pos = cuReal3(
						(i + 0.5) * h.x + M.rect.s.x - M_Bot.rect.s.x,
						(j + 0.5) * h.y + M.rect.s.y - M_Bot.rect.s.y,
						M_Bot.rect.e.z - M_Bot.rect.s.z - M_Bot.h.z / 2);

					//can't couple to an empty cell
					if (!M_Bot.rect.contains(cell_rel_pos + M_Bot.rect.s) || M_Bot.is_empty(cell_rel_pos)) continue;

					//Surface exchange field from an antiferromagnetic mesh (exchange bias)

					calculate_mm_AFM_coupling(
						M, Mnew, spin_index,
						*ppMeshAFM_Bot[mesh_idx], cell_rel_pos,
						J1, J2, h.z,
						energy_old, energy_new);

					//for each cell, either it's not coupled to any other mesh cell (so we never get here), or else it's coupled to exactly one cell on this surface (thus can stop looping over meshes now)
					break;
				}
			}

			if (!cell_coupled) {

				//next atomistic meshes
				for (int mesh_idx = 0; mesh_idx < (int)pMeshAtom_Bot_size; mesh_idx++) {

					//coupling rectangle in atomistic mesh in absolute coordinates
					mcuVEC_Managed<cuVEC_VC<cuReal3>, cuReal3>& M1 = ppMeshAtom_Bot[mesh_idx]->pM1->mcuvec();
					cuRect rect_c = cuRect(
						cuReal3(i * h.x, j * h.y, M1.rect.e.z - M1.h.z),
						cuReal3((i + 1) * h.x, (j + 1) * h.y, M1.rect.e.z));
					rect_c += cuReal3(M.rect.s.x, M.rect.s.y, 0.0);

					calculate_atom_coupling(
						M, Mnew, spin_index,
						*ppMeshAtom_Bot[mesh_idx], rect_c,
						J1, J2, h.z,
						energy_old, energy_new);

					//for each cell, either it's not coupled to any other mesh cell (so we never get here), or else it's coupled to exactly one cell on this surface (thus can stop looping over meshes now)
					break;
				}
			}
		}
	}

	//------------------ SURFACE COUPLING FOR OVERLAPPING MESHES

	if (pMeshFM_Bulk_size + pMeshAFM_Bulk_size + paMesh_Bulk_size) {

		if (M.is_not_empty(spin_index) && (*pbulk_coupling_mask)[spin_index] != cuINT3()) {

			cuBReal energy_bulk_new = 0, energy_bulk_old = 0;

			cuBReal J1 = *pJ1;
			cuBReal J2 = *pJ2;
			update_parameters_mcoarse(spin_index, *pJ1, J1, *pJ2, J2);

			int num_couplings = 0;

			cuReal3 abs_pos = M.cellidx_to_position(spin_index) + M.rect.s;

			cuReal3 cell_rel_pos;
			cuRect rect_c;
			int mesh_idx = -1;
			cuBReal cell_size = 0.0;

			for (int nidx = 0; nidx < 6; nidx++) {

				//+x coupling direction
				if (nidx == 0 && (*pbulk_coupling_mask)[spin_index].x & 0x0000ffff) {

					mesh_idx = ((*pbulk_coupling_mask)[spin_index].x & 0x0000ffff) - 1;
					cell_size = M.h.x;
					if (mesh_idx < pMeshFM_Bulk_size) cell_rel_pos = abs_pos - ppMeshFM_Bulk[mesh_idx]->pM->origin + cuReal3((M.h.x + ppMeshFM_Bulk[mesh_idx]->pM->h.x) / 2, 0, 0);
					else if (mesh_idx < pMeshFM_Bulk_size + pMeshAFM_Bulk_size) cell_rel_pos = abs_pos - ppMeshAFM_Bulk[mesh_idx - pMeshFM_Bulk_size]->pM->origin + cuReal3((M.h.x + ppMeshAFM_Bulk[mesh_idx - pMeshFM_Bulk_size]->pM->h.x) / 2, 0, 0);
					else if (mesh_idx < pMeshFM_Bulk_size + pMeshAFM_Bulk_size + paMesh_Bulk_size) {

						cuVEC_VC<cuReal3>& M1 = *ppaMesh_Bulk[mesh_idx - pMeshFM_Bulk_size - pMeshAFM_Bulk_size]->pM1;
						rect_c = cuRect(
							cuReal3(abs_pos.x + M.h.x / 2, abs_pos.y - M.h.y / 2, abs_pos.z - M.h.z / 2),
							cuReal3(abs_pos.x + M.h.x / 2 + M1.h.x, abs_pos.y + M.h.y / 2, abs_pos.z + M.h.z / 2));
					}
				}

				//-x coupling direction
				else if (nidx == 1 && (*pbulk_coupling_mask)[spin_index].x & 0xffff0000) {

					mesh_idx = mesh_idx = ((*pbulk_coupling_mask)[spin_index].x >> 16) - 1;
					cell_size = M.h.x;
					//coupling for micromagnetic FM mesh
					if (mesh_idx < pMeshFM_Bulk_size) cell_rel_pos = abs_pos - ppMeshFM_Bulk[mesh_idx]->pM->origin + cuReal3(-(M.h.x + ppMeshFM_Bulk[mesh_idx]->pM->h.x) / 2, 0, 0);
					//coupling for micromagnetic AFM mesh
					else if (mesh_idx < pMeshFM_Bulk_size + pMeshAFM_Bulk_size) cell_rel_pos = abs_pos - ppMeshAFM_Bulk[mesh_idx - pMeshFM_Bulk_size]->pM->origin + cuReal3(-(M.h.x + ppMeshAFM_Bulk[mesh_idx - pMeshFM_Bulk_size]->pM->h.x) / 2, 0, 0);
					else if (mesh_idx < pMeshFM_Bulk_size + pMeshAFM_Bulk_size + paMesh_Bulk_size) {

						cuVEC_VC<cuReal3>& M1 = *ppaMesh_Bulk[mesh_idx - pMeshFM_Bulk_size - pMeshAFM_Bulk_size]->pM1;
						rect_c = cuRect(
							cuReal3(abs_pos.x - M.h.x / 2 - M1.h.x, abs_pos.y - M.h.y / 2, abs_pos.z - M.h.z / 2),
							cuReal3(abs_pos.x - M.h.x / 2, abs_pos.y + M.h.y / 2, abs_pos.z + M.h.z / 2));
					}
				}

				//+y coupling direction
				else if (nidx == 2 && (*pbulk_coupling_mask)[spin_index].y & 0x0000ffff) {

					mesh_idx = ((*pbulk_coupling_mask)[spin_index].y & 0x0000ffff) - 1;
					cell_size = M.h.y;
					//coupling for micromagnetic mesh
					if (mesh_idx < pMeshFM_Bulk_size) cell_rel_pos = abs_pos - ppMeshFM_Bulk[mesh_idx]->pM->origin + cuReal3(0, (M.h.y + ppMeshFM_Bulk[mesh_idx]->pM->h.y) / 2, 0);
					//coupling for micromagnetic AFM mesh
					else if (mesh_idx < pMeshFM_Bulk_size + pMeshAFM_Bulk_size) cell_rel_pos = abs_pos - ppMeshAFM_Bulk[mesh_idx - pMeshFM_Bulk_size]->pM->origin + cuReal3(0, (M.h.y + ppMeshAFM_Bulk[mesh_idx - pMeshFM_Bulk_size]->pM->h.y) / 2, 0);
					else if (mesh_idx < pMeshFM_Bulk_size + pMeshAFM_Bulk_size + paMesh_Bulk_size) {

						cuVEC_VC<cuReal3>& M1 = *ppaMesh_Bulk[mesh_idx - pMeshFM_Bulk_size - pMeshAFM_Bulk_size]->pM1;
						rect_c = cuRect(
							cuReal3(abs_pos.x - M.h.x / 2, abs_pos.y + M.h.y / 2, abs_pos.z - M.h.z / 2),
							cuReal3(abs_pos.x + M.h.x / 2, abs_pos.y + M.h.y / 2 + M1.h.y, abs_pos.z + M.h.z / 2));
					}
				}

				//-y coupling direction
				else if (nidx == 3 && (*pbulk_coupling_mask)[spin_index].y & 0xffff0000) {

					mesh_idx = ((*pbulk_coupling_mask)[spin_index].y >> 16) - 1;
					cell_size = M.h.y;
					//coupling for micromagnetic mesh
					if (mesh_idx < pMeshFM_Bulk_size) cell_rel_pos = abs_pos - ppMeshFM_Bulk[mesh_idx]->pM->origin + cuReal3(0, -(M.h.y + ppMeshFM_Bulk[mesh_idx]->pM->h.y) / 2, 0);
					//coupling for micromagnetic AFM mesh
					else if (mesh_idx < pMeshFM_Bulk_size + pMeshAFM_Bulk_size) cell_rel_pos = abs_pos - ppMeshAFM_Bulk[mesh_idx - pMeshFM_Bulk_size]->pM->origin + cuReal3(0, -(M.h.y + ppMeshAFM_Bulk[mesh_idx - pMeshFM_Bulk_size]->pM->h.y) / 2, 0);
					else if (mesh_idx < pMeshFM_Bulk_size + pMeshAFM_Bulk_size + paMesh_Bulk_size) {

						cuVEC_VC<cuReal3>& M1 = *ppaMesh_Bulk[mesh_idx - pMeshFM_Bulk_size - pMeshAFM_Bulk_size]->pM1;
						rect_c = cuRect(
							cuReal3(abs_pos.x - M.h.x / 2, abs_pos.y - M.h.y / 2 - M1.h.y, abs_pos.z - M.h.z / 2),
							cuReal3(abs_pos.x + M.h.x / 2, abs_pos.y - M.h.y / 2, abs_pos.z + M.h.z / 2));
					}
				}

				//+z coupling direction
				else if (nidx == 4 && (*pbulk_coupling_mask)[spin_index].z & 0x0000ffff) {

					mesh_idx = ((*pbulk_coupling_mask)[spin_index].z & 0x0000ffff) - 1;
					cell_size = M.h.z;
					//coupling for micromagnetic mesh
					if (mesh_idx < pMeshFM_Bulk_size) cell_rel_pos = abs_pos - ppMeshFM_Bulk[mesh_idx]->pM->origin + cuReal3(0, 0, (M.h.z + ppMeshFM_Bulk[mesh_idx]->pM->h.z) / 2);
					//coupling for micromagnetic AFM mesh
					else if (mesh_idx < pMeshFM_Bulk_size + pMeshAFM_Bulk_size) cell_rel_pos = abs_pos - ppMeshAFM_Bulk[mesh_idx - pMeshFM_Bulk_size]->pM->origin + cuReal3(0, 0, (M.h.z + ppMeshAFM_Bulk[mesh_idx - pMeshFM_Bulk_size]->pM->h.z) / 2);
					else if (mesh_idx < pMeshFM_Bulk_size + pMeshAFM_Bulk_size + paMesh_Bulk_size) {

						cuVEC_VC<cuReal3>& M1 = *ppaMesh_Bulk[mesh_idx - pMeshFM_Bulk_size - pMeshAFM_Bulk_size]->pM1;
						rect_c = cuRect(
							cuReal3(abs_pos.x - M.h.x / 2, abs_pos.y - M.h.y / 2, abs_pos.z + M.h.z / 2),
							cuReal3(abs_pos.x + M.h.x / 2, abs_pos.y + M.h.y / 2, abs_pos.z + M.h.z / 2 + M1.h.z));
					}
				}

				//-z coupling direction
				else if (nidx == 5 && (*pbulk_coupling_mask)[spin_index].z & 0xffff0000) {

					mesh_idx = ((*pbulk_coupling_mask)[spin_index].z >> 16) - 1;
					cell_size = M.h.z;
					//coupling for micromagnetic mesh
					if (mesh_idx < pMeshFM_Bulk_size) cell_rel_pos = abs_pos - ppMeshFM_Bulk[mesh_idx]->pM->origin + cuReal3(0, 0, -(M.h.z + ppMeshFM_Bulk[mesh_idx]->pM->h.z) / 2);
					//coupling for micromagnetic AFM mesh
					else if (mesh_idx < pMeshFM_Bulk_size + pMeshAFM_Bulk_size) cell_rel_pos = abs_pos - ppMeshAFM_Bulk[mesh_idx - pMeshFM_Bulk_size]->pM->origin + cuReal3(0, 0, -(M.h.z + ppMeshAFM_Bulk[mesh_idx - pMeshFM_Bulk_size]->pM->h.z) / 2);
					else if (mesh_idx < pMeshFM_Bulk_size + pMeshAFM_Bulk_size + paMesh_Bulk_size) {

						cuVEC_VC<cuReal3>& M1 = *ppaMesh_Bulk[mesh_idx - pMeshFM_Bulk_size - pMeshAFM_Bulk_size]->pM1;
						rect_c = cuRect(
							cuReal3(abs_pos.x - M.h.x / 2, abs_pos.y - M.h.y / 2, abs_pos.z - M.h.z / 2 - M1.h.z),
							cuReal3(abs_pos.x + M.h.x / 2, abs_pos.y + M.h.y / 2, abs_pos.z - M.h.z / 2));
					}
				}

				if (mesh_idx >= 0) {

					num_couplings++;

					//coupling for micromagnetic FM mesh
					if (mesh_idx < pMeshFM_Bulk_size) {

						calculate_mm_FM_coupling(
							M, Mnew, spin_index,
							*ppMeshFM_Bulk[mesh_idx], cell_rel_pos,
							J1, J2, cell_size,
							energy_bulk_old, energy_bulk_new);
					}
					else if (mesh_idx < pMeshFM_Bulk_size + pMeshAFM_Bulk_size) {
						calculate_mm_AFM_coupling(
							M, Mnew, spin_index,
							*ppMeshAFM_Bulk[mesh_idx - pMeshFM_Bulk_size], cell_rel_pos,
							J1, J2, cell_size,
							energy_bulk_old, energy_bulk_new);
					}
					//coupling for atomistic mesh
					else if (mesh_idx < pMeshFM_Bulk_size + pMeshAFM_Bulk_size + paMesh_Bulk_size) {
						calculate_atom_coupling(
							M, Mnew, spin_index,
							*ppaMesh_Bulk[mesh_idx - pMeshFM_Bulk_size - pMeshAFM_Bulk_size], rect_c,
							J1, J2, cell_size,
							energy_bulk_old, energy_bulk_new);
					}
				}
				mesh_idx = -1;
			}

			if (num_couplings) {

				energy_old += energy_bulk_old / num_couplings;
				energy_new += energy_bulk_new / num_couplings;
			}
		}
	}

	//------------------

	if (Mnew != cuReal3()) return M.h.dim() * (energy_new - energy_old);
	else return M.h.dim() * energy_old;
}

//Antiferromagnetic


#endif

