#include "hip/hip_runtime.h"
#include "Atom_DiffEqCubicCUDA.h"

#if COMPILECUDA == 1
#ifdef ODE_EVAL_COMPILATION_RK4
#ifdef MESH_COMPILATION_ATOM_CUBIC

#include "Atom_MeshParamsControlCUDA.h"

#include "Reduction.cuh"

//defines evaluation methods kernel launchers

//----------------------------------------- EVALUATIONS : RK4

__global__ void RunRK4_Step0_withReductions_Kernel(ManagedAtom_DiffEqCubicCUDA& cuaDiffEq, ManagedAtom_MeshCUDA& cuaMesh)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;

	cuBReal dT = *cuaDiffEq.pdT;
	
	cuBReal mxh = 0.0;

	//multiplicative conversion factor from atomic moment (units of muB) to A/m
	cuBReal conversion = (cuBReal)MUB / cuaMesh.pM1->h.dim();

	if (idx < cuaMesh.pM1->linear_size()) {

		if (cuaMesh.pM1->is_not_empty(idx)) {

			//Save current moment for later use
			(*cuaDiffEq.psM1)[idx] = (*cuaMesh.pM1)[idx];

			if (!cuaMesh.pM1->is_skipcell(idx)) {

				//obtain maximum normalized torque term
				cuBReal Mnorm = (*cuaMesh.pM1)[idx].norm();
				mxh = cu_GetMagnitude((*cuaMesh.pM1)[idx] ^ (*cuaMesh.pHeff1)[idx]) / (conversion * Mnorm * Mnorm);

				(*cuaDiffEq.psEval0)[idx] = (cuaDiffEq.*(cuaDiffEq.pODEFunc))(idx);

				//Now estimate moment using RK4 midle step
				(*cuaMesh.pM1)[idx] += (*cuaDiffEq.psEval0)[idx] * (dT / 2);
			}
		}
	}

	if (cuaMesh.pgrel->get0()) reduction_max(0, 1, &mxh, *cuaDiffEq.pmxh);
}

__global__ void RunRK4_Step0_withAverageReductions_Kernel(ManagedAtom_DiffEqCubicCUDA& cuaDiffEq, ManagedAtom_MeshCUDA& cuaMesh)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;

	cuBReal dT = *cuaDiffEq.pdT;

	cuReal3 mxh = cuReal3();
	bool include_in_average = false;

	//multiplicative conversion factor from atomic moment (units of muB) to A/m
	cuBReal conversion = (cuBReal)MUB / cuaMesh.pM1->h.dim();

	if (idx < cuaMesh.pM1->linear_size()) {

		if (cuaMesh.pM1->is_not_empty(idx)) {

			//Save current moment for later use
			(*cuaDiffEq.psM1)[idx] = (*cuaMesh.pM1)[idx];

			if (!cuaMesh.pM1->is_skipcell(idx)) {

				//obtain maximum normalized torque term
				cuBReal Mnorm = (*cuaMesh.pM1)[idx].norm();
				mxh = cu_GetMagnitude((*cuaMesh.pM1)[idx] ^ (*cuaMesh.pHeff1)[idx]) / (conversion * Mnorm * Mnorm);

				(*cuaDiffEq.psEval0)[idx] = (cuaDiffEq.*(cuaDiffEq.pODEFunc))(idx);

				//Now estimate moment using RK4 midle step
				(*cuaMesh.pM1)[idx] += (*cuaDiffEq.psEval0)[idx] * (dT / 2);
			}
		}
	}

	if (cuaMesh.pgrel->get0()) reduction_avg(0, 1, &mxh, *cuaDiffEq.pmxh_av, *cuaDiffEq.pavpoints, include_in_average);
}

__global__ void RunRK4_Step0_Kernel(ManagedAtom_DiffEqCubicCUDA& cuaDiffEq, ManagedAtom_MeshCUDA& cuaMesh)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;

	cuBReal dT = *cuaDiffEq.pdT;

	if (idx < cuaMesh.pM1->linear_size()) {

		if (cuaMesh.pM1->is_not_empty(idx)) {

			//Save current moment for later use
			(*cuaDiffEq.psM1)[idx] = (*cuaMesh.pM1)[idx];

			if (!cuaMesh.pM1->is_skipcell(idx)) {

				(*cuaDiffEq.psEval0)[idx] = (cuaDiffEq.*(cuaDiffEq.pODEFunc))(idx);

				//Now estimate moment using RK4 midle step
				(*cuaMesh.pM1)[idx] += (*cuaDiffEq.psEval0)[idx] * (dT / 2);
			}
		}
	}
}

__global__ void RunRK4_Step1_Kernel(ManagedAtom_DiffEqCubicCUDA& cuaDiffEq, ManagedAtom_MeshCUDA& cuaMesh)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;

	cuBReal dT = *cuaDiffEq.pdT;

	if (idx < cuaMesh.pM1->linear_size()) {

		if (cuaMesh.pM1->is_not_empty(idx) && !cuaMesh.pM1->is_skipcell(idx)) {

			(*cuaDiffEq.psEval1)[idx] = (cuaDiffEq.*(cuaDiffEq.pODEFunc))(idx);

			//Now estimate moment using RK4 midle step
			(*cuaMesh.pM1)[idx] = (*cuaDiffEq.psM1)[idx] + (*cuaDiffEq.psEval1)[idx] * (dT / 2);
		}
	}
}

__global__ void RunRK4_Step2_Kernel(ManagedAtom_DiffEqCubicCUDA& cuaDiffEq, ManagedAtom_MeshCUDA& cuaMesh)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;

	cuBReal dT = *cuaDiffEq.pdT;

	if (idx < cuaMesh.pM1->linear_size()) {

		if (cuaMesh.pM1->is_not_empty(idx) && !cuaMesh.pM1->is_skipcell(idx)) {

			(*cuaDiffEq.psEval2)[idx] = (cuaDiffEq.*(cuaDiffEq.pODEFunc))(idx);

			//Now estimate moment using RK4 last step
			(*cuaMesh.pM1)[idx] = (*cuaDiffEq.psM1)[idx] + (*cuaDiffEq.psEval2)[idx] * dT;
		}
	}
}

__global__ void RunRK4_Step3_withReductions_Kernel(ManagedAtom_DiffEqCubicCUDA& cuaDiffEq, ManagedAtom_MeshCUDA& cuaMesh)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;

	cuBReal dT = *cuaDiffEq.pdT;

	cuBReal dmdt = 0.0;

	//multiplicative conversion factor from atomic moment (units of muB) to A/m
	cuBReal conversion = (cuBReal)MUB / cuaMesh.pM1->h.dim();

	if (idx < cuaMesh.pM1->linear_size()) {

		if (cuaMesh.pM1->is_not_empty(idx)) {

			if (!cuaMesh.pM1->is_skipcell(idx)) {

				//First evaluate RHS of set equation at the current time step
				cuReal3 rhs = (cuaDiffEq.*(cuaDiffEq.pODEFunc))(idx);

				//Now estimate moment using previous RK4 evaluations
				(*cuaMesh.pM1)[idx] = (*cuaDiffEq.psM1)[idx] + ((*cuaDiffEq.psEval0)[idx] + 2 * (*cuaDiffEq.psEval1)[idx] + 2 * (*cuaDiffEq.psEval2)[idx] + rhs) * (dT / 6);

				if (*cuaDiffEq.prenormalize) {

					cuBReal mu_s = *cuaMesh.pmu_s;
					cuaMesh.update_parameters_mcoarse(idx, *cuaMesh.pmu_s, mu_s);
					(*cuaMesh.pM1)[idx].renormalize(mu_s);
				}

				//obtain maximum normalized dmdt term
				cuBReal Mnorm = (*cuaMesh.pM1)[idx].norm();
				dmdt = cu_GetMagnitude((*cuaMesh.pM1)[idx] - (*cuaDiffEq.psM1)[idx]) / (dT * (cuBReal)GAMMA * conversion * Mnorm * Mnorm);
			}
		}
	}

	if (cuaMesh.pgrel->get0()) reduction_max(0, 1, &dmdt, *cuaDiffEq.pdmdt);
}

__global__ void RunRK4_Step3_withAverageReductions_Kernel(ManagedAtom_DiffEqCubicCUDA& cuaDiffEq, ManagedAtom_MeshCUDA& cuaMesh)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;

	cuBReal dT = *cuaDiffEq.pdT;

	cuReal3 dmdt = cuReal3();
	bool include_in_average = false;

	//multiplicative conversion factor from atomic moment (units of muB) to A/m
	cuBReal conversion = (cuBReal)MUB / cuaMesh.pM1->h.dim();

	if (idx < cuaMesh.pM1->linear_size()) {

		if (cuaMesh.pM1->is_not_empty(idx)) {

			if (!cuaMesh.pM1->is_skipcell(idx)) {

				//First evaluate RHS of set equation at the current time step
				cuReal3 rhs = (cuaDiffEq.*(cuaDiffEq.pODEFunc))(idx);

				//Now estimate moment using previous RK4 evaluations
				(*cuaMesh.pM1)[idx] = (*cuaDiffEq.psM1)[idx] + ((*cuaDiffEq.psEval0)[idx] + 2 * (*cuaDiffEq.psEval1)[idx] + 2 * (*cuaDiffEq.psEval2)[idx] + rhs) * (dT / 6);

				if (*cuaDiffEq.prenormalize) {

					cuBReal mu_s = *cuaMesh.pmu_s;
					cuaMesh.update_parameters_mcoarse(idx, *cuaMesh.pmu_s, mu_s);
					(*cuaMesh.pM1)[idx].renormalize(mu_s);
				}

				//obtain maximum normalized dmdt term
				cuBReal Mnorm = (*cuaMesh.pM1)[idx].norm();
				dmdt = ((*cuaMesh.pM1)[idx] - (*cuaDiffEq.psM1)[idx]) / (dT * (cuBReal)GAMMA * conversion * Mnorm * Mnorm);
				include_in_average = true;
			}
		}
	}

	if (cuaMesh.pgrel->get0()) reduction_avg(0, 1, &dmdt, *cuaDiffEq.pdmdt_av, *cuaDiffEq.pavpoints2, include_in_average);
}

__global__ void RunRK4_Step3_Kernel(ManagedAtom_DiffEqCubicCUDA& cuaDiffEq, ManagedAtom_MeshCUDA& cuaMesh)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;

	cuBReal dT = *cuaDiffEq.pdT;

	if (idx < cuaMesh.pM1->linear_size()) {

		if (cuaMesh.pM1->is_not_empty(idx)) {

			if (!cuaMesh.pM1->is_skipcell(idx)) {

				//First evaluate RHS of set equation at the current time step
				cuReal3 rhs = (cuaDiffEq.*(cuaDiffEq.pODEFunc))(idx);

				//Now estimate moment using previous RK4 evaluations
				(*cuaMesh.pM1)[idx] = (*cuaDiffEq.psM1)[idx] + ((*cuaDiffEq.psEval0)[idx] + 2 * (*cuaDiffEq.psEval1)[idx] + 2 * (*cuaDiffEq.psEval2)[idx] + rhs) * (dT / 6);

				if (*cuaDiffEq.prenormalize) {

					cuBReal mu_s = *cuaMesh.pmu_s;
					cuaMesh.update_parameters_mcoarse(idx, *cuaMesh.pmu_s, mu_s);
					(*cuaMesh.pM1)[idx].renormalize(mu_s);
				}
			}
		}
	}
}

//----------------------------------------- DifferentialEquationCUDA Launchers

//RUNGE KUTTA 4th order

void Atom_DifferentialEquationCubicCUDA::RunRK4(int step, bool calculate_mxh, bool calculate_dmdt, bool stochastic)
{
	switch (step) {

	case 0:

		if (calculate_mxh) {

			if (stochastic) {

				for (mGPU.device_begin(); mGPU != mGPU.device_end(); mGPU++) {

					RunRK4_Step0_withAverageReductions_Kernel <<< (paMeshCUDA->M1.device_size(mGPU) + CUDATHREADS) / CUDATHREADS, CUDATHREADS >>> 
						(cuaDiffEq.get_deviceobject(mGPU), paMeshCUDA->cuaMesh.get_deviceobject(mGPU));
				}
			}
			else {

				for (mGPU.device_begin(); mGPU != mGPU.device_end(); mGPU++) {

					RunRK4_Step0_withReductions_Kernel <<< (paMeshCUDA->M1.device_size(mGPU) + CUDATHREADS) / CUDATHREADS, CUDATHREADS >>> 
						(cuaDiffEq.get_deviceobject(mGPU), paMeshCUDA->cuaMesh.get_deviceobject(mGPU));
				}
			}
		}
		else {

			for (mGPU.device_begin(); mGPU != mGPU.device_end(); mGPU++) {

				RunRK4_Step0_Kernel <<< (paMeshCUDA->M1.device_size(mGPU) + CUDATHREADS) / CUDATHREADS, CUDATHREADS >>> 
					(cuaDiffEq.get_deviceobject(mGPU), paMeshCUDA->cuaMesh.get_deviceobject(mGPU));
			}
		}

		break;

	case 1:

		for (mGPU.device_begin(); mGPU != mGPU.device_end(); mGPU++) {

			RunRK4_Step1_Kernel <<< (paMeshCUDA->M1.device_size(mGPU) + CUDATHREADS) / CUDATHREADS, CUDATHREADS >>> 
				(cuaDiffEq.get_deviceobject(mGPU), paMeshCUDA->cuaMesh.get_deviceobject(mGPU));
		}

		break;

	case 2:

		for (mGPU.device_begin(); mGPU != mGPU.device_end(); mGPU++) {

			RunRK4_Step2_Kernel <<< (paMeshCUDA->M1.device_size(mGPU) + CUDATHREADS) / CUDATHREADS, CUDATHREADS >>> 
				(cuaDiffEq.get_deviceobject(mGPU), paMeshCUDA->cuaMesh.get_deviceobject(mGPU));
		}

		break;

	case 3:

		if (calculate_dmdt) {

			if (stochastic) {

				for (mGPU.device_begin(); mGPU != mGPU.device_end(); mGPU++) {

					RunRK4_Step3_withAverageReductions_Kernel <<< (paMeshCUDA->M1.device_size(mGPU) + CUDATHREADS) / CUDATHREADS, CUDATHREADS >>> 
						(cuaDiffEq.get_deviceobject(mGPU), paMeshCUDA->cuaMesh.get_deviceobject(mGPU));
				}
			}
			else {

				for (mGPU.device_begin(); mGPU != mGPU.device_end(); mGPU++) {

					RunRK4_Step3_withReductions_Kernel <<< (paMeshCUDA->M1.device_size(mGPU) + CUDATHREADS) / CUDATHREADS, CUDATHREADS >>> 
						(cuaDiffEq.get_deviceobject(mGPU), paMeshCUDA->cuaMesh.get_deviceobject(mGPU));
				}
			}
		}
		else {

			for (mGPU.device_begin(); mGPU != mGPU.device_end(); mGPU++) {

				RunRK4_Step3_Kernel <<< (paMeshCUDA->M1.device_size(mGPU) + CUDATHREADS) / CUDATHREADS, CUDATHREADS >>> 
					(cuaDiffEq.get_deviceobject(mGPU), paMeshCUDA->cuaMesh.get_deviceobject(mGPU));
			}
		}

		break;
	}
}

#endif
#endif
#endif