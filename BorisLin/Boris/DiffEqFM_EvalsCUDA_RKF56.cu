#include "hip/hip_runtime.h"
#include "DiffEqFMCUDA.h"

#if COMPILECUDA == 1
#ifdef ODE_EVAL_COMPILATION_RKF56
#ifdef MESH_COMPILATION_FERROMAGNETIC

#include "MeshParamsControlCUDA.h"

#include "Reduction.cuh"

//defines evaluation methods kernel launchers

//----------------------------------------- EVALUATIONS : RKF56

__global__ void RunRKF56_Step0_withReductions_Kernel(ManagedDiffEqFMCUDA& cuDiffEq, ManagedMeshCUDA& cuMesh)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;

	cuBReal dT = *cuDiffEq.pdT;

	cuBReal mxh = 0.0;

	if (idx < cuMesh.pM->linear_size()) {

		if (cuMesh.pM->is_not_empty(idx)) {

			//Save current magnetization for later use
			(*cuDiffEq.psM1)[idx] = (*cuMesh.pM)[idx];

			if (!cuMesh.pM->is_skipcell(idx)) {

				//obtain maximum normalized torque term
				cuBReal Mnorm = (*cuMesh.pM)[idx].norm();
				mxh = cu_GetMagnitude((*cuMesh.pM)[idx] ^ (*cuMesh.pHeff)[idx]) / (Mnorm * Mnorm);

				//First evaluate RHS of set equation at the current time step
				(*cuDiffEq.psEval0)[idx] = (cuDiffEq.*(cuDiffEq.pODEFunc))(idx);

				//Now estimate magnetization using RKF first step
				(*cuMesh.pM)[idx] += (*cuDiffEq.psEval0)[idx] * (dT / 6);
			}
		}
	}

	//only reduce for mxh if grel is not zero (if it's zero this means magnetization dynamics is disabled in this mesh)
	if (cuMesh.pgrel->get0()) {

		reduction_max(0, 1, &mxh, *cuDiffEq.pmxh);
	}
}

__global__ void RunRKF56_Step0_Kernel(ManagedDiffEqFMCUDA& cuDiffEq, ManagedMeshCUDA& cuMesh)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;

	cuBReal dT = *cuDiffEq.pdT;

	if (idx < cuMesh.pM->linear_size()) {

		if (cuMesh.pM->is_not_empty(idx)) {

			//Save current magnetization for later use
			(*cuDiffEq.psM1)[idx] = (*cuMesh.pM)[idx];

			if (!cuMesh.pM->is_skipcell(idx)) {

				//First evaluate RHS of set equation at the current time step
				(*cuDiffEq.psEval0)[idx] = (cuDiffEq.*(cuDiffEq.pODEFunc))(idx);

				//Now estimate magnetization using RKF first step
				(*cuMesh.pM)[idx] += (*cuDiffEq.psEval0)[idx] * (dT / 6);
			}
		}
	}
}

__global__ void RunRKF56_Step1_Kernel(ManagedDiffEqFMCUDA& cuDiffEq, ManagedMeshCUDA& cuMesh)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;

	cuBReal dT = *cuDiffEq.pdT;

	if (idx < cuMesh.pM->linear_size()) {

		if (cuMesh.pM->is_not_empty(idx) && !cuMesh.pM->is_skipcell(idx)) {

			//First evaluate RHS of set equation at the current time step
			(*cuDiffEq.psEval1)[idx] = (cuDiffEq.*(cuDiffEq.pODEFunc))(idx);

			//Now estimate magnetization using RKF midle step 1
			(*cuMesh.pM)[idx] = (*cuDiffEq.psM1)[idx] + (4 * (*cuDiffEq.psEval0)[idx] + 16 * (*cuDiffEq.psEval1)[idx]) * dT / 75;
		}
	}
}

__global__ void RunRKF56_Step2_Kernel(ManagedDiffEqFMCUDA& cuDiffEq, ManagedMeshCUDA& cuMesh)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;

	cuBReal dT = *cuDiffEq.pdT;

	if (idx < cuMesh.pM->linear_size()) {

		if (cuMesh.pM->is_not_empty(idx) && !cuMesh.pM->is_skipcell(idx)) {

			//First evaluate RHS of set equation at the current time step
			(*cuDiffEq.psEval2)[idx] = (cuDiffEq.*(cuDiffEq.pODEFunc))(idx);

			//Now estimate magnetization using RKF midle step 2
			(*cuMesh.pM)[idx] = (*cuDiffEq.psM1)[idx] + (5 * (*cuDiffEq.psEval0)[idx] / 6 - 8 * (*cuDiffEq.psEval1)[idx] / 3 + 5 * (*cuDiffEq.psEval2)[idx] / 2) * dT;
		}
	}
}

__global__ void RunRKF56_Step3_Kernel(ManagedDiffEqFMCUDA& cuDiffEq, ManagedMeshCUDA& cuMesh)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;

	cuBReal dT = *cuDiffEq.pdT;

	if (idx < cuMesh.pM->linear_size()) {

		if (cuMesh.pM->is_not_empty(idx) && !cuMesh.pM->is_skipcell(idx)) {

			//First evaluate RHS of set equation at the current time step
			(*cuDiffEq.psEval3)[idx] = (cuDiffEq.*(cuDiffEq.pODEFunc))(idx);

			//Now estimate magnetization using RKF midle step 3
			(*cuMesh.pM)[idx] = (*cuDiffEq.psM1)[idx] + (-8 * (*cuDiffEq.psEval0)[idx] / 5 + 144 * (*cuDiffEq.psEval1)[idx] / 25 - 4 * (*cuDiffEq.psEval2)[idx] + 16 * (*cuDiffEq.psEval3)[idx] / 25) * dT;
		}
	}
}

__global__ void RunRKF56_Step4_Kernel(ManagedDiffEqFMCUDA& cuDiffEq, ManagedMeshCUDA& cuMesh)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;

	cuBReal dT = *cuDiffEq.pdT;

	if (idx < cuMesh.pM->linear_size()) {

		if (cuMesh.pM->is_not_empty(idx) && !cuMesh.pM->is_skipcell(idx)) {

			//First evaluate RHS of set equation at the current time step
			(*cuDiffEq.psEval4)[idx] = (cuDiffEq.*(cuDiffEq.pODEFunc))(idx);

			//Now estimate magnetization using RKF midle step 4
			(*cuMesh.pM)[idx] = (*cuDiffEq.psM1)[idx] + (361 * (*cuDiffEq.psEval0)[idx] / 320 - 18 * (*cuDiffEq.psEval1)[idx] / 5 + 407 * (*cuDiffEq.psEval2)[idx] / 128 - 11 * (*cuDiffEq.psEval3)[idx] / 80 + 55 * (*cuDiffEq.psEval4)[idx] / 128) * dT;
		}
	}
}

__global__ void RunRKF56_Step5_Kernel(ManagedDiffEqFMCUDA& cuDiffEq, ManagedMeshCUDA& cuMesh)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;

	cuBReal dT = *cuDiffEq.pdT;

	if (idx < cuMesh.pM->linear_size()) {

		if (cuMesh.pM->is_not_empty(idx) && !cuMesh.pM->is_skipcell(idx)) {

			//First evaluate RHS of set equation at the current time step
			(*cuDiffEq.psEval5)[idx] = (cuDiffEq.*(cuDiffEq.pODEFunc))(idx);

			//Now estimate magnetization using RKF midle step 4
			(*cuMesh.pM)[idx] = (*cuDiffEq.psM1)[idx] + (-11 * (*cuDiffEq.psEval0)[idx] / 640 + 11 * (*cuDiffEq.psEval2)[idx] / 256 - 11 * (*cuDiffEq.psEval3)[idx] / 160 + 11 * (*cuDiffEq.psEval4)[idx] / 256) * dT;
		}
	}
}

__global__ void RunRKF56_Step6_Kernel(ManagedDiffEqFMCUDA& cuDiffEq, ManagedMeshCUDA& cuMesh)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;

	cuBReal dT = *cuDiffEq.pdT;

	if (idx < cuMesh.pM->linear_size()) {

		if (cuMesh.pM->is_not_empty(idx) && !cuMesh.pM->is_skipcell(idx)) {

			//First evaluate RHS of set equation at the current time step
			(*cuDiffEq.psEval6)[idx] = (cuDiffEq.*(cuDiffEq.pODEFunc))(idx);

			//Now estimate magnetization using RKF midle step 4
			(*cuMesh.pM)[idx] = (*cuDiffEq.psM1)[idx] + (93 * (*cuDiffEq.psEval0)[idx] / 640 - 18 * (*cuDiffEq.psEval1)[idx] / 5 + 803 * (*cuDiffEq.psEval2)[idx] / 256 - 11 * (*cuDiffEq.psEval3)[idx] / 160 + 99 * (*cuDiffEq.psEval4)[idx] / 256 + (*cuDiffEq.psEval6)[idx]) * dT;
		}
	}
}

__global__ void RunRKF56_Step7_withReductions_Kernel(ManagedDiffEqFMCUDA& cuDiffEq, ManagedMeshCUDA& cuMesh)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;

	cuBReal dT = *cuDiffEq.pdT;

	cuBReal dmdt = 0.0;
	cuBReal lte = 0.0;

	if (idx < cuMesh.pM->linear_size()) {

		if (cuMesh.pM->is_not_empty(idx)) {

			if (!cuMesh.pM->is_skipcell(idx)) {

				//First evaluate RHS of set equation at the current time step
				cuReal3 rhs = (cuDiffEq.*(cuDiffEq.pODEFunc))(idx);

				//5th order evaluation
				(*cuMesh.pM)[idx] = (*cuDiffEq.psM1)[idx] + (31 * (*cuDiffEq.psEval0)[idx] / 384 + 1125 * (*cuDiffEq.psEval2)[idx] / 2816 + 9 * (*cuDiffEq.psEval3)[idx] / 32 + 125 * (*cuDiffEq.psEval4)[idx] / 768 + 5 * (*cuDiffEq.psEval5)[idx] / 66) * dT;

				//local truncation error from 5th order evaluation and 6th order evaluation
				cuReal3 lte_diff = 5 * ((*cuDiffEq.psEval0)[idx] + (*cuDiffEq.psEval5)[idx] - (*cuDiffEq.psEval6)[idx] - rhs) * dT / 66;

				if (*cuDiffEq.prenormalize) {

					cuBReal Ms = *cuMesh.pMs;
					cuMesh.update_parameters_mcoarse(idx, *cuMesh.pMs, Ms);
					(*cuMesh.pM)[idx].renormalize(Ms);
				}

				//obtain maximum normalized dmdt term
				cuBReal Mnorm = (*cuMesh.pM)[idx].norm();
				dmdt = cu_GetMagnitude((*cuMesh.pM)[idx] - (*cuDiffEq.psM1)[idx]) / (dT * (cuBReal)GAMMA * Mnorm * Mnorm);

				//local truncation error (between predicted and corrected)
				lte = cu_GetMagnitude(lte_diff) / (*cuMesh.pM)[idx].norm();
			}
			else {

				cuBReal Ms = *cuMesh.pMs;
				cuMesh.update_parameters_mcoarse(idx, *cuMesh.pMs, Ms);
				(*cuMesh.pM)[idx].renormalize(Ms);		//re-normalize the skipped cells no matter what - temperature can change
			}
		}
	}

	//only reduce for dmdt if grel is not zero (if it's zero this means magnetization dynamics is disabled in this mesh)
	if (cuMesh.pgrel->get0()) {

		reduction_max(0, 1, &dmdt, *cuDiffEq.pdmdt);
	}

	reduction_max(0, 1, &lte, *cuDiffEq.plte);
}

__global__ void RunRKF56_Step7_Kernel(ManagedDiffEqFMCUDA& cuDiffEq, ManagedMeshCUDA& cuMesh)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;

	cuBReal dT = *cuDiffEq.pdT;

	cuBReal lte = 0.0;

	if (idx < cuMesh.pM->linear_size()) {

		if (cuMesh.pM->is_not_empty(idx)) {

			if (!cuMesh.pM->is_skipcell(idx)) {

				//First evaluate RHS of set equation at the current time step
				cuReal3 rhs = (cuDiffEq.*(cuDiffEq.pODEFunc))(idx);

				//5th order evaluation
				(*cuMesh.pM)[idx] = (*cuDiffEq.psM1)[idx] + (31 * (*cuDiffEq.psEval0)[idx] / 384 + 1125 * (*cuDiffEq.psEval2)[idx] / 2816 + 9 * (*cuDiffEq.psEval3)[idx] / 32 + 125 * (*cuDiffEq.psEval4)[idx] / 768 + 5 * (*cuDiffEq.psEval5)[idx] / 66) * dT;

				//local truncation error from 5th order evaluation and 6th order evaluation
				cuReal3 lte_diff = 5 * ((*cuDiffEq.psEval0)[idx] + (*cuDiffEq.psEval5)[idx] - (*cuDiffEq.psEval6)[idx] - rhs) * dT / 66;

				if (*cuDiffEq.prenormalize) {

					cuBReal Ms = *cuMesh.pMs;
					cuMesh.update_parameters_mcoarse(idx, *cuMesh.pMs, Ms);
					(*cuMesh.pM)[idx].renormalize(Ms);
				}

				//local truncation error (between predicted and corrected)
				lte = cu_GetMagnitude(lte_diff) / (*cuMesh.pM)[idx].norm();
			}
			else {

				cuBReal Ms = *cuMesh.pMs;
				cuMesh.update_parameters_mcoarse(idx, *cuMesh.pMs, Ms);
				(*cuMesh.pM)[idx].renormalize(Ms);		//re-normalize the skipped cells no matter what - temperature can change
			}
		}
	}

	reduction_max(0, 1, &lte, *cuDiffEq.plte);
}

//----------------------------------------- DifferentialEquationCUDA Launchers

//RUNGE KUTTA FEHLBERG 5(6)

void DifferentialEquationFMCUDA::RunRKF56(int step, bool calculate_mxh, bool calculate_dmdt)
{
	switch (step) {

	case 0:

		for (mGPU.device_begin(); mGPU != mGPU.device_end(); mGPU++) {

			if (calculate_mxh) {

				RunRKF56_Step0_withReductions_Kernel <<< (pMeshCUDA->M.device_size(mGPU) + CUDATHREADS) / CUDATHREADS, CUDATHREADS >>> 
					(cuDiffEq.get_deviceobject(mGPU), pMeshCUDA->cuMesh.get_deviceobject(mGPU));
			}
			else {

				RunRKF56_Step0_Kernel <<< (pMeshCUDA->M.device_size(mGPU) + CUDATHREADS) / CUDATHREADS, CUDATHREADS >>> 
					(cuDiffEq.get_deviceobject(mGPU), pMeshCUDA->cuMesh.get_deviceobject(mGPU));
			}
		}

		break;

	case 1:

		for (mGPU.device_begin(); mGPU != mGPU.device_end(); mGPU++) {

			RunRKF56_Step1_Kernel <<< (pMeshCUDA->M.device_size(mGPU) + CUDATHREADS) / CUDATHREADS, CUDATHREADS >>> 
				(cuDiffEq.get_deviceobject(mGPU), pMeshCUDA->cuMesh.get_deviceobject(mGPU));
		}

		break;

	case 2:

		for (mGPU.device_begin(); mGPU != mGPU.device_end(); mGPU++) {

			RunRKF56_Step2_Kernel <<< (pMeshCUDA->M.device_size(mGPU) + CUDATHREADS) / CUDATHREADS, CUDATHREADS >>> 
				(cuDiffEq.get_deviceobject(mGPU), pMeshCUDA->cuMesh.get_deviceobject(mGPU));
		}

		break;

	case 3:

		for (mGPU.device_begin(); mGPU != mGPU.device_end(); mGPU++) {

			RunRKF56_Step3_Kernel <<< (pMeshCUDA->M.device_size(mGPU) + CUDATHREADS) / CUDATHREADS, CUDATHREADS >>> 
				(cuDiffEq.get_deviceobject(mGPU), pMeshCUDA->cuMesh.get_deviceobject(mGPU));
		}

		break;

	case 4:

		for (mGPU.device_begin(); mGPU != mGPU.device_end(); mGPU++) {

			RunRKF56_Step4_Kernel <<< (pMeshCUDA->M.device_size(mGPU) + CUDATHREADS) / CUDATHREADS, CUDATHREADS >>> 
				(cuDiffEq.get_deviceobject(mGPU), pMeshCUDA->cuMesh.get_deviceobject(mGPU));
		}

		break;

	case 5:

		for (mGPU.device_begin(); mGPU != mGPU.device_end(); mGPU++) {

			RunRKF56_Step5_Kernel <<< (pMeshCUDA->M.device_size(mGPU) + CUDATHREADS) / CUDATHREADS, CUDATHREADS >>> 
				(cuDiffEq.get_deviceobject(mGPU), pMeshCUDA->cuMesh.get_deviceobject(mGPU));
		}

		break;

	case 6:

		for (mGPU.device_begin(); mGPU != mGPU.device_end(); mGPU++) {

			RunRKF56_Step6_Kernel <<< (pMeshCUDA->M.device_size(mGPU) + CUDATHREADS) / CUDATHREADS, CUDATHREADS >>> 
				(cuDiffEq.get_deviceobject(mGPU), pMeshCUDA->cuMesh.get_deviceobject(mGPU));
		}

		break;

	case 7:

		for (mGPU.device_begin(); mGPU != mGPU.device_end(); mGPU++) {

			if (calculate_dmdt) {

				RunRKF56_Step7_withReductions_Kernel <<< (pMeshCUDA->M.device_size(mGPU) + CUDATHREADS) / CUDATHREADS, CUDATHREADS >>> 
					(cuDiffEq.get_deviceobject(mGPU), pMeshCUDA->cuMesh.get_deviceobject(mGPU));
			}
			else {

				RunRKF56_Step7_Kernel <<< (pMeshCUDA->M.device_size(mGPU) + CUDATHREADS) / CUDATHREADS, CUDATHREADS >>> 
					(cuDiffEq.get_deviceobject(mGPU), pMeshCUDA->cuMesh.get_deviceobject(mGPU));
			}
		}

		break;
	}
}

#endif
#endif
#endif