#include "hip/hip_runtime.h"
#include "Atom_Mesh_CubicCUDA.h"

#if COMPILECUDA == 1

#ifdef MESH_COMPILATION_ATOM_CUBIC

#include "Reduction.cuh"

#include "Atom_MeshParamsControlCUDA.h"

//////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
//////////////////////////////////////////////////////////////////////////////////////////////////////////////////////

__global__ void GetTopologicalCharge_Cubic_Kernel(ManagedAtom_MeshCUDA& cuaMesh, cuRect rectangle, cuBReal& Q)
{
	cuVEC_VC<cuReal3>& M1 = *cuaMesh.pM1;

	int idx = blockIdx.x * blockDim.x + threadIdx.x;

	cuBReal Q_ = 0.0;
	bool include_in_reduction = false;

	cuReal3 pos;

	if (idx < M1.linear_size()) {

		if (M1.is_not_empty(idx)) {

			pos = M1.cellidx_to_position(idx);

			cuBReal Mnorm = M1[idx].norm();

			cuReal33 M_grad = M1.grad_neu(idx);

			cuReal3 dm_dx = M_grad.x / Mnorm;
			cuReal3 dm_dy = M_grad.y / Mnorm;

			Q_ = (M1[idx] / Mnorm) * (dm_dx ^ dm_dy) * M1.h.x * M1.h.y / (4 * (cuBReal)PI * M1.n.z);

			include_in_reduction = true;
		}
	}

	reduction_sum(0, 1, &Q_, Q, include_in_reduction && rectangle.contains(pos));
}

//get topological charge using formula Q = Integral(m.(dm/dx x dm/dy) dxdy) / 4PI
cuBReal Atom_Mesh_CubicCUDA::GetTopologicalCharge(cuRect rectangle)
{
	if (rectangle.IsNull()) rectangle = meshRect;

	Zero_aux_values();

	for (mGPU.device_begin(); mGPU != mGPU.device_end(); mGPU++) {

		GetTopologicalCharge_Cubic_Kernel <<< (M1.device_size(mGPU) + CUDATHREADS) / CUDATHREADS, CUDATHREADS >>> (cuaMesh.get_deviceobject(mGPU), rectangle, aux_real(mGPU));
	}

	return aux_real.to_cpu_sum();
}

//////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
//////////////////////////////////////////////////////////////////////////////////////////////////////////////////////

__global__ void Compute_TopoChargeDensity_Cubic_Kernel(ManagedAtom_MeshCUDA& cuaMesh, cuVEC<cuBReal>& auxVEC_cuBReal)
{
	cuVEC_VC<cuReal3>& M1 = *cuaMesh.pM1;

	int idx = blockIdx.x * blockDim.x + threadIdx.x;

	if (idx < M1.linear_size()) {

		if (M1.is_not_empty(idx)) {

			cuBReal Mnorm = M1[idx].norm();

			cuReal33 M_grad = M1.grad_neu(idx);

			cuReal3 dm_dx = M_grad.x / Mnorm;
			cuReal3 dm_dy = M_grad.y / Mnorm;

			auxVEC_cuBReal[idx] = (M1[idx] / Mnorm) * (dm_dx ^ dm_dy) * M1.h.x * M1.h.y / (4 * (cuBReal)PI * M1.n.z);
		}
		else auxVEC_cuBReal[idx] = 0.0;
	}
}

//compute topological charge density spatial dependence and have it available in auxVEC_cuBReal
//Use formula Qdensity = m.(dm/dx x dm/dy) / 4PI
void Atom_Mesh_CubicCUDA::Compute_TopoChargeDensity(void)
{
	auxVEC_cuBReal.resize(h, meshRect);

	for (mGPU.device_begin(); mGPU != mGPU.device_end(); mGPU++) {

		Compute_TopoChargeDensity_Cubic_Kernel <<< (M1.device_size(mGPU) + CUDATHREADS) / CUDATHREADS, CUDATHREADS >>> (cuaMesh.get_deviceobject(mGPU), auxVEC_cuBReal.get_deviceobject(mGPU));
	}
}

#endif

#endif