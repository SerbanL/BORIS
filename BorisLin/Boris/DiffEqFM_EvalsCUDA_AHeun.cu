#include "hip/hip_runtime.h"
#include "DiffEqFMCUDA.h"

#if COMPILECUDA == 1
#ifdef ODE_EVAL_COMPILATION_AHEUN
#ifdef MESH_COMPILATION_FERROMAGNETIC

#include "MeshParamsControlCUDA.h"

#include "Reduction.cuh"

//defines evaluation methods kernel launchers

//----------------------------------------- EVALUATIONS : Trapezoidal Euler

__global__ void RunAHeun_Step0_withReductions_Kernel(ManagedDiffEqFMCUDA& cuDiffEq, ManagedMeshCUDA& cuMesh)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;

	cuBReal dT = *cuDiffEq.pdT;

	cuReal3 mxh = cuReal3();
	bool include_in_average = false;

	if (idx < cuMesh.pM->linear_size()) {

		if (cuMesh.pM->is_not_empty(idx)) {

			//obtain average normalized torque term
			cuBReal Mnorm = (*cuMesh.pM)[idx].norm();
			mxh = ((*cuMesh.pM)[idx] ^ (*cuMesh.pHeff)[idx]) / (Mnorm * Mnorm);
			include_in_average = true;

			//Save current magnetization for the next step
			(*cuDiffEq.psM1)[idx] = (*cuMesh.pM)[idx];

			if (!cuMesh.pM->is_skipcell(idx)) {

				//First evaluate RHS of set equation at the current time step
				cuReal3 rhs = (cuDiffEq.*(cuDiffEq.pODEFunc))(idx);

				//Now estimate magnetization for the next time step
				(*cuMesh.pM)[idx] += rhs * dT;
			}
		}
	}

	//only reduce for mxh if grel is not zero (if it's zero this means magnetization dynamics is disabled in this mesh)
	if (cuMesh.pgrel->get0()) {

		reduction_avg(0, 1, &mxh, *cuDiffEq.pmxh_av, *cuDiffEq.pavpoints, include_in_average);
	}
}

__global__ void RunAHeun_Step0_Kernel(ManagedDiffEqFMCUDA& cuDiffEq, ManagedMeshCUDA& cuMesh)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;

	cuBReal dT = *cuDiffEq.pdT;

	if (idx < cuMesh.pM->linear_size()) {

		if (cuMesh.pM->is_not_empty(idx)) {

			//Save current magnetization for the next step
			(*cuDiffEq.psM1)[idx] = (*cuMesh.pM)[idx];

			if (!cuMesh.pM->is_skipcell(idx)) {

				//First evaluate RHS of set equation at the current time step
				cuReal3 rhs = (cuDiffEq.*(cuDiffEq.pODEFunc))(idx);

				//Now estimate magnetization for the next time step
				(*cuMesh.pM)[idx] += rhs * dT;
			}
		}
	}
}

__global__ void RunAHeun_Step1_withReductions_Kernel(ManagedDiffEqFMCUDA& cuDiffEq, ManagedMeshCUDA& cuMesh)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;

	cuBReal dT = *cuDiffEq.pdT;

	cuReal3 dmdt = cuReal3();
	cuBReal lte = 0.0;
	bool include_in_average = false;

	if (idx < cuMesh.pM->linear_size()) {

		if (cuMesh.pM->is_not_empty(idx)) {

			if (!cuMesh.pM->is_skipcell(idx)) {

				//First evaluate RHS of set equation at the current time step
				cuReal3 rhs = (cuDiffEq.*(cuDiffEq.pODEFunc))(idx);

				//First save predicted magnetization for lte calculation
				cuReal3 saveM = (*cuMesh.pM)[idx];

				//Now estimate magnetization using the second trapezoidal Euler step equation
				(*cuMesh.pM)[idx] = ((*cuDiffEq.psM1)[idx] + (*cuMesh.pM)[idx] + rhs * dT) / 2;

				if (*cuDiffEq.prenormalize) {

					cuBReal Ms = *cuMesh.pMs;
					cuMesh.update_parameters_mcoarse(idx, *cuMesh.pMs, Ms);
					(*cuMesh.pM)[idx].renormalize(Ms);
				}

				//obtain maximum normalized dmdt term
				cuBReal Mnorm = (*cuMesh.pM)[idx].norm();
				dmdt = ((*cuMesh.pM)[idx] - (*cuDiffEq.psM1)[idx]) / (dT * (cuBReal)GAMMA * Mnorm * Mnorm);
				include_in_average = true;

				//local truncation error (between predicted and corrected)
				lte = cu_GetMagnitude((*cuMesh.pM)[idx] - saveM) / (*cuMesh.pM)[idx].norm();
			}
			else {

				cuBReal Ms = *cuMesh.pMs;
				cuMesh.update_parameters_mcoarse(idx, *cuMesh.pMs, Ms);
				(*cuMesh.pM)[idx].renormalize(Ms);		//re-normalize the skipped cells no matter what - temperature can change
			}
		}
	}

	//only reduce for dmdt (and mxh) if grel is not zero (if it's zero this means magnetization dynamics is disabled in this mesh)
	if (cuMesh.pgrel->get0()) {

		reduction_avg(0, 1, &dmdt, *cuDiffEq.pdmdt_av, *cuDiffEq.pavpoints2, include_in_average);
	}

	reduction_max(0, 1, &lte, *cuDiffEq.plte);
}

__global__ void RunAHeun_Step1_Kernel(ManagedDiffEqFMCUDA& cuDiffEq, ManagedMeshCUDA& cuMesh)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;

	cuBReal dT = *cuDiffEq.pdT;

	cuBReal lte = 0.0;

	if (idx < cuMesh.pM->linear_size()) {

		if (cuMesh.pM->is_not_empty(idx)) {

			if (!cuMesh.pM->is_skipcell(idx)) {

				//First evaluate RHS of set equation at the current time step
				cuReal3 rhs = (cuDiffEq.*(cuDiffEq.pODEFunc))(idx);

				//First save predicted magnetization for lte calculation
				cuReal3 saveM = (*cuMesh.pM)[idx];

				//Now estimate magnetization using the second trapezoidal Euler step equation
				(*cuMesh.pM)[idx] = ((*cuDiffEq.psM1)[idx] + (*cuMesh.pM)[idx] + rhs * dT) / 2;

				if (*cuDiffEq.prenormalize) {

					cuBReal Ms = *cuMesh.pMs;
					cuMesh.update_parameters_mcoarse(idx, *cuMesh.pMs, Ms);
					(*cuMesh.pM)[idx].renormalize(Ms);
				}

				//local truncation error (between predicted and corrected)
				lte = cu_GetMagnitude((*cuMesh.pM)[idx] - saveM) / (*cuMesh.pM)[idx].norm();
			}
			else {

				cuBReal Ms = *cuMesh.pMs;
				cuMesh.update_parameters_mcoarse(idx, *cuMesh.pMs, Ms);
				(*cuMesh.pM)[idx].renormalize(Ms);		//re-normalize the skipped cells no matter what - temperature can change
			}
		}
	}

	reduction_max(0, 1, &lte, *cuDiffEq.plte);
}

//----------------------------------------- DifferentialEquationCUDA Launchers

//TRAPEZOIDAL EULER

void DifferentialEquationFMCUDA::RunAHeun(int step, bool calculate_mxh, bool calculate_dmdt)
{
	if (step == 0) {

		for (mGPU.device_begin(); mGPU != mGPU.device_end(); mGPU++) {

			if (calculate_mxh) {

				RunAHeun_Step0_withReductions_Kernel <<< (pMeshCUDA->M.device_size(mGPU) + CUDATHREADS) / CUDATHREADS, CUDATHREADS >>>
					(cuDiffEq.get_deviceobject(mGPU), pMeshCUDA->cuMesh.get_deviceobject(mGPU));
			}
			else {

				RunAHeun_Step0_Kernel <<< (pMeshCUDA->M.device_size(mGPU) + CUDATHREADS) / CUDATHREADS, CUDATHREADS >>>
					(cuDiffEq.get_deviceobject(mGPU), pMeshCUDA->cuMesh.get_deviceobject(mGPU));
			}
		}
	}
	else {

		for (mGPU.device_begin(); mGPU != mGPU.device_end(); mGPU++) {

			if (calculate_dmdt) {

				RunAHeun_Step1_withReductions_Kernel <<< (pMeshCUDA->M.device_size(mGPU) + CUDATHREADS) / CUDATHREADS, CUDATHREADS >>>
					(cuDiffEq.get_deviceobject(mGPU), pMeshCUDA->cuMesh.get_deviceobject(mGPU));
			}
			else {

				RunAHeun_Step1_Kernel <<< (pMeshCUDA->M.device_size(mGPU) + CUDATHREADS) / CUDATHREADS, CUDATHREADS >>>
					(cuDiffEq.get_deviceobject(mGPU), pMeshCUDA->cuMesh.get_deviceobject(mGPU));
			}
		}
	}
}

#endif
#endif
#endif