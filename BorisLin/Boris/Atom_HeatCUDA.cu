#include "hip/hip_runtime.h"
#include "Atom_HeatCUDA.h"

#if COMPILECUDA == 1

#if defined(MODULE_COMPILATION_HEAT) && ATOMISTIC == 1

#include "Atom_MeshCUDA.h"
#include "Atom_MeshParamsControlCUDA.h"

#include "mcuVEC_halo.cuh"

//-------------------Calculation Methods

//////////////////////////////////////////////////////////////////////////////////////////////////
///////////////////////////////// 1-TEMPERATURE MODEL ////////////////////////////////////////////
//////////////////////////////////////////////////////////////////////////////////////////////////

__global__ void IterateHeatEquation_1TM_Kernel(ManagedAtom_MeshCUDA& cuaMesh, cuVEC<cuBReal>& heatEq_RHS)
{
	cuVEC_VC<cuBReal>& Temp = *cuaMesh.pTemp;
	cuVEC_VC<cuReal3>& E = *cuaMesh.pE;
	cuVEC_VC<cuBReal>& elC = *cuaMesh.pelC;

	int idx = blockIdx.x * blockDim.x + threadIdx.x;

	if (idx < Temp.linear_size()) {

		if (!Temp.is_not_empty(idx) || !Temp.is_not_cmbnd(idx)) return;

		cuBReal density = *cuaMesh.pdensity;
		cuBReal shc = *cuaMesh.pshc;
		cuBReal thermCond = *cuaMesh.pthermCond;
		cuaMesh.update_parameters_tcoarse(idx, *cuaMesh.pdensity, density, *cuaMesh.pshc, shc, *cuaMesh.pthermCond, thermCond);

		cuBReal cro = density * shc;
		cuBReal K = thermCond;

		//heat equation with Robin boundaries (based on Newton's law of cooling)
		heatEq_RHS[idx] = Temp.delsq_robin(idx, K) * K / cro;

		//add Joule heating if set
		if (E.linear_size()) {

			cuBReal joule_eff = *cuaMesh.pjoule_eff;
			cuaMesh.update_parameters_tcoarse(idx, *cuaMesh.pjoule_eff, joule_eff);

			if (cuIsNZ(joule_eff)) {

				cuReal3 position = Temp.cellidx_to_position(idx);

				cuReal3 E_value = E.weighted_average(position, Temp.h);
				cuBReal elC_value = elC.weighted_average(position, Temp.h);

				//add Joule heating source term
				heatEq_RHS[idx] += joule_eff * (elC_value * E_value * E_value) / cro;
			}
		}

		//add heat source contribution if set
		if (cuIsNZ(cuaMesh.pQ->get0())) {
			
			cuBReal Q = *cuaMesh.pQ;
			cuaMesh.update_parameters_tcoarse(idx, *cuaMesh.pQ, Q);

			heatEq_RHS[idx] += Q / cro;
		}
	}
}

__global__ void IterateHeatEquation_1TM_Equation_Kernel(
	ManagedAtom_MeshCUDA& cuaMesh, cuVEC<cuBReal>& heatEq_RHS,
	ManagedFunctionCUDA<cuBReal, cuBReal, cuBReal, cuBReal>& Q_equation,
	cuBReal time)
{
	cuVEC_VC<cuBReal>& Temp = *cuaMesh.pTemp;
	cuVEC_VC<cuReal3>& E = *cuaMesh.pE;
	cuVEC_VC<cuBReal>& elC = *cuaMesh.pelC;

	int idx = blockIdx.x * blockDim.x + threadIdx.x;

	if (idx < Temp.linear_size()) {

		if (!Temp.is_not_empty(idx) || !Temp.is_not_cmbnd(idx)) return;

		cuBReal density = *cuaMesh.pdensity;
		cuBReal shc = *cuaMesh.pshc;
		cuBReal thermCond = *cuaMesh.pthermCond;
		cuaMesh.update_parameters_tcoarse(idx, *cuaMesh.pdensity, density, *cuaMesh.pshc, shc, *cuaMesh.pthermCond, thermCond);

		cuBReal cro = density * shc;
		cuBReal K = thermCond;

		//heat equation with Robin boundaries (based on Newton's law of cooling)
		heatEq_RHS[idx] = Temp.delsq_robin(idx, K) * K / cro;

		//add Joule heating if set
		if (E.linear_size()) {

			cuBReal joule_eff = *cuaMesh.pjoule_eff;
			cuaMesh.update_parameters_tcoarse(idx, *cuaMesh.pjoule_eff, joule_eff);

			if (cuIsNZ(joule_eff)) {

				cuReal3 position = Temp.cellidx_to_position(idx);

				cuReal3 E_value = E.weighted_average(position, Temp.h);
				cuBReal elC_value = elC.weighted_average(position, Temp.h);

				//add Joule heating source term
				heatEq_RHS[idx] += joule_eff * (elC_value * E_value * E_value) / cro;
			}
		}

		//add heat source contribution
		//when evaluating equation must use mrelpos not relpos, as equation set by user expects position to be relative to mcu_VEC origin
		cuReal3 crelpos = Temp.get_crelpos_from_relpos(Temp.cellidx_to_position(idx));
		cuBReal Q = Q_equation.evaluate(crelpos.x, crelpos.y, crelpos.z, time);

		heatEq_RHS[idx] += Q / cro;
	}
}

__global__ void TemperatureFTCS_Atom_Kernel(cuVEC_VC<cuBReal>& Temp, cuVEC<cuBReal>& heatEq_RHS, cuBReal dT)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;

	if (idx < Temp.linear_size()) {

		if (!Temp.is_not_empty(idx) || !Temp.is_not_cmbnd(idx)) return;

		Temp[idx] += dT * heatEq_RHS[idx];
	}
}

void Atom_HeatCUDA::IterateHeatEquation_1TM(cuBReal dT)
{
	paMeshCUDA->Temp.exchange_halos();

	/////////////////////////////////////////
	// Fixed Q set (which could be zero)
	/////////////////////////////////////////

	if (!Q_equation.is_set()) {

		//1. First solve the RHS of the heat equation (centered space) : dT/dt = k del_sq T + j^2, where k = K/ c*ro , j^2 = Jc^2 / (c*ro*sigma)
		for (mGPU.device_begin(); mGPU != mGPU.device_end(); mGPU++) {

			IterateHeatEquation_1TM_Kernel <<< (paMeshCUDA->Temp.device_size(mGPU) + CUDATHREADS) / CUDATHREADS, CUDATHREADS >>>
				(paMeshCUDA->cuaMesh.get_deviceobject(mGPU), heatEq_RHS.get_deviceobject(mGPU));
		}
	}

	/////////////////////////////////////////
	// Q set using text equation
	/////////////////////////////////////////

	else {

		//1. First solve the RHS of the heat equation (centered space) : dT/dt = k del_sq T + j^2, where k = K/ c*ro , j^2 = Jc^2 / (c*ro*sigma)
		for (mGPU.device_begin(); mGPU != mGPU.device_end(); mGPU++) {

			IterateHeatEquation_1TM_Equation_Kernel <<< (paMeshCUDA->Temp.device_size(mGPU) + CUDATHREADS) / CUDATHREADS, CUDATHREADS >>>
				(paMeshCUDA->cuaMesh.get_deviceobject(mGPU), heatEq_RHS.get_deviceobject(mGPU),
				Q_equation.get_x(mGPU), paMeshCUDA->GetStageTime());
		}
	}

	//kernel launches asynchronous so must synchronize here since kernel below updated Temp data (which would otherwise lead to a data race between different devices)
	mGPU.synchronize_if_multi();
	
	//2. Now use forward time to advance by dT
	for (mGPU.device_begin(); mGPU != mGPU.device_end(); mGPU++) {

		TemperatureFTCS_Atom_Kernel <<< (paMeshCUDA->Temp.device_size(mGPU) + CUDATHREADS) / CUDATHREADS, CUDATHREADS >>>
			(paMeshCUDA->Temp.get_deviceobject(mGPU), heatEq_RHS.get_deviceobject(mGPU), dT);
	}
}

//////////////////////////////////////////////////////////////////////////////////////////////////
///////////////////////////////// 2-TEMPERATURE MODEL ////////////////////////////////////////////
//////////////////////////////////////////////////////////////////////////////////////////////////

__global__ void IterateHeatEquation_2TM_Kernel(ManagedAtom_MeshCUDA& cuaMesh, cuVEC<cuBReal>& heatEq_RHS, cuBReal dT)
{
	cuVEC_VC<cuBReal>& Temp = *cuaMesh.pTemp;
	cuVEC_VC<cuBReal>& Temp_l = *cuaMesh.pTemp_l;
	cuVEC_VC<cuReal3>& E = *cuaMesh.pE;
	cuVEC_VC<cuBReal>& elC = *cuaMesh.pelC;

	int idx = blockIdx.x * blockDim.x + threadIdx.x;

	if (idx < Temp.linear_size()) {

		if (!Temp.is_not_empty(idx)) return;

		cuBReal density = *cuaMesh.pdensity;
		cuBReal shc = *cuaMesh.pshc;
		cuBReal shc_e = *cuaMesh.pshc_e;
		cuBReal G_el = *cuaMesh.pG_e;
		cuBReal thermCond = *cuaMesh.pthermCond;
		cuaMesh.update_parameters_tcoarse(idx, *cuaMesh.pdensity, density, *cuaMesh.pshc, shc, *cuaMesh.pshc_e, shc_e, *cuaMesh.pG_e, G_el, *cuaMesh.pthermCond, thermCond);

		cuBReal cro_e = density * shc_e;
		cuBReal K = thermCond;

		//1. Itinerant Electrons Temperature

		if (Temp.is_not_cmbnd(idx)) {

			//heat equation with Robin boundaries (based on Newton's law of cooling) and coupling to lattice
			heatEq_RHS[idx] = (Temp.delsq_robin(idx, K) * K - G_el * (Temp[idx] - Temp_l[idx])) / cro_e;

			//add Joule heating if set
			if (E.linear_size()) {

				cuBReal joule_eff = *cuaMesh.pjoule_eff;
				cuaMesh.update_parameters_tcoarse(idx, *cuaMesh.pjoule_eff, joule_eff);

				if (cuIsNZ(joule_eff)) {

					cuReal3 position = Temp.cellidx_to_position(idx);

					cuBReal elC_value = elC.weighted_average(position, Temp.h);
					cuReal3 E_value = E.weighted_average(position, Temp.h);

					//add Joule heating source term
					heatEq_RHS[idx] += joule_eff * (elC_value * E_value * E_value) / cro_e;
				}
			}

			//add heat source contribution if set
			if (cuIsNZ(cuaMesh.pQ->get0())) {

				cuBReal Q = *cuaMesh.pQ;
				cuaMesh.update_parameters_tcoarse(idx, *cuaMesh.pQ, Q);

				heatEq_RHS[idx] += Q / cro_e;
			}
		}

		//2. Lattice Temperature

		//lattice specific heat capacity + electron specific heat capacity gives the total specific heat capacity
		cuBReal cro_l = density * (shc - shc_e);

		Temp_l[idx] += dT * G_el * (Temp[idx] - Temp_l[idx]) / cro_l;
	}
}

__global__ void IterateHeatEquation_2TM_Equation_Kernel(
	ManagedAtom_MeshCUDA& cuaMesh, cuVEC<cuBReal>& heatEq_RHS,
	ManagedFunctionCUDA<cuBReal, cuBReal, cuBReal, cuBReal>& Q_equation,
	cuBReal time, cuBReal dT)
{
	cuVEC_VC<cuBReal>& Temp = *cuaMesh.pTemp;
	cuVEC_VC<cuBReal>& Temp_l = *cuaMesh.pTemp_l;
	cuVEC_VC<cuReal3>& E = *cuaMesh.pE;
	cuVEC_VC<cuBReal>& elC = *cuaMesh.pelC;

	int idx = blockIdx.x * blockDim.x + threadIdx.x;

	if (idx < Temp.linear_size()) {

		if (!Temp.is_not_empty(idx)) return;

		cuBReal density = *cuaMesh.pdensity;
		cuBReal shc = *cuaMesh.pshc;
		cuBReal shc_e = *cuaMesh.pshc_e;
		cuBReal G_el = *cuaMesh.pG_e;
		cuBReal thermCond = *cuaMesh.pthermCond;
		cuaMesh.update_parameters_tcoarse(idx, *cuaMesh.pdensity, density, *cuaMesh.pshc, shc, *cuaMesh.pshc_e, shc_e, *cuaMesh.pG_e, G_el, *cuaMesh.pthermCond, thermCond);

		cuBReal cro_e = density * shc_e;
		cuBReal K = thermCond;

		//1. Itinerant Electrons Temperature

		if (Temp.is_not_cmbnd(idx)) {

			//heat equation with Robin boundaries (based on Newton's law of cooling) and coupling to lattice
			heatEq_RHS[idx] = (Temp.delsq_robin(idx, K) * K - G_el * (Temp[idx] - Temp_l[idx])) / cro_e;

			//add Joule heating if set
			if (E.linear_size()) {

				cuBReal joule_eff = *cuaMesh.pjoule_eff;
				cuaMesh.update_parameters_tcoarse(idx, *cuaMesh.pjoule_eff, joule_eff);

				if (cuIsNZ(joule_eff)) {

					cuReal3 position = Temp.cellidx_to_position(idx);

					cuBReal elC_value = elC.weighted_average(position, Temp.h);
					cuReal3 E_value = E.weighted_average(position, Temp.h);

					//add Joule heating source term
					heatEq_RHS[idx] += joule_eff * (elC_value * E_value * E_value) / cro_e;
				}
			}

			//add heat source contribution
			//when evaluating equation must use mrelpos not relpos, as equation set by user expects position to be relative to mcu_VEC origin
			cuReal3 crelpos = Temp.get_crelpos_from_relpos(Temp.cellidx_to_position(idx));
			cuBReal Q = Q_equation.evaluate(crelpos.x, crelpos.y, crelpos.z, time);

			heatEq_RHS[idx] += Q / cro_e;
		}

		//2. Lattice Temperature

		//lattice specific heat capacity + electron specific heat capacity gives the total specific heat capacity
		cuBReal cro_l = density * (shc - shc_e);

		Temp_l[idx] += dT * G_el * (Temp[idx] - Temp_l[idx]) / cro_l;
	}
}

void Atom_HeatCUDA::IterateHeatEquation_2TM(cuBReal dT)
{
	paMeshCUDA->Temp.exchange_halos();

	/////////////////////////////////////////
	// Fixed Q set (which could be zero)
	/////////////////////////////////////////

	if (!Q_equation.is_set()) {

		//1. First solve the RHS of the heat equation (centered space) : dT/dt = k del_sq T + j^2, where k = K/ c*ro , j^2 = Jc^2 / (c*ro*sigma)
		for (mGPU.device_begin(); mGPU != mGPU.device_end(); mGPU++) {

			IterateHeatEquation_2TM_Kernel <<< (paMeshCUDA->Temp.device_size(mGPU) + CUDATHREADS) / CUDATHREADS, CUDATHREADS >>>
				(paMeshCUDA->cuaMesh.get_deviceobject(mGPU), heatEq_RHS.get_deviceobject(mGPU), dT);
		}
	}

	/////////////////////////////////////////
	// Q set using text equation
	/////////////////////////////////////////

	else {

		//1. First solve the RHS of the heat equation (centered space) : dT/dt = k del_sq T + j^2, where k = K/ c*ro , j^2 = Jc^2 / (c*ro*sigma)
		for (mGPU.device_begin(); mGPU != mGPU.device_end(); mGPU++) {

			IterateHeatEquation_2TM_Equation_Kernel <<< (paMeshCUDA->Temp.device_size(mGPU) + CUDATHREADS) / CUDATHREADS, CUDATHREADS >>> 
				(paMeshCUDA->cuaMesh.get_deviceobject(mGPU), heatEq_RHS.get_deviceobject(mGPU),
				Q_equation.get_x(mGPU), paMeshCUDA->GetStageTime(), dT);
		}
	}

	//kernel launches asynchronous so must synchronize here since kernel below updated Temp data (which would otherwise lead to a data race between different devices)
	mGPU.synchronize_if_multi();

	//2. Now use forward time to advance by dT
	for (mGPU.device_begin(); mGPU != mGPU.device_end(); mGPU++) {

		TemperatureFTCS_Atom_Kernel <<< (paMeshCUDA->Temp.device_size(mGPU) + CUDATHREADS) / CUDATHREADS, CUDATHREADS >>> 
			(paMeshCUDA->Temp.get_deviceobject(mGPU), heatEq_RHS.get_deviceobject(mGPU), dT);
	}
}

//-------------------Setters

//non-uniform temperature setting
__global__ void SetBaseTemperature_Nonuniform_Kernel(ManagedAtom_MeshCUDA& cuaMesh, cuBReal Temperature)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;

	cuVEC_VC<cuBReal>& Temp = *cuaMesh.pTemp;
	cuVEC_VC<cuBReal>& Temp_l = *cuaMesh.pTemp_l;

	if (idx < Temp.linear_size()) {

		if (Temp.is_not_empty(idx)) {

			cuBReal cT = *cuaMesh.pcT;
			cuaMesh.update_parameters_tcoarse(idx, *cuaMesh.pcT, cT);

			Temp[idx] = cT * Temperature;

			if (Temp_l.linear_size()) Temp_l[idx] = cT * Temperature;
		}
	}
}

//set Temp non-uniformly as specified through the cT mesh parameter
void Atom_HeatCUDA::SetBaseTemperature_Nonuniform(cuBReal Temperature)
{
	for (mGPU.device_begin(); mGPU != mGPU.device_end(); mGPU++) {

		SetBaseTemperature_Nonuniform_Kernel <<< (paMeshCUDA->Temp.device_size(mGPU) + CUDATHREADS) / CUDATHREADS, CUDATHREADS >>>
			(paMeshCUDA->cuaMesh.get_deviceobject(mGPU), Temperature);
	}
}

//set Temp uniformly to base temperature
void Atom_HeatCUDA::SetBaseTemperature(cuBReal Temperature)
{
	paMeshCUDA->Temp.setnonempty(Temperature);
	paMeshCUDA->Temp_l.setnonempty(Temperature);
}

__global__ void SetFromGlobalTemperature_Kernel(ManagedAtom_MeshCUDA& cuaMesh, cuVEC_VC<cuBReal>& globalTemp)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;

	cuVEC_VC<cuBReal>& Temp = *cuaMesh.pTemp;
	cuVEC_VC<cuBReal>& Temp_l = *cuaMesh.pTemp_l;
	cuBReal& base_temperature = *cuaMesh.pbase_temperature;

	if (idx < Temp.linear_size()) {

		if (Temp.is_not_empty(idx)) {

			cuBReal cT = *cuaMesh.pcT;
			cuaMesh.update_parameters_tcoarse(idx, *cuaMesh.pcT, cT);

			cuReal3 abs_pos = Temp.cellidx_to_position(idx) + Temp.rect.s;

			if (globalTemp.rect.contains(abs_pos)) {

				Temp[idx] = base_temperature + globalTemp[abs_pos - globalTemp.rect.s] * cT;
			}
			else {

				Temp[idx] = base_temperature;
			}

			if (Temp_l.linear_size()) Temp_l[idx] = Temp[idx];
		}
	}
}

//transfer values from globalTemp to Temp in this mesh
//globalTemp values are scaled by cT, and then added to base temperature
void Atom_HeatCUDA::SetFromGlobalTemperature(mcu_VEC_VC(cuBReal)& globalTemp)
{
	for (mGPU.device_begin(); mGPU != mGPU.device_end(); mGPU++) {

		SetFromGlobalTemperature_Kernel <<< (paMeshCUDA->Temp.device_size(mGPU) + CUDATHREADS) / CUDATHREADS, CUDATHREADS >>>
			(paMeshCUDA->cuaMesh.get_deviceobject(mGPU), globalTemp.get_deviceobject(mGPU));
	}
}

#endif

#endif