#include "hip/hip_runtime.h"
#include "TransportBaseCUDA.h"

#if COMPILECUDA == 1

#ifdef MODULE_COMPILATION_TRANSPORT

#include "Reduction.cuh"

#include "MeshBaseCUDA.h"

__global__ void ZeroAux_kernel(cuBReal& auxReal)
{
	if (threadIdx.x == 0) auxReal = 0.0;
}

//--------------------------------------------------------------- Electrode Current

__global__ void CalculateElectrodeCurrent_X_Kernel(cuVEC_VC<cuBReal>& elC, cuVEC_VC<cuReal3>& E, cuBReal& current, cuBox electrode_box, int sign)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;

	int stride = electrode_box.e.j - electrode_box.s.j;

	//negative x side of box, parse j and k
	cuINT3 ijk = cuINT3(electrode_box.s.i, (idx % stride) + electrode_box.s.j, (idx / stride) + electrode_box.s.k);

	cuReal3 h_e = E.h;

	cuBReal current_ = 0.0;

	if (idx < stride * (electrode_box.e.k - electrode_box.s.k)) {

		current_ = sign * elC[ijk] * E[ijk].x * h_e.y * h_e.z;
	}

	reduction_sum(0, 1, &current_, current);
}

__global__ void CalculateElectrodeCurrent_Y_Kernel(cuVEC_VC<cuBReal>& elC, cuVEC_VC<cuReal3>& E, cuBReal& current, cuBox electrode_box, int sign)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;

	int stride = electrode_box.e.i - electrode_box.s.i;

	//negative y side of box, parse i and k
	cuINT3 ijk = cuINT3((idx % stride) + electrode_box.s.i, electrode_box.s.j, (idx / stride) + electrode_box.s.k);

	cuReal3 h_e = E.h;

	cuBReal current_ = 0.0;

	if (idx < stride * (electrode_box.e.k - electrode_box.s.k)) {

		current_ = sign * elC[ijk] * E[ijk].y * h_e.x * h_e.z;
	}

	reduction_sum(0, 1, &current_, current);
}

__global__ void CalculateElectrodeCurrent_Z_Kernel(cuVEC_VC<cuBReal>& elC, cuVEC_VC<cuReal3>& E, cuBReal& current, cuBox electrode_box, int sign)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;

	int stride = electrode_box.e.i - electrode_box.s.i;

	//negative z side of box, parse i and j
	cuINT3 ijk = cuINT3((idx % stride) + electrode_box.s.i, (idx / stride) + electrode_box.s.j, electrode_box.s.k);

	cuReal3 h_e = E.h;

	cuBReal current_ = 0.0;

	if (idx < stride * (electrode_box.e.j - electrode_box.s.j)) {

		current_ = sign * elC[ijk] * E[ijk].z * h_e.x * h_e.y;
	}

	reduction_sum(0, 1, &current_, current);
}

cuBReal TransportBaseCUDA::CalculateElectrodeCurrent(cuBox electrode_box, cuINT3 sign)
{
	//calculate current from current density in cells just next to the box
	//Normally there is only one side of the box we can use so it's easier to separate into multiple kernels - one per side.

	//Obtain the current by reduction in the energy value

	for (mGPU.device_begin(); mGPU != mGPU.device_end(); mGPU++) {

		ZeroAux_kernel <<< 1, CUDATHREADS >>>
			(auxReal(mGPU));
	}

	//cells on -x side
	if (sign.x) {

		for (mGPU.device_begin(); mGPU != mGPU.device_end(); mGPU++) {

			cuBox electrode_sbox = pMeshBaseCUDA->elC.device_sub_box(electrode_box, mGPU);
			size_t ker_size = (electrode_sbox.e.j - electrode_sbox.s.j) * (electrode_sbox.e.k - electrode_sbox.s.k);

			CalculateElectrodeCurrent_X_Kernel <<< (ker_size + CUDATHREADS) / CUDATHREADS, CUDATHREADS >>> 
				(pMeshBaseCUDA->elC.get_deviceobject(mGPU), pMeshBaseCUDA->E.get_deviceobject(mGPU), auxReal(mGPU), electrode_sbox, sign.x);
		}
	}

	//cells on -y side
	if (sign.y) {

		for (mGPU.device_begin(); mGPU != mGPU.device_end(); mGPU++) {

			cuBox electrode_sbox = pMeshBaseCUDA->elC.device_sub_box(electrode_box, mGPU);
			size_t ker_size = (electrode_sbox.e.i - electrode_sbox.s.i) * (electrode_sbox.e.k - electrode_sbox.s.k);

			CalculateElectrodeCurrent_Y_Kernel <<< (ker_size + CUDATHREADS) / CUDATHREADS, CUDATHREADS >>> 
				(pMeshBaseCUDA->elC.get_deviceobject(mGPU), pMeshBaseCUDA->E.get_deviceobject(mGPU), auxReal(mGPU), electrode_sbox, sign.y);
		}
	}

	//cells on -z side
	if (sign.z) {

		for (mGPU.device_begin(); mGPU != mGPU.device_end(); mGPU++) {

			cuBox electrode_sbox = pMeshBaseCUDA->elC.device_sub_box(electrode_box, mGPU);
			size_t ker_size = (electrode_sbox.e.i - electrode_sbox.s.i) * (electrode_sbox.e.j - electrode_sbox.s.j);

			CalculateElectrodeCurrent_Z_Kernel <<< (ker_size + CUDATHREADS) / CUDATHREADS, CUDATHREADS >>> 
				(pMeshBaseCUDA->elC.get_deviceobject(mGPU), pMeshBaseCUDA->E.get_deviceobject(mGPU), auxReal(mGPU), electrode_sbox, sign.z);
		}
	}

	//energy has the current value; reset it after as we don't want to count it to the total energy density
	double current = auxReal.to_cpu_sum();

	return current;
}

#endif

#endif