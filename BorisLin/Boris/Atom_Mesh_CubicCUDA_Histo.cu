#include "hip/hip_runtime.h"
#include "Atom_Mesh_CubicCUDA.h"

#if COMPILECUDA == 1

#ifdef MESH_COMPILATION_ATOM_CUBIC

#include "Reduction.cuh"

#include "Atom_MeshParamsControlCUDA.h"

__global__ void SetField_SCHisto_CUDA(ManagedAtom_MeshCUDA& cuaMesh, cuReal3& Ha)
{
	if (threadIdx.x == 0) {

		//set applied field for MC
		cuaMesh.Ha_MC = Ha;
	}
}

__global__ void ZeroField_SCHisto_CUDA(ManagedAtom_MeshCUDA& cuaMesh)
{
	if (threadIdx.x == 0) {

		//set applied field for MC
		cuaMesh.Ha_MC = cuReal3();
	}
}

//////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
//////////////////////////////////////////////////////////////////////////////////////////////////////////////////////

__global__ void GetThermodynamicAverageMagnetization_kernel(ManagedAtom_MeshCUDA& cuaMesh, cuRect rectangle, cuBReal& Z, cuReal3& Mthav)
{
	cuVEC_VC<cuReal3>& M1 = *cuaMesh.pM1;
	cuVEC_VC<cuBReal>& Temp = *cuaMesh.pTemp;

	int idx = blockIdx.x * blockDim.x + threadIdx.x;

	cuBReal Z_value = 0.0;
	cuReal3 Mthav_value = cuReal3();
	bool include_in_reduction = false;

	cuReal3 pos;

	if (idx < M1.linear_size()) {

		if (M1.is_not_empty(idx)) {

			pos = M1.cellidx_to_position(idx);

			cuBReal J = *cuaMesh.pJ;
			cuaMesh.update_parameters_mcoarse(idx, *cuaMesh.pJ, J);
			cuBReal E0 = -6 * J;

			cuBReal Ei = 0.0;
			for (int midx = 0; midx < cuaMesh.num_SC_MCFuncs; midx++)
				if (cuaMesh.pSC_MCFuncs[midx]) Ei += (cuaMesh.*(cuaMesh.pSC_MCFuncs[midx]))(idx, cuReal3());

			cuBReal Temperature;
			if (Temp.linear_size()) Temperature = Temp[M1.cellidx_to_position(idx)];
			else Temperature = *cuaMesh.pbase_temperature;

			cuBReal w = exp(-(Ei - E0) / ((cuBReal)BOLTZMANN * Temperature));
			Z_value = w;

			Mthav_value = w * M1[idx];

			include_in_reduction = true;
		}
	}

	reduction_sum(0, 1, &Z_value, Z, include_in_reduction && rectangle.contains(pos));
	reduction_sum(0, 1, &Mthav_value, Mthav, include_in_reduction && rectangle.contains(pos));
}

//calculate thermodynamic average of magnetization
cuReal3 Atom_Mesh_CubicCUDA::GetThermodynamicAverageMagnetization(cuRect rectangle)
{
	if (rectangle.IsNull()) rectangle = meshRect;

	Zero_aux_values();

	if (pHa) {

		for (mGPU.device_begin(); mGPU != mGPU.device_end(); mGPU++) {

			SetField_SCHisto_CUDA <<< 1, CUDATHREADS >>> (cuaMesh.get_deviceobject(mGPU), (*pHa)(mGPU));
		}
	}
	else {

		for (mGPU.device_begin(); mGPU != mGPU.device_end(); mGPU++) {

			ZeroField_SCHisto_CUDA <<< 1, CUDATHREADS >>> (cuaMesh.get_deviceobject(mGPU));
		}
	}

	for (mGPU.device_begin(); mGPU != mGPU.device_end(); mGPU++) {

		GetThermodynamicAverageMagnetization_kernel <<< (M1.device_size(mGPU) + CUDATHREADS) / CUDATHREADS, CUDATHREADS >>>
			(cuaMesh.get_deviceobject(mGPU), rectangle, aux_real(mGPU), aux_real3(mGPU));
	}

	return aux_real3.to_cpu() * (MUB / h.dim()) / aux_real.to_cpu();
}

//////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
//////////////////////////////////////////////////////////////////////////////////////////////////////////////////////

__global__ void Get_ThAvHistogram_preaverage_kernel(
	ManagedAtom_MeshCUDA& cuaMesh, cuVEC<cuReal3>& auxVEC_w, cuVEC<cuBReal>& auxVEC_Z, cuINT3 num_av_cells, cuINT3 av_cell_dims)
{
	//launched with num_av_cells.dim(), (1024 or CUDATHREADS) kernel dimensions
	//i.e there are num_av_cells.dim() segments, each of size of av_cell_dims.dim()

	cuVEC_VC<cuReal3>& M1 = *cuaMesh.pM1;
	cuVEC_VC<cuBReal>& Temp = *cuaMesh.pTemp;

	//segment size
	size_t K = av_cell_dims.dim();

	//linear index in this segment, starting at threadIdx.x value
	int linear_idx = threadIdx.x;

	//partial segment sum in this thread
	cuReal3 sum_w = cuReal3();
	cuBReal sum_Z = 0.0;

	//each segment receives up to 1024 worker threads. first use them to load all input data in current segment.
	while (linear_idx < K) {

		//segment ijk values
		int i_seg = blockIdx.x % num_av_cells.x;
		int j_seg = (blockIdx.x / num_av_cells.x) % num_av_cells.y;
		int k_seg = blockIdx.x / (num_av_cells.x * num_av_cells.y);

		//convert linear segment index to cuvec ijk index for this segment
		int i = linear_idx % av_cell_dims.x;
		int j = (linear_idx / av_cell_dims.x) % av_cell_dims.y;
		int k = linear_idx / (av_cell_dims.x * av_cell_dims.y);

		//finally required ijk index in cuvec
		cuINT3 ijk = cuINT3(i_seg * av_cell_dims.i + i, j_seg * av_cell_dims.j + j, k_seg * av_cell_dims.k + k);
		int idx = ijk.i + ijk.j * M1.n.x + ijk.k * M1.n.x*M1.n.y;

		if (idx < M1.linear_size() && M1.is_not_empty(idx)) {
			
			cuBReal J = *cuaMesh.pJ;
			cuaMesh.update_parameters_mcoarse(idx, *cuaMesh.pJ, J);
			cuBReal E0 = -6 * J;

			cuBReal Ei = 0.0;
			for (int midx = 0; midx < cuaMesh.num_SC_MCFuncs; midx++)
				if (cuaMesh.pSC_MCFuncs[midx]) Ei += (cuaMesh.*(cuaMesh.pSC_MCFuncs[midx]))(idx, cuReal3());

			cuBReal Temperature;
			if (Temp.linear_size()) Temperature = Temp[M1.cellidx_to_position(idx)];
			else Temperature = *cuaMesh.pbase_temperature;

			cuBReal w = exp(-(Ei - E0) / ((cuBReal)BOLTZMANN * Temperature));
			
			sum_Z += w;
			
			sum_w += w * M1[idx];
		}

		linear_idx += blockDim.x;
	}

	//now reduced all partial segment sums in this block
	reduction_sum(0, 1, &sum_w, auxVEC_w[blockIdx.x]);
	reduction_sum(0, 1, &sum_Z, auxVEC_Z[blockIdx.x]);
}

__global__ void Get_ThAvHistogram_preaverage_finish_kernel(cuVEC<cuReal3>& auxVEC_w, cuVEC<cuBReal>& auxVEC_Z)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;

	if (idx < auxVEC_w.linear_size()) {

		if (auxVEC_Z[idx]) auxVEC_w[idx] /= auxVEC_Z[idx];
	}
}

bool Atom_Mesh_CubicCUDA::Get_ThAvHistogram(std::vector<double>& histogram_x, std::vector<double>& histogram_p, int num_bins, double& min, double& max, cuINT3 macrocell_dims)
{
	//First do thermal cell-wise pre-averaging

	//allocate required memory for auxVEC
	cuINT3 num_av_cells = round((cuReal3)n / macrocell_dims);
	auxVEC_cuReal3.assign(num_av_cells, cuReal3());
	auxVEC_cuBReal.assign(num_av_cells, 0.0);

	if (pHa) {

		for (mGPU.device_begin(); mGPU != mGPU.device_end(); mGPU++) {

			SetField_SCHisto_CUDA <<< 1, CUDATHREADS >>> (cuaMesh.get_deviceobject(mGPU), (*pHa)(mGPU));
		}
	}
	else {

		for (mGPU.device_begin(); mGPU != mGPU.device_end(); mGPU++) {

			ZeroField_SCHisto_CUDA <<< 1, CUDATHREADS >>> (cuaMesh.get_deviceobject(mGPU));
		}
	}

	//cell-wise pre-averaging
	for (mGPU.device_begin(); mGPU != mGPU.device_end(); mGPU++) {

		Get_ThAvHistogram_preaverage_kernel <<< auxVEC_cuReal3.device_size(mGPU), CUDATHREADS >>>
			(cuaMesh.get_deviceobject(mGPU), auxVEC_cuReal3.get_deviceobject(mGPU), auxVEC_cuBReal.get_deviceobject(mGPU), num_av_cells, macrocell_dims);
	}

	for (mGPU.device_begin(); mGPU != mGPU.device_end(); mGPU++) {

		Get_ThAvHistogram_preaverage_finish_kernel <<< (auxVEC_cuReal3.device_size(mGPU) + CUDATHREADS) / CUDATHREADS, CUDATHREADS >>> 
			(auxVEC_cuReal3.get_deviceobject(mGPU), auxVEC_cuBReal.get_deviceobject(mGPU));
	}

	//get histogram from auxVEC
	return auxVEC_cuReal3.get_mag_histogram(histogram_x, histogram_p, num_bins, min, max, num_av_cells.dim());
}

//As for Get_AngHistogram, but use thermal averaging in each macrocell
bool Atom_Mesh_CubicCUDA::Get_ThAvAngHistogram(std::vector<double>& histogram_x, std::vector<double>& histogram_p, int num_bins, double& min, double& max, cuINT3 macrocell_dims, cuReal3 ndir)
{
	//First do thermal cell-wise pre-averaging

	//allocate required memory for auxVEC
	cuINT3 num_av_cells = round((cuReal3)n / macrocell_dims);
	auxVEC_cuReal3.assign(num_av_cells, cuReal3());
	auxVEC_cuBReal.assign(num_av_cells, 0.0);

	//cell-wise pre-averaging
	for (mGPU.device_begin(); mGPU != mGPU.device_end(); mGPU++) {

		Get_ThAvHistogram_preaverage_kernel <<< auxVEC_cuReal3.device_size(mGPU), CUDATHREADS >>>
			(cuaMesh.get_deviceobject(mGPU), auxVEC_cuReal3.get_deviceobject(mGPU), auxVEC_cuBReal.get_deviceobject(mGPU), num_av_cells, macrocell_dims);
	}

	for (mGPU.device_begin(); mGPU != mGPU.device_end(); mGPU++) {

		Get_ThAvHistogram_preaverage_finish_kernel <<< (auxVEC_cuReal3.device_size(mGPU) + CUDATHREADS) / CUDATHREADS, CUDATHREADS >>> 
			(auxVEC_cuReal3.get_deviceobject(mGPU), auxVEC_cuBReal.get_deviceobject(mGPU));
	}

	if (ndir.IsNull()) ndir = GetThermodynamicAverageMagnetization(cuRect()).normalized();

	//get histogram from auxVEC
	return auxVEC_cuReal3.get_ang_histogram(histogram_x, histogram_p, num_bins, min, max, num_av_cells.dim(), cuINT3(1), ndir);
}

#endif

#endif