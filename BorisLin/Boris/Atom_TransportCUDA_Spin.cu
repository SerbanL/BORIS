#include "hip/hip_runtime.h"
#include "Atom_TransportCUDA.h"

#if COMPILECUDA == 1

#if defined(MODULE_COMPILATION_TRANSPORT) && ATOMISTIC == 1

#include "mcuVEC_solve.cuh"
#include "mcuVEC_halo.cuh"

#include "Atom_MeshCUDA.h"
#include "SuperMeshCUDA.h"
#include "Atom_MeshParamsControlCUDA.h"
#include "MeshParamsControlCUDA.h"

#include "ManagedAtom_DiffEqPolicyCubicCUDA.h"

//-------------------Calculation Methods : Iterate Spin-Charge Solver

void Atom_TransportCUDA::IterateSpinSolver_Charge_SOR(mcu_val<cuBReal>& damping, mcu_val<cuBReal>& max_error, mcu_val<cuBReal>& max_value, bool use_NNeu)
{
	paMeshCUDA->elC.exchange_halos();
	paMeshCUDA->S.exchange_halos();
	paMeshCUDA->M1.exchange_halos();

	//use_NNeu not needed here, but implementing TransportBaseCUDA interface
	paMeshCUDA->V.IteratePoisson_SOR(poisson_Spin_V, damping, max_error, max_value);
}

//------------------- PRIME SPIN-CHARGE SOLVER

__global__ void Atom_Get_dM_dt_Kernel(cuVEC_VC<cuReal3>& dM_dt, ManagedAtom_DiffEqCubicCUDA& cuaDiffEq)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;

	if (idx < dM_dt.linear_size()) {

		if (dM_dt.is_not_empty(idx)) {

			dM_dt[idx] = cuaDiffEq.dMdt(idx);
		}
	}
}

__global__ void Atom_PrimeSpinSolver_Charge_Kernel(ManagedAtom_MeshCUDA& cuaMesh, TransportCUDA_Spin_V_Funcs& poisson_Spin_V, cuVEC_VC<cuReal3>& dM_dt)
{
	cuVEC_VC<cuBReal>& V = *cuaMesh.pV;
	cuVEC_VC<cuBReal>& elC = *cuaMesh.pelC;
	cuVEC_VC<cuReal3>& S = *cuaMesh.pS;
	cuVEC_VC<cuReal3>& M1 = *cuaMesh.pM1;

	cuVEC<cuBReal>& delsq_V_fixed = *poisson_Spin_V.pdelsq_V_fixed;

	int idx = blockIdx.x * blockDim.x + threadIdx.x;

	if (idx < delsq_V_fixed.linear_size()) {

		delsq_V_fixed[idx] = 0.0;

		//only calculate current on non-empty cells - empty cells have already been assigned 0 at UpdateConfiguration
		if (V.is_not_empty(idx)) {

			bool cppgmr_enabled = cuIsNZ(cuaMesh.pbetaD->get0());
			bool cpump_enabled = cuIsNZ(cuaMesh.pcpump_eff->get0());

			if (cppgmr_enabled || cpump_enabled) {

				cuBReal mu_s = *cuaMesh.pmu_s;
				cuaMesh.update_parameters_ecoarse(idx, *cuaMesh.pmu_s, mu_s);

				int idx_M = M1.position_to_cellidx(V.cellidx_to_position(idx));
				cuReal33 grad_m = M1.grad_neu(idx_M) / mu_s;
				cuReal3 m = M1[idx_M] / mu_s;

				//CPP-GMR contribution
				if (cppgmr_enabled) {

					cuBReal De = *cuaMesh.pDe;
					cuBReal betaD = *cuaMesh.pbetaD;
					cuaMesh.update_parameters_ecoarse(idx, *cuaMesh.pDe, De, *cuaMesh.pbetaD, betaD);

					cuReal33 grad_S = S.grad_neu(idx);
					cuReal3 delsq_S = S.delsq_neu(idx);
					cuBReal div_grad_S_m = (grad_S.i * grad_m.i) + (grad_S.j * grad_m.j) + (grad_S.k * grad_m.k) + (m * delsq_S);

					delsq_V_fixed[idx] += div_grad_S_m * betaD * De / ((cuBReal)MUB_E * elC[idx]);
				}

				//Charge pumping pre-calculation
				if (cpump_enabled) {

					cuReal33 grad_dm_dt = dM_dt.grad_neu(idx_M) / mu_s;
					cuReal3 dm_dt = dM_dt[idx_M] / mu_s;

					cuBReal P = *cuaMesh.pP;
					cuaMesh.update_parameters_ecoarse(idx, *cuaMesh.pP, P);

					cuReal3 dx_m = grad_m.x;
					cuReal3 dy_m = grad_m.y;
					cuReal3 dxx_m = M1.dxx_neu(idx_M) / mu_s;
					cuReal3 dyy_m = M1.dyy_neu(idx_M) / mu_s;

					delsq_V_fixed[idx] += (cuaMesh.pcpump_eff->get0() * P * (cuBReal)HBAR_E / 2) * ((grad_dm_dt.x ^ dx_m) + (grad_dm_dt.y ^ dy_m) + (dm_dt ^ (dxx_m + dyy_m))) * m;
				}
			}
		}
	}
}

//before iterating the spin solver (charge part) we need to prime it : pre-compute values which do not change as the spin solver relaxes.
void Atom_TransportCUDA::PrimeSpinSolver_Charge(void)
{
	//Update dM_dt values if needed
	if (Need_dM_dt_Calculation()) {

		//dM_dt diff ops not needed in Atom_Get_dM_dt_Kernel, but will be needed in Atom_PrimeSpinSolver_Charge_Kernel
		dM_dt.exchange_halos();

		for (mGPU.device_begin(); mGPU != mGPU.device_end(); mGPU++) {

			Atom_Get_dM_dt_Kernel <<< (paMeshCUDA->M1.device_size(mGPU) + CUDATHREADS) / CUDATHREADS, CUDATHREADS >>> 
				(dM_dt.get_deviceobject(mGPU), paMeshCUDA->Get_ManagedAtom_DiffEqCubicCUDA().get_deviceobject(mGPU));
		}
	}

	//the rest are terms to calculate in delsq_V_fixed
	if (Need_delsq_V_fixed_Precalculation()) {

		paMeshCUDA->S.exchange_halos();
		paMeshCUDA->M1.exchange_halos();

		for (mGPU.device_begin(); mGPU != mGPU.device_end(); mGPU++) {

			Atom_PrimeSpinSolver_Charge_Kernel <<< (paMeshCUDA->elC.device_size(mGPU) + CUDATHREADS) / CUDATHREADS, CUDATHREADS >>> 
				(paMeshCUDA->cuaMesh.get_deviceobject(mGPU), poisson_Spin_V.get_deviceobject(mGPU), dM_dt.get_deviceobject(mGPU));
		}
	}
}

//-------------------Calculation Methods : Iterate Spin-Spin Solver

//solve for spin accumulation using Poisson equation for delsq_S, solved using SOR algorithm
void Atom_TransportCUDA::IterateSpinSolver_Spin_SOR(mcu_val<cuBReal>& damping, mcu_val<cuBReal>& max_error, mcu_val<cuBReal>& max_value, bool use_NNeu)
{
	//no halo exchanges needed
	
	//use_NNeu not needed here, but implementing TransportBaseCUDA interface
	paMeshCUDA->S.IteratePoisson_SOR(poisson_Spin_S, damping, max_error, max_value);
}

//------------------- PRIME SPIN-SPIN SOLVER

__global__ void Atom_PrimeSpinSolver_Spin_Kernel(ManagedAtom_MeshCUDA& cuaMesh, TransportCUDA_Spin_S_Funcs& poisson_Spin_S, cuVEC_VC<cuReal3>& dM_dt)
{
	cuVEC_VC<cuBReal>& V = *cuaMesh.pV;
	cuVEC_VC<cuBReal>& elC = *cuaMesh.pelC;
	cuVEC_VC<cuReal3>& E = *cuaMesh.pE;
	cuVEC_VC<cuReal3>& M1 = *cuaMesh.pM1;

	cuVEC<cuReal3>& delsq_S_fixed = *poisson_Spin_S.pdelsq_S_fixed;

	int idx = blockIdx.x * blockDim.x + threadIdx.x;

	if (idx < delsq_S_fixed.linear_size()) {

		delsq_S_fixed[idx] = 0.0;

		//only calculate current on non-empty cells - empty cells have already been assigned 0 at UpdateConfiguration
		if (V.is_not_empty(idx)) {

			bool cpump_enabled = cuIsNZ(cuaMesh.pcpump_eff->get0());
			bool the_enabled = cuIsNZ(cuaMesh.pthe_eff->get0());
			bool she_enabled = cuIsNZ(cuaMesh.pSHA->get0());

			if (poisson_Spin_S.stsolve == STSOLVE_FERROMAGNETIC_ATOM) {

				//magnetic mesh

				cuBReal mu_s = *cuaMesh.pmu_s;
				cuBReal P = *cuaMesh.pP;
				cuBReal De = *cuaMesh.pDe;
				cuaMesh.update_parameters_ecoarse(idx, *cuaMesh.pmu_s, mu_s, *cuaMesh.pP, P, *cuaMesh.pDe, De);

				//term due to drift (non-uniformity of M term, and delsq V contribution - non-uniformity of E term)

				//find grad M and M at the M cell in which the current S cell center is
				int idx_M = M1.position_to_cellidx(V.cellidx_to_position(idx));

				cuReal3 m = M1[idx_M] / mu_s;
				cuReal33 grad_m = M1.grad_neu(idx_M) / mu_s;
				cuReal3 E_dot_del_m = grad_m | E[idx];

				//E_dot_del_m term is very important, but Evaluate_SpinSolver_delsqV_RHS term could be neglected in most cases especially if E is uniform.
				delsq_S_fixed[idx] += (P * (cuBReal)MUB_E * elC[idx] / De) * (poisson_Spin_S.pPoisson_Spin_V->Poisson_RHS(idx) * m - E_dot_del_m);

				//charge pumping and topological Hall effect
				if (cpump_enabled || the_enabled) {

					cuReal3 dx_m = grad_m.x;
					cuReal3 dy_m = grad_m.y;
					cuReal3 dxy_m = M1.dxy_neu(idx_M) / mu_s;
					cuReal3 dxx_m = M1.dxx_neu(idx_M) / mu_s;
					cuReal3 dyy_m = M1.dyy_neu(idx_M) / mu_s;

					if (cpump_enabled) {

						cuReal3 dmdt = dM_dt[idx_M] / mu_s;
						cuReal33 grad_dm_dt = dM_dt.grad_neu(idx_M) / mu_s;

						delsq_S_fixed[idx] += cuaMesh.pcpump_eff->get0() * (elC[idx] * (cuBReal)HBAR_E * (cuBReal)MUB_E / (2 * De)) * ((grad_dm_dt.x ^ dx_m) + (grad_dm_dt.y ^ dy_m) + (dmdt ^ (dxx_m + dyy_m)));
					}

					if (the_enabled) {

						cuBReal n_density = *cuaMesh.pn_density;
						cuaMesh.update_parameters_ecoarse(idx, *cuaMesh.pn_density, n_density);

						delsq_S_fixed[idx] += cuaMesh.pthe_eff->get0() * ((cuBReal)HBAR_E * (cuBReal)MUB_E * elC[idx] * elC[idx] / ((cuBReal)ECHARGE * n_density * De)) * (E[idx].x * ((dxy_m ^ dy_m) + (dx_m ^ dyy_m)) - E[idx].y * ((dxx_m ^ dy_m) + (dx_m ^ dxy_m)));
					}
				}
			}
		}
	}
}

//before iterating the spin solver (spin part) we need to prime it : pre-compute values which do not change as the spin solver relaxes.
void Atom_TransportCUDA::PrimeSpinSolver_Spin(void)
{
	if (Need_delsq_S_fixed_Precalculation()) {

		paMeshCUDA->V.exchange_halos();
		paMeshCUDA->S.exchange_halos();
		paMeshCUDA->M1.exchange_halos();
		paMeshCUDA->elC.exchange_halos();
		paMeshCUDA->E.exchange_halos();
		dM_dt.exchange_halos();

		for (mGPU.device_begin(); mGPU != mGPU.device_end(); mGPU++) {

			Atom_PrimeSpinSolver_Spin_Kernel <<< (paMeshCUDA->elC.device_size(mGPU) + CUDATHREADS) / CUDATHREADS, CUDATHREADS >>>
				(paMeshCUDA->cuaMesh.get_deviceobject(mGPU), poisson_Spin_S.get_deviceobject(mGPU), dM_dt.get_deviceobject(mGPU));
		}
	}
}

//--------------------------------------------------------------- Effective field from spin accumulation

__global__ void Atom_CalculateSAField_Kernel(ManagedAtom_MeshCUDA& cuaMesh)
{
	cuVEC<cuReal3>& Heff1 = *cuaMesh.pHeff1;
	cuVEC_VC<cuReal3>& M1 = *cuaMesh.pM1;
	cuVEC_VC<cuReal3>& S = *cuaMesh.pS;

	int idx = blockIdx.x * blockDim.x + threadIdx.x;

	if (idx < M1.linear_size()) {

		if (M1.is_not_empty(idx)) {

			cuBReal De = *cuaMesh.pDe;
			cuBReal ts_eff = *cuaMesh.pts_eff;
			cuBReal grel = *cuaMesh.pgrel;
			cuBReal mu_s = *cuaMesh.pmu_s;
			cuBReal l_ex = *cuaMesh.pl_ex;
			cuBReal l_ph = *cuaMesh.pl_ph;
			cuaMesh.update_parameters_mcoarse(idx, *cuaMesh.pgrel, grel, *cuaMesh.pmu_s, mu_s, *cuaMesh.pDe, De, *cuaMesh.pts_eff, ts_eff, *cuaMesh.pl_ex, l_ex, *cuaMesh.pl_ph, l_ph);

			if (cuIsNZ((cuBReal)grel)) {

				cuReal3 Sa = S.weighted_average(M1.cellidx_to_position(idx), M1.h);

				cuBReal conv = M1.h.dim() / (cuBReal)MUB;
				Heff1[idx] += (conv * De * ts_eff / ((cuBReal)GAMMA * grel * mu_s)) * (Sa / (l_ex * l_ex) + (M1[idx] ^ Sa) / (l_ph * l_ph * mu_s));
			}
		}
	}
}

//Spin accumulation field
void Atom_TransportCUDA::CalculateSAField(void)
{
	if (stsolve == STSOLVE_FERROMAGNETIC_ATOM) {

		for (mGPU.device_begin(); mGPU != mGPU.device_end(); mGPU++) {

			Atom_CalculateSAField_Kernel <<< (paMeshCUDA->M1.device_size(mGPU) + CUDATHREADS) / CUDATHREADS, CUDATHREADS >>> 
				(paMeshCUDA->cuaMesh.get_deviceobject(mGPU));
		}
	}
}

//--------------------------------------------------------------- Effective field from interface spin accumulation drop

__global__ void Atom_CalculateSAInterfaceField_Kernel(CMBNDInfoCUDA& contact, TransportCUDA_Spin_S_CMBND_Sec& cmbndFuncs_sec, TransportCUDA_Spin_S_CMBND_Pri& cmbndFuncs_pri)
{
	//primary mesh is atomistic (cmbndFuncs_pri.pcuaMesh), and secondary is MeshCUDA (cmbndFuncs_sec.pcuMesh) since secondary is non-magnetic

	cuVEC<cuReal3>& Heff1 = *cmbndFuncs_pri.pcuaMesh->pHeff1;
	cuVEC_VC<cuReal3>& M1 = *cmbndFuncs_pri.pcuaMesh->pM1;
	cuVEC_VC<cuReal3>& S_pri = *cmbndFuncs_pri.pcuaMesh->pS;
	//access S on first device, which contains origin of entire mcuVEC
	cuVEC_VC<cuReal3>& S0_sec = *cmbndFuncs_sec.ppcuMesh[0]->pS;

	int box_idx = blockIdx.x * blockDim.x + threadIdx.x;

	//interface conductance method with F being the primary mesh : calculate and set spin torque

	//convert the cells box from S mesh to M mesh
	cuINT3 mbox_start = M1.cellidx_from_position(S_pri.cellidx_to_position(contact.cells_box.s) + M1.rect.s);
	cuINT3 mbox_end = M1.cellidx_from_position(S_pri.cellidx_to_position(contact.cells_box.e - cuINT3(1)) + M1.rect.s);

	if ((mbox_end.i - mbox_start.i) == 0) mbox_end.i = mbox_start.i + 1;
	if ((mbox_end.j - mbox_start.j) == 0) mbox_end.j = mbox_start.j + 1;
	if ((mbox_end.k - mbox_start.k) == 0) mbox_end.k = mbox_start.k + 1;

	cuINT3 box_sizes = mbox_end - mbox_start;
	
	if (box_idx < box_sizes.dim()) {

		//the cellsize perpendicular to the contact (in the M mesh)
		cuBReal dh = (cuReal3(contact.cell_shift) & M1.h).norm();

		int i = (box_idx % box_sizes.x) + mbox_start.i;
		int j = ((box_idx / box_sizes.x) % box_sizes.y) + mbox_start.j;
		int k = (box_idx / (box_sizes.x * box_sizes.y)) + mbox_start.k;

		//index of magnetic cell 1
		int mcell1_idx = i + j * M1.n.x + k * M1.n.x*M1.n.y;

		if (M1.is_empty(mcell1_idx)) return;

		cuBReal grel = *cmbndFuncs_pri.pcuaMesh->pgrel;
		cuBReal mu_s = *cmbndFuncs_pri.pcuaMesh->pmu_s;
		cuBReal tsi_eff = *cmbndFuncs_pri.pcuaMesh->ptsi_eff;
		cmbndFuncs_pri.pcuaMesh->update_parameters_mcoarse(mcell1_idx, *cmbndFuncs_pri.pcuaMesh->pgrel, grel, *cmbndFuncs_pri.pcuaMesh->pmu_s, mu_s, *cmbndFuncs_pri.pcuaMesh->ptsi_eff, tsi_eff);

		if (cuIsNZ((cuBReal)grel)) {

			//position at interface relative to primary mesh
			cuReal3 mhshift_primary = contact.hshift_primary.normalized() & M1.h;
			cuReal3 relpos_interf = ((cuReal3(i, j, k) + cuReal3(0.5)) & M1.h) + mhshift_primary / 2;

			cuReal3 relpos_1 = relpos_interf - contact.hshift_primary / 2;

			cuReal3 relpos_m1 = S_pri.rect.s - S0_sec.rect.s + relpos_interf + contact.hshift_secondary / 2;
			//get device and device-relative position on secondary side
			int device = 0;
			cuReal3 devrelpos_m1 = cmbndFuncs_sec.global_relpos_to_device_relpos(relpos_m1, device);
			cuVEC_VC<cuReal3>& S_sec = *cmbndFuncs_sec.ppcuMesh[device]->pS;

			cuReal3 stencil_pri = M1.h - cu_mod(mhshift_primary) + cu_mod(contact.hshift_primary);
			cuReal3 stencil_sec = M1.h - cu_mod(mhshift_primary) + cu_mod(contact.hshift_secondary);

			//S values
			cuReal3 S_1 = S_pri.weighted_average(relpos_1, stencil_pri);
			cuReal3 S_2 = S_pri.weighted_average(relpos_1 - contact.hshift_primary, stencil_pri);
			cuReal3 S_m1 = S_sec.weighted_average(devrelpos_m1, stencil_sec);
			cuReal3 S_m2 = S_sec.weighted_average(devrelpos_m1 + contact.hshift_secondary, stencil_sec);

			//c values
			cuBReal c_1 = cmbndFuncs_pri.c_func_sec(relpos_1, stencil_pri);
			cuBReal c_2 = cmbndFuncs_pri.c_func_sec(relpos_1 - contact.hshift_primary, stencil_pri);
			cuBReal c_m1 = cmbndFuncs_sec.c_func_sec(relpos_m1, stencil_sec);
			cuBReal c_m2 = cmbndFuncs_sec.c_func_sec(relpos_m1 + contact.hshift_secondary, stencil_sec);

			//Calculate S drop at the interface
			cuReal3 Vs_F = 1.5 * c_1 * S_1 - 0.5 * c_2 * S_2;
			cuReal3 Vs_N = 1.5 * c_m1 * S_m1 - 0.5 * c_m2 * S_m2;
			cuReal3 dVs = Vs_F - Vs_N;

			//Get G values from top contacting mesh
			cuReal2 Gmix;
			if (contact.IsPrimaryTop()) {

				Gmix = *cmbndFuncs_pri.pcuaMesh->pGmix;
				cmbndFuncs_pri.pcuaMesh->update_parameters_mcoarse(mcell1_idx, *cmbndFuncs_pri.pcuaMesh->pGmix, Gmix);
			}
			else {

				Gmix = *cmbndFuncs_sec.ppcuMesh[device]->pGmix;
				cmbndFuncs_sec.ppcuMesh[device]->update_parameters_atposition(relpos_m1, *cmbndFuncs_sec.ppcuMesh[device]->pGmix, Gmix);
			}

			cuBReal conv = M1.h.dim() / (cuBReal)MUB;
			cuBReal gI = (conv * 2.0 * (cuBReal)GMUB_2E / dh) * cuReal2(Gmix).j / (-(cuBReal)GAMMA * grel * mu_s);
			cuBReal gR = (conv * 2.0 * (cuBReal)GMUB_2E / dh) * cuReal2(Gmix).i / (-(cuBReal)GAMMA * grel * mu_s);

			Heff1[mcell1_idx] += tsi_eff * (gI * dVs + gR * (M1[mcell1_idx] ^ dVs) / mu_s);
		}
	}
}

//Calculate the field resulting from interface spin accumulation torque for a given contact (in magnetic meshes for NF interfaces with G interface conductance set)
void Atom_TransportCUDA::CalculateSAInterfaceField(TransportBaseCUDA* ptrans_sec, mCMBNDInfoCUDA& contactCUDA, bool primary_top)
{
	//the top contacting mesh sets G values
	bool isGInterface_Enabled = ((primary_top && GInterface_Enabled()) || (!primary_top && ptrans_sec->GInterface_Enabled()));

	if (isGInterface_Enabled && stsolve == STSOLVE_FERROMAGNETIC_ATOM && (ptrans_sec->Get_STSolveType() == STSOLVE_NORMALMETAL || ptrans_sec->Get_STSolveType() == STSOLVE_TUNNELING)) {

		for (mGPU.device_begin(); mGPU != mGPU.device_end(); mGPU++) {

			//primary mesh is atomistic (poisson_Spin_S), and secondary is MeshCUDA (ptrans_sec->poisson_Spin_S) since secondary is non-magnetic
			Atom_CalculateSAInterfaceField_Kernel <<< (paMeshCUDA->M1.device_size(mGPU) + CUDATHREADS) / CUDATHREADS, CUDATHREADS >>>
				(contactCUDA.get_deviceobject(mGPU), ptrans_sec->spin_S_cmbnd_funcs_sec.get_deviceobject(mGPU), spin_S_cmbnd_funcs_pri.get_deviceobject(mGPU));
		}
	}
}

#endif

#endif